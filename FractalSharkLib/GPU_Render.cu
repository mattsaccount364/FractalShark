// TODO: 2x32 perturb is busted, do git diff
// Re-run  profile on current default view
#include <stdio.h>
#include <iostream>

#include "GPU_Render.h"
#include "dbldbl.cuh"
#include "dblflt.cuh"
#include "QuadDouble/gqd_basic.cuh"
#include "QuadFloat/gqf_basic.cuh"

#include "CudaDblflt.h"

#include "GPU_BLAS.h"

#include "HDRFloatComplex.h"
#include "BLA.h"
#include "HDRFloat.h"

#include "GPU_LAReference.h"

#include "GPU_LAInfoDeep.h"
#include "LAReference.h"

#include <type_traits>
#include <stdint.h>
//#include <cuda/pipeline>
//#include <cuda_pipeline.h>

enum FractalSharkError : int32_t {
    Error1 = 10000,
    Error2,
    Error3,
    Error4,
    Error5,
    Error6,
    Error7,
    Error8,
    Error9,
};

constexpr static bool Default = true;
constexpr static bool ForceEnable = true;

constexpr static bool EnableGpu1x32 = Default;
constexpr static bool EnableGpu2x32 = Default;
constexpr static bool EnableGpu4x32 = Default;
constexpr static bool EnableGpu1x64 = Default;
constexpr static bool EnableGpu2x64 = Default;
constexpr static bool EnableGpu4x64 = Default;
constexpr static bool EnableGpuHDRx32 = Default;

constexpr static bool EnableGpu1x32PerturbedScaled = Default;
constexpr static bool EnableGpu2x32PerturbedScaled = Default;
constexpr static bool EnableGpuHDRx32PerturbedScaled = Default;

constexpr static bool EnableGpu1x32PerturbedScaledBLA = Default;
constexpr static bool EnableGpu1x64PerturbedBLA = Default;
constexpr static bool EnableGpuHDRx32PerturbedBLA = Default;
constexpr static bool EnableGpuHDRx64PerturbedBLA = Default;

constexpr static bool EnableGpu1x32PerturbedLAv2 = ForceEnable;
constexpr static bool EnableGpu1x32PerturbedLAv2PO = ForceEnable;
constexpr static bool EnableGpu1x32PerturbedLAv2LAO = ForceEnable;
constexpr static bool EnableGpu2x32PerturbedLAv2 = ForceEnable;
constexpr static bool EnableGpu2x32PerturbedLAv2PO = ForceEnable;
constexpr static bool EnableGpu2x32PerturbedLAv2LAO = ForceEnable;
constexpr static bool EnableGpu1x64PerturbedLAv2 = ForceEnable;
constexpr static bool EnableGpu1x64PerturbedLAv2PO = ForceEnable;
constexpr static bool EnableGpu1x64PerturbedLAv2LAO = ForceEnable;
constexpr static bool EnableGpuHDRx32PerturbedLAv2 = ForceEnable;
constexpr static bool EnableGpuHDRx32PerturbedLAv2PO = ForceEnable;
constexpr static bool EnableGpuHDRx32PerturbedLAv2LAO = ForceEnable;
constexpr static bool EnableGpuHDRx2x32PerturbedLAv2 = ForceEnable;
constexpr static bool EnableGpuHDRx2x32PerturbedLAv2PO = ForceEnable;
constexpr static bool EnableGpuHDRx2x32PerturbedLAv2LAO = ForceEnable;
constexpr static bool EnableGpuHDRx64PerturbedLAv2 = ForceEnable;
constexpr static bool EnableGpuHDRx64PerturbedLAv2PO = ForceEnable;
constexpr static bool EnableGpuHDRx64PerturbedLAv2LAO = ForceEnable;

//#define DEFAULT_KERNEL_LAUNCH_PARAMS nb_blocks, threads_per_block, 0, hipStreamPerThread
#define DEFAULT_KERNEL_LAUNCH_PARAMS nb_blocks, threads_per_block

__device__
size_t
ConvertLocToIndex(size_t X, size_t Y, size_t OriginalWidth) {
    auto RoundedBlocks = OriginalWidth / GPURenderer::NB_THREADS_W + (OriginalWidth % GPURenderer::NB_THREADS_W != 0);
    auto RoundedWidth = RoundedBlocks * GPURenderer::NB_THREADS_W;
    return Y * RoundedWidth + X;
}

#include "InitStatics.cuh"
#include "BLA.cuh"
#include "Perturb.cuh"
#include "PerturbResultsCollection.cuh"
#include "LowPrecisionKernels.cuh"
#include "BLAKernels.cuh"
#include "ReductionKernels.cuh"
#include "AntialiasingKernel.cuh"
#include "LAKernel.cuh"
#include "DisabledKernels.cuh"
#include "ScaledKernels.cuh"

GPURenderer::GPURenderer() {
    ClearLocals();
}

GPURenderer::~GPURenderer() {
    ResetMemory(ResetLocals::Yes, ResetPalettes::Yes, ResetPerturb::Yes);

    if (m_Stream1Initialized) {
        hipStreamDestroy(m_Stream1);
    }
}

void GPURenderer::ResetPalettesOnly() {
    if (Pals.local_pal != nullptr) {
        hipFree(Pals.local_pal);
        Pals.local_pal = nullptr;
    }
}

void GPURenderer::ResetMemory(
    ResetLocals locals,
    ResetPalettes palettes,
    ResetPerturb perturb) {

    if (OutputIterMatrix != nullptr) {
        hipFree(OutputIterMatrix);
        OutputIterMatrix = nullptr;
    }

    if (OutputReductionResults != nullptr) {
        hipFree(OutputReductionResults);
        OutputReductionResults = nullptr;
    }

    if (OutputColorMatrix.aa_colors != nullptr) {
        hipFree(OutputColorMatrix.aa_colors);
        OutputColorMatrix.aa_colors = nullptr;
    }

    if (palettes == ResetPalettes::Yes) {
        ResetPalettesOnly();
    }

    if (perturb == ResetPerturb::Yes) {
        m_PerturbResults.DeleteAll();
    }

    if (locals == ResetLocals::Yes) {
        ClearLocals();
    }
}

void GPURenderer::ClearLocals() {
    // This function assumes memory is freed!
    OutputIterMatrix = nullptr;
    OutputReductionResults = nullptr;
    OutputColorMatrix = {};

    m_Width = 0;
    m_Height = 0;
    local_color_width = 0;
    local_color_height = 0;
    m_Antialiasing = 0;
    m_IterTypeSize = 0;
    w_block = 0;
    h_block = 0;
    w_color_block = 0;
    h_color_block = 0;
    N_cu = 0;
    N_color_cu = 0;
    
    m_Stream1Initialized = false;

    Pals = {};

    m_PerturbResults = {};
}

template<typename IterType>
void GPURenderer::ClearMemory() {
    if (OutputIterMatrix != nullptr) {
        hipMemset(OutputIterMatrix, 0, N_cu * sizeof(IterType));
    }

    if (OutputReductionResults != nullptr) {
        hipMemset(OutputReductionResults, 0, sizeof(IterType));
    }

    if (OutputColorMatrix.aa_colors != nullptr) {
        hipMemset(OutputColorMatrix.aa_colors, 0, N_color_cu * sizeof(Color16));
    }
}

template
void GPURenderer::ClearMemory<uint32_t>();
template
void GPURenderer::ClearMemory<uint64_t>();

template<typename IterType>
uint32_t GPURenderer::InitializeMemory(
    uint32_t antialias_width, // screen width
    uint32_t antialias_height, // screen height
    uint32_t antialiasing,
    const uint16_t* palR,
    const uint16_t* palG,
    const uint16_t* palB,
    uint32_t palIters,
    uint32_t paletteAuxDepth)
{
    if (Pals.palette_aux_depth != paletteAuxDepth) {
        Pals.palette_aux_depth = paletteAuxDepth;
    }

    // Re-do palettes only.
    if ((Pals.cached_hostPalR != palR) ||
        (Pals.cached_hostPalG != palG) ||
        (Pals.cached_hostPalB != palB)) {

        ResetPalettesOnly();

        Pals = Palette(
            nullptr,
            palIters,
            paletteAuxDepth,
            palR,
            palG,
            palB);

        // Palettes:
        hipError_t err = hipMallocManaged(
            &Pals.local_pal,
            Pals.local_palIters * sizeof(Color16),
            hipMemAttachGlobal);
        if (err != hipSuccess) {
            ResetMemory(ResetLocals::Yes, ResetPalettes::Yes, ResetPerturb::Yes);
            return err;
        }

        // TODO the incoming data should be rearranged so we can memcpy
        // Copy in palettes
        for (uint32_t i = 0; i < Pals.local_palIters; i++) {
            Pals.local_pal[i].r = palR[i];
            Pals.local_pal[i].g = palG[i];
            Pals.local_pal[i].b = palB[i];
        }
    }

    if ((m_Width == antialias_width) &&
        (m_Height == antialias_height) &&
        (m_Antialiasing == antialiasing) &&
        (m_IterTypeSize == sizeof(IterType))) {
        return 0;
    }

    //if (w % NB_THREADS_W != 0) {
    //    return FractalSharkError::Error1;
    //}

    //if (h % NB_THREADS_H != 0) {
    //    return FractalSharkError::Error2;
    //}

    if (antialiasing > 4 || antialiasing < 1) {
        return FractalSharkError::Error3;
    }

    if (antialias_width % antialiasing != 0) {
        return FractalSharkError::Error4;
    }

    if (antialias_height % antialiasing != 0) {
        return FractalSharkError::Error5;
    }

    w_block =
        antialias_width / GPURenderer::NB_THREADS_W +
        (antialias_width % GPURenderer::NB_THREADS_W != 0);
    h_block =
        antialias_height / GPURenderer::NB_THREADS_H +
        (antialias_height % GPURenderer::NB_THREADS_H != 0);
    m_Width = antialias_width;
    m_Height = antialias_height;
    m_Antialiasing = antialiasing;
    m_IterTypeSize = sizeof(IterType);
    N_cu = w_block * NB_THREADS_W * h_block * NB_THREADS_H;

    const auto no_antialias_width = antialias_width / antialiasing;
    const auto no_antialias_height = antialias_height / antialiasing;
    w_color_block =
        no_antialias_width / GPURenderer::NB_THREADS_W_AA +
        (no_antialias_width % GPURenderer::NB_THREADS_W_AA != 0);
    h_color_block =
        no_antialias_height / GPURenderer::NB_THREADS_H_AA +
        (no_antialias_height % GPURenderer::NB_THREADS_H_AA != 0);
    local_color_width = no_antialias_width;
    local_color_height = no_antialias_height;
    N_color_cu = w_color_block * NB_THREADS_W_AA * h_color_block * NB_THREADS_H_AA;

    ResetMemory(ResetLocals::No, ResetPalettes::No, ResetPerturb::No);

    {
        IterType* tempiter = nullptr;
        hipError_t err = hipMallocManaged(
            &tempiter,
            N_cu * sizeof(IterType),
            hipMemAttachGlobal);
        if (err != hipSuccess) {
            ResetMemory(ResetLocals::Yes, ResetPalettes::Yes, ResetPerturb::Yes);
            return err;
        }

        OutputIterMatrix = tempiter;
    }

    {
        // Unconditionally allocate uint64_t
        ReductionResults* tempreduction = nullptr;
        hipError_t err = hipMallocManaged(
            &tempreduction,
            sizeof(ReductionResults));
        if (err != hipSuccess) {
            ResetMemory(ResetLocals::Yes, ResetPalettes::Yes, ResetPerturb::Yes);
            return err;
        }

        OutputReductionResults = tempreduction;
    }

    {
        Color16* tempaa = nullptr;

        hipError_t err = hipMallocManaged(
            &tempaa,
            N_color_cu * sizeof(Color16),
            hipMemAttachGlobal);
        if (err != hipSuccess) {
            ResetMemory(ResetLocals::Yes, ResetPalettes::Yes, ResetPerturb::Yes);
            return err;
        }

        OutputColorMatrix.aa_colors = tempaa;
    }

    ClearMemory<IterType>();

    if (m_Stream1Initialized == false) {
        hipError_t err = hipDeviceGetStreamPriorityRange(&m_StreamPriorityLow, &m_StreamPriorityHigh);
        if (err != hipSuccess) {
            ResetMemory(ResetLocals::Yes, ResetPalettes::Yes, ResetPerturb::Yes);
            return err;
        }

        // hipStreamNonBlocking
        err = hipStreamCreateWithPriority(&m_Stream1, hipStreamNonBlocking, m_StreamPriorityHigh);
        if (err != hipSuccess) {
            ResetMemory(ResetLocals::Yes, ResetPalettes::Yes, ResetPerturb::Yes);
            return err;
        }

        m_Stream1Initialized = true;
    }

    return 0;
}

template
uint32_t GPURenderer::InitializeMemory<uint32_t>(
    uint32_t antialias_width, // screen width
    uint32_t antialias_height, // screen height
    uint32_t antialiasing,
    const uint16_t* palR,
    const uint16_t* palG,
    const uint16_t* palB,
    uint32_t palIters,
    uint32_t paletteAuxDepth);

template
uint32_t GPURenderer::InitializeMemory<uint64_t>(
    uint32_t antialias_width, // screen width
    uint32_t antialias_height, // screen height
    uint32_t antialiasing,
    const uint16_t* palR,
    const uint16_t* palG,
    const uint16_t* palB,
    uint32_t palIters,
    uint32_t paletteAuxDepth);

template<typename IterType, class T1, class SubType, CalcBad Bad, class T2>
uint32_t GPURenderer::InitializePerturb(
    size_t GenerationNumber1,
    const MattPerturbResults<IterType, T1, Bad>* Perturb1,
    size_t GenerationNumber2,
    const MattPerturbResults<IterType, T2, Bad>* Perturb2,
    const LAReference<IterType, T1, SubType>* LaReferenceHost,
    size_t LaGenerationNumber)
{
    if (GenerationNumber1 != m_PerturbResults.GetHostGenerationNumber1() ||
        GenerationNumber2 != m_PerturbResults.GetHostGenerationNumber2() ||
        LaGenerationNumber != m_PerturbResults.GetHostLaGenerationNumber1()) {
        m_PerturbResults.DeleteAll();
    }

    if (GenerationNumber1 != m_PerturbResults.GetHostGenerationNumber1()) {
        auto *CudaResults1 = new MattPerturbSingleResults<IterType, T1, Bad>{
            Perturb1->size,
            Perturb1->PeriodMaybeZero,
            Perturb1->iters
        };

        auto result = CudaResults1->CheckValid();
        if (result != 0) {
            ResetMemory(ResetLocals::Yes, ResetPalettes::Yes, ResetPerturb::Yes);
            return result;
        }

        m_PerturbResults.SetPtr1(GenerationNumber1, CudaResults1);
    }

    if (GenerationNumber2 != m_PerturbResults.GetHostGenerationNumber2()) {
        auto* CudaResults2 = new MattPerturbSingleResults<IterType, T2, Bad>{
            Perturb2->size,
            Perturb2->PeriodMaybeZero,
            Perturb2->iters
        };

        auto result = CudaResults2->CheckValid();
        if (result != 0) {
            ResetMemory(ResetLocals::Yes, ResetPalettes::Yes, ResetPerturb::Yes);
            return result;
        }

        m_PerturbResults.SetPtr2(GenerationNumber2, CudaResults2);
    }

    if (LaGenerationNumber != m_PerturbResults.GetHostLaGenerationNumber1()) {
        auto* LaReferenceCuda = new GPU_LAReference<IterType, T1, SubType>{ *LaReferenceHost };
        auto result = LaReferenceCuda->CheckValid();
        if (result != 0) {
            ResetMemory(ResetLocals::Yes, ResetPalettes::Yes, ResetPerturb::Yes);
            return result;
        }

        m_PerturbResults.SetLaReference1(LaGenerationNumber, LaReferenceCuda);
    }

    return hipSuccess;
}

template
uint32_t GPURenderer::InitializePerturb<uint32_t, float, float, CalcBad::Disable, float>(
    size_t GenerationNumber1,
    const MattPerturbResults<uint32_t, float, CalcBad::Disable>* Perturb1,
    size_t GenerationNumber2,
    const MattPerturbResults<uint32_t, float, CalcBad::Disable>* Perturb2,
    const LAReference<uint32_t, float, float>* LaReferenceHost,
    size_t LaGenerationNumber1);
template
uint32_t GPURenderer::InitializePerturb<uint32_t, double, double, CalcBad::Disable, double>(
    size_t GenerationNumber1,
    const MattPerturbResults<uint32_t, double, CalcBad::Disable>* Perturb1,
    size_t GenerationNumber2,
    const MattPerturbResults<uint32_t, double, CalcBad::Disable>* Perturb2,
    const LAReference<uint32_t, double, double>* LaReferenceHost,
    size_t LaGenerationNumber1);
template
uint32_t GPURenderer::InitializePerturb<uint32_t, CudaDblflt<MattDblflt>, CudaDblflt<MattDblflt>, CalcBad::Disable, CudaDblflt<MattDblflt>>(
    size_t GenerationNumber1,
    const MattPerturbResults<uint32_t, CudaDblflt<MattDblflt>, CalcBad::Disable>* Perturb1,
    size_t GenerationNumber2,
    const MattPerturbResults<uint32_t, CudaDblflt<MattDblflt>, CalcBad::Disable>* Perturb2,
    const LAReference<uint32_t, CudaDblflt<MattDblflt>, CudaDblflt<MattDblflt>>* LaReferenceHost,
    size_t LaGenerationNumber1);
template
uint32_t GPURenderer::InitializePerturb<uint32_t, class HDRFloat<float>, float, CalcBad::Disable, HDRFloat<float>>(
    size_t GenerationNumber1,
    const MattPerturbResults<uint32_t, HDRFloat<float>, CalcBad::Disable>* Perturb1,
    size_t GenerationNumber2,
    const MattPerturbResults<uint32_t, HDRFloat<float>, CalcBad::Disable>* Perturb2,
    const LAReference<uint32_t, HDRFloat<float>, float>* LaReferenceHost,
    size_t LaGenerationNumber1);
template
uint32_t GPURenderer::InitializePerturb<uint32_t, class HDRFloat<double>, double, CalcBad::Disable, HDRFloat<double>>(
    size_t GenerationNumber1,
    const MattPerturbResults<uint32_t, HDRFloat<double>, CalcBad::Disable>* Perturb1,
    size_t GenerationNumber2,
    const MattPerturbResults<uint32_t, HDRFloat<double>, CalcBad::Disable>* Perturb2,
    const LAReference<uint32_t, HDRFloat<double>, double>* LaReferenceHost,
    size_t LaGenerationNumber1);
template
uint32_t GPURenderer::InitializePerturb<uint32_t, class HDRFloat<CudaDblflt<MattDblflt>>, CudaDblflt<MattDblflt>, CalcBad::Disable, HDRFloat<CudaDblflt<MattDblflt>>>(
    size_t GenerationNumber1,
    const MattPerturbResults<uint32_t, HDRFloat<CudaDblflt<MattDblflt>>, CalcBad::Disable>* Perturb1,
    size_t GenerationNumber2,
    const MattPerturbResults<uint32_t, HDRFloat<CudaDblflt<MattDblflt>>, CalcBad::Disable>* Perturb2,
    const LAReference<uint32_t, HDRFloat<CudaDblflt<MattDblflt>>, CudaDblflt<MattDblflt>>* LaReferenceHost,
    size_t LaGenerationNumber1);

template
uint32_t GPURenderer::InitializePerturb<uint64_t, float, float, CalcBad::Disable, float>(
    size_t GenerationNumber1,
    const MattPerturbResults<uint64_t, float, CalcBad::Disable>* Perturb1,
    size_t GenerationNumber2,
    const MattPerturbResults<uint64_t, float, CalcBad::Disable>* Perturb2,
    const LAReference<uint64_t, float, float>* LaReferenceHost,
    size_t LaGenerationNumber1);
template
uint32_t GPURenderer::InitializePerturb<uint64_t, double, double, CalcBad::Disable, double>(
    size_t GenerationNumber1,
    const MattPerturbResults<uint64_t, double, CalcBad::Disable>* Perturb1,
    size_t GenerationNumber2,
    const MattPerturbResults<uint64_t, double, CalcBad::Disable>* Perturb2,
    const LAReference<uint64_t, double, double>* LaReferenceHost,
    size_t LaGenerationNumber1);
template
uint32_t GPURenderer::InitializePerturb<uint64_t, CudaDblflt<MattDblflt>, CudaDblflt<MattDblflt>, CalcBad::Disable, CudaDblflt<MattDblflt>>(
    size_t GenerationNumber1,
    const MattPerturbResults<uint64_t, CudaDblflt<MattDblflt>, CalcBad::Disable>* Perturb1,
    size_t GenerationNumber2,
    const MattPerturbResults<uint64_t, CudaDblflt<MattDblflt>, CalcBad::Disable>* Perturb2,
    const LAReference<uint64_t, CudaDblflt<MattDblflt>, CudaDblflt<MattDblflt>>* LaReferenceHost,
    size_t LaGenerationNumber1);
template
uint32_t GPURenderer::InitializePerturb<uint64_t, class HDRFloat<float>, float, CalcBad::Disable, HDRFloat<float>>(
    size_t GenerationNumber1,
    const MattPerturbResults<uint64_t, HDRFloat<float>, CalcBad::Disable>* Perturb1,
    size_t GenerationNumber2,
    const MattPerturbResults<uint64_t, HDRFloat<float>, CalcBad::Disable>* Perturb2,
    const LAReference<uint64_t, HDRFloat<float>, float>* LaReferenceHost,
    size_t LaGenerationNumber1);
template
uint32_t GPURenderer::InitializePerturb<uint64_t, class HDRFloat<double>, double, CalcBad::Disable, HDRFloat<double>>(
    size_t GenerationNumber1,
    const MattPerturbResults<uint64_t, HDRFloat<double>, CalcBad::Disable>* Perturb1,
    size_t GenerationNumber2,
    const MattPerturbResults<uint64_t, HDRFloat<double>, CalcBad::Disable>* Perturb2,
    const LAReference<uint64_t, HDRFloat<double>, double>* LaReferenceHost,
    size_t LaGenerationNumber1);
template
uint32_t GPURenderer::InitializePerturb<uint64_t, class HDRFloat<CudaDblflt<MattDblflt>>, CudaDblflt<MattDblflt>, CalcBad::Disable, HDRFloat<CudaDblflt<MattDblflt>>>(
    size_t GenerationNumber1,
    const MattPerturbResults<uint64_t, HDRFloat<CudaDblflt<MattDblflt>>, CalcBad::Disable>* Perturb1,
    size_t GenerationNumber2,
    const MattPerturbResults<uint64_t, HDRFloat<CudaDblflt<MattDblflt>>, CalcBad::Disable>* Perturb2,
    const LAReference<uint64_t, HDRFloat<CudaDblflt<MattDblflt>>, CudaDblflt<MattDblflt>>* LaReferenceHost,
    size_t LaGenerationNumber1);

bool GPURenderer::MemoryInitialized() const {
    if (OutputIterMatrix == nullptr) {
        return false;
    }

    if (OutputReductionResults == nullptr) {
        return false;
    }

    if (OutputColorMatrix.aa_colors == nullptr) {
        return false;
    }

    return true;
}

// Not the same as OnlyAA
template<typename IterType>
uint32_t GPURenderer::RenderAsNeeded(
    IterType n_iterations,
    IterType* iter_buffer,
    Color16* color_buffer) {

    uint32_t result = hipSuccess;

    // TODO
    //result = RunAntialiasing(n_iterations, hipStreamDefault);
    //if (!result) {
    //    result = ExtractItersAndColors<IterType, false>(iter_buffer, color_buffer);
    //}

    return result;
}

template<typename IterType>
void GPURenderer::RenderAsNeeded(
    uint32_t &result,
    IterType n_iterations,
    IterType* iter_buffer,
    Color16* color_buffer) {

    result = RenderAsNeeded(n_iterations, iter_buffer, color_buffer);
}

template<typename IterType>
uint32_t GPURenderer::RenderCurrent(
    IterType n_iterations,
    IterType* iter_buffer,
    Color16* color_buffer,
    ReductionResults* reduction_results) {

    if (!MemoryInitialized()) {
        return hipSuccess;
    }

    uint32_t result = hipSuccess;

    if (iter_buffer == nullptr) {
        result = RunAntialiasing(n_iterations, &m_Stream1);

        if (!result) {
            result = ExtractItersAndColors<IterType, true>(
                iter_buffer,
                color_buffer,
                reduction_results);
        }
    }
    else {
        hipStream_t stream = 0;
        result = RunAntialiasing(n_iterations, &stream);

        if (!result) {
            result = ExtractItersAndColors<IterType, false>(
                iter_buffer,
                color_buffer,
                reduction_results);
        }
    }

    return result;
}

template uint32_t GPURenderer::RenderCurrent(
    uint32_t n_iterations,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    ReductionResults* reduction_results);
template uint32_t GPURenderer::RenderCurrent(
    uint64_t n_iterations,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    ReductionResults* reduction_results);

uint32_t GPURenderer::SyncStream(bool altStream) {
    if (altStream) {
        return hipStreamSynchronize(m_Stream1);
    }
    else {
        return hipStreamSynchronize(hipStreamDefault);
    }
}

template<typename IterType, class T>
uint32_t GPURenderer::Render(
    RenderAlgorithm algorithm,
    IterType* iter_buffer,
    Color16* color_buffer,
    T cx,
    T cy,
    T dx,
    T dy,
    IterType n_iterations,
    int iteration_precision)
{
    if (!MemoryInitialized()) {
        return hipSuccess;
    }

    dim3 nb_blocks(w_block, h_block, 1);
    dim3 threads_per_block(NB_THREADS_W, NB_THREADS_H, 1);

    if (algorithm == RenderAlgorithm::Gpu1x64) {
        // all are doubleOnly
        if constexpr (EnableGpu1x64 && std::is_same<T, double>::value) {
            switch (iteration_precision) {
            case 1:
                mandel_1x_double<IterType, 1> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                    static_cast<IterType*>(OutputIterMatrix),
                    OutputColorMatrix,
                    m_Width, m_Height, cx, cy, dx, dy,
                    n_iterations);
                break;
            case 4:
                mandel_1x_double<IterType, 4> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                    static_cast<IterType*>(OutputIterMatrix),
                    OutputColorMatrix,
                    m_Width, m_Height, cx, cy, dx, dy,
                    n_iterations);
                break;
            case 8:
                mandel_1x_double<IterType, 8> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                    static_cast<IterType*>(OutputIterMatrix),
                    OutputColorMatrix,
                    m_Width, m_Height, cx, cy, dx, dy,
                    n_iterations);
                break;
            case 16:
                mandel_1x_double<IterType, 16> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                    static_cast<IterType*>(OutputIterMatrix),
                    OutputColorMatrix,
                    m_Width, m_Height, cx, cy, dx, dy,
                    n_iterations);
                break;
            default:
                break;
            }
        }
    }
    else if (algorithm == RenderAlgorithm::Gpu2x64) {
        if constexpr (EnableGpu2x64 && std::is_same<T, MattDbldbl>::value) {
            dbldbl cx2{ cx.head, cx.tail };
            dbldbl cy2{ cy.head, cy.tail };
            dbldbl dx2{ dx.head, dx.tail };
            dbldbl dy2{ dy.head, dy.tail };

            mandel_2x_double<IterType> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                static_cast<IterType*>(OutputIterMatrix),
                OutputColorMatrix,
                m_Width, m_Height, cx2, cy2, dx2, dy2,
                n_iterations);
        }
    }
    else if (algorithm == RenderAlgorithm::Gpu4x64) {
        // qdbl
        if constexpr (EnableGpu4x64 && std::is_same<T, MattQDbldbl>::value) {
            using namespace GQD;
            gqd_real cx2;
            cx2 = make_qd(cx.x, cx.y, cx.z, cx.w);

            gqd_real cy2;
            cy2 = make_qd(cy.x, cy.y, cy.z, cy.w);

            gqd_real dx2;
            dx2 = make_qd(dx.x, dx.y, dx.z, dx.w);

            gqd_real dy2;
            dy2 = make_qd(dy.x, dy.y, dy.z, dy.w);

            mandel_4x_double<IterType> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                static_cast<IterType*>(OutputIterMatrix),
                OutputColorMatrix,
                m_Width, m_Height, cx2, cy2, dx2, dy2,
                n_iterations);
        }
    }
    else if (algorithm == RenderAlgorithm::Gpu1x32) {
        if constexpr (EnableGpu1x32 && std::is_same<T, float>::value) {
            // floatOnly
            switch (iteration_precision) {
            case 1:
                mandel_1x_float<IterType, 1> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                    static_cast<IterType*>(OutputIterMatrix),
                    OutputColorMatrix,
                    m_Width, m_Height, cx, cy, dx, dy,
                    n_iterations);
                break;
            case 4:
                mandel_1x_float<IterType, 4> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                    static_cast<IterType*>(OutputIterMatrix),
                    OutputColorMatrix,
                    m_Width, m_Height, cx, cy, dx, dy,
                    n_iterations);
                break;
            case 8:
                mandel_1x_float<IterType, 8> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                    static_cast<IterType*>(OutputIterMatrix),
                    OutputColorMatrix,
                    m_Width, m_Height, cx, cy, dx, dy,
                    n_iterations);
                break;
            case 16:
                mandel_1x_float<IterType, 16> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                    static_cast<IterType*>(OutputIterMatrix),
                    OutputColorMatrix,
                    m_Width, m_Height, cx, cy, dx, dy,
                    n_iterations);
                break;
            default:
                break;
            }
        }
    }
    else if (algorithm == RenderAlgorithm::Gpu2x32) {
        // flt
        if constexpr (EnableGpu2x32 && std::is_same<T, MattDblflt>::value) {
            dblflt cx2{ cx.head, cx.tail };
            dblflt cy2{ cy.head, cy.tail };
            dblflt dx2{ dx.head, dx.tail };
            dblflt dy2{ dy.head, dy.tail };

            switch (iteration_precision) {
            case 1:
                mandel_2x_float<IterType, 1> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                    static_cast<IterType*>(OutputIterMatrix),
                    OutputColorMatrix,
                    m_Width, m_Height, cx2, cy2, dx2, dy2,
                    n_iterations);
                break;
            case 4:
                mandel_2x_float<IterType, 4> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                    static_cast<IterType*>(OutputIterMatrix),
                    OutputColorMatrix,
                    m_Width, m_Height, cx2, cy2, dx2, dy2,
                    n_iterations);
                break;
            case 8:
                mandel_2x_float<IterType, 8> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                    static_cast<IterType*>(OutputIterMatrix),
                    OutputColorMatrix,
                    m_Width, m_Height, cx2, cy2, dx2, dy2,
                    n_iterations);
                break;
            case 16:
                mandel_2x_float<IterType, 16> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                    static_cast<IterType*>(OutputIterMatrix),
                    OutputColorMatrix,
                    m_Width, m_Height, cx2, cy2, dx2, dy2,
                    n_iterations);
                break;
            default:
                break;
            }
        }
    }
    else if (algorithm == RenderAlgorithm::Gpu4x32) {
        // qflt
        if constexpr (EnableGpu4x32 && std::is_same<T, MattQFltflt>::value) {
            using namespace GQF;
            gqf_real cx2;
            cx2 = make_qf(cx.x, cx.y, cx.z, cx.w);

            gqf_real cy2;
            cy2 = make_qf(cy.x, cy.y, cy.z, cy.w);

            gqf_real dx2;
            dx2 = make_qf(dx.x, dx.y, dx.z, dx.w);

            gqf_real dy2;
            dy2 = make_qf(dy.x, dy.y, dy.z, dy.w);

            mandel_4x_float<IterType> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                static_cast<IterType*>(OutputIterMatrix),
                OutputColorMatrix,
                m_Width, m_Height, cx2, cy2, dx2, dy2,
                n_iterations);
        }
    }
    else if (algorithm == RenderAlgorithm::GpuHDRx32) {
        if constexpr (EnableGpuHDRx32 && std::is_same<T, HDRFloat<double>>::value) {
            HDRFloat<CudaDblflt<dblflt>> cx2{ cx };
            HDRFloat<CudaDblflt<dblflt>> cy2{ cy };
            HDRFloat<CudaDblflt<dblflt>> dx2{ dx };
            HDRFloat<CudaDblflt<dblflt>> dy2{ dy };

            mandel_1xHDR_InitStatics << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > ();

            switch(iteration_precision) {
                case 1:
                    mandel_hdr_float<IterType, 1> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                        static_cast<IterType*>(OutputIterMatrix),
                        OutputColorMatrix,
                        m_Width, m_Height, cx2, cy2, dx2, dy2,
                        n_iterations);
                    break;
                case 4:
                    mandel_hdr_float<IterType, 4> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                        static_cast<IterType*>(OutputIterMatrix),
                        OutputColorMatrix,
                        m_Width, m_Height, cx2, cy2, dx2, dy2,
                        n_iterations);
                    break;
                case 8:
                    mandel_hdr_float<IterType, 8> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                        static_cast<IterType*>(OutputIterMatrix),
                        OutputColorMatrix,
                        m_Width, m_Height, cx2, cy2, dx2, dy2,
                        n_iterations);
                    break;
                case 16:
                    mandel_hdr_float<IterType, 16> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                        static_cast<IterType*>(OutputIterMatrix),
                        OutputColorMatrix,
                        m_Width, m_Height, cx2, cy2, dx2, dy2,
                        n_iterations);
                    break;
                default:
                    break;
            }
        }
    }
    else {
        return hipSuccess;
    }

    return RenderAsNeeded(n_iterations, iter_buffer, color_buffer);
}

//////////////////////////////////////////////////
template uint32_t GPURenderer::Render(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    double cx,
    double cy,
    double dx,
    double dy,
    uint32_t n_iterations,
    int iteration_precision);

template uint32_t GPURenderer::Render(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    float cx,
    float cy,
    float dx,
    float dy,
    uint32_t n_iterations,
    int iteration_precision);

template uint32_t GPURenderer::Render(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    MattDbldbl cx,
    MattDbldbl cy,
    MattDbldbl dx,
    MattDbldbl dy,
    uint32_t n_iterations,
    int iteration_precision);

template uint32_t GPURenderer::Render(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    MattQDbldbl cx,
    MattQDbldbl cy,
    MattQDbldbl dx,
    MattQDbldbl dy,
    uint32_t n_iterations,
    int iteration_precision);

template uint32_t GPURenderer::Render(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    MattDblflt cx,
    MattDblflt cy,
    MattDblflt dx,
    MattDblflt dy,
    uint32_t n_iterations,
    int iteration_precision);

template uint32_t GPURenderer::Render(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    MattQFltflt cx,
    MattQFltflt cy,
    MattQFltflt dx,
    MattQFltflt dy,
    uint32_t n_iterations,
    int iteration_precision);

template uint32_t GPURenderer::Render(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    CudaDblflt<MattDblflt> cx,
    CudaDblflt<MattDblflt> cy,
    CudaDblflt<MattDblflt> dx,
    CudaDblflt<MattDblflt> dy,
    uint32_t n_iterations,
    int iteration_precision);

template uint32_t GPURenderer::Render(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    HDRFloat<double> cx,
    HDRFloat<double> cy,
    HDRFloat<double> dx,
    HDRFloat<double> dy,
    uint32_t n_iterations,
    int iteration_precision);
//////////////////////////////////////////////////
template uint32_t GPURenderer::Render(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    double cx,
    double cy,
    double dx,
    double dy,
    uint64_t n_iterations,
    int iteration_precision);

template uint32_t GPURenderer::Render(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    float cx,
    float cy,
    float dx,
    float dy,
    uint64_t n_iterations,
    int iteration_precision);

template uint32_t GPURenderer::Render(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    MattDbldbl cx,
    MattDbldbl cy,
    MattDbldbl dx,
    MattDbldbl dy,
    uint64_t n_iterations,
    int iteration_precision);

template uint32_t GPURenderer::Render(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    MattQDbldbl cx,
    MattQDbldbl cy,
    MattQDbldbl dx,
    MattQDbldbl dy,
    uint64_t n_iterations,
    int iteration_precision);

template uint32_t GPURenderer::Render(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    MattDblflt cx,
    MattDblflt cy,
    MattDblflt dx,
    MattDblflt dy,
    uint64_t n_iterations,
    int iteration_precision);

template uint32_t GPURenderer::Render(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    MattQFltflt cx,
    MattQFltflt cy,
    MattQFltflt dx,
    MattQFltflt dy,
    uint64_t n_iterations,
    int iteration_precision);

template uint32_t GPURenderer::Render(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    CudaDblflt<MattDblflt> cx,
    CudaDblflt<MattDblflt> cy,
    CudaDblflt<MattDblflt> dx,
    CudaDblflt<MattDblflt> dy,
    uint64_t n_iterations,
    int iteration_precision);

template uint32_t GPURenderer::Render(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    HDRFloat<double> cx,
    HDRFloat<double> cy,
    HDRFloat<double> dx,
    HDRFloat<double> dy,
    uint64_t n_iterations,
    int iteration_precision);
/////////////////////////////////////////////////////////


template<typename IterType, class T, class SubType, LAv2Mode Mode>
uint32_t GPURenderer::RenderPerturbLAv2(
    RenderAlgorithm algorithm,
    IterType* iter_buffer,
    Color16* color_buffer,
    T cx,
    T cy,
    T dx,
    T dy,
    T centerX,
    T centerY,
    IterType n_iterations)
{
    uint32_t result = hipSuccess;

    if (!MemoryInitialized()) {
        return hipSuccess;
    }

    dim3 nb_blocks(w_block, h_block, 1);
    dim3 threads_per_block(NB_THREADS_W, NB_THREADS_H, 1);

    auto* cudaResults = m_PerturbResults.GetPtr1<IterType, T>();
    if (!cudaResults) {
        return FractalSharkError::Error6;
    }

    auto* laReferenceCuda = m_PerturbResults.GetLaReference1<IterType, T, SubType>();
    if (!cudaResults) {
        return FractalSharkError::Error7;
    }

    if ((algorithm == RenderAlgorithm::Gpu1x32PerturbedLAv2) ||
        (algorithm == RenderAlgorithm::Gpu1x32PerturbedLAv2PO) ||
        (algorithm == RenderAlgorithm::Gpu1x32PerturbedLAv2LAO)) {
        if constexpr (
            (EnableGpu1x32PerturbedLAv2 || EnableGpu1x32PerturbedLAv2PO || EnableGpu1x32PerturbedLAv2LAO)
            && std::is_same<float, T>::value) {
            // hdrflt
            mandel_1xHDR_float_perturb_lav2<IterType, float, float, Mode> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                static_cast<IterType*>(OutputIterMatrix),
                OutputColorMatrix,
                *cudaResults, *laReferenceCuda,
                m_Width, m_Height, m_Antialiasing, cx, cy, dx, dy,
                centerX, centerY,
                n_iterations);

            RenderAsNeeded(result, n_iterations, iter_buffer, color_buffer);
        }
    } else if ((algorithm == RenderAlgorithm::Gpu2x32PerturbedLAv2) ||
        (algorithm == RenderAlgorithm::Gpu2x32PerturbedLAv2PO) ||
        (algorithm == RenderAlgorithm::Gpu2x32PerturbedLAv2LAO)) {
        if constexpr (
            (EnableGpu2x32PerturbedLAv2 || EnableGpu2x32PerturbedLAv2PO || EnableGpu2x32PerturbedLAv2LAO)
            && std::is_same<CudaDblflt<MattDblflt>, T>::value) {
            // hdrflt

            CudaDblflt<dblflt> cx2{ cx };
            CudaDblflt<dblflt> cy2{ cy };
            CudaDblflt<dblflt> dx2{ dx };
            CudaDblflt<dblflt> dy2{ dy };

            CudaDblflt<dblflt> centerX2{ centerX };
            CudaDblflt<dblflt> centerY2{ centerY };

            mandel_1xHDR_float_perturb_lav2<IterType, CudaDblflt<dblflt>, CudaDblflt<dblflt>, Mode> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                static_cast<IterType*>(OutputIterMatrix),
                OutputColorMatrix,
                *cudaResults, *laReferenceCuda,
                m_Width, m_Height, m_Antialiasing, cx2, cy2, dx2, dy2,
                centerX2, centerY2,
                n_iterations);

            RenderAsNeeded(result, n_iterations, iter_buffer, color_buffer);
        }
    } else if ((algorithm == RenderAlgorithm::Gpu1x64PerturbedLAv2) ||
        (algorithm == RenderAlgorithm::Gpu1x64PerturbedLAv2PO) ||
        (algorithm == RenderAlgorithm::Gpu1x64PerturbedLAv2LAO)) {
        if constexpr (
            (EnableGpu1x64PerturbedLAv2 || EnableGpu1x64PerturbedLAv2PO || EnableGpu1x64PerturbedLAv2LAO)
            && std::is_same<double, T>::value) {
            // hdrflt
            mandel_1xHDR_float_perturb_lav2<IterType, double, double, Mode> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                static_cast<IterType*>(OutputIterMatrix),
                OutputColorMatrix,
                *cudaResults, *laReferenceCuda,
                m_Width, m_Height, m_Antialiasing, cx, cy, dx, dy,
                centerX, centerY,
                n_iterations);

            RenderAsNeeded(result, n_iterations, iter_buffer, color_buffer);
        }
    }
    else if ((algorithm == RenderAlgorithm::GpuHDRx32PerturbedLAv2) ||
        (algorithm == RenderAlgorithm::GpuHDRx32PerturbedLAv2PO) ||
        (algorithm == RenderAlgorithm::GpuHDRx32PerturbedLAv2LAO)) {
        if constexpr (
            (EnableGpuHDRx32PerturbedLAv2 || EnableGpuHDRx32PerturbedLAv2PO || EnableGpuHDRx32PerturbedLAv2LAO)
            && std::is_same<HDRFloat<float>, T>::value) {
            mandel_1xHDR_InitStatics << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > ();

            // hdrflt
            mandel_1xHDR_float_perturb_lav2<IterType, HDRFloat<float>, float, Mode> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                static_cast<IterType*>(OutputIterMatrix),
                OutputColorMatrix,
                *cudaResults, *laReferenceCuda,
                m_Width, m_Height, m_Antialiasing, cx, cy, dx, dy,
                centerX, centerY,
                n_iterations);

            RenderAsNeeded(result, n_iterations, iter_buffer, color_buffer);
        }
    } else if ((algorithm == RenderAlgorithm::GpuHDRx64PerturbedLAv2) ||
               (algorithm == RenderAlgorithm::GpuHDRx64PerturbedLAv2PO) ||
               (algorithm == RenderAlgorithm::GpuHDRx64PerturbedLAv2LAO)) {
        // hdrdbl
        if constexpr (
            (EnableGpuHDRx64PerturbedLAv2 || EnableGpuHDRx64PerturbedLAv2PO || EnableGpuHDRx64PerturbedLAv2LAO)
            && std::is_same<HDRFloat<double>, T>::value) {
            mandel_1xHDR_InitStatics << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > ();

            mandel_1xHDR_float_perturb_lav2<IterType, HDRFloat<double>, double, Mode> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                static_cast<IterType*>(OutputIterMatrix),
                OutputColorMatrix,
                *cudaResults, *laReferenceCuda,
                m_Width, m_Height, m_Antialiasing, cx, cy, dx, dy,
                centerX, centerY,
                n_iterations);

            RenderAsNeeded(result, n_iterations, iter_buffer, color_buffer);
        }
    } else if ((algorithm == RenderAlgorithm::GpuHDRx2x32PerturbedLAv2) ||
               (algorithm == RenderAlgorithm::GpuHDRx2x32PerturbedLAv2PO) ||
               (algorithm == RenderAlgorithm::GpuHDRx2x32PerturbedLAv2LAO)) {
        if constexpr (
            (EnableGpuHDRx2x32PerturbedLAv2 || EnableGpuHDRx2x32PerturbedLAv2PO || EnableGpuHDRx2x32PerturbedLAv2LAO) &&
            std::is_same<HDRFloat<CudaDblflt<MattDblflt>>, T>::value) {
            HDRFloat<CudaDblflt<dblflt>> cx2{ cx };
            HDRFloat<CudaDblflt<dblflt>> cy2{ cy };
            HDRFloat<CudaDblflt<dblflt>> dx2{ dx };
            HDRFloat<CudaDblflt<dblflt>> dy2{ dy };

            HDRFloat<CudaDblflt<dblflt>> centerX2{ centerX };
            HDRFloat<CudaDblflt<dblflt>> centerY2{ centerY };

            mandel_1xHDR_InitStatics << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > ();

            // hdrflt
            mandel_1xHDR_float_perturb_lav2<IterType, HDRFloat<CudaDblflt<dblflt>>, CudaDblflt<dblflt>, Mode> << < DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                static_cast<IterType*>(OutputIterMatrix),
                OutputColorMatrix,
                *cudaResults, *laReferenceCuda,
                m_Width, m_Height, m_Antialiasing, cx2, cy2, dx2, dy2,
                centerX2, centerY2,
                n_iterations);

            RenderAsNeeded(result, n_iterations, iter_buffer, color_buffer);
        }
    }

    return result;
}

////////////////////////////////////////////////////////

template
uint32_t GPURenderer::RenderPerturbLAv2<uint32_t, float, float, LAv2Mode::Full>(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    float cx,
    float cy,
    float dx,
    float dy,
    float centerX,
    float centerY,
    uint32_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint32_t, float, float, LAv2Mode::PO>(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    float cx,
    float cy,
    float dx,
    float dy,
    float centerX,
    float centerY,
    uint32_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint32_t, float, float, LAv2Mode::LAO>(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    float cx,
    float cy,
    float dx,
    float dy,
    float centerX,
    float centerY,
    uint32_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint32_t, double, double, LAv2Mode::Full>(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    double cx,
    double cy,
    double dx,
    double dy,
    double centerX,
    double centerY,
    uint32_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint32_t, double, double, LAv2Mode::PO>(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    double cx,
    double cy,
    double dx,
    double dy,
    double centerX,
    double centerY,
    uint32_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint32_t, double, double, LAv2Mode::LAO>(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    double cx,
    double cy,
    double dx,
    double dy,
    double centerX,
    double centerY,
    uint32_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint32_t, HDRFloat<CudaDblflt<MattDblflt>>, CudaDblflt<MattDblflt>, LAv2Mode::Full>(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    HDRFloat<CudaDblflt<MattDblflt>> cx,
    HDRFloat<CudaDblflt<MattDblflt>> cy,
    HDRFloat<CudaDblflt<MattDblflt>> dx,
    HDRFloat<CudaDblflt<MattDblflt>> dy,
    HDRFloat<CudaDblflt<MattDblflt>> centerX,
    HDRFloat<CudaDblflt<MattDblflt>> centerY,
    uint32_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint32_t, HDRFloat<CudaDblflt<MattDblflt>>, CudaDblflt<MattDblflt>, LAv2Mode::PO>(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    HDRFloat<CudaDblflt<MattDblflt>> cx,
    HDRFloat<CudaDblflt<MattDblflt>> cy,
    HDRFloat<CudaDblflt<MattDblflt>> dx,
    HDRFloat<CudaDblflt<MattDblflt>> dy,
    HDRFloat<CudaDblflt<MattDblflt>> centerX,
    HDRFloat<CudaDblflt<MattDblflt>> centerY,
    uint32_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint32_t, HDRFloat<CudaDblflt<MattDblflt>>, CudaDblflt<MattDblflt>, LAv2Mode::LAO>(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    HDRFloat<CudaDblflt<MattDblflt>> cx,
    HDRFloat<CudaDblflt<MattDblflt>> cy,
    HDRFloat<CudaDblflt<MattDblflt>> dx,
    HDRFloat<CudaDblflt<MattDblflt>> dy,
    HDRFloat<CudaDblflt<MattDblflt>> centerX,
    HDRFloat<CudaDblflt<MattDblflt>> centerY,
    uint32_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint32_t, HDRFloat<float>, float, LAv2Mode::Full>(
    RenderAlgorithm algorithm,
    uint32_t * iter_buffer,
    Color16 * color_buffer,
    HDRFloat<float> cx,
    HDRFloat<float> cy,
    HDRFloat<float> dx,
    HDRFloat<float> dy,
    HDRFloat<float> centerX,
    HDRFloat<float> centerY,
    uint32_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint32_t, HDRFloat<float>, float, LAv2Mode::PO>(
    RenderAlgorithm algorithm,
    uint32_t * iter_buffer,
    Color16 * color_buffer,
    HDRFloat<float> cx,
    HDRFloat<float> cy,
    HDRFloat<float> dx,
    HDRFloat<float> dy,
    HDRFloat<float> centerX,
    HDRFloat<float> centerY,
    uint32_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint32_t, HDRFloat<float>, float, LAv2Mode::LAO>(
    RenderAlgorithm algorithm,
    uint32_t * iter_buffer,
    Color16 * color_buffer,
    HDRFloat<float> cx,
    HDRFloat<float> cy,
    HDRFloat<float> dx,
    HDRFloat<float> dy,
    HDRFloat<float> centerX,
    HDRFloat<float> centerY,
    uint32_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint32_t, CudaDblflt<MattDblflt>, CudaDblflt<MattDblflt>, LAv2Mode::Full>(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    CudaDblflt<MattDblflt> cx,
    CudaDblflt<MattDblflt> cy,
    CudaDblflt<MattDblflt> dx,
    CudaDblflt<MattDblflt> dy,
    CudaDblflt<MattDblflt> centerX,
    CudaDblflt<MattDblflt> centerY,
    uint32_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint32_t, CudaDblflt<MattDblflt>, CudaDblflt<MattDblflt>, LAv2Mode::PO>(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    CudaDblflt<MattDblflt> cx,
    CudaDblflt<MattDblflt> cy,
    CudaDblflt<MattDblflt> dx,
    CudaDblflt<MattDblflt> dy,
    CudaDblflt<MattDblflt> centerX,
    CudaDblflt<MattDblflt> centerY,
    uint32_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint32_t, CudaDblflt<MattDblflt>, CudaDblflt<MattDblflt>, LAv2Mode::LAO>(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    CudaDblflt<MattDblflt> cx,
    CudaDblflt<MattDblflt> cy,
    CudaDblflt<MattDblflt> dx,
    CudaDblflt<MattDblflt> dy,
    CudaDblflt<MattDblflt> centerX,
    CudaDblflt<MattDblflt> centerY,
    uint32_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint32_t, HDRFloat<double>, double, LAv2Mode::Full>(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    HDRFloat<double> cx,
    HDRFloat<double> cy,
    HDRFloat<double> dx,
    HDRFloat<double> dy,
    HDRFloat<double> centerX,
    HDRFloat<double> centerY,
    uint32_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint32_t, HDRFloat<double>, double, LAv2Mode::PO>(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    HDRFloat<double> cx,
    HDRFloat<double> cy,
    HDRFloat<double> dx,
    HDRFloat<double> dy,
    HDRFloat<double> centerX,
    HDRFloat<double> centerY,
    uint32_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint32_t, HDRFloat<double>, double, LAv2Mode::LAO>(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    HDRFloat<double> cx,
    HDRFloat<double> cy,
    HDRFloat<double> dx,
    HDRFloat<double> dy,
    HDRFloat<double> centerX,
    HDRFloat<double> centerY,
    uint32_t n_iterations);

////////////////////////////////////////////////////////

template
uint32_t GPURenderer::RenderPerturbLAv2<uint64_t, float, float, LAv2Mode::Full>(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    float cx,
    float cy,
    float dx,
    float dy,
    float centerX,
    float centerY,
    uint64_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint64_t, float, float, LAv2Mode::PO>(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    float cx,
    float cy,
    float dx,
    float dy,
    float centerX,
    float centerY,
    uint64_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint64_t, float, float, LAv2Mode::LAO>(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    float cx,
    float cy,
    float dx,
    float dy,
    float centerX,
    float centerY,
    uint64_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint64_t, double, double, LAv2Mode::Full>(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    double cx,
    double cy,
    double dx,
    double dy,
    double centerX,
    double centerY,
    uint64_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint64_t, double, double, LAv2Mode::PO>(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    double cx,
    double cy,
    double dx,
    double dy,
    double centerX,
    double centerY,
    uint64_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint64_t, double, double, LAv2Mode::LAO>(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    double cx,
    double cy,
    double dx,
    double dy,
    double centerX,
    double centerY,
    uint64_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint64_t, CudaDblflt<MattDblflt>, CudaDblflt<MattDblflt>, LAv2Mode::Full>(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    CudaDblflt<MattDblflt> cx,
    CudaDblflt<MattDblflt> cy,
    CudaDblflt<MattDblflt> dx,
    CudaDblflt<MattDblflt> dy,
    CudaDblflt<MattDblflt> centerX,
    CudaDblflt<MattDblflt> centerY,
    uint64_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint64_t, CudaDblflt<MattDblflt>, CudaDblflt<MattDblflt>, LAv2Mode::PO>(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    CudaDblflt<MattDblflt> cx,
    CudaDblflt<MattDblflt> cy,
    CudaDblflt<MattDblflt> dx,
    CudaDblflt<MattDblflt> dy,
    CudaDblflt<MattDblflt> centerX,
    CudaDblflt<MattDblflt> centerY,
    uint64_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint64_t, CudaDblflt<MattDblflt>, CudaDblflt<MattDblflt>, LAv2Mode::LAO>(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    CudaDblflt<MattDblflt> cx,
    CudaDblflt<MattDblflt> cy,
    CudaDblflt<MattDblflt> dx,
    CudaDblflt<MattDblflt> dy,
    CudaDblflt<MattDblflt> centerX,
    CudaDblflt<MattDblflt> centerY,
    uint64_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint64_t, HDRFloat<float>, float, LAv2Mode::Full>(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    HDRFloat<float> cx,
    HDRFloat<float> cy,
    HDRFloat<float> dx,
    HDRFloat<float> dy,
    HDRFloat<float> centerX,
    HDRFloat<float> centerY,
    uint64_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint64_t, HDRFloat<float>, float, LAv2Mode::PO>(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    HDRFloat<float> cx,
    HDRFloat<float> cy,
    HDRFloat<float> dx,
    HDRFloat<float> dy,
    HDRFloat<float> centerX,
    HDRFloat<float> centerY,
    uint64_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint64_t, HDRFloat<float>, float, LAv2Mode::LAO>(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    HDRFloat<float> cx,
    HDRFloat<float> cy,
    HDRFloat<float> dx,
    HDRFloat<float> dy,
    HDRFloat<float> centerX,
    HDRFloat<float> centerY,
    uint64_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint64_t, HDRFloat<CudaDblflt<MattDblflt>>, CudaDblflt<MattDblflt>, LAv2Mode::Full>(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    HDRFloat<CudaDblflt<MattDblflt>> cx,
    HDRFloat<CudaDblflt<MattDblflt>> cy,
    HDRFloat<CudaDblflt<MattDblflt>> dx,
    HDRFloat<CudaDblflt<MattDblflt>> dy,
    HDRFloat<CudaDblflt<MattDblflt>> centerX,
    HDRFloat<CudaDblflt<MattDblflt>> centerY,
    uint64_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint64_t, HDRFloat<CudaDblflt<MattDblflt>>, CudaDblflt<MattDblflt>, LAv2Mode::PO>(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    HDRFloat<CudaDblflt<MattDblflt>> cx,
    HDRFloat<CudaDblflt<MattDblflt>> cy,
    HDRFloat<CudaDblflt<MattDblflt>> dx,
    HDRFloat<CudaDblflt<MattDblflt>> dy,
    HDRFloat<CudaDblflt<MattDblflt>> centerX,
    HDRFloat<CudaDblflt<MattDblflt>> centerY,
    uint64_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint64_t, HDRFloat<CudaDblflt<MattDblflt>>, CudaDblflt<MattDblflt>, LAv2Mode::LAO>(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    HDRFloat<CudaDblflt<MattDblflt>> cx,
    HDRFloat<CudaDblflt<MattDblflt>> cy,
    HDRFloat<CudaDblflt<MattDblflt>> dx,
    HDRFloat<CudaDblflt<MattDblflt>> dy,
    HDRFloat<CudaDblflt<MattDblflt>> centerX,
    HDRFloat<CudaDblflt<MattDblflt>> centerY,
    uint64_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint64_t, HDRFloat<double>, double, LAv2Mode::Full>(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    HDRFloat<double> cx,
    HDRFloat<double> cy,
    HDRFloat<double> dx,
    HDRFloat<double> dy,
    HDRFloat<double> centerX,
    HDRFloat<double> centerY,
    uint64_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint64_t, HDRFloat<double>, double, LAv2Mode::PO>(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    HDRFloat<double> cx,
    HDRFloat<double> cy,
    HDRFloat<double> dx,
    HDRFloat<double> dy,
    HDRFloat<double> centerX,
    HDRFloat<double> centerY,
    uint64_t n_iterations);

template
uint32_t GPURenderer::RenderPerturbLAv2<uint64_t, HDRFloat<double>, double, LAv2Mode::LAO>(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    HDRFloat<double> cx,
    HDRFloat<double> cy,
    HDRFloat<double> dx,
    HDRFloat<double> dy,
    HDRFloat<double> centerX,
    HDRFloat<double> centerY,
    uint64_t n_iterations);

////////////////////////////////////////////////////////

template<typename IterType, class T>
uint32_t GPURenderer::RenderPerturbBLAScaled(
    RenderAlgorithm algorithm,
    IterType* iter_buffer,
    Color16* color_buffer,
    const MattPerturbResults<IterType, T, CalcBad::Enable>* double_perturb,
    const MattPerturbResults<IterType, float, CalcBad::Enable>* float_perturb,
    BLAS<IterType, T, CalcBad::Enable>* blas,
    T cx,
    T cy,
    T dx,
    T dy,
    T centerX,
    T centerY,
    IterType n_iterations,
    int /*iteration_precision*/)
{
    uint32_t result = hipSuccess;

    if (!MemoryInitialized()) {
        return hipSuccess;
    }

    dim3 nb_blocks(w_block, h_block, 1);
    dim3 threads_per_block(NB_THREADS_W, NB_THREADS_H, 1);

    MattPerturbSingleResults<IterType, float, CalcBad::Enable> cudaResults(
        float_perturb->size,
        float_perturb->PeriodMaybeZero,
        float_perturb->iters);

    result = cudaResults.CheckValid();
    if (result != 0) {
        return result;
    }

    MattPerturbSingleResults<IterType, T, CalcBad::Enable> cudaResultsDouble(
        double_perturb->size,
        float_perturb->PeriodMaybeZero,
        double_perturb->iters);

    result = cudaResultsDouble.CheckValid();
    if (result != 0) {
        return result;
    }

    if (algorithm == RenderAlgorithm::Gpu1x32PerturbedScaled) {
        if constexpr (EnableGpu1x32PerturbedScaled && std::is_same<T, double>::value) {
            // doubleOnly
            mandel_1x_float_perturb_scaled<IterType, T> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                static_cast<IterType*>(OutputIterMatrix),
                OutputColorMatrix,
                cudaResults, cudaResultsDouble,
                m_Width, m_Height, cx, cy, dx, dy,
                centerX, centerY,
                n_iterations);

            RenderAsNeeded(result, n_iterations, iter_buffer, color_buffer);
        }
    } else if (algorithm == RenderAlgorithm::GpuHDRx32PerturbedScaled) {
        if constexpr (EnableGpuHDRx32PerturbedScaled && std::is_same<T, HDRFloat<float>>::value) {
            // hdrflt
            mandel_1xHDR_InitStatics << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > ();

            mandel_1x_float_perturb_scaled<IterType, T> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                static_cast<IterType*>(OutputIterMatrix),
                OutputColorMatrix,
                cudaResults, cudaResultsDouble,
                m_Width, m_Height, cx, cy, dx, dy,
                centerX, centerY,
                n_iterations);

            RenderAsNeeded(result, n_iterations, iter_buffer, color_buffer);
        }
    }
    else if (algorithm == RenderAlgorithm::Gpu1x32PerturbedScaledBLA) {
        if constexpr (EnableGpu1x32PerturbedScaledBLA && std::is_same<T, double>::value) {
            // doubleOnly
            auto Run = [&]<int32_t LM2>() -> uint32_t {
                GPU_BLAS<IterType, double, BLA<double>, LM2> doubleGpuBlas(blas->m_B);
                result = doubleGpuBlas.CheckValid();
                if (result != 0) {
                    return result;
                }

                mandel_1xHDR_InitStatics << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > ();

                mandel_1x_float_perturb_scaled_bla<IterType, LM2> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                    static_cast<IterType*>(OutputIterMatrix),
                    OutputColorMatrix,
                    cudaResults, cudaResultsDouble, doubleGpuBlas,
                    m_Width, m_Height, cx, cy, dx, dy,
                    centerX, centerY,
                    n_iterations);

                RenderAsNeeded(result, n_iterations, iter_buffer, color_buffer);
                return result;
            };

            LargeSwitch
        }
    }

    return result;
}

//////////////////////////////////////////////////////////////////

template uint32_t GPURenderer::RenderPerturbBLAScaled<uint32_t, double>(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    const MattPerturbResults<uint32_t, double, CalcBad::Enable>* double_perturb,
    const MattPerturbResults<uint32_t, float, CalcBad::Enable>* float_perturb,
    BLAS<uint32_t, double, CalcBad::Enable>* blas,
    double cx,
    double cy,
    double dx,
    double dy,
    double centerX,
    double centerY,
    uint32_t n_iterations,
    int /*iteration_precision*/
);

template uint32_t GPURenderer::RenderPerturbBLAScaled<uint32_t, HDRFloat<float>>(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    const MattPerturbResults<uint32_t, HDRFloat<float>, CalcBad::Enable>* double_perturb,
    const MattPerturbResults<uint32_t, float, CalcBad::Enable>* float_perturb,
    BLAS<uint32_t, HDRFloat<float>, CalcBad::Enable>* blas,
    HDRFloat<float> cx,
    HDRFloat<float> cy,
    HDRFloat<float> dx,
    HDRFloat<float> dy,
    HDRFloat<float> centerX,
    HDRFloat<float> centerY,
    uint32_t n_iterations,
    int /*iteration_precision*/
);

//////////////////////////////////////////////////////////////////

template uint32_t GPURenderer::RenderPerturbBLAScaled<uint64_t, double>(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    const MattPerturbResults<uint64_t, double, CalcBad::Enable>* double_perturb,
    const MattPerturbResults<uint64_t, float, CalcBad::Enable>* float_perturb,
    BLAS<uint64_t, double, CalcBad::Enable>* blas,
    double cx,
    double cy,
    double dx,
    double dy,
    double centerX,
    double centerY,
    uint64_t n_iterations,
    int /*iteration_precision*/
);

template uint32_t GPURenderer::RenderPerturbBLAScaled<uint64_t, HDRFloat<float>>(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    const MattPerturbResults<uint64_t, HDRFloat<float>, CalcBad::Enable>* double_perturb,
    const MattPerturbResults<uint64_t, float, CalcBad::Enable>* float_perturb,
    BLAS<uint64_t, HDRFloat<float>, CalcBad::Enable>* blas,
    HDRFloat<float> cx,
    HDRFloat<float> cy,
    HDRFloat<float> dx,
    HDRFloat<float> dy,
    HDRFloat<float> centerX,
    HDRFloat<float> centerY,
    uint64_t n_iterations,
    int /*iteration_precision*/
);

//////////////////////////////////////////////////////////////////

template<typename IterType, class T>
uint32_t GPURenderer::RenderPerturbBLA(
    RenderAlgorithm algorithm,
    IterType* iter_buffer,
    Color16* color_buffer,
    const MattPerturbResults<IterType, T>* perturb,
    BLAS<IterType, T>* blas,
    T cx,
    T cy,
    T dx,
    T dy,
    T centerX,
    T centerY,
    IterType n_iterations,
    int /*iteration_precision*/)
{
    uint32_t result = hipSuccess;

    if (!MemoryInitialized()) {
        return hipSuccess;
    }

    dim3 nb_blocks(w_block, h_block, 1);
    dim3 threads_per_block(NB_THREADS_W, NB_THREADS_H, 1);

    if (algorithm == RenderAlgorithm::GpuHDRx32PerturbedBLA) {
        if constexpr (EnableGpuHDRx32PerturbedBLA && std::is_same<T, HDRFloat<float>>::value) {
            MattPerturbSingleResults<IterType, HDRFloat<float>> cudaResults(
                perturb->size,
                perturb->PeriodMaybeZero,
                perturb->iters);

            result = cudaResults.CheckValid();
            if (result != 0) {
                return result;
            }

            auto Run = [&]<int32_t LM2>() -> uint32_t {
                GPU_BLAS<IterType, HDRFloat<float>, BLA<HDRFloat<float>>, LM2> gpu_blas(blas->m_B);
                result = gpu_blas.CheckValid();
                if (result != 0) {
                    return result;
                }

                mandel_1xHDR_InitStatics << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > ();

                // hdrflt
                mandel_1xHDR_float_perturb_bla<IterType, HDRFloat<float>, LM2> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                    static_cast<IterType*>(OutputIterMatrix),
                    OutputColorMatrix,
                    cudaResults,
                    gpu_blas,
                    m_Width, m_Height, cx, cy, dx, dy,
                    centerX, centerY,
                    n_iterations);

                RenderAsNeeded(result, n_iterations, iter_buffer, color_buffer);
                return result;
            };

            LargeSwitch
        }
    }
    else if (algorithm == RenderAlgorithm::GpuHDRx64PerturbedBLA) {
        if constexpr (EnableGpuHDRx64PerturbedBLA && std::is_same<T, HDRFloat<double>>::value) {
            MattPerturbSingleResults<IterType, HDRFloat<double>> cudaResults(
                perturb->size,
                perturb->PeriodMaybeZero,
                perturb->iters);

            result = cudaResults.CheckValid();
            if (result != 0) {
                return result;
            }

            auto Run = [&]<int32_t LM2>() -> uint32_t {
                GPU_BLAS<IterType, HDRFloat<double>, BLA<HDRFloat<double>>, LM2> gpu_blas(blas->m_B);
                result = gpu_blas.CheckValid();
                if (result != 0) {
                    return result;
                }

                mandel_1xHDR_InitStatics << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > ();

                // hdrflt -- that looks like a bug and probably should be hdrdbl
                mandel_1xHDR_float_perturb_bla<IterType, HDRFloat<double>, LM2> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                    static_cast<IterType*>(OutputIterMatrix),
                    OutputColorMatrix,
                    cudaResults,
                    gpu_blas,
                    m_Width, m_Height, cx, cy, dx, dy,
                    centerX, centerY,
                    n_iterations);

                RenderAsNeeded(result, n_iterations, iter_buffer, color_buffer);
                return result;
            };

            LargeSwitch
        }
    } else if (algorithm == RenderAlgorithm::Gpu1x64PerturbedBLA) {
        if constexpr (EnableGpu1x64PerturbedBLA && std::is_same<T, double>::value) {
            MattPerturbSingleResults<IterType, double> cudaResults(
                perturb->size,
                perturb->PeriodMaybeZero,
                perturb->iters);

            result = cudaResults.CheckValid();
            if (result != 0) {
                return result;
            }

            auto Run = [&]<int32_t LM2>() -> uint32_t {
                GPU_BLAS<IterType, double, BLA<double>, LM2> gpu_blas(blas->m_B);
                result = gpu_blas.CheckValid();
                if (result != 0) {
                    return result;
                }

                mandel_1xHDR_InitStatics << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > ();

                // doubleOnly
                mandel_1x_double_perturb_bla<IterType, LM2> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
                    static_cast<IterType*>(OutputIterMatrix),
                    OutputColorMatrix,
                    cudaResults,
                    gpu_blas,
                    m_Width, m_Height, cx, cy, dx, dy,
                    centerX, centerY,
                    n_iterations);

                RenderAsNeeded(result, n_iterations, iter_buffer, color_buffer);
                return result;
            };

            LargeSwitch
        }
    }
    else if (algorithm == RenderAlgorithm::Gpu2x32PerturbedScaled) {
        if constexpr (EnableGpu2x32PerturbedScaled && std::is_same<T, dblflt>::value) {
            //MattPerturbSingleResults<IterType, dblflt> cudaResults(
            //    Perturb->size,
            //    Perturb->PeriodMaybeZero,
            //    Perturb->iters);

            //result = cudaResults.CheckValid();
            //if (result != 0) {
            //    return result;
            //}

            //MattPerturbSingleResults<IterType, double> cudaResultsDouble(
            //    Perturb->size,
            //    Perturb->PeriodMaybeZero,
            //    Perturb->iters);

            //result = cudaResultsDouble.CheckValid();
            //if (result != 0) {
            //    return result;
            //}

            //// doubleOnly
            //mandel_2x_float_perturb_setup << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (cudaResults);

            //mandel_2x_float_perturb_scaled<IterType> << <DEFAULT_KERNEL_LAUNCH_PARAMS >> > (
            //    static_cast<IterType*>(OutputIterMatrix),
            //    OutputColorMatrix,
            //    cudaResults, cudaResultsDouble,
            //    m_Width, m_Height, cx, cy, dx, dy,
            //    centerX, centerY,
            //    n_iterations);

            // RenderAsNeeded(result, n_iterations, iter_buffer, color_buffer);
        }
    }

    return result;
}

//////////////////////////////////////////////////////////
template
uint32_t GPURenderer::RenderPerturbBLA(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    const MattPerturbResults<uint32_t, HDRFloat<float>>* perturb,
    BLAS<uint32_t, HDRFloat<float>>* blas,
    HDRFloat<float> cx,
    HDRFloat<float> cy,
    HDRFloat<float> dx,
    HDRFloat<float> dy,
    HDRFloat<float> centerX,
    HDRFloat<float> centerY,
    uint32_t n_iterations,
    int /*iteration_precision*/);

template
uint32_t GPURenderer::RenderPerturbBLA(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    const MattPerturbResults<uint32_t, HDRFloat<double>>* perturb,
    BLAS<uint32_t, HDRFloat<double>>* blas,
    HDRFloat<double> cx,
    HDRFloat<double> cy,
    HDRFloat<double> dx,
    HDRFloat<double> dy,
    HDRFloat<double> centerX,
    HDRFloat<double> centerY,
    uint32_t n_iterations,
    int /*iteration_precision*/);

template
uint32_t GPURenderer::RenderPerturbBLA(
    RenderAlgorithm algorithm,
    uint32_t* iter_buffer,
    Color16* color_buffer,
    const MattPerturbResults<uint32_t, double>* perturb,
    BLAS<uint32_t, double>* blas,
    double cx,
    double cy,
    double dx,
    double dy,
    double centerX,
    double centerY,
    uint32_t n_iterations,
    int /*iteration_precision*/);
//////////////////////////////////////////////////////////
template
uint32_t GPURenderer::RenderPerturbBLA(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    const MattPerturbResults<uint64_t, HDRFloat<float>>* perturb,
    BLAS<uint64_t, HDRFloat<float>>* blas,
    HDRFloat<float> cx,
    HDRFloat<float> cy,
    HDRFloat<float> dx,
    HDRFloat<float> dy,
    HDRFloat<float> centerX,
    HDRFloat<float> centerY,
    uint64_t n_iterations,
    int /*iteration_precision*/);

template
uint32_t GPURenderer::RenderPerturbBLA(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    const MattPerturbResults<uint64_t, HDRFloat<double>>* perturb,
    BLAS<uint64_t, HDRFloat<double>>* blas,
    HDRFloat<double> cx,
    HDRFloat<double> cy,
    HDRFloat<double> dx,
    HDRFloat<double> dy,
    HDRFloat<double> centerX,
    HDRFloat<double> centerY,
    uint64_t n_iterations,
    int /*iteration_precision*/);

template
uint32_t GPURenderer::RenderPerturbBLA(
    RenderAlgorithm algorithm,
    uint64_t* iter_buffer,
    Color16* color_buffer,
    const MattPerturbResults<uint64_t, double>* perturb,
    BLAS<uint64_t, double>* blas,
    double cx,
    double cy,
    double dx,
    double dy,
    double centerX,
    double centerY,
    uint64_t n_iterations,
    int /*iteration_precision*/);
//////////////////////////////////////////////////////////

//// Not the same as RenderAsNeeded
//template<typename IterType>
//__host__
//uint32_t
//GPURenderer::OnlyAA(
//    Color16* color_buffer,
//    IterType n_iterations) {
//
//    auto result = RunAntialiasing(n_iterations, hipStreamDefault);
//    if (result != hipSuccess) {
//        return result;
//    }
//
//    result = ExtractItersAndColors<IterType, false>(nullptr, color_buffer);
//    if (result != hipSuccess) {
//        return result;
//    }
//
//    return hipSuccess;
//}

//template
//__host__
//uint32_t
//GPURenderer::OnlyAA<uint32_t>(
//    Color16* color_buffer,
//    uint32_t n_iterations);
//
//template
//__host__
//uint32_t
//GPURenderer::OnlyAA(
//    Color16* color_buffer,
//    uint64_t n_iterations);

template<typename IterType>
__host__
uint32_t
GPURenderer::RunAntialiasing(IterType n_iterations, hipStream_t *stream) {
    dim3 aa_blocks(w_color_block, h_color_block, 1);
    dim3 aa_threads_per_block(NB_THREADS_W_AA, NB_THREADS_H_AA, 1);
    
    switch (m_Antialiasing) {
    case 1:
        antialiasing_kernel<IterType, 1, true> << <aa_blocks, aa_threads_per_block, 0, *stream >> > (
            static_cast<IterType*>(OutputIterMatrix),
            m_Width,
            m_Height,
            OutputColorMatrix,
            Pals,
            local_color_width,
            local_color_height,
            n_iterations);
        break;
    case 2:
        antialiasing_kernel<IterType, 2, true> << <aa_blocks, aa_threads_per_block, 0, *stream >> > (
            static_cast<IterType*>(OutputIterMatrix),
            m_Width,
            m_Height,
            OutputColorMatrix,
            Pals,
            local_color_width,
            local_color_height,
            n_iterations);
        break;
    case 3:
        antialiasing_kernel<IterType, 3, true> << <aa_blocks, aa_threads_per_block, 0, *stream >> > (
            static_cast<IterType*>(OutputIterMatrix),
            m_Width,
            m_Height,
            OutputColorMatrix,
            Pals,
            local_color_width,
            local_color_height,
            n_iterations);
        break;
    case 4:
    default:
        antialiasing_kernel<IterType, 4, true> << <aa_blocks, aa_threads_per_block, 0, *stream >> > (
            static_cast<IterType*>(OutputIterMatrix),
            m_Width,
            m_Height,
            OutputColorMatrix,
            Pals,
            local_color_width,
            local_color_height,
            n_iterations);
        break;
    }

    dim3 max_blocks(16, 16, 1);
    max_kernel<IterType> << <max_blocks, aa_threads_per_block, 0, *stream >> > (
        static_cast<IterType*>(OutputIterMatrix),
        m_Width,
        m_Height,
        OutputReductionResults);
    return hipSuccess;
}

template<typename IterType, bool Async>
uint32_t GPURenderer::ExtractItersAndColors(
    IterType* iter_buffer,
    Color16 *color_buffer,
    ReductionResults* reduction_results) {

    const size_t ERROR_COLOR = 255;
    hipError_t result = hipSuccess;

    if (!Async) {
        result = hipStreamSynchronize(hipStreamDefault);
        if (result != hipSuccess) {
            if (iter_buffer) {
                hipMemset(iter_buffer, ERROR_COLOR, sizeof(IterType) * m_Width * m_Height);
            }

            if (color_buffer) {
                hipMemset(color_buffer, ERROR_COLOR, sizeof(Color16) * local_color_width * local_color_height);
            }
            return result;
        }
    }

    if (iter_buffer) {
        if constexpr (Async) {
            result = hipMemcpyAsync(
                iter_buffer,
                static_cast<IterType*>(OutputIterMatrix),
                sizeof(IterType) * N_cu,
                hipMemcpyDefault,
                m_Stream1);
            if (result != hipSuccess) {
                return result;
            }
        } else {
            result = hipMemcpy(
                iter_buffer,
                static_cast<IterType*>(OutputIterMatrix),
                sizeof(IterType) * N_cu,
                hipMemcpyDefault);
            if (result != hipSuccess) {
                return result;
            }
        }
    }

    if (color_buffer) {
        if constexpr (Async) {
            result = hipMemcpyAsync(
                color_buffer,
                OutputColorMatrix.aa_colors,
                sizeof(Color16) * N_color_cu,
                hipMemcpyDefault,
                m_Stream1);
            if (result != hipSuccess) {
                return result;
            }
        } else {
            result = hipMemcpy(
                color_buffer,
                OutputColorMatrix.aa_colors,
                sizeof(Color16) * N_color_cu,
                hipMemcpyDefault);
            if (result != hipSuccess) {
                return result;
            }
        }
    }

    if (reduction_results != nullptr) {
        if constexpr (Async) {
            result = hipMemcpyAsync(
                reduction_results,
                OutputReductionResults,
                sizeof(ReductionResults),
                hipMemcpyDefault,
                m_Stream1);
            if (result != hipSuccess) {
                return result;
            }
        }
        else {
            result = hipMemcpy(
                reduction_results,
                OutputReductionResults,
                sizeof(ReductionResults),
                hipMemcpyDefault);
            if (result != hipSuccess) {
                return result;
            }
        }
    }

    return hipSuccess;
}

template
uint32_t GPURenderer::ExtractItersAndColors<uint32_t, false>(
    uint32_t* iter_buffer,
    Color16* color_buffer,
    ReductionResults* reduction_results);
template
uint32_t GPURenderer::ExtractItersAndColors<uint64_t, false>(
    uint64_t* iter_buffer,
    Color16* color_buffer,
    ReductionResults* reduction_results);

template
uint32_t GPURenderer::ExtractItersAndColors<uint32_t, true>(
    uint32_t* iter_buffer,
    Color16* color_buffer,
    ReductionResults* reduction_results);
template
uint32_t GPURenderer::ExtractItersAndColors<uint64_t, true>(
    uint64_t* iter_buffer,
    Color16* color_buffer,
    ReductionResults* reduction_results);

const char* GPURenderer::ConvertErrorToString(uint32_t err) {
    auto typeNotExposedOutSideHere = static_cast<hipError_t>(err);
    return hipGetErrorString(typeNotExposedOutSideHere);
}