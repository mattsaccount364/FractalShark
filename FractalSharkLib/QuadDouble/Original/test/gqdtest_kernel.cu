#include "hip/hip_runtime.h"
#ifndef __GDD_TEST_CU__
#define __GDD_TEST_CU__

#include "test_common.h"
#include "gqd.cu"

#define TEST_SINCOS


template<class T>
__global__
void device_basic_kernel(const T* d_in1, const T* d_in2, 
			 const unsigned int numElement,
                         T* d_out) {
        const unsigned numTotalThread = NUM_TOTAL_THREAD;
        const unsigned globalThreadOffset = GLOBAL_THREAD_OFFSET;

        for(unsigned int i = globalThreadOffset; i < numElement; i += numTotalThread) {
                d_out[i] = BASIC_FUNC(d_in1[i], d_in2[i]); 
        }
}


template<class T>
__global__
void device_add_kernel(const T* d_in1, const T* d_in2,
                         const unsigned int numElement,
                         T* d_out) {
        const unsigned numTotalThread = NUM_TOTAL_THREAD;
        const unsigned globalThreadOffset = GLOBAL_THREAD_OFFSET;

        for(unsigned int i = globalThreadOffset; i < numElement; i += numTotalThread) {
                d_out[i] = d_in1[i] + d_in2[i];
        }
}


template<class T>
__global__
void device_mul_kernel(const T* d_in1, const T* d_in2,
                         const unsigned int numElement,
                         T* d_out) {
        const unsigned numTotalThread = NUM_TOTAL_THREAD;
        const unsigned globalThreadOffset = GLOBAL_THREAD_OFFSET;

        for(unsigned int i = globalThreadOffset; i < numElement; i += numTotalThread) {
                d_out[i] = d_in1[i] * d_in2[i];
        }
}


template<class T>
__global__
void device_div_kernel(const T* d_in1, const T* d_in2,
                         const unsigned int numElement,
                         T* d_out) {
        const unsigned numTotalThread = NUM_TOTAL_THREAD;
        const unsigned globalThreadOffset = GLOBAL_THREAD_OFFSET;

        for(unsigned int i = globalThreadOffset; i < numElement; i += numTotalThread) {
                d_out[i] = d_in1[i] / d_in2[i];
        }
}


template<class T>
__global__
void device_sqrt_kernel(const T* d_in, const unsigned int numElement,
                          T* d_out) {
        const unsigned numTotalThread = NUM_TOTAL_THREAD;
        const unsigned globalThreadOffset = GLOBAL_THREAD_OFFSET;

        for(unsigned int i = globalThreadOffset; i < numElement; i += numTotalThread) {
                d_out[i] = sqrt(d_in[i]);
        }
}


template<class T>
__global__
void device_sqr_kernel(const T* d_in, const unsigned int numElement,
                          T* d_out) {
        const unsigned numTotalThread = NUM_TOTAL_THREAD;
        const unsigned globalThreadOffset = GLOBAL_THREAD_OFFSET;

        for(unsigned int i = globalThreadOffset; i < numElement; i += numTotalThread) {
                d_out[i] = sqr(d_in[i]);
        }
}


template<class T>
__global__
void device_exp_kernel(const T* d_in, const unsigned int numElement, 
			  T* d_out) {
	const unsigned numTotalThread = NUM_TOTAL_THREAD;
	const unsigned globalThreadOffset = GLOBAL_THREAD_OFFSET;
	
	for(unsigned int i = globalThreadOffset; i < numElement; i += numTotalThread) {
		d_out[i] = exp(d_in[i]);
	}
}


template<class T>
__global__
void device_log_kernel(const T* d_in, const unsigned int numElement,
                          T* d_out) {
        const unsigned numTotalThread = NUM_TOTAL_THREAD;
        const unsigned globalThreadOffset = GLOBAL_THREAD_OFFSET;

        for(unsigned int i = globalThreadOffset; i < numElement; i += numTotalThread) {
                d_out[i] = log(d_in[i]);
        }
}


template<class T>
__global__
void device_sin_kernel(const T* d_in, const unsigned int numElement,
                          T* d_out) {

        const unsigned numTotalThread = NUM_TOTAL_THREAD;
        const unsigned globalThreadOffset = GLOBAL_THREAD_OFFSET;

        for(unsigned int i = globalThreadOffset; i < numElement; i += numTotalThread) {
                d_out[i] = sin(d_in[i]);
        }
}


template<class T>
__global__
void device_cos_kernel(const T* d_in, const unsigned int numElement,
                          T* d_out) {
	const unsigned numTotalThread = NUM_TOTAL_THREAD;
        const unsigned globalThreadOffset = GLOBAL_THREAD_OFFSET;

        for(unsigned int i = globalThreadOffset; i < numElement; i += numTotalThread) {
                d_out[i] = cos(d_in[i]);
        }
}


template<class T>
__global__
void device_tan_kernel(const T* d_in, const unsigned int numElement,
                          T* d_out) {
        const unsigned numTotalThread = NUM_TOTAL_THREAD;
        const unsigned globalThreadOffset = GLOBAL_THREAD_OFFSET;

        for(unsigned int i = globalThreadOffset; i < numElement; i += numTotalThread) {
                d_out[i] = tan(d_in[i]);
        }
}



template<class T>
__global__
void device_atan_kernel(const T* d_in, const unsigned int numElement,
                          T* d_out) {
/*
#ifndef TEST_SINCOS
        const unsigned numTotalThread = NUM_TOTAL_THREAD;
        const unsigned globalThreadOffset = GLOBAL_THREAD_OFFSET;

        for(unsigned int i = globalThreadOffset; i < numElement; i += numTotalThread) {
                d_out[i] = atan(d_in[i]);
        }
#endif
*/
}



template<class T>
float device_basic_template(T* h_in1, T* h_in2, T* h_out, const unsigned int numElement,
			OPERATOR op = ADD, 
                        const unsigned int numBlock = 128, 
                        const unsigned int numThread = 128) {

        T* d_in1 = NULL;
	T* d_in2 = NULL;
        T* d_out = NULL;
        GPUMALLOC((void**)&d_in1, sizeof(T)*numElement);
	GPUMALLOC((void**)&d_in2, sizeof(T)*numElement);
        GPUMALLOC((void**)&d_out, sizeof(T)*numElement);
        TOGPU(d_in1, h_in1, sizeof(T)*numElement);
	TOGPU(d_in2, h_in2, sizeof(T)*numElement);

        unsigned int timer = 0;
	float elapsedTime = 0.0;

	if(op == ADD) {
	        startTimer(&timer);
	        device_add_kernel<<<numBlock, numThread>>>(d_in1, d_in2, numElement, d_out);
	        cutilCheckMsg("device_add_kernel");
	        cutilSafeCall(hipDeviceSynchronize());
	        elapsedTime = endTimer(&timer, "device_add_kernel");
	} else if (op == MUL) {
                startTimer(&timer);
                device_mul_kernel<<<numBlock, numThread>>>(d_in1, d_in2, numElement, d_out);
                cutilCheckMsg("device_mul_kernel");
                cutilSafeCall(hipDeviceSynchronize());
                elapsedTime = endTimer(&timer, "device_mul_kernel");
	} else if(op == DIV) {
                startTimer(&timer);
                device_div_kernel<<<numBlock, numThread>>>(d_in1, d_in2, numElement, d_out);
                cutilCheckMsg("device_div_kernel");
                cutilSafeCall(hipDeviceSynchronize());
                elapsedTime = endTimer(&timer, "device_div_kernel");
	} else {
		printf("!!!Never here!\n");
		exit(EXIT_FAILURE);
	}

        FROMGPU(h_out, d_out, sizeof(T)*numElement);

        GPUFREE(d_in1);
	GPUFREE(d_in2);
        GPUFREE(d_out);

	return elapsedTime;
}

float device_basic(gdd_real* h_in1, gdd_real* h_in2, gdd_real* h_out, const unsigned int numElement,
                        OPERATOR op = ADD,
                        const unsigned int numBlock = 128,
                        const unsigned int numThread = 128) {
	return device_basic_template(h_in1, h_in2, h_out, numElement, op, numBlock, numThread);
}


float device_basic(gqd_real* h_in1, gqd_real* h_in2, gqd_real* h_out, const unsigned int numElement,
                        OPERATOR op = ADD,
                        const unsigned int numBlock = 128,
                        const unsigned int numThread = 128) {
        return device_basic_template(h_in1, h_in2, h_out, numElement, op, numBlock, numThread);
}


template<class T>
float device_math_template(T* h_in, const unsigned int numElement, T* h_out,
		   MATH math, const unsigned int numBlock, const unsigned int numThread) {

        T* d_in = NULL;
        T* d_out = NULL;
        GPUMALLOC((void**)&d_in, sizeof(T)*numElement);
        GPUMALLOC((void**)&d_out, sizeof(T)*numElement);
        TOGPU(d_in, h_in, sizeof(T)*numElement);
        unsigned int timer = 0;
	float elapsedTime = 0.0;

	if(math == SQRT) {
        	startTimer(&timer);
	        device_sqrt_kernel<<<numBlock, numThread>>>(d_in, numElement, d_out);
	        cutilCheckMsg("device_sqrt_kernel");
	        cutilSafeCall(hipDeviceSynchronize());
	        elapsedTime = endTimer(&timer, "device_sqrt_kernel");
	} else if(math == SQR) {
                startTimer(&timer);
                device_sqr_kernel<<<numBlock, numThread>>>(d_in, numElement, d_out);
                cutilCheckMsg("device_sqr_kernel");
                cutilSafeCall(hipDeviceSynchronize());
                elapsedTime = endTimer(&timer, "device_sqr_kernel");
	} else if(math == EXP) {
                startTimer(&timer);
                device_exp_kernel<<<numBlock, numThread>>>(d_in, numElement, d_out);
                cutilCheckMsg("device_exp_kernel");
                cutilSafeCall(hipDeviceSynchronize());
                elapsedTime = endTimer(&timer, "device_exp_kernel");
	} else if(math == LOG) {
                startTimer(&timer);
                device_log_kernel<<<numBlock, numThread>>>(d_in, numElement, d_out);
                cutilCheckMsg("device_log_kernel");
                cutilSafeCall(hipDeviceSynchronize());
                elapsedTime = endTimer(&timer, "device_log_kernel");
	} 
	else if(math == SIN) {
                startTimer(&timer);
                device_sin_kernel<<<numBlock, numThread>>>(d_in, numElement, d_out);
                cutilCheckMsg("device_sin_kernel");
                cutilSafeCall(hipDeviceSynchronize());
                elapsedTime = endTimer(&timer, "device_sin_kernel");
	} 
	else if(math == COS) {
                startTimer(&timer);
                device_cos_kernel<<<numBlock, numThread>>>(d_in, numElement, d_out);
                cutilCheckMsg("device_cos_kernel");
                cutilSafeCall(hipDeviceSynchronize());
                elapsedTime = endTimer(&timer, "device_cos_kernel");
	} else if(math == TAN) {
                startTimer(&timer);
                device_tan_kernel<<<numBlock, numThread>>>(d_in, numElement, d_out);
                cutilCheckMsg("device_tan_kernel");
                cutilSafeCall(hipDeviceSynchronize());
                elapsedTime = endTimer(&timer, "device_tan_kernel");
	} else if(math == ATAN) {
                /*startTimer(&timer);
                device_atan_kernel<<<numBlock, numThread>>>(d_in, numElement, d_out);
                cutilCheckMsg("device_atan_kernel");
                cutilSafeCall(hipDeviceSynchronize());
                elapsedTime = endTimer(&timer, "device_atan_kernel");*/
        } 
	else {
		printf("!!! Never here.\n");
		exit(EXIT_FAILURE);
	}

        FROMGPU(h_out, d_out, sizeof(T)*numElement);

        GPUFREE(d_in);
        GPUFREE(d_out);

	return elapsedTime;
}


float device_math(gdd_real* h_in, const unsigned int numElement, gdd_real* h_out,
                   MATH math, const unsigned int numBlock, const unsigned int numThread) {
	return device_math_template(h_in, numElement, h_out, math, numBlock, numThread);
}


float device_math(gqd_real* h_in, const unsigned int numElement, gqd_real* h_out,
                   MATH math, const unsigned int numBlock, const unsigned int numThread) {
	return device_math_template(h_in, numElement, h_out, math, numBlock, numThread);
}


template<class T>
__global__
void device_defined_kernel(const T* d_in, const unsigned int numElement, T* d_out) {
        const unsigned numTotalThread = NUM_TOTAL_THREAD;
        const unsigned globalThreadOffset = GLOBAL_THREAD_OFFSET;

        for(unsigned int i = globalThreadOffset; i < numElement; i += numTotalThread) {
                //d_out[i] = (sin(d_in[i]));
        }
}


template<class T>
float device_defined_template(T* h_in, const unsigned int numElement, T* h_out,
                   	      const unsigned int numBlock, const unsigned int numThread) {

        T* d_in = NULL;
        T* d_out = NULL;
        GPUMALLOC((void**)&d_in, sizeof(T)*numElement);
        GPUMALLOC((void**)&d_out, sizeof(T)*numElement);
        TOGPU(d_in, h_in, sizeof(T)*numElement);
        unsigned int timer = 0;
        float elapsedTime = 0.0;

        startTimer(&timer);
        device_defined_kernel<<<numBlock, numThread>>>(d_in, numElement, d_out);
        cutilCheckMsg("device_defined_kernel");
        cutilSafeCall(hipDeviceSynchronize());
        elapsedTime = endTimer(&timer, "device_defined_kernel");

        FROMGPU(h_out, d_out, sizeof(T)*numElement);

        GPUFREE(d_in);
        GPUFREE(d_out);

        return elapsedTime;
}


float device_defined(gdd_real* h_in, const unsigned int numElement, gdd_real* h_out,
                        const unsigned int numBlock, const unsigned int numThread) {
	return device_defined_template(h_in, numElement, h_out, numBlock, numThread);
}

float device_defined(gqd_real* h_in, const unsigned int numElement, gqd_real* h_out,
                        const unsigned int numBlock, const unsigned int numThread) {
	return device_defined_template(h_in, numElement, h_out, numBlock, numThread);
}


/* the QRS map kernel */
__global__
void gpu_fx_map_kernel1( const gqd_real* d_x, gqd_real* d_c, const unsigned int N ) { 
	const unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int delta = blockDim.x*gridDim.x;
	gqd_real x = d_x[0];
	gqd_real c;
	
	for( unsigned int i = index; i < N; i += delta ) {
		c = sqrt(1.0 - sqr( ((N - x)/N) * (1.0 - 2.0*(i)/(N - 1.0)) ));
		d_c[i] = 2.0*c - 1.0/c;
	}
}

void device_qrsmap(const unsigned int N, const int numBlock, const int numThread) {
	printf("numBlock = %d, numThread = %d\n", numBlock, numThread);

	gqd_real h_x;
	h_x.x = 0.6; 
	h_x.y = 0.0; 
	h_x.z = 0.0; 
	h_x.w = 0.0;
	
	gqd_real* d_x = NULL;
	GPUMALLOC((void**)&d_x, sizeof(gqd_real));
	TOGPU( d_x, &h_x, sizeof(gqd_real) );
	gqd_real* d_c = NULL;
	GPUMALLOC((void**)&d_c, sizeof(gqd_real)*N);
	unsigned int timer = 0;

	startTimer(&timer);	
	gpu_fx_map_kernel1<<<numBlock, numThread>>>(d_x, d_c, N);
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg("gpu_fx_map_kernel1");
	endTimer(&timer, "gpu_fx_map_kernel");

	GPUFREE(d_x);
	GPUFREE(d_c);
}


#endif /* __GDD_TEST_CU__ */


