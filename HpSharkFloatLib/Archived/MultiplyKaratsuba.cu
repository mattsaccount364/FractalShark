#include "hip/hip_runtime.h"
﻿#include "MultiplyKaratsuba.cuh"

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "HpSharkFloat.cuh"
#include "BenchmarkTimer.h"
#include "DebugChecksum.cuh"

#include <iostream>
#include <vector>
#include <gmp.h>
#include <cstring>
#include <sstream>
#include <iomanip>
#include <cmath>
#include <algorithm>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/barrier>


namespace cg = cooperative_groups;

static constexpr auto
CalcAlign16Bytes64BitIndex(uint64_t Sixty4BitIndex) {
    return Sixty4BitIndex % 2 == 0 ? 0 : 1;
}

static constexpr auto
CalcAlign16Bytes32BitIndex(uint64_t Thirty2BitIndex) {
    return 4 - (Thirty2BitIndex % 4);
}


// Initialize the random number generator state.  Note that
// this uses a constant seed.  This is lame and we should be
// using a different seed for each thread.
static void
__device__ DebugInitRandom (
    cg::thread_block &block,
    hiprandState *state)
{
    int index = block.group_index().x * block.dim_threads().x + block.thread_index().x;
    hiprand_init(1234, index, 0, &state[index]);
}

//// Introduce a random delay.  This delay is per-thread and is
//// intended to exacerbate any races.  Note that you may want
//// instead a block-level delay.  This isn't it.
//static void
//__device__ DebugRandomDelay (
//    cg::thread_block &block,
//    hiprandState *state)
//{
//    int idx = block.group_index().x * block.dim_threads().x + block.thread_index().x;
//
//    static constexpr int maxIters = 1000;
//    float myrandf = hiprand_uniform(&state[idx]);
//    myrandf *= (maxIters + 0.999999);
//    int myrand = (int)truncf(myrandf);
//
//    volatile int dummy = 0;
//    for (auto i = 0; i < myrand; ++i) {
//        auto orig = dummy;
//        dummy = orig + 1;
//    }
//}

// Shared-memory “bands” prefetch model (used by ProcessConvolutionBatchPipelined):
// We cache four bands per tile: A_idx[], B_idx[], A_mirr[], B_mirr[].
// Worst-case unique values per tile (in uint32): 
//   values = 2*B + 2*(K + B - 1) = 2*K + 4*B - 2
// where K = threads per block, B = batch size cap per tile.
// For double buffering, we need 2 * values * sizeof(uint32_t).

template<class SharkFloatParams, int RecursionDepth>
struct SharedMemoryLayout {
    // ---- Explanatory constants (no more magic numbers) ----
    static constexpr int kBytesPerU32 = 4;   // sizeof(uint32_t)
    static constexpr int kIdxBands = 2;   // A_idx, B_idx
    static constexpr int kMirrBands = 2;   // A_mirr, B_mirr
    static constexpr int kDoubleBuffers = 2;   // ping & pong
    static constexpr int kBatchCapPipelined = 8;   // B_max we support in the tiler (tunable)

    // Threads per block (compile-time for the params)
    static constexpr int K = SharkFloatParams::GlobalThreadsPerBlock;
    static constexpr int Bmax = kBatchCapPipelined;

    // Worst-case unique uint32 values needed for one tile (single buffer)
    // values = 2*B + 2*(K + B - 1) = 2*K + 4*B - 2
    static constexpr int kWorstCaseTileU32 = (kIdxBands * Bmax) + (kMirrBands * (K + Bmax - 1));

    // Single-buffer bytes for one tile
    static constexpr int kSingleBufferBytes = kWorstCaseTileU32 * kBytesPerU32;

    // Double-buffer bytes (ping-pong) for overlap
    static constexpr int kDoubleBufferBytes = kDoubleBuffers * kSingleBufferBytes;

    // ---- Existing shared layout components (unchanged here) ----
    static constexpr int SyncVarsSize = 2 * sizeof(uint32_t);

    static constexpr int BaseSharedMemory = HpShark::LoadAllInShared
        ? (SharkFloatParams::GlobalNumUint32 * 4 * 2  // aDigits + bDigits
            + SharkFloatParams::GlobalNumUint32 * 2    // x_diff_abs + y_diff_abs
            + 1024)                                    // misc padding/other
        : (2048 + SyncVarsSize);                      // minimum when not loading all in shared + sync vars

    // ---- New pipeline buffer size expressed via the constants above ----
    static constexpr int PipelineBufferSize = !HpShark::LoadAllInShared
        ? kDoubleBufferBytes                 // double-buffered worst-case tile
        : 0;

    // Offsets & totals by recursion
    static constexpr int RecursionOffset = RecursionDepth * (BaseSharedMemory + PipelineBufferSize);
    static constexpr int TotalSharedMemory = RecursionOffset + BaseSharedMemory + PipelineBufferSize;

    static_assert(TotalSharedMemory <= 48 * 1024, "Shared memory exceeds 48KB limit");
};

// Get sync variables for specific recursion depth
template<class SharkFloatParams, int RecursionDepth>
__device__ uint32_t *GetRecursionSyncVars(uint32_t *base_shared_data) {
    constexpr int base_offset = SharedMemoryLayout<SharkFloatParams, RecursionDepth>::RecursionOffset;
    return base_shared_data + (base_offset / sizeof(uint32_t));
}

// Get pipeline buffer for specific recursion depth (when not loading all in shared)
template<class SharkFloatParams, int RecursionDepth>
__device__ uint32_t *GetRecursionPipelineBuffer(uint32_t *base_shared_data) {
    if constexpr (HpShark::LoadAllInShared) {
        return nullptr; // No pipeline buffer needed
    } else {
        constexpr int base_offset = SharedMemoryLayout<SharkFloatParams, RecursionDepth>::RecursionOffset;
        constexpr int base_size = SharedMemoryLayout<SharkFloatParams, RecursionDepth>::BaseSharedMemory;
        return base_shared_data + ((base_offset + base_size) / sizeof(uint32_t));
    }
}

// Compare two digit arrays, returning 1 if a > b, -1 if a < b, and 0 if equal
template<int n1, int n2>
static __device__ int CompareDigits(
    const uint32_t *SharkRestrict highArray,
    const uint32_t *SharkRestrict lowArray)
{
    // The biggest possible "digit index" is one less
    // than the max of the two sizes.
    int maxLen = std::max(n1, n2);

    // Compare top-down, from maxLen-1 down to 0
    for (int i = maxLen - 1; i >= 0; --i) {
        // Treat out-of-range as zero
        uint32_t a_val = (i < n1) ? highArray[i] : 0u;
        uint32_t b_val = (i < n2) ? lowArray[i] : 0u;

        if (a_val > b_val) {
            return 1;  // A is bigger
        } else if (a_val < b_val) {
            return -1; // B is bigger
        }
    }
    return 0;
}

// Subtract two digit arrays, returning the result.
// This is a serial implementation.
template<int n1, int n2>
__device__ static void
SubtractDigitsSerial(const uint32_t *a, const uint32_t *b, uint32_t *result) {
    uint64_t borrow = 0;
    for (int i = 0; i < n1; ++i) {
        uint64_t ai;
        uint64_t bi;

        ai = a[i];

        if (i >= n2) {
            bi = 0;
        } else {
            bi = b[i];
        }

        uint64_t temp = ai - bi - borrow;
        if (ai < bi + borrow) {
            borrow = 1;
            temp += ((uint64_t)1 << 32);
        } else {
            borrow = 0;
        }
        result[i] = (uint32_t)temp;
    }
}

#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

/**
 * Parallel subtraction (a1 - b1) and (a2 - b2), stored in global_(x|y)_diff_abs,
 * using a multi-pass approach to propagate borrows.
 *
 * The function attempts to subtract each digit of 'b' from 'a' in parallel,
 * then uses repeated passes (do/while) to handle newly introduced borrows
 * until no more remain or a maximum pass count is reached.
 * 
 * Corrupts x_diff_abs shared memory intentionally
 */
template<
    class SharkFloatParams,
    int a1n,
    int b1n,
    int a2n,
    int b2n,
    int ExecutionBlockBase,
    int ExecutionNumBlocks>
__device__ SharkForceInlineReleaseOnly static void
SubtractDigitsParallel(
    uint32_t *SharkRestrict x_diff_abs,
    uint32_t *SharkRestrict y_diff_abs,
    const uint32_t *SharkRestrict a1,
    const uint32_t *SharkRestrict b1,
    const uint32_t *SharkRestrict a2,
    const uint32_t *SharkRestrict b2,
    uint32_t *SharkRestrict subtractionBorrows1a,
    uint32_t *SharkRestrict subtractionBorrows1b,
    uint32_t *SharkRestrict subtractionBorrows2a,
    uint32_t *SharkRestrict subtractionBorrows2b,
    uint32_t *SharkRestrict global_x_diff_abs,
    uint32_t *SharkRestrict global_y_diff_abs,
    uint32_t *SharkRestrict globalBorrowAny,
    cg::grid_group &grid,
    cg::thread_block &block
) {
    // Note: steps on this.
    auto *SharkRestrict sharedBorrowAny = x_diff_abs;

    // Note: not ExecutionBlockBase
    if (block.group_index().x == 0 && block.thread_index().x == 0) {
        *globalBorrowAny = 0;
    }

    if (block.thread_index().x == 0) {
        *sharedBorrowAny = 0;
    }

    // Constants 
    constexpr int MaxPasses = 5000;     // maximum number of multi-pass sweeps

    // We'll define a grid-stride range covering [0..n) for each pass
    // 1) global thread id
    int tid = (block.group_index().x - ExecutionBlockBase) * block.dim_threads().x + block.thread_index().x;
    // 2) stride
    int stride = block.dim_threads().x * ExecutionNumBlocks;

    constexpr auto n1max = std::max(a1n, b1n);
    constexpr auto n2max = std::max(a2n, b2n);
    constexpr auto nmax = std::max(n1max, n2max);

    // (1) First pass: naive partial difference (a[i] - b[i]) and set borrowBit
    // Instead of dividing digits among blocks, each thread does a grid-stride loop:
    for (int idx = tid; idx < nmax; idx += stride) {
        uint32_t ai1;
        uint32_t bi1;
        uint32_t ai2;
        uint32_t bi2;

        // Fill in with 0s if idx is out of bounds
        if (idx < a1n) {
            ai1 = a1[idx];
        } else {
            ai1 = 0;
        }

        if (idx < a2n) {
            ai2 = a2[idx];
        } else {
            ai2 = 0;
        }

        if (idx < b1n) {
            bi1 = b1[idx];
        } else {
            bi1 = 0;
        }

        if (idx < b2n) {
            bi2 = b2[idx];
        } else {
            bi2 = 0;
        }

        // naive difference
        uint64_t diff1 = (uint64_t)ai1 - (uint64_t)bi1;
        uint64_t diff2 = (uint64_t)ai2 - (uint64_t)bi2;

        uint32_t borrow1 = (ai1 < bi1) ? 1u : 0u;
        uint32_t borrow2 = (ai2 < bi2) ? 1u : 0u;

        global_x_diff_abs[idx] = static_cast<uint32_t>(diff1 & 0xFFFFFFFFu);
        subtractionBorrows1a[idx] = borrow1;

        global_y_diff_abs[idx] = static_cast<uint32_t>(diff2 & 0xFFFFFFFFu);
        subtractionBorrows2a[idx] = borrow2;
    }

    // We'll do repeated passes to fix newly introduced borrows
    uint32_t *curBorrow1 = subtractionBorrows1a;
    uint32_t *newBorrow1 = subtractionBorrows1b;
    uint32_t *curBorrow2 = subtractionBorrows2a;
    uint32_t *newBorrow2 = subtractionBorrows2b;
    int pass = 0;
    uint32_t initialBorrowAny = 0;

    // sync the entire grid before multi-pass fixes
    grid.sync();

    do {
        // (2) For each digit, apply the borrow from the previous digit
        for (int idx = tid; idx < nmax; idx += stride) {
            uint64_t borrow_in1 = 0ULL;
            uint64_t borrow_in2 = 0ULL;
            if (idx > 0) {   // borrow_in is from digit (idx-1)
                borrow_in1 = (uint64_t)(curBorrow1[idx - 1]);
                borrow_in2 = (uint64_t)(curBorrow2[idx - 1]);
            }

            uint32_t digit1 = global_x_diff_abs[idx];
            uint32_t digit2 = global_y_diff_abs[idx];

            // subtract the borrow
            uint64_t sum1 = (uint64_t)digit1 - borrow_in1;
            uint64_t sum2 = (uint64_t)digit2 - borrow_in2;

            // store updated digit
            global_x_diff_abs[idx] = static_cast<uint32_t>(sum1 & 0xFFFFFFFFULL);
            global_y_diff_abs[idx] = static_cast<uint32_t>(sum2 & 0xFFFFFFFFULL);

            // If sum is negative => top bit is 1 => new borrow
            if (sum1 & 0x8000'0000'0000'0000ULL) {
                newBorrow1[idx] = 1;
                atomicAdd(sharedBorrowAny, 1);
            } else {
                newBorrow1[idx] = 0;
            }

            if (sum2 & 0x8000'0000'0000'0000ULL) {
                newBorrow2[idx] = 1;
                atomicAdd(sharedBorrowAny, 1);
            } else {
                newBorrow2[idx] = 0;
            }
        }

        // (a) Block-level synchronization (so all threads see final sharedBorrowAny)
        block.sync();

        // The block's thread 0 aggregates once into globalBorrowAny
        if (block.thread_index().x == 0) {
            // Add sharedBorrowAny to the global counter
            atomicAdd(globalBorrowAny, *sharedBorrowAny);

            // Reset local aggregator for the next pass
            *sharedBorrowAny = 0;
        }

        // sync before checking if any new borrows remain
        grid.sync();

        auto tempCopyGlobalBorrowAny = *globalBorrowAny;
        if (tempCopyGlobalBorrowAny == initialBorrowAny) {
            break;  // no new borrows => done
        }

        grid.sync();
        initialBorrowAny = tempCopyGlobalBorrowAny;

        // swap curBorrow, newBorrow
        uint32_t *tmp = curBorrow1;
        curBorrow1 = newBorrow1;
        newBorrow1 = tmp;

        tmp = curBorrow2;
        curBorrow2 = newBorrow2;
        newBorrow2 = tmp;

        pass++;
    } while (pass < MaxPasses);

    if constexpr (HpShark::Debug) {
        if (pass == MaxPasses && block.group_index().x == 0) {
            // This will deadlock the kernel because this problem is hard to diagnose
            grid.sync();
        }
    }
}


// This implementation is pretty ameteur hour since we're not doing blelloch scan
// Next version maybe I'll try something in that direction.  For now since we're
// mostly dealing with random numbers anyway and not weird cases where every digit
// generates a borrow or something, this should be good enough.
template<
    class SharkFloatParams,
    int a1n, int b1n,
    int a2n, int b2n,
    int ExecutionBlockBase,
    int ExecutionNumBlocks>
__device__ SharkForceInlineReleaseOnly static void
SubtractDigitsParallelImproved3(
    // Working arrays (which may be in shared memory)
    uint32_t *SharkRestrict x_diff_abs,
    uint32_t *SharkRestrict y_diff_abs,
    // Input digit arrays (for the two halves)
    const uint32_t *SharkRestrict a1,
    const uint32_t *SharkRestrict b1,
    const uint32_t *SharkRestrict a2,
    const uint32_t *SharkRestrict b2,
    // Two borrow arrays (one for each half)
    uint32_t *SharkRestrict subtractionBorrows1a,
    uint32_t *SharkRestrict subtractionBorrows2a,
    uint32_t *SharkRestrict subtractionBorrows1b,
    uint32_t *SharkRestrict subtractionBorrows2b,
    // An array (of size ExecutionNumBlocks) for storing each block's final borrow
    uint32_t *SharkRestrict blockBorrow1,
    uint32_t *SharkRestrict blockBorrow2,
    // Global buffers to hold the "working" differences
    uint32_t *SharkRestrict global_x_diff_abs,
    uint32_t *SharkRestrict global_y_diff_abs,
    // A single global counter to indicate if any borrow remains
    uint32_t *SharkRestrict globalBorrowAny,
    cg::grid_group &grid,
    cg::thread_block &block) {

    // Note: steps on this.
    auto *SharkRestrict sharedBorrowAny =
        HpShark::LoadAllInShared ?
        x_diff_abs :
        &x_diff_abs[block.group_index().x];

    if constexpr (ExecutionNumBlocks > 1) {
        // Compute maximum digit count from the two halves.
        constexpr int n1max = (a1n > b1n) ? a1n : b1n;
        constexpr int n2max = (a2n > b2n) ? a2n : b2n;
        constexpr int nmax = (n1max > n2max) ? n1max : n2max;

        // Use the same mapping as the original:
        const int tid = (block.group_index().x - ExecutionBlockBase) * block.dim_threads().x + block.thread_index().x;
        const int stride = block.dim_threads().x * ExecutionNumBlocks;

        // Reset the global borrow counter.
        if (block.group_index().x == 0 && block.thread_index().x == 0) {
            *globalBorrowAny = 0;
        }

        // Erase the per-block borrow arrays.
        if (block.thread_index().x == 0) {
            blockBorrow1[block.group_index().x] = 0;
            blockBorrow2[block.group_index().x] = 0;
        }

        auto *SharkRestrict curSubtract1 = subtractionBorrows1a;
        auto *SharkRestrict curSubtract2 = subtractionBorrows2a;
        auto *SharkRestrict newSubtract1 = subtractionBorrows1b;
        auto *SharkRestrict newSubtract2 = subtractionBorrows2b;

        // === (1) INITIAL SUBTRACTION: Process all digits using a grid-stride loop.
        // Only active blocks (those with group_index().x in [ExecutionBlockBase, ExecutionBlockBase+ExecutionNumBlocks))
        // participate.

        // Compute chunk size for each block.
        const int blockIdx = block.group_index().x - ExecutionBlockBase;

        const int baseSize = nmax / ExecutionNumBlocks;         // integer division
        const int remainder = nmax % ExecutionNumBlocks;          // extra digits to distribute
        const int chunkSize = (blockIdx < remainder) ? (baseSize + 1) : baseSize;
        const int blockStart = blockIdx * baseSize + min(blockIdx, remainder);
        const int blockEnd = blockStart + chunkSize;

        {
            // Each thread in the block processes its assigned indices in the contiguous chunk.
            for (int idx = blockStart + block.thread_index().x;
                idx < blockEnd;
                idx += block.dim_threads().x) {

                uint32_t a1_val = (idx < a1n) ? a1[idx] : 0;
                uint32_t b1_val = (idx < b1n) ? b1[idx] : 0;
                uint32_t a2_val = (idx < a2n) ? a2[idx] : 0;
                uint32_t b2_val = (idx < b2n) ? b2[idx] : 0;

                uint64_t diff1 = (uint64_t)a1_val - b1_val;
                uint64_t diff2 = (uint64_t)a2_val - b2_val;

                uint32_t borrow1 = (a1_val < b1_val) ? 1u : 0u;
                uint32_t borrow2 = (a2_val < b2_val) ? 1u : 0u;

                global_x_diff_abs[idx] = static_cast<uint32_t>(diff1);
                global_y_diff_abs[idx] = static_cast<uint32_t>(diff2);

                curSubtract1[idx] = borrow1;
                curSubtract2[idx] = borrow2;

                // Initialize newSubtract as well
                newSubtract1[idx] = 0;
                newSubtract2[idx] = 0;
            }
        }

        grid.sync();

        // === (2b) Each block's last thread writes its final borrow.
        if (block.thread_index().x == block.dim_threads().x - 1) {
            // Each block processes a contiguous chunk.
            // const int blockStart = (block.group_index().x - ExecutionBlockBase) * block.dim_threads().x;
            // const int blockEnd = blockStart + block.dim_threads().x; // exclusive

            const uint32_t finalBorrow1 = curSubtract1[blockEnd - 1];
            blockBorrow1[block.group_index().x] = finalBorrow1;
            curSubtract1[blockEnd - 1] = 0;

            const uint32_t finalBorrow2 = curSubtract2[blockEnd - 1];
            blockBorrow2[block.group_index().x] = finalBorrow2;
            curSubtract2[blockEnd - 1] = 0;
        }

        grid.sync();  // Ensure all initial differences and borrows are computed

        uint32_t initialBorrowAny = 0;

        // === (2) OUTER LOOP: Propagate borrows across blocks.
        const int MaxPasses = 500; // Adjust as needed.
        int outerPass = 0;
        do {
            uint32_t injection1 = 0, injection2 = 0;

            if (block.thread_index().x == 0) {
                injection1 = (block.group_index().x > ExecutionBlockBase)
                    ? blockBorrow1[block.group_index().x - 1]
                    : 0;
                injection2 = (block.group_index().x > ExecutionBlockBase)
                    ? blockBorrow2[block.group_index().x - 1]
                    : 0;
            }
            block.sync();

            // === (2a) LOCAL PROPAGATION WITHIN THE BLOCK.
            // Iterate blockDim.x times so that a borrow created at the block's start
            // immediately cascades through.

            for (int pass = 0; pass < nmax; ++pass) {

                if (block.thread_index().x == 0) {
                    *sharedBorrowAny = 0;
                }

                block.sync();

                for (int localIdx = blockStart + block.thread_index().x;
                    localIdx < blockEnd;
                    localIdx += block.dim_threads().x) {

                    // For x_diff_abs:
                    uint32_t borrow1;
                    if (block.thread_index().x == 0) {
                        // Only on the first pass do we subtract the injected borrow.
                        if (pass == 0 && localIdx == blockStart) {
                            borrow1 = injection1;
                        } else {
                            if (localIdx == blockStart) {
                                borrow1 = 0;
                            } else {
                                // This path occurs when there are few threads but extra digits.
                                // Thread 0 might iterate the inner loop once, and in that case
                                // propagation needs to happen.
                                borrow1 = curSubtract1[localIdx - 1];
                            }
                        }
                    } else {
                        borrow1 = curSubtract1[localIdx - 1];
                    }
                    const uint64_t newVal1 = (uint64_t)global_x_diff_abs[localIdx] - borrow1;
                    global_x_diff_abs[localIdx] = static_cast<uint32_t>(newVal1 & 0xFFFFFFFFULL);

                    // Last thread in the block that actually did anything
                    if (newVal1 & 0x8000000000000000ULL) {
                        atomicAdd(sharedBorrowAny, 1);

                        if (localIdx == blockEnd - 1) {
                            newSubtract1[localIdx] |= 1u;
                            curSubtract1[localIdx] |= 1u;
                        } else {
                            newSubtract1[localIdx] = 1u;
                        }
                    } else {
                        if (localIdx < blockEnd - 1) {
                            newSubtract1[localIdx] = 0u;
                        }
                    }

                    // For y_diff_abs:
                    uint32_t borrow2;
                    if (block.thread_index().x == 0) {
                        // Only on the first pass do we subtract the injected borrow.
                        if (pass == 0 && localIdx == blockStart) {
                            borrow2 = injection2;
                        } else {
                            if (localIdx == blockStart) {
                                borrow2 = 0;
                            } else {
                                // This path occurs when there are few threads but extra digits.
                                // Thread 0 might iterate the inner loop once, and in that case
                                // propagation needs to happen.
                                borrow2 = curSubtract2[localIdx - 1];
                            }
                        }
                    } else {
                        borrow2 = curSubtract2[localIdx - 1];
                    }
                    const uint64_t newVal2 = (uint64_t)global_y_diff_abs[localIdx] - borrow2;
                    global_y_diff_abs[localIdx] = static_cast<uint32_t>(newVal2 & 0xFFFFFFFFULL);

                    // Last thread in the block that actually did anything
                    if (newVal2 & 0x8000000000000000ULL) {
                        atomicAdd(sharedBorrowAny, 1);

                        if (localIdx == blockEnd - 1) {
                            newSubtract2[localIdx] |= 1u;
                            curSubtract2[localIdx] |= 1u;
                        } else {
                            newSubtract2[localIdx] = 1u;
                        }
                    } else {
                        if (localIdx < blockEnd - 1) {
                            newSubtract2[localIdx] = 0u;
                        }
                    }
                }

                // Swap curSubtract and newSubtract.
                auto *SharkRestrict tmp = curSubtract1;
                curSubtract1 = newSubtract1;
                newSubtract1 = tmp;

                tmp = curSubtract2;
                curSubtract2 = newSubtract2;
                newSubtract2 = tmp;

                block.sync();

                auto tmpBorrow = *sharedBorrowAny;

                block.sync();

                if (tmpBorrow == 0)
                    break;  // no new borrows
            }
            grid.sync();

            // === (2b) Each block's last thread writes its final borrow.
            if (block.thread_index().x == block.dim_threads().x - 1) {
                const uint32_t finalBorrow1 = curSubtract1[blockEnd - 1];
                blockBorrow1[block.group_index().x] = finalBorrow1;
                curSubtract1[blockEnd - 1] = 0;
                newSubtract1[blockEnd - 1] = 0;

                const uint32_t finalBorrow2 = curSubtract2[blockEnd - 1];
                blockBorrow2[block.group_index().x] = finalBorrow2;
                curSubtract2[blockEnd - 1] = 0;
                newSubtract2[blockEnd - 1] = 0;
            }
            grid.sync();

            // === (2c) Global aggregation: One designated block sums the per-block borrows.
            if (block.group_index().x == ExecutionBlockBase &&
                block.thread_index().x == 0) {

                uint32_t totalBorrow = 0;
                for (int i = ExecutionBlockBase; i < ExecutionBlockBase + ExecutionNumBlocks; ++i) {
                    totalBorrow += blockBorrow1[i];
                    totalBorrow += blockBorrow2[i];
                }

                atomicAdd(globalBorrowAny, totalBorrow);  // Overwrite with the new total.
            }
            grid.sync();

            uint32_t tempCopyGlobalBorrowAny = *globalBorrowAny;
            if (tempCopyGlobalBorrowAny == initialBorrowAny)
                break;  // no new borrows --> done

            grid.sync();

            initialBorrowAny = tempCopyGlobalBorrowAny;
            outerPass++;
        } while (outerPass < MaxPasses);

        grid.sync();  // Final grid sync to guarantee all blocks are done.


    } else { //////////////////////////////////////////////////////////////////////

        // Compute maximum digit count.
        constexpr int n1max = (a1n > b1n) ? a1n : b1n;
        constexpr int n2max = (a2n > b2n) ? a2n : b2n;
        constexpr int nmax = (n1max > n2max) ? n1max : n2max;

        // For one block, block.group_index().x == ExecutionBlockBase.
        // Define a simple block-stride loop over the contiguous chunk: [0, nmax).
        const int tid = block.thread_index().x;
        const int stride = block.dim_threads().x;
        const int blockStart = 0;
        const int blockEnd = nmax; // all digits in [0, nmax)

        auto *SharkRestrict curSubtract1 = subtractionBorrows1a;
        auto *SharkRestrict curSubtract2 = subtractionBorrows2a;
        auto *SharkRestrict newSubtract1 = subtractionBorrows1b;
        auto *SharkRestrict newSubtract2 = subtractionBorrows2b;

        // INITIAL SUBTRACTION: each thread processes its assigned digits.
        for (int idx = blockStart + tid; idx < blockEnd; idx += stride) {
            uint32_t a1_val = (idx < a1n) ? a1[idx] : 0;
            uint32_t b1_val = (idx < b1n) ? b1[idx] : 0;
            uint32_t a2_val = (idx < a2n) ? a2[idx] : 0;
            uint32_t b2_val = (idx < b2n) ? b2[idx] : 0;

            uint64_t diff1 = (uint64_t)a1_val - b1_val;
            uint64_t diff2 = (uint64_t)a2_val - b2_val;

            uint32_t borrow1 = (a1_val < b1_val) ? 1u : 0u;
            uint32_t borrow2 = (a2_val < b2_val) ? 1u : 0u;

            global_x_diff_abs[idx] = static_cast<uint32_t>(diff1);
            global_y_diff_abs[idx] = static_cast<uint32_t>(diff2);

            curSubtract1[idx] = borrow1;
            curSubtract2[idx] = borrow2;
        }

        block.sync();

        // LOCAL PROPAGATION: For each pass, each thread processes its indices in [blockStart, blockEnd)
        // using a block-stride loop.
        // We run (blockEnd - blockStart + 1) passes to ensure complete propagation.

        for (int pass = 0; pass < (blockEnd - blockStart + 1); ++pass) {
            if (block.thread_index().x == 0) {
                *sharedBorrowAny = 0;
            }

            block.sync();

            for (int idx = blockStart + tid; idx < blockEnd; idx += stride) {
                // For the first digit, there is no previous digit, so the borrow is 0.
                uint32_t borrow1 = (idx == blockStart) ? 0 : curSubtract1[idx - 1];
                uint64_t newVal1 = static_cast<uint64_t>(global_x_diff_abs[idx]) - borrow1;
                global_x_diff_abs[idx] = static_cast<uint32_t>(newVal1 & 0xFFFFFFFFULL);

                if (newVal1 & 0x8000000000000000ULL) {
                    newSubtract1[idx] = 1;
                    atomicAdd(sharedBorrowAny, 1);
                } else {
                    newSubtract1[idx] = 0;
                }

                uint32_t borrow2 = (idx == blockStart) ? 0 : curSubtract2[idx - 1];
                uint64_t newVal2 = static_cast<uint64_t>(global_y_diff_abs[idx]) - borrow2;
                global_y_diff_abs[idx] = static_cast<uint32_t>(newVal2 & 0xFFFFFFFFULL);

                if (newVal2 & 0x8000000000000000ULL) {
                    newSubtract2[idx] = 1;
                    atomicAdd(sharedBorrowAny, 1);
                } else {
                    newSubtract2[idx] = 0;
                }
            }

            // All threads synchronize after processing the entire chunk.
            block.sync();

            auto tmpBorrow = *sharedBorrowAny;
            block.sync();
            if (tmpBorrow == 0) {
                break;  // no new borrows
            }

            // Swap curSubtract and newSubtract.
            auto *SharkRestrict tmp = curSubtract1;
            curSubtract1 = newSubtract1;
            newSubtract1 = tmp;

            tmp = curSubtract2;
            curSubtract2 = newSubtract2;
            newSubtract2 = tmp;
        }
    }
}



// Function to perform addition with carry
__device__ SharkForceInlineReleaseOnly static void
Add128(
    uint64_t a_low, uint64_t a_high,
    uint64_t b_low, uint64_t b_high,
    uint64_t &result_low, uint64_t &result_high) {

    result_low = a_low + b_low;
    uint64_t carry = (result_low < a_low) ? 1 : 0;
    result_high = a_high + b_high + carry;
}

__device__ SharkForceInlineReleaseOnly static void
Subtract128(
    uint64_t a_low, uint64_t a_high,
    uint64_t b_low, uint64_t b_high,
    uint64_t &result_low, uint64_t &result_high) {

    uint64_t borrow = 0;

    // Subtract low parts
    result_low = a_low - b_low;
    borrow = (a_low < b_low) ? 1 : 0;

    // Subtract high parts with borrow
    result_high = a_high - b_high - borrow;
}

template<class SharkFloatParams>
__device__ SharkForceInlineReleaseOnly static void
SerialCarryPropagationThread0 (
    uint64_t *SharkRestrict shared_data,
    cg::grid_group &grid,
    cg::thread_block &block,
    int thread_start_idx,
    int thread_end_idx,
    const uint64_t *SharkRestrict final128XX,
    const uint64_t *SharkRestrict final128XY,
    const uint64_t *SharkRestrict final128YY,
    uint64_t *SharkRestrict resultXX,
    uint64_t *SharkRestrict resultXY,
    uint64_t *SharkRestrict resultYY)
{
    constexpr int total_result_digits = 2 * SharkFloatParams::GlobalNumUint32;

    // Stateless lambda to process one component (XX, XY, or YY)
    auto ProcessComponent = [](
        const uint64_t *final128,
        uint64_t *result,
        int total_digits) -> uint64_t {

            uint64_t local_carry = 0;

            // Process all digits serially from lowest to highest
            for (int idx = 0; idx < total_digits; ++idx) {
                int sum_low_idx = idx * 2;
                int sum_high_idx = sum_low_idx + 1;

                uint64_t sum_low = final128[sum_low_idx];
                uint64_t sum_high = final128[sum_high_idx];

                // Add local carry to sum_low
                bool new_sum_low_negative = false;
                uint64_t new_sum_low = sum_low + local_carry;

                // Extract one 32-bit digit from new_sum_low
                auto digit = static_cast<uint32_t>(new_sum_low & 0xFFFFFFFFULL);
                result[idx] = digit;

                bool local_carry_negative = ((local_carry & (1ULL << 63)) != 0);
                local_carry = 0ULL;

                if (!local_carry_negative && new_sum_low < sum_low) {
                    local_carry = 1ULL << 32;
                } else if (local_carry_negative && new_sum_low > sum_low) {
                    new_sum_low_negative = (new_sum_low & 0x8000'0000'0000'0000ULL) != 0;
                }

                // Update local_carry
                if (new_sum_low_negative) {
                    // Shift sum_high by 32 bits and add carry_from_low
                    uint64_t upper_new_sum_low = new_sum_low >> 32;
                    upper_new_sum_low |= 0xFFFF'FFFF'0000'0000ULL;
                    local_carry += upper_new_sum_low;
                    local_carry += sum_high << 32;
                } else {
                    local_carry += new_sum_low >> 32;
                    local_carry += sum_high << 32;
                }
            }

            return local_carry;
        };

    // Process each component using the lambda
    uint64_t final_carry_xx = ProcessComponent(final128XX, resultXX, total_result_digits);
    uint64_t final_carry_xy = ProcessComponent(final128XY, resultXY, total_result_digits);
    uint64_t final_carry_yy = ProcessComponent(final128YY, resultYY, total_result_digits);

    // Store final carries
    if (final_carry_xx > 0) {
        resultXX[total_result_digits] = static_cast<uint32_t>(final_carry_xx & 0xFFFFFFFFULL);
    }
    if (final_carry_xy > 0) {
        resultXY[total_result_digits] = static_cast<uint32_t>(final_carry_xy & 0xFFFFFFFFULL);
    }
    if (final_carry_yy > 0) {
        resultYY[total_result_digits] = static_cast<uint32_t>(final_carry_yy & 0xFFFFFFFFULL);
    }
}

template<class SharkFloatParams>
__device__ SharkForceInlineReleaseOnly static void
SerialCarryPropagation (
    uint64_t *SharkRestrict shared_data,
    cg::grid_group &grid,
    cg::thread_block &block,
    int thread_start_idx,
    int thread_end_idx,
    const uint64_t *SharkRestrict final128XX,
    const uint64_t *SharkRestrict final128XY,
    const uint64_t *SharkRestrict final128YY,
    uint64_t *SharkRestrict resultXX,
    uint64_t *SharkRestrict resultXY,
    uint64_t *SharkRestrict resultYY,
    uint64_t *SharkRestrict block_carry_outs,
    uint64_t *SharkRestrict /*globalCarryCheck*/) {

    // Only execute on a single thread to maintain serial behavior
    if (block.thread_index().x == 0 && block.group_index().x == 0) {
        SerialCarryPropagationThread0<SharkFloatParams>(
            shared_data,
            grid,
            block,
            thread_start_idx,
            thread_end_idx,
            final128XX,
            final128XY,
            final128YY,
            resultXX,
            resultXY,
            resultYY);
    }
}

static __device__ int
CarryGlobalToIndex (
    const bool PriorIndex,
    const int block_idx)
{
    return (block_idx - (PriorIndex ? 1 : 0)) * 3;
}

static __device__ int
CarrySharedToIndex (
    const bool PriorIndex,
    const int thread_idx)
{
    return (thread_idx - (PriorIndex ? 1 : 0)) * 3;
}

template<class SharkFloatParams>
__device__ SharkForceInlineReleaseOnly static void
CarryPropagation (
    uint64_t *SharkRestrict shared_data,
    cg::grid_group &grid,
    cg::thread_block &block,
    int thread_start_idx,
    int thread_end_idx,
    const uint64_t *SharkRestrict final128XX,
    const uint64_t *SharkRestrict final128XY,
    const uint64_t *SharkRestrict final128YY,
    uint64_t *SharkRestrict resultXX,
    uint64_t *SharkRestrict resultXY,
    uint64_t *SharkRestrict resultYY,
    uint64_t *SharkRestrict block_carry_outs,
    uint64_t *SharkRestrict globalCarryCheck) {

    auto *SharkRestrict shared_carries = shared_data;

    // TODO: Ensure we allocate a minimum amount of shared memory to support shared_carries use
    // TODO: Ensure we allocate a minimum amount of global memory to support block_carry_outs use

    // First Pass: Process convolution results to compute initial digits and local carries
    // Initialize local carry
    uint64_t local_carry_xx = 0;
    uint64_t local_carry_xy = 0;
    uint64_t local_carry_yy = 0;

    // TODO: Remove this:
    constexpr auto sharedMem = CalculateMultiplySharedMemorySize<SharkFloatParams>() / sizeof(uint64_t);
    {
        // Erase shared_carries
        for (int i = block.thread_index().x; i < sharedMem; i += block.dim_threads().x) {
            shared_carries[i] = 0;
        }
    }

    grid.sync();

    const auto MaxBlocks = grid.group_dim().x;
    const auto MaxThreads = block.dim_threads().x;

    // Constants and offsets
    constexpr int MaxPasses = 5000; // Maximum number of carry propagation passes
    constexpr int total_result_digits = 2 * SharkFloatParams::GlobalNumUint32;

    uint64_t *carries_remaining_global = globalCarryCheck;

    for (int idx = thread_start_idx; idx < thread_end_idx; ++idx) {
        const int sum_low_idx = idx * 2;
        const int sum_high_idx = sum_low_idx + 1;

        const uint64_t xx_sum_low = final128XX[sum_low_idx];     // Lower 64 bits
        const uint64_t xx_sum_high = final128XX[sum_high_idx];   // Higher 64 bits

        const uint64_t xy_sum_low = final128XY[sum_low_idx];     // Lower 64 bits
        const uint64_t xy_sum_high = final128XY[sum_high_idx];   // Higher 64 bits

        const uint64_t yy_sum_low = final128YY[sum_low_idx];     // Lower 64 bits
        const uint64_t yy_sum_high = final128YY[sum_high_idx];   // Higher 64 bits

        // Add local carry to sum_low
        auto LocalCarry = [](
            uint64_t &local_carry,
            uint64_t sum_low,
            uint64_t sum_high,
            uint64_t *result,
            int idx) {

            bool new_sum_low_negative = false;
            const uint64_t new_sum_low = sum_low + local_carry;

            // Extract one 32-bit digit from new_sum_low
            const auto digit = static_cast<uint32_t>(new_sum_low & 0xFFFFFFFFULL);
            result[idx] = digit;

            const bool local_carry_negative = ((local_carry & (1ULL << 63)) != 0);
            local_carry = 0ULL;

            if (!local_carry_negative && new_sum_low < sum_low) {
                local_carry = 1ULL << 32;
            } else if (local_carry_negative && new_sum_low > sum_low) {
                new_sum_low_negative = (new_sum_low & 0x8000'0000'0000'0000) != 0;
            }

            // Update local_carry
            if (new_sum_low_negative) {
                // Shift sum_high by 32 bits and add carry_from_low
                uint64_t upper_new_sum_low = new_sum_low >> 32;
                upper_new_sum_low |= 0xFFFF'FFFF'0000'0000;
                local_carry += upper_new_sum_low;
                local_carry += sum_high << 32;
            } else {
                local_carry += new_sum_low >> 32;
                local_carry += sum_high << 32;
            }
        };

        // Process xx_sum
        LocalCarry(local_carry_xx, xx_sum_low, xx_sum_high, resultXX, idx);

        // Process xy_sum
        LocalCarry(local_carry_xy, xy_sum_low, xy_sum_high, resultXY, idx);

        // Process yy_sum
        LocalCarry(local_carry_yy, yy_sum_low, yy_sum_high, resultYY, idx);
    }

    const auto threadIndexInBlock = block.thread_index().x;
    const auto blockIndexInGrid = block.group_index().x;
    const auto carrySharedToIndex = CarrySharedToIndex(
        false,
        threadIndexInBlock);
    if (threadIndexInBlock == SharkFloatParams::GlobalThreadsPerBlock - 1) {
        const auto carryGlobalToIndex = CarryGlobalToIndex(
            false,
            block.group_index().x);

        block_carry_outs[carryGlobalToIndex + 0] = local_carry_xx;
        block_carry_outs[carryGlobalToIndex + 1] = local_carry_xy;
        block_carry_outs[carryGlobalToIndex + 2] = local_carry_yy;

        shared_carries[carrySharedToIndex + 0] = local_carry_xx;
        shared_carries[carrySharedToIndex + 1] = local_carry_xy;
        shared_carries[carrySharedToIndex + 2] = local_carry_yy;
    } else {
        shared_carries[carrySharedToIndex + 0] = local_carry_xx;
        shared_carries[carrySharedToIndex + 1] = local_carry_xy;

        shared_carries[carrySharedToIndex + 2] = local_carry_yy;
    }

    // Inter-Block Carry Propagation
    int pass = 0;

    do {
        // Synchronize all blocks
        grid.sync();

        // Zero out the global carry count for the current pass
        if (block.group_index().x == 0 && block.thread_index().x == 0) {
            *carries_remaining_global = 0;
        }

        // Get carry-in from the previous block
        // The warning here is about the constant template parameter not being used in
        // the parameter list.  I don't understand why it's even a warning?  Maybe because
        // it cannot be inferred?  It's clearly being used in the body?

#pragma nv_diag_suppress 445
        auto LocalCarryIn = []<int XX_XY_YY>(
            cg::thread_block & block,
            const int MaxBlocks,
            const int MaxThreads,
            uint64_t &local_carry,
            uint64_t *SharkRestrict block_carry_outs,
            uint64_t *SharkRestrict shared_carries) {

            local_carry = 0;
            if (block.thread_index().x == 0 && block.group_index().x > 0) {
                const auto block_carry_outs_idx = CarryGlobalToIndex(
                    true,
                    block.group_index().x) + XX_XY_YY;
                local_carry = block_carry_outs[block_carry_outs_idx];
            } else {
                if (block.thread_index().x > 0) {
                    const auto shared_carries_idx = CarrySharedToIndex(
                        true,
                        block.thread_index().x) + XX_XY_YY;
                    local_carry = shared_carries[shared_carries_idx];
                }
            }
            };
#pragma nv_diag_default 445

        // Initialize local carry for this pass
        LocalCarryIn.template operator()<0>(
            block,
            MaxBlocks,
            MaxThreads,
            local_carry_xx,
            block_carry_outs,
            shared_carries);
        LocalCarryIn.template operator()<1>(
            block,
            MaxBlocks,
            MaxThreads,
            local_carry_xy,
            block_carry_outs,
            shared_carries);
        LocalCarryIn.template operator()<2>(
            block,
            MaxBlocks,
            MaxThreads,
            local_carry_yy,
            block_carry_outs,
            shared_carries);

        auto LocalCarry = [](uint64_t *SharkRestrict resultXY, uint64_t &local_carry, int idx) {
            // Read the previously stored digit
            const uint32_t digit = resultXY[idx];

            // Add local_carry to digit
            const uint64_t sum = static_cast<uint64_t>(digit) + local_carry;

            // Update digit
            resultXY[idx] = static_cast<uint32_t>(sum & 0xFFFFFFFFULL);

            local_carry = 0;

            // Check negativity of the 64-bit sum
            // If "sum" is negative, its top bit is set. 
            const bool sum_is_negative = ((sum & (1ULL << 63)) != 0ULL);
            if (sum_is_negative) {
                // sign-extend the top 32 bits
                uint64_t upper_bits = (sum >> 32);
                upper_bits |= 0xFFFF'FFFF'0000'0000ULL;  // set top 32 bits to 1
                local_carry += upper_bits;               // incorporate sign-extended bits
            } else {
                // normal path: just add top 32 bits
                local_carry += (sum >> 32);
            }
            };

        // Each thread processes its assigned digits
        for (int idx = thread_start_idx; idx < thread_end_idx; ++idx) {
            // Process xx_sum
            LocalCarry(resultXX, local_carry_xx, idx);

            // Process xy_sum
            LocalCarry(resultXY, local_carry_xy, idx);

            // Process yy_sum
            LocalCarry(resultYY, local_carry_yy, idx);
        }

        // TODO should we interleave each of these instead of separating them?  probably?
        const auto carrySharedToCurIndex = CarrySharedToIndex(
            false,
            block.thread_index().x);
        const auto carryGlobalToCurIndex = CarryGlobalToIndex(
            false,
            block.group_index().x);

        shared_carries[carrySharedToCurIndex + 0] = local_carry_xx;
        shared_carries[carrySharedToCurIndex + 1] = local_carry_xy;
        shared_carries[carrySharedToCurIndex + 2] = local_carry_yy;

        // This sync is required to address the *carries_remaining_global = 0; assignment
        // racing with the atomicAdd below.
        grid.sync();

        // The block's carry-out is the carry from the last thread
        const auto temp_xx = shared_carries[carrySharedToCurIndex + 0];
        if (block.thread_index().x == SharkFloatParams::GlobalThreadsPerBlock - 1) {
            block_carry_outs[carryGlobalToCurIndex + 0] = temp_xx;
        }

        const auto temp_xy = shared_carries[carrySharedToCurIndex + 1];
        if (block.thread_index().x == SharkFloatParams::GlobalThreadsPerBlock - 1) {
            block_carry_outs[carryGlobalToCurIndex + 1] = temp_xy;
        }

        const auto temp_yy = shared_carries[carrySharedToCurIndex + 2];
        if (block.thread_index().x == SharkFloatParams::GlobalThreadsPerBlock - 1) {
            block_carry_outs[carryGlobalToCurIndex + 2] = temp_yy;
        }

        if (temp_xx != 0 || temp_xy != 0 || temp_yy != 0) {
            atomicAdd(carries_remaining_global, 1);
        }

        // Synchronize all blocks before checking if carries remain
        grid.sync();

        // If no carries remain, exit the loop
        if (*carries_remaining_global == 0) {
            break;
        }

        pass++;
    } while (pass < MaxPasses);

    // ---- Handle Final Carry-Out ----

    // TODO is this correct?  remove?
    // Handle final carry-out
    if (block.thread_index().x == 0 && block.group_index().x == grid.dim_blocks().x - 1) {
        const auto carryGlobalToIndex = CarryGlobalToIndex(
            false,
            block.group_index().x);
        const auto block_idx_xx = carryGlobalToIndex + 0;
        uint64_t final_carry_xx = block_carry_outs[block_idx_xx];
        if (final_carry_xx > 0) {
            // Store the final carry as an additional digit
            resultXX[total_result_digits] = static_cast<uint32_t>(final_carry_xx & 0xFFFFFFFFULL);
        }

        const auto block_idx_xy = carryGlobalToIndex + 1;
        uint64_t final_carry_xy = block_carry_outs[block_idx_xy];
        if (final_carry_xy > 0) {
            // Store the final carry as an additional digit
            resultXY[total_result_digits] = static_cast<uint32_t>(final_carry_xy & 0xFFFFFFFFULL);
        }

        const auto block_idx_yy = carryGlobalToIndex + 2;
        uint64_t final_carry_yy = block_carry_outs[block_idx_yy];
        if (final_carry_yy > 0) {
            // Store the final carry as an additional digit
            resultYY[total_result_digits] = static_cast<uint32_t>(final_carry_yy & 0xFFFFFFFFULL);
        }
    }
}

// Look for CalculateKaratsubaFrameSize and ScratchMemoryArraysForMultiply
// and make sure the number of NewN arrays we're using here fits within that limit.
// The list here should go up to ScratchMemoryArraysForMultiply.
static_assert(AdditionalUInt64PerFrame == 256, "See below");
#define DefineTempProductsOffsets(CallIndex) \
    const int threadIdxGlobal = block.group_index().x * SharkFloatParams::GlobalThreadsPerBlock + block.thread_index().x; \
    constexpr int TestMultiplier = 1; \
    constexpr auto Multiplies_offset = AdditionalGlobalSyncSpace; \
    constexpr auto Checksum_offset = Multiplies_offset + AdditionalGlobalMultipliesPerThread; \
    /* Start from AdditionalUInt64PerFrame next, global state is above */ \
    constexpr auto CallOffset = Checksum_offset + AdditionalGlobalChecksumSpace + CallIndex * CalculateKaratsubaFrameSize<SharkFloatParams>(); \
    constexpr auto TempBase = 0; \
    constexpr auto TempBaseOffset = TempBase + CallOffset; \
    constexpr auto Z0_offsetXX = TempBaseOffset + AdditionalUInt64PerFrame + CalcAlign16Bytes64BitIndex(TempBaseOffset + AdditionalUInt64PerFrame); /* 0 */ \
    constexpr auto Z0_offsetXY = Z0_offsetXX + 4 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(4 * NewN * TestMultiplier);            /* 4 */ \
    constexpr auto Z0_offsetYY = Z0_offsetXY + 4 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(4 * NewN * TestMultiplier);            /* 8 */ \
    constexpr auto Z2_offsetXX = Z0_offsetYY + 4 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(4 * NewN * TestMultiplier);       /* 12 */ \
    constexpr auto Z2_offsetXY = Z2_offsetXX + 4 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(4 * NewN * TestMultiplier); /* 16 */ \
    constexpr auto Z2_offsetYY = Z2_offsetXY + 4 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(4 * NewN * TestMultiplier); /* 20 */ \
    constexpr auto Z1_temp_offsetXX = Z2_offsetYY + 4 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(4 * NewN * TestMultiplier); /* 24 */ \
    constexpr auto Z1_temp_offsetXY = Z1_temp_offsetXX + 4 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(4 * NewN * TestMultiplier); /* 28 */ \
    constexpr auto Z1_temp_offsetYY = Z1_temp_offsetXY + 4 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(4 * NewN * TestMultiplier); /* 32 */ \
    constexpr auto Z1_offsetXX = Z1_temp_offsetYY + 4 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(4 * NewN * TestMultiplier); /* 36 */ \
    constexpr auto Z1_offsetXY = Z1_offsetXX + 4 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(4 * NewN * TestMultiplier); /* 40 */ \
    constexpr auto Z1_offsetYY = Z1_offsetXY + 4 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(4 * NewN * TestMultiplier); /* 44 */ \
    constexpr auto Convolution_offsetXX = Z1_offsetYY + 4 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(4 * NewN * TestMultiplier); /* 48 */ \
    constexpr auto Convolution_offsetXY = Convolution_offsetXX + 4 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(4 * NewN * TestMultiplier); /* 52 */ \
    constexpr auto Convolution_offsetYY = Convolution_offsetXY + 4 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(4 * NewN * TestMultiplier); /* 56 */ \
    constexpr auto Result_offsetXX = Convolution_offsetYY + 4 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(4 * NewN * TestMultiplier); /* 60 */ \
    constexpr auto Result_offsetXY = Result_offsetXX + 4 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(4 * NewN * TestMultiplier); /* 64 */ \
    constexpr auto Result_offsetYY = Result_offsetXY + 4 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(4 * NewN * TestMultiplier); /* 68 */ \
    constexpr auto XDiff_offset = Result_offsetYY + 2 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(2 * NewN * TestMultiplier); /* 70 */ \
    constexpr auto YDiff_offset = XDiff_offset + 1 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(1 * NewN * TestMultiplier); /* 71 */ \
    constexpr auto GlobalCarryOffset = YDiff_offset + 1 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(1 * NewN * TestMultiplier); /* 72 */ \
    constexpr auto SubtractionOffset1 = GlobalCarryOffset + 1 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(1 * NewN * TestMultiplier); /* 73 */ \
    constexpr auto SubtractionOffset2 = SubtractionOffset1 + 1 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(1 * NewN * TestMultiplier); /* 74 */ \
    constexpr auto SubtractionOffset3 = SubtractionOffset2 + 1 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(1 * NewN * TestMultiplier); /* 75 */ \
    constexpr auto SubtractionOffset4 = SubtractionOffset3 + 1 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(1 * NewN * TestMultiplier); /* 76 */ \
    constexpr auto SubtractionOffset5 = SubtractionOffset4 + 1 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(1 * NewN * TestMultiplier); /* 77 */ \
    constexpr auto SubtractionOffset6 = SubtractionOffset5 + 1 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(1 * NewN * TestMultiplier); /* 78 */ \
    constexpr auto CarryInsOffset = SubtractionOffset6 + 1 * NewN * TestMultiplier + CalcAlign16Bytes64BitIndex(1 * NewN * TestMultiplier); /* requires 3xNewN 79 */ \
    constexpr auto CarryInsEnd = CarryInsOffset + 3 * NewN + CalcAlign16Bytes64BitIndex(3 * NewN); \


#define TempProductsGlobals(TempBase, CallIndex) \
    constexpr auto BorrowGlobalOffset = 0; \
    constexpr auto BorrowBlockLevelOffset1 = MaxBlocks; \
    constexpr auto BorrowBlockLevelOffset2 = MaxBlocks * 2; \

#define DefineExtraDefinitions() \
    const auto RelativeBlockIndex = block.group_index().x - ExecutionBlockBase; \
    constexpr int total_result_digits = 2 * NewN; \
    constexpr auto digits_per_block = NewN * 2 / ExecutionNumBlocks; \
    const auto block_start_idx = block.group_index().x * digits_per_block; \
    const auto block_end_idx = min(block_start_idx + digits_per_block, total_result_digits); \
    const int digits_per_thread = (digits_per_block + block.dim_threads().x - 1) / block.dim_threads().x; \
    const int thread_start_idx = block_start_idx + block.thread_index().x * digits_per_thread; \
    const int thread_end_idx = min(thread_start_idx + digits_per_thread, block_end_idx);

#define DefineCarryDefinitions() \
    constexpr auto total_result_digits = 2 * NewN; \
    constexpr auto per_thread_multiplier = 1; \
    constexpr auto total_threads = SharkFloatParams::GlobalThreadsPerBlock * SharkFloatParams::GlobalNumBlocks * per_thread_multiplier; \
    const int digits_per_thread = (total_result_digits + total_threads - 1) / total_threads; \
    const int thread_idx = block.group_index().x * SharkFloatParams::GlobalThreadsPerBlock + block.thread_index().x; \
    const int thread_start_idx = thread_idx * digits_per_thread; \
    const int thread_end_idx = min(thread_start_idx + digits_per_thread, total_result_digits);


template<
    class SharkFloatParams,
    int RecursionDepth,
    int CallIndex,
    DebugStatePurpose Purpose>
__device__ SharkForceInlineReleaseOnly static void
EraseCurrentDebugState(
    RecordIt record,
    DebugState<SharkFloatParams> *debugStates,
    cooperative_groups::grid_group &grid,
    cooperative_groups::thread_block &block) {

    constexpr auto maxPurposes = static_cast<int>(DebugStatePurpose::NumPurposes);
    constexpr auto curPurpose = static_cast<int>(Purpose);
    debugStates[CallIndex * maxPurposes + curPurpose].Erase(
        record, grid, block, Purpose, RecursionDepth, CallIndex);
}

template<
    class SharkFloatParams,
    int RecursionDepth,
    int CallIndex,
    DebugStatePurpose Purpose,
    typename ArrayType>
__device__ SharkForceInlineReleaseOnly static void
StoreCurrentDebugState (
    RecordIt record,
    UseConvolution useConvolution,
    DebugState<SharkFloatParams> *debugStates,
    cooperative_groups::grid_group &grid,
    cooperative_groups::thread_block &block,
    const ArrayType *arrayToChecksum,
    size_t arraySize)
{
    constexpr auto maxPurposes = static_cast<int>(DebugStatePurpose::NumPurposes);
    constexpr auto curPurpose = static_cast<int>(Purpose);
    debugStates[CallIndex * maxPurposes + curPurpose].Reset(
        record, useConvolution, grid, block, arrayToChecksum, arraySize, Purpose, RecursionDepth, CallIndex);
}

enum class ConditionalAccess {
    False,
    True
};

// Unified scalar accumulator used for BOTH prologue and epilogue
template<
    class SharkFloatParams,
    ConditionalAccess Cond
>
__device__ SharkForceInlineReleaseOnly static void
accumulate_scalar_span (
    cg::grid_group &grid,
    cg::thread_block &block,
    DebugMultiplyCount<SharkFloatParams> *debugMultiplyCounts,
    int i_lo, int i_hi, int k,
    const uint32_t *__restrict__ aDigits_base,
    const uint32_t *__restrict__ bDigits_base,
    int a_offset, int b_offset,
    const uint32_t *__restrict__ x_diff_abs,
    const uint32_t *__restrict__ y_diff_abs,
    uint64_t &xx_low, uint64_t &xx_high,
    uint64_t &xy_low, uint64_t &xy_high,
    uint64_t &yy_low, uint64_t &yy_high) {
    if (i_lo > i_hi) return;
#pragma unroll
    for (int idx = i_lo; idx <= i_hi; ++idx) {
        const int idx2 = k - idx;
        uint64_t xx_a, xx_b, xy_a, xy_b, yy_a, yy_b;
        if constexpr (Cond == ConditionalAccess::True) {
            xx_a = x_diff_abs[idx];     xx_b = x_diff_abs[idx2];
            xy_a = x_diff_abs[idx];     xy_b = y_diff_abs[idx2];
            yy_a = y_diff_abs[idx];     yy_b = y_diff_abs[idx2];
        } else {
            xx_a = aDigits_base[idx + a_offset];
            xx_b = aDigits_base[idx2 + a_offset];
            xy_a = aDigits_base[idx + a_offset];
            xy_b = bDigits_base[idx2 + b_offset];
            yy_a = bDigits_base[idx + b_offset];
            yy_b = bDigits_base[idx2 + b_offset];
        }
        uint64_t p;
        p = xx_a * xx_b; xx_low += p; if (xx_low < p) xx_high += 1;
        p = xy_a * xy_b; xy_low += p; if (xy_low < p) xy_high += 1;
        p = yy_a * yy_b; yy_low += p; if (yy_low < p) yy_high += 1;

        DebugMultiplyIncrement<SharkFloatParams>(debugMultiplyCounts, grid, block, 3);
    }
}

__device__ __forceinline__ int warpMinI32(int v) {
    unsigned m = __activemask();
    v = min(v, __shfl_down_sync(m, v, 16));
    v = min(v, __shfl_down_sync(m, v, 8));
    v = min(v, __shfl_down_sync(m, v, 4));
    v = min(v, __shfl_down_sync(m, v, 2));
    v = min(v, __shfl_down_sync(m, v, 1));
    return __shfl_sync(m, v, 0);
}

#include "MultiplyHelperVectorLoads.cuh"
#include "MultiplyHelperGlobalUnaligned.cuh"

template<
    class SharkFloatParams,
    int BatchSize,
    ConditionalAccess UseConditionalAccess,
    int RecursionDepth,
    int ExecutionBlockBase,
    int ExecutionNumBlocks>
__device__ SharkForceInlineReleaseOnly static void
ProcessConvolutionBatch (
    cg::grid_group &grid,
    cg::thread_block &block,
    DebugMultiplyCount<SharkFloatParams> *debugMultiplyCounts,
    const int RelativeBlockIndex,
    const int outerIteration,
    const int k,
    const int total_k,
    const int i_start,
    const int i_end,
    const int n_limit,
    const uint32_t *aDigits_base,
    const uint32_t *bDigits_base,
    const int a_offset,
    const int b_offset,
    uint64_t &xx_sum_low,
    uint64_t &xx_sum_high,
    uint64_t &xy_sum_low,
    uint64_t &xy_sum_high,
    uint64_t &yy_sum_low,
    uint64_t &yy_sum_high,
    uint32_t *shared_data,
    const uint32_t *x_diff_abs = nullptr,
    const uint32_t *y_diff_abs = nullptr) {


    // ---------------- scalar-only fast path when BatchSize==1 ----------------
    if constexpr (BatchSize == 1) {
        accumulate_scalar_span<SharkFloatParams, UseConditionalAccess>(
            grid,
            block,
            debugMultiplyCounts,
            i_start, i_end, k,
            aDigits_base, bDigits_base, a_offset, b_offset,
            x_diff_abs, y_diff_abs,
            xx_sum_low, xx_sum_high,
            xy_sum_low, xy_sum_high,
            yy_sum_low, yy_sum_high);
        return;
    }

    // ---------------- main path (BatchSize>=2; vectorized steady-state when 4) ----------------
    int i = i_start;

    // If all data already in shared (HpShark::LoadAllInShared), we can skip alignment steering
    // and simply use the 4-stage register pipeline with scalar loads from shared.
    if constexpr (HpShark::LoadAllInShared) {
        // Prologue: scalar until we have >= BatchSize items
        const int pro_end = min(i_end, i + ((i_end - i + 1) % BatchSize) - 1);
        if (pro_end >= i) {
            accumulate_scalar_span<SharkFloatParams, UseConditionalAccess>(
                grid,
                block,
                debugMultiplyCounts,
                i, pro_end, k,
                aDigits_base, bDigits_base, a_offset, b_offset,
                x_diff_abs, y_diff_abs,
                xx_sum_low, xx_sum_high, xy_sum_low, xy_sum_high, yy_sum_low, yy_sum_high);
            i = pro_end + 1;
        }

        // Steady-state: consume as many full batches as possible (scalar loads from shared)
        while (i + BatchSize - 1 <= i_end) {
            const int remaining = i_end - i + 1;
            const int nFull = remaining / BatchSize;    // >=1

            // Four fixed buffers (compile-time indices)
            uint32_t ax0[BatchSize], bx0[BatchSize], ay0[BatchSize], by0[BatchSize], cy0[BatchSize], dy0[BatchSize];
            uint32_t ax1[BatchSize], bx1[BatchSize], ay1[BatchSize], by1[BatchSize], cy1[BatchSize], dy1[BatchSize];
            uint32_t ax2[BatchSize], bx2[BatchSize], ay2[BatchSize], by2[BatchSize], cy2[BatchSize], dy2[BatchSize];
            uint32_t ax3[BatchSize], bx3[BatchSize], ay3[BatchSize], by3[BatchSize], cy3[BatchSize], dy3[BatchSize];

            // scalar loaders (from shared inputs x_diff_abs/y_diff_abs or a/b + offsets)
            auto load_buf_shared = [&](int base_i, uint32_t(&ax)[BatchSize], uint32_t(&bx)[BatchSize],
                uint32_t(&ay)[BatchSize], uint32_t(&by)[BatchSize],
                uint32_t(&cy)[BatchSize], uint32_t(&dy)[BatchSize]) {
#pragma unroll
                    for (int j = 0; j < BatchSize; ++j) {
                        const int idx = base_i + j;
                        const int idx2 = k - idx;
                        if constexpr (UseConditionalAccess == ConditionalAccess::True) {
                            ax[j] = x_diff_abs[idx];
                            bx[j] = x_diff_abs[idx2];
                            ay[j] = x_diff_abs[idx];
                            by[j] = y_diff_abs[idx2];
                            cy[j] = y_diff_abs[idx];
                            dy[j] = y_diff_abs[idx2];
                        } else {
                            ax[j] = aDigits_base[idx + a_offset];
                            bx[j] = aDigits_base[idx2 + a_offset];
                            ay[j] = aDigits_base[idx + a_offset];
                            by[j] = bDigits_base[idx2 + b_offset];
                            cy[j] = bDigits_base[idx + b_offset];
                            dy[j] = bDigits_base[idx2 + b_offset];
                        }
                    }
                };

            auto compute_buf = [&](uint32_t(&ax)[BatchSize], uint32_t(&bx)[BatchSize],
                uint32_t(&ay)[BatchSize], uint32_t(&by)[BatchSize],
                uint32_t(&cy)[BatchSize], uint32_t(&dy)[BatchSize]) {
#pragma unroll
                    for (int j = 0; j < BatchSize; ++j) {
                        const uint64_t xx_a = (uint64_t)ax[j], xx_b = (uint64_t)bx[j];
                        const uint64_t xy_a = (uint64_t)ay[j], xy_b = (uint64_t)by[j];
                        const uint64_t yy_a = (uint64_t)cy[j], yy_b = (uint64_t)dy[j];
                        uint64_t p;
                        p = xx_a * xx_b; xx_sum_low += p; if (xx_sum_low < p) xx_sum_high += 1;
                        p = xy_a * xy_b; xy_sum_low += p; if (xy_sum_low < p) xy_sum_high += 1;
                        p = yy_a * yy_b; yy_sum_low += p; if (yy_sum_low < p) yy_sum_high += 1;

                        DebugMultiplyIncrement<SharkFloatParams>(debugMultiplyCounts, grid, block, 3);
                    }
                };

            if (nFull == 1) {
                load_buf_shared(i, ax0, bx0, ay0, by0, cy0, dy0);
                compute_buf(ax0, bx0, ay0, by0, cy0, dy0);
                i += BatchSize;
            } else if (nFull == 2) {
                load_buf_shared(i, ax0, bx0, ay0, by0, cy0, dy0);
                load_buf_shared(i + BatchSize, ax1, bx1, ay1, by1, cy1, dy1);
                compute_buf(ax0, bx0, ay0, by0, cy0, dy0);
                compute_buf(ax1, bx1, ay1, by1, cy1, dy1);
                i += 2 * BatchSize;
            } else if (nFull == 3) {
                load_buf_shared(i, ax0, bx0, ay0, by0, cy0, dy0);
                load_buf_shared(i + BatchSize, ax1, bx1, ay1, by1, cy1, dy1);
                load_buf_shared(i + 2 * BatchSize, ax2, bx2, ay2, by2, cy2, dy2);
                compute_buf(ax0, bx0, ay0, by0, cy0, dy0);
                compute_buf(ax1, bx1, ay1, by1, cy1, dy1);
                compute_buf(ax2, bx2, ay2, by2, cy2, dy2);
                i += 3 * BatchSize;
            } else {
                // 4-stage steady-state
                load_buf_shared(i + 0 * BatchSize, ax0, bx0, ay0, by0, cy0, dy0);
                load_buf_shared(i + 1 * BatchSize, ax1, bx1, ay1, by1, cy1, dy1);
                load_buf_shared(i + 2 * BatchSize, ax2, bx2, ay2, by2, cy2, dy2);
                load_buf_shared(i + 3 * BatchSize, ax3, bx3, ay3, by3, cy3, dy3);

                int next = 4;
                const int nf = nFull;
                const int rounds = nf / 4;
                const int rem = nf % 4;

                for (int r = 0; r < rounds; ++r) {
                    compute_buf(ax0, bx0, ay0, by0, cy0, dy0);
                    if (next < nf) load_buf_shared(i + next * BatchSize, ax0, bx0, ay0, by0, cy0, dy0), ++next;

                    compute_buf(ax1, bx1, ay1, by1, cy1, dy1);
                    if (next < nf) load_buf_shared(i + next * BatchSize, ax1, bx1, ay1, by1, cy1, dy1), ++next;

                    compute_buf(ax2, bx2, ay2, by2, cy2, dy2);
                    if (next < nf) load_buf_shared(i + next * BatchSize, ax2, bx2, ay2, by2, cy2, dy2), ++next;

                    compute_buf(ax3, bx3, ay3, by3, cy3, dy3);
                    if (next < nf) load_buf_shared(i + next * BatchSize, ax3, bx3, ay3, by3, cy3, dy3), ++next;
                }
                if (rem >= 1) compute_buf(ax0, bx0, ay0, by0, cy0, dy0);
                if (rem >= 2) compute_buf(ax1, bx1, ay1, by1, cy1, dy1);
                if (rem >= 3) compute_buf(ax2, bx2, ay2, by2, cy2, dy2);

                i += nf * BatchSize;
            }
        }

        // Epilogue: scalar tail
        if (i <= i_end) {
            accumulate_scalar_span<SharkFloatParams, UseConditionalAccess>(
                grid,
                block,
                debugMultiplyCounts,
                i, i_end, k,
                aDigits_base, bDigits_base, a_offset, b_offset,
                x_diff_abs, y_diff_abs,
                xx_sum_low, xx_sum_high, xy_sum_low, xy_sum_high, yy_sum_low, yy_sum_high);
        }
        return;
    }

    // ---------------- global-memory path with alignment steering (BatchSize==4) ----------------
    if constexpr (
        SharkInnerLoopOption == HpShark::InnerLoopOption::TryUnalignedLoads2 ||
        SharkInnerLoopOption == HpShark::InnerLoopOption::TryUnalignedLoads2Shared) {

        ProcessConvolutionDirectLoad_Unaligned2<
            SharkFloatParams,
            16,
            UseConditionalAccess,
            RecursionDepth,
            ExecutionBlockBase,
            ExecutionNumBlocks>(
                grid,
                block,
                debugMultiplyCounts,
                i, i_start, i_end, k,
                aDigits_base, bDigits_base,
                a_offset, b_offset,
                xx_sum_low, xx_sum_high,
                xy_sum_low, xy_sum_high,
                yy_sum_low, yy_sum_high,
                x_diff_abs, y_diff_abs);
        return;
    }

    if constexpr (SharkInnerLoopOption == HpShark::InnerLoopOption::TryVectorLoads) {
        //
        // This implementation uses BatchSize==8 and vector loads.  It
        // imperfectly handles weird cases where digits don't match number of
        // threads and some weird digit lengths, but should work for e.g.
        // 108 blocks, 128 threads per block, and 7776 total digits.
        // 
        // Performance experiments are showing this approach is worse than direct
        // unaligned load-from-global-to-registers and it's far more complex.
        // It seems like we should be able to get a win in here but the logic for
        // dealing with the unaligned cases probably is what's slowing it down.
        //

        ProcessConvolutionDirectLoad_BS8_FwdAligned<
            SharkFloatParams,
            8,
            UseConditionalAccess,
            RecursionDepth,
            ExecutionBlockBase,
            ExecutionNumBlocks>(
                grid,
                block,
                debugMultiplyCounts,
                i, i_end, k,
                aDigits_base, bDigits_base,
                a_offset, b_offset,
                xx_sum_low, xx_sum_high,
                xy_sum_low, xy_sum_high,
                yy_sum_low, yy_sum_high,
                x_diff_abs, y_diff_abs);
        return;
    }

    if constexpr (SharkInnerLoopOption == HpShark::InnerLoopOption::TryUnalignedLoads) {
        //
        // This implementation uses BatchSize==8 and unaligned loads.
        //

        ProcessConvolutionDirectLoad_Unaligned<SharkFloatParams, 8, UseConditionalAccess,
            RecursionDepth, ExecutionBlockBase, ExecutionNumBlocks>(
                grid,
                block,
                debugMultiplyCounts,
                i, i_end, k,
                aDigits_base, bDigits_base,
                a_offset, b_offset,
                xx_sum_low, xx_sum_high,
                xy_sum_low, xy_sum_high,
                yy_sum_low, yy_sum_high,
                x_diff_abs, y_diff_abs);
        return;
    }
}


template<
    class SharkFloatParams,
    int RecursionDepth,
    int CallIndex,
    int NewN,
    int n1,
    int n2,
    int ExecutionBlockBase,
    int ExecutionNumBlocks,
    int NewNumBlocks,
    int TempBase>
__device__ SharkForceInlineReleaseOnly static void
MultiplyDigitsOnly(
    uint32_t *SharkRestrict shared_data,
    const HpSharkFloat<SharkFloatParams> *SharkRestrict A,
    const HpSharkFloat<SharkFloatParams> *SharkRestrict B,
    const uint32_t *SharkRestrict aDigits,
    const uint32_t *SharkRestrict bDigits,
    uint32_t *SharkRestrict x_diff_abs,
    uint32_t *SharkRestrict y_diff_abs,
    uint64_t *SharkRestrict final128XX,
    uint64_t *SharkRestrict final128XY,
    uint64_t *SharkRestrict final128YY,
    cg::grid_group &grid,
    cg::thread_block &block,
    uint64_t *SharkRestrict tempProducts) {

    if ((ExecutionBlockBase > 0 && block.group_index().x < ExecutionBlockBase) ||
        block.group_index().x >= ExecutionBlockBase + ExecutionNumBlocks) {

        return;
    }

    DefineTempProductsOffsets(CallIndex);
    TempProductsGlobals(TempBase, CallIndex);

    auto *SharkRestrict debugMultiplyCounts = reinterpret_cast<DebugMultiplyCount<SharkFloatParams>*>(&tempProducts[Multiplies_offset]);
    auto *SharkRestrict debugStates = reinterpret_cast<DebugState<SharkFloatParams>*>(&tempProducts[Checksum_offset]);

    constexpr auto MaxHalfN = std::max(n1, n2);
    constexpr int total_k = MaxHalfN * 2 - 1; // Total number of k values
    constexpr bool UseConvolutionBool =
        (NewNumBlocks <= std::max(SharkFloatParams::GlobalNumBlocks / SharkFloatParams::ConvolutionLimit, 1) ||
        (NewNumBlocks % 3 != 0));
    constexpr UseConvolution UseConvolutionHere = UseConvolutionBool ? UseConvolution::Yes : UseConvolution::No;
    constexpr bool UseParallelSubtract = true;

    using DebugState = DebugState<SharkFloatParams>;

    const RecordIt record =
        (block.thread_index().x == 0 && block.group_index().x == ExecutionBlockBase) ?
        RecordIt::Yes :
        RecordIt::No;

    if constexpr (HpShark::DebugChecksums) {
        grid.sync();

        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Invalid>(
            record, debugStates, grid, block);
        StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::ADigits, uint32_t>(
            record, UseConvolutionHere, debugStates, grid, block, aDigits, NewN);
        StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::BDigits, uint32_t>(
            record, UseConvolutionHere, debugStates, grid, block, bDigits, NewN);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::CDigits>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::DDigits>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::EDigits>(
            record, debugStates, grid, block);

        grid.sync();
    }

    auto *SharkRestrict Z0_OutDigitsXX = &tempProducts[Z0_offsetXX];
    auto *SharkRestrict Z0_OutDigitsXY = &tempProducts[Z0_offsetXY];
    auto *SharkRestrict Z0_OutDigitsYY = &tempProducts[Z0_offsetYY];

    auto *SharkRestrict Z1_temp_digitsXX = &tempProducts[Z1_temp_offsetXX];
    auto *SharkRestrict Z1_temp_digitsXY = &tempProducts[Z1_temp_offsetXY];
    auto *SharkRestrict Z1_temp_digitsYY = &tempProducts[Z1_temp_offsetYY];

    auto *SharkRestrict Z2_OutDigitsXX = &tempProducts[Z2_offsetXX];
    auto *SharkRestrict Z2_OutDigitsXY = &tempProducts[Z2_offsetXY];
    auto *SharkRestrict Z2_OutDigitsYY = &tempProducts[Z2_offsetYY];

    // Arrays to hold the absolute differences (size n)
    auto *SharkRestrict global_x_diff_abs = reinterpret_cast<uint32_t *>(&tempProducts[XDiff_offset]);
    auto *SharkRestrict global_y_diff_abs = reinterpret_cast<uint32_t *>(&tempProducts[YDiff_offset]);

    // ---- Compute Differences x_diff = A1 - A0 and y_diff = B1 - B0 ----

    DefineExtraDefinitions();

    int x_diff_sign = 0; // 0 if positive, 1 if negative
    int y_diff_sign = 0; // 0 if positive, 1 if negative

    // Compute x_diff_abs and x_diff_sign
    auto *SharkRestrict subtractionBorrows = reinterpret_cast<uint32_t *>(&tempProducts[SubtractionOffset1]);
    auto *SharkRestrict subtractionBorrows2 = reinterpret_cast<uint32_t *>(&tempProducts[SubtractionOffset2]);
    auto *SharkRestrict subtractionBorrows3 = reinterpret_cast<uint32_t *>(&tempProducts[SubtractionOffset3]);
    auto *SharkRestrict subtractionBorrows4 = reinterpret_cast<uint32_t *>(&tempProducts[SubtractionOffset4]);
    auto *SharkRestrict globalBorrowAny = reinterpret_cast<uint32_t *>(&tempProducts[BorrowGlobalOffset]);
    auto *SharkRestrict globalBlockBorrow1 = reinterpret_cast<uint32_t *>(&tempProducts[BorrowBlockLevelOffset1]);
    auto *SharkRestrict globalBlockBorrow2 = reinterpret_cast<uint32_t *>(&tempProducts[BorrowBlockLevelOffset2]);

    const auto *SharkRestrict a_high = aDigits + n1;
    const auto *SharkRestrict b_high = bDigits + n1;
    const auto *SharkRestrict a_low = aDigits;
    const auto *SharkRestrict b_low = bDigits;

    if constexpr (HpShark::DebugChecksums) {
        grid.sync();

        StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::AHalfHigh>(
            record, UseConvolutionHere, debugStates, grid, block, a_high, n2);
        StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::AHalfLow>(
            record, UseConvolutionHere, debugStates, grid, block, a_low, n1);
        StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::BHalfHigh>(
            record, UseConvolutionHere, debugStates, grid, block, b_high, n2);
        StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::BHalfLow>(
            record, UseConvolutionHere, debugStates, grid, block, b_low, n1);

        grid.sync();
    }

    if constexpr (!SharkFloatParams::DisableSubtraction) {
        if constexpr (UseParallelSubtract) {
            int x_compare = CompareDigits<n2, n1>(a_high, a_low);
            int y_compare = CompareDigits<n2, n1>(b_high, b_low);

            if (x_compare >= 0 && y_compare >= 0) {
                x_diff_sign = 0;
                y_diff_sign = 0;
                SubtractDigitsParallelImproved3<
                    SharkFloatParams,
                    n2,
                    n1,
                    n2,
                    n1,
                    ExecutionBlockBase,
                    ExecutionNumBlocks>(
                        x_diff_abs,
                        y_diff_abs,
                        a_high,
                        a_low,
                        b_high,
                        b_low,
                        subtractionBorrows,
                        subtractionBorrows2,
                        subtractionBorrows3,
                        subtractionBorrows4,
                        globalBlockBorrow1,
                        globalBlockBorrow2,
                        global_x_diff_abs,
                        global_y_diff_abs,
                        globalBorrowAny,
                        grid,
                        block);
            } else if (x_compare < 0 && y_compare < 0) {
                x_diff_sign = 1;
                y_diff_sign = 1;
                SubtractDigitsParallelImproved3<
                    SharkFloatParams,
                    n1,
                    n2,
                    n1,
                    n2,
                    ExecutionBlockBase,
                    ExecutionNumBlocks>(
                        x_diff_abs,
                        y_diff_abs,
                        a_low,
                        a_high,
                        b_low,
                        b_high,
                        subtractionBorrows,
                        subtractionBorrows2,
                        subtractionBorrows3,
                        subtractionBorrows4,
                        globalBlockBorrow1,
                        globalBlockBorrow2,
                        global_x_diff_abs,
                        global_y_diff_abs,
                        globalBorrowAny,
                        grid,
                        block);
            } else if (x_compare >= 0 && y_compare < 0) {
                x_diff_sign = 0;
                y_diff_sign = 1;
                SubtractDigitsParallelImproved3<
                    SharkFloatParams,
                    n2,
                    n1,
                    n1,
                    n2,
                    ExecutionBlockBase,
                    ExecutionNumBlocks>(
                        x_diff_abs,
                        y_diff_abs,
                        a_high,
                        a_low,
                        b_low,
                        b_high,
                        subtractionBorrows,
                        subtractionBorrows2,
                        subtractionBorrows3,
                        subtractionBorrows4,
                        globalBlockBorrow1,
                        globalBlockBorrow2,
                        global_x_diff_abs,
                        global_y_diff_abs,
                        globalBorrowAny,
                        grid,
                        block);
            } else {
                x_diff_sign = 1;
                y_diff_sign = 0;
                SubtractDigitsParallelImproved3<
                    SharkFloatParams,
                    n1,
                    n2,
                    n2,
                    n1,
                    ExecutionBlockBase,
                    ExecutionNumBlocks>(
                        x_diff_abs,
                        y_diff_abs,
                        a_low,
                        a_high,
                        b_high,
                        b_low,
                        subtractionBorrows,
                        subtractionBorrows2,
                        subtractionBorrows3,
                        subtractionBorrows4,
                        globalBlockBorrow1,
                        globalBlockBorrow2,
                        global_x_diff_abs,
                        global_y_diff_abs,
                        globalBorrowAny,
                        grid,
                        block);
            }
        } else {
            if (block.thread_index().x == 0 && block.group_index().x == ExecutionBlockBase) {
                int x_compare = CompareDigits<n1, n2>(a_high, a_low);

                if (x_compare >= 0) {
                    x_diff_sign = 0;
                    SubtractDigitsSerial<n2, n1>(a_high, a_low, global_x_diff_abs); // x_diff = A1 - A0
                } else {
                    x_diff_sign = 1;
                    SubtractDigitsSerial<n1, n2>(a_low, a_high, global_x_diff_abs); // x_diff = A0 - A1
                }

                // Compute y_diff_abs and y_diff_sign
                int y_compare = CompareDigits<n1, n2>(b_high, b_low);
                if (y_compare >= 0) {
                    y_diff_sign = 0;
                    SubtractDigitsSerial<n2, n1>(b_high, b_low, global_y_diff_abs); // y_diff = B1 - B0
                } else {
                    y_diff_sign = 1;
                    SubtractDigitsSerial<n1, n2>(b_low, b_high, global_y_diff_abs); // y_diff = B0 - B1
                }
            }
        }
    }

    grid.sync();

    if constexpr (HpShark::DebugChecksums) {
        StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::XDiff>(
            record, UseConvolutionHere, debugStates, grid, block, global_x_diff_abs, MaxHalfN);
        StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::YDiff>(
            record, UseConvolutionHere, debugStates, grid, block, global_y_diff_abs, MaxHalfN);

        grid.sync();
    }

    constexpr auto SubNewNRoundUp = (NewN + 1) / 2;
    constexpr auto SubNewN2a = SubNewNRoundUp / 2;
    constexpr auto SubNewN1a = SubNewNRoundUp - SubNewN2a;   /* n1 is larger or same */

    constexpr auto SubRemainingNewN = NewN - SubNewNRoundUp;
    constexpr auto SubNewN2b = SubRemainingNewN / 2;
    constexpr auto SubNewN1b = SubRemainingNewN - SubNewN2b;   /* n1 is larger or same */

    // Determine the sign of Z1_temp
    // int z1_sign = x_diff_sign ^ y_diff_sign;

    const int z1_signXX = (x_diff_sign ^ x_diff_sign) ? 1 : 0; // TODO obviously can be simplified
    const int z1_signXY = (x_diff_sign ^ y_diff_sign) ? 1 : 0;
    const int z1_signYY = (y_diff_sign ^ y_diff_sign) ? 1 : 0;

    constexpr auto FinalZ0Size =
        (UseConvolutionHere == UseConvolution::Yes) ?
        (total_k * 2) :
        (SubNewNRoundUp * 2 * 2);
    constexpr auto FinalZ2Size =
        (UseConvolutionHere == UseConvolution::Yes) ?
        (total_k * 2) :
        (SubRemainingNewN * 2 * 2);
    constexpr auto FinalZ1TempSize =
        (UseConvolutionHere == UseConvolution::Yes) ?
        (total_k * 2) :
        (SubNewNRoundUp * 2 * 2);

    if constexpr (UseConvolutionHere == UseConvolution::Yes) {
        // Replace A and B in shared memory with their absolute differences
        if constexpr (HpShark::LoadAllInShared) {
            cg::memcpy_async(block, const_cast<uint32_t *>(x_diff_abs), global_x_diff_abs, sizeof(uint32_t) * MaxHalfN);
            cg::memcpy_async(block, const_cast<uint32_t *>(y_diff_abs), global_y_diff_abs, sizeof(uint32_t) * MaxHalfN);
        }

        const int tid = RelativeBlockIndex * block.dim_threads().x + block.thread_index().x;
        const int stride = block.dim_threads().x * ExecutionNumBlocks;

        if constexpr (HpShark::LoadAllInShared) {
            // Wait for the first batch of A to be loaded
            cg::wait(block);
        }

        constexpr int outerIteration = 0;
        for (int idx = tid; idx < total_k * 3; idx += stride) {

            // Check if idx < total_k => handle Z0, else handle Z2
            if (idx < total_k) {
                // Z0 partial sums
                const int k_base = idx;
                int k = k_base; // shift to [0..total_k-1]
                uint64_t xx_sum_low = 0ULL, xx_sum_high = 0ULL;
                uint64_t xy_sum_low = 0ULL, xy_sum_high = 0ULL;
                uint64_t yy_sum_low = 0ULL, yy_sum_high = 0ULL;

                int i_start = (k < n1) ? 0 : (k - (n1 - 1));
                int i_end = (k < n1) ? k : (n1 - 1);

                ProcessConvolutionBatch<
                    SharkFloatParams,
                    SharkKaratsubaBatchSize,
                    ConditionalAccess::False,
                    RecursionDepth,
                    ExecutionBlockBase,
                    ExecutionNumBlocks>(

                    grid,
                    block,
                    debugMultiplyCounts,
                    RelativeBlockIndex,
                    outerIteration,
                    k,
                    total_k,
                    i_start,
                    i_end,
                    n1,
                    aDigits,
                    bDigits,
                    0,
                    0,  // Z0 uses base arrays with no offset
                    xx_sum_low,
                    xx_sum_high,
                    xy_sum_low,
                    xy_sum_high,
                    yy_sum_low,
                    yy_sum_high,
                    shared_data);

                int out_idx = k * 2;
                Z0_OutDigitsXX[out_idx] = xx_sum_low;
                Z0_OutDigitsXX[out_idx + 1] = xx_sum_high;
                Z0_OutDigitsXY[out_idx] = xy_sum_low;
                Z0_OutDigitsXY[out_idx + 1] = xy_sum_high;
                Z0_OutDigitsYY[out_idx] = yy_sum_low;
                Z0_OutDigitsYY[out_idx + 1] = yy_sum_high;
            } else if (idx < 2 * total_k) {
                // Z2 partial sums
                const int k_base = idx - total_k; // shift to [0..total_k-1]
                //int k = (k_base + total_k / 3) % total_k;
                int k = k_base;
                uint64_t xx_sum_low = 0ULL, xx_sum_high = 0ULL;
                uint64_t xy_sum_low = 0ULL, xy_sum_high = 0ULL;
                uint64_t yy_sum_low = 0ULL, yy_sum_high = 0ULL;

                int i_start = (k < n2) ? 0 : (k - (n2 - 1));
                int i_end = (k < n2) ? k : (n2 - 1);

                ProcessConvolutionBatch<
                    SharkFloatParams,
                    SharkKaratsubaBatchSize,
                    ConditionalAccess::False,
                    RecursionDepth,
                    ExecutionBlockBase,
                    ExecutionNumBlocks>(

                    grid,
                    block,
                    debugMultiplyCounts,
                    RelativeBlockIndex,
                    outerIteration,
                    k,
                    total_k,
                    i_start,
                    i_end,
                    n2,
                    aDigits,
                    bDigits,
                    n1,
                    n1,  // Z2 uses arrays with n1 offset
                    xx_sum_low,
                    xx_sum_high,
                    xy_sum_low,
                    xy_sum_high,
                    yy_sum_low,
                    yy_sum_high,
                    shared_data);

                int out_idx = k * 2;
                Z2_OutDigitsXX[out_idx] = xx_sum_low;
                Z2_OutDigitsXX[out_idx + 1] = xx_sum_high;
                Z2_OutDigitsXY[out_idx] = xy_sum_low;
                Z2_OutDigitsXY[out_idx + 1] = xy_sum_high;
                Z2_OutDigitsYY[out_idx] = yy_sum_low;
                Z2_OutDigitsYY[out_idx + 1] = yy_sum_high;
            } else {
                const int k_base = idx - 2 * total_k; // shift to [0..total_k-1]
                //int k = (k_base + 2 * total_k / 3) % total_k;
                int k = k_base;
                uint64_t xx_sum_low = 0ULL, xx_sum_high = 0ULL;
                uint64_t xy_sum_low = 0ULL, xy_sum_high = 0ULL;
                uint64_t yy_sum_low = 0ULL, yy_sum_high = 0ULL;

                int i_start = (k < MaxHalfN) ? 0 : (k - (MaxHalfN - 1));
                int i_end = (k < MaxHalfN) ? k : (MaxHalfN - 1);

                ProcessConvolutionBatch<
                    SharkFloatParams,
                    SharkKaratsubaBatchSize,
                    ConditionalAccess::True,
                    RecursionDepth,
                    ExecutionBlockBase,
                    ExecutionNumBlocks>(

                    grid,
                    block,
                    debugMultiplyCounts,
                    RelativeBlockIndex,
                    outerIteration,
                    k,
                    total_k,
                    i_start,
                    i_end,
                    MaxHalfN,
                    nullptr,
                    nullptr,
                    0,
                    0,  // Not used for Z1_temp
                    xx_sum_low,
                    xx_sum_high,
                    xy_sum_low,
                    xy_sum_high,
                    yy_sum_low,
                    yy_sum_high,
                    shared_data,
                    HpShark::LoadAllInShared ? x_diff_abs : global_x_diff_abs,
                    HpShark::LoadAllInShared ? y_diff_abs : global_y_diff_abs);

                int out_idx = k * 2;
                Z1_temp_digitsXX[out_idx] = xx_sum_low;
                Z1_temp_digitsXX[out_idx + 1] = xx_sum_high;
                Z1_temp_digitsXY[out_idx] = xy_sum_low;
                Z1_temp_digitsXY[out_idx + 1] = xy_sum_high;
                Z1_temp_digitsYY[out_idx] = yy_sum_low;
                Z1_temp_digitsYY[out_idx + 1] = yy_sum_high;
            }
        }
    } else {
        static_assert(RecursionDepth <= 5, "Unexpected recursion depth");

        MultiplyDigitsOnly<
            SharkFloatParams,
            RecursionDepth + 1,
            CallIndex * 3 - 1,
            SubNewNRoundUp,
            SubNewN1a,
            SubNewN2a,
            ExecutionBlockBase,
            ExecutionNumBlocks / 3,
            NewNumBlocks / 3,
            TempBase>(
            shared_data,
            A,
            B,
            aDigits,
            bDigits,
            x_diff_abs,
            y_diff_abs,
            Z0_OutDigitsXX,
            Z0_OutDigitsXY,
            Z0_OutDigitsYY,
            grid,
            block,
            tempProducts);

        MultiplyDigitsOnly<
            SharkFloatParams,
            RecursionDepth + 1,
            CallIndex * 3,
            SubRemainingNewN,
            SubNewN1b,
            SubNewN2b,
            ExecutionBlockBase + ExecutionNumBlocks / 3,
            ExecutionNumBlocks / 3,
            NewNumBlocks / 3,
            TempBase>(
            shared_data,
            A,
            B,
            aDigits + n1,
            bDigits + n1,
            x_diff_abs,
            y_diff_abs,
            Z2_OutDigitsXX,
            Z2_OutDigitsXY,
            Z2_OutDigitsYY,
            grid,
            block,
            tempProducts);

        //grid.sync();

        {
            constexpr auto NewExecutionBlockBase = ExecutionBlockBase + 2 * ExecutionNumBlocks / 3;
            constexpr auto NewExecutionNumBlocks = ExecutionNumBlocks / 3;

            const bool ExecuteAtAll =
                !((NewExecutionBlockBase > 0 && block.group_index().x < NewExecutionBlockBase) ||
                    block.group_index().x >= NewExecutionBlockBase + NewExecutionNumBlocks);
            constexpr auto MaxSubNewN = std::max(SubNewN1a, SubNewN2a);

            if (ExecuteAtAll) {
                // Replace A and B in shared memory with their absolute differences
                if constexpr (HpShark::LoadAllInShared) {
                    cg::memcpy_async(block,
                        const_cast<uint32_t *>(aDigits),
                        global_x_diff_abs,
                        sizeof(uint32_t) * MaxHalfN);
                    cg::memcpy_async(block,
                        const_cast<uint32_t *>(bDigits),
                        global_y_diff_abs,
                        sizeof(uint32_t) * MaxHalfN);
                    cg::wait(block);
                }

                MultiplyDigitsOnly<
                    SharkFloatParams,
                    RecursionDepth + 1,
                    CallIndex * 3 + 1,
                    SubNewNRoundUp,
                    SubNewN1a,
                    SubNewN2a,
                    NewExecutionBlockBase,
                    NewExecutionNumBlocks,
                    NewNumBlocks / 3,
                    TempBase>(
                        shared_data,
                        A,
                        B,
                        HpShark::LoadAllInShared ? aDigits : global_x_diff_abs,
                        HpShark::LoadAllInShared ? bDigits : global_y_diff_abs,
                        x_diff_abs,
                        y_diff_abs,
                        Z1_temp_digitsXX,
                        Z1_temp_digitsXY,
                        Z1_temp_digitsYY,
                        grid,
                        block,
                        tempProducts);

                if constexpr (HpShark::LoadAllInShared) {
                    cg::memcpy_async(block,
                        const_cast<uint32_t *>(aDigits),
                        A->Digits,
                        sizeof(uint32_t) * SharkFloatParams::GlobalNumUint32);
                    cg::memcpy_async(block,
                        const_cast<uint32_t *>(bDigits),
                        B->Digits,
                        sizeof(uint32_t) * SharkFloatParams::GlobalNumUint32);
                    cg::wait(block);
                }
            }
        }
    }

    grid.sync();

    if constexpr (HpShark::DebugChecksums) {
        StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z0XX>(
            record, UseConvolutionHere, debugStates, grid, block, Z0_OutDigitsXX, FinalZ0Size);
        StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z0XY>(
            record, UseConvolutionHere, debugStates, grid, block, Z0_OutDigitsXY, FinalZ0Size);
        StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z0YY>(
            record, UseConvolutionHere, debugStates, grid, block, Z0_OutDigitsYY, FinalZ0Size);

        StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z2XX>(
            record, UseConvolutionHere, debugStates, grid, block, Z2_OutDigitsXX, FinalZ2Size);
        StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z2XY>(
            record, UseConvolutionHere, debugStates, grid, block, Z2_OutDigitsXY, FinalZ2Size);
        StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z2YY>(
            record, UseConvolutionHere, debugStates, grid, block, Z2_OutDigitsYY, FinalZ2Size);

        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z2_Perm1>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z2_Perm2>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z2_Perm3>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z2_Perm4>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z2_Perm5>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z2_Perm6>(
            record, debugStates, grid, block);

        StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z1_offsetXX>(
            record, UseConvolutionHere, debugStates, grid, block, Z1_temp_digitsXX, FinalZ1TempSize);
        StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z1_offsetXY>(
            record, UseConvolutionHere, debugStates, grid, block, Z1_temp_digitsXY, FinalZ1TempSize);
        StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z1_offsetYY>(
            record, UseConvolutionHere, debugStates, grid, block, Z1_temp_digitsYY, FinalZ1TempSize);

        grid.sync();
    }

    auto *SharkRestrict Z1_digitsXX = &tempProducts[Z1_offsetXX];
    auto *SharkRestrict Z1_digitsXY = &tempProducts[Z1_offsetXY];
    auto *SharkRestrict Z1_digitsYY = &tempProducts[Z1_offsetYY];

    if constexpr (!SharkFloatParams::DisableAllAdditions) {

        // After computing Z1_temp (Z1'), we now form Z1 directly:
        // If z1_sign == 0: Z1 = Z2 + Z0 - Z1_temp
        // If z1_sign == 1: Z1 = Z2 + Z0 + Z1_temp

        const int tid = RelativeBlockIndex * block.dim_threads().x + block.thread_index().x;
        const int stride = block.dim_threads().x * ExecutionNumBlocks;

        for (int i = tid; i < total_k; i += stride) {
            // Retrieve Z0
            int z0_idx = i * 2;
            const uint64_t xx_z0_low = Z0_OutDigitsXX[z0_idx];
            const uint64_t xx_z0_high = Z0_OutDigitsXX[z0_idx + 1];

            const uint64_t xy_z0_low = Z0_OutDigitsXY[z0_idx];
            const uint64_t xy_z0_high = Z0_OutDigitsXY[z0_idx + 1];

            const uint64_t yy_z0_low = Z0_OutDigitsYY[z0_idx];
            const uint64_t yy_z0_high = Z0_OutDigitsYY[z0_idx + 1];

            // Retrieve Z2
            int z2_idx = i * 2;
            const uint64_t xx_z2_low = z2_idx < FinalZ2Size ? Z2_OutDigitsXX[z2_idx] : 0;
            const uint64_t xx_z2_high = z2_idx < FinalZ2Size ? Z2_OutDigitsXX[z2_idx + 1] : 0;

            const uint64_t xy_z2_low = z2_idx < FinalZ2Size ? Z2_OutDigitsXY[z2_idx] : 0;
            const uint64_t xy_z2_high = z2_idx < FinalZ2Size ? Z2_OutDigitsXY[z2_idx + 1] : 0;

            const uint64_t yy_z2_low = z2_idx < FinalZ2Size ? Z2_OutDigitsYY[z2_idx] : 0;
            const uint64_t yy_z2_high = z2_idx < FinalZ2Size ? Z2_OutDigitsYY[z2_idx + 1] : 0;

            // Retrieve Z1_temp (Z1')
            int z1_temp_idx = i * 2;
            const uint64_t xx_z1_temp_low = Z1_temp_digitsXX[z1_temp_idx];
            const uint64_t xx_z1_temp_high = Z1_temp_digitsXX[z1_temp_idx + 1];

            const uint64_t xy_z1_temp_low = Z1_temp_digitsXY[z1_temp_idx];
            const uint64_t xy_z1_temp_high = Z1_temp_digitsXY[z1_temp_idx + 1];

            const uint64_t yy_z1_temp_low = Z1_temp_digitsYY[z1_temp_idx];
            const uint64_t yy_z1_temp_high = Z1_temp_digitsYY[z1_temp_idx + 1];

            // Combine Z2 + Z0 first
            uint64_t xx_temp_low, xx_temp_high;
            uint64_t xy_temp_low, xy_temp_high;
            uint64_t yy_temp_low, yy_temp_high;

            Add128(xx_z2_low, xx_z2_high, xx_z0_low, xx_z0_high, xx_temp_low, xx_temp_high);
            Add128(xy_z2_low, xy_z2_high, xy_z0_low, xy_z0_high, xy_temp_low, xy_temp_high);
            Add128(yy_z2_low, yy_z2_high, yy_z0_low, yy_z0_high, yy_temp_low, yy_temp_high);

            // Now combine with Z1_temp
            // Z1 = (Z2 + Z0) +/- Z1_temp
            uint64_t xx_z1_low, xx_z1_high;
            uint64_t xy_z1_low, xy_z1_high;
            uint64_t yy_z1_low, yy_z1_high;

            if (z1_signXX == 0) {
                // same sign: Z1 = (Z2 + Z0) - Z1_temp
                Subtract128(xx_temp_low, xx_temp_high, xx_z1_temp_low, xx_z1_temp_high, xx_z1_low, xx_z1_high);
            } else {
                // opposite signs: Z1 = (Z2 + Z0) + Z1_temp
                Add128(xx_temp_low, xx_temp_high, xx_z1_temp_low, xx_z1_temp_high, xx_z1_low, xx_z1_high);
            }

            if (z1_signXY == 0) {
                // same sign: Z1 = (Z2 + Z0) - Z1_temp
                Subtract128(xy_temp_low, xy_temp_high, xy_z1_temp_low, xy_z1_temp_high, xy_z1_low, xy_z1_high);
            } else {
                // opposite signs: Z1 = (Z2 + Z0) + Z1_temp
                Add128(xy_temp_low, xy_temp_high, xy_z1_temp_low, xy_z1_temp_high, xy_z1_low, xy_z1_high);
            }

            if (z1_signYY == 0) {
                // same sign: Z1 = (Z2 + Z0) - Z1_temp
                Subtract128(yy_temp_low, yy_temp_high, yy_z1_temp_low, yy_z1_temp_high, yy_z1_low, yy_z1_high);
            } else {
                // opposite signs: Z1 = (Z2 + Z0) + Z1_temp
                Add128(yy_temp_low, yy_temp_high, yy_z1_temp_low, yy_z1_temp_high, yy_z1_low, yy_z1_high);
            }

            // Store fully formed Z1
            int z1_idx = i * 2;
            Z1_digitsXX[z1_idx] = xx_z1_low;
            Z1_digitsXX[z1_idx + 1] = xx_z1_high;

            Z1_digitsXY[z1_idx] = xy_z1_low;
            Z1_digitsXY[z1_idx + 1] = xy_z1_high;

            Z1_digitsYY[z1_idx] = yy_z1_low;
            Z1_digitsYY[z1_idx + 1] = yy_z1_high;
        }

        if constexpr (HpShark::DebugChecksums) {
            grid.sync();

            StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z1XX>(
                record, UseConvolutionHere, debugStates, grid, block, Z1_digitsXX, total_k * 2);
            StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z1XY>(
                record, UseConvolutionHere, debugStates, grid, block, Z1_digitsXY, total_k * 2);
            StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z1YY>(
                record, UseConvolutionHere, debugStates, grid, block, Z1_digitsYY, total_k * 2);
        }

        // Synchronize before final combination
        grid.sync();

        // Now the final combination is just:
        // final = Z0 + (Z1 << (32*n)) + (Z2 << (64*n))
        for (int i = tid; i < total_result_digits; i += stride) {
            uint64_t xx_sum_low = 0;
            uint64_t xx_sum_high = 0;

            uint64_t xy_sum_low = 0;
            uint64_t xy_sum_high = 0;

            uint64_t yy_sum_low = 0;
            uint64_t yy_sum_high = 0;

            // Add Z0
            if (i < 2 * n1 - 1) {
                int z0_idx = i * 2;

                const uint64_t xx_z0_low = Z0_OutDigitsXX[z0_idx];
                const uint64_t xx_z0_high = Z0_OutDigitsXX[z0_idx + 1];

                const uint64_t xy_z0_low = Z0_OutDigitsXY[z0_idx];
                const uint64_t xy_z0_high = Z0_OutDigitsXY[z0_idx + 1];

                const uint64_t yy_z0_low = Z0_OutDigitsYY[z0_idx];
                const uint64_t yy_z0_high = Z0_OutDigitsYY[z0_idx + 1];
                
                Add128(xx_sum_low, xx_sum_high, xx_z0_low, xx_z0_high, xx_sum_low, xx_sum_high);
                Add128(xy_sum_low, xy_sum_high, xy_z0_low, xy_z0_high, xy_sum_low, xy_sum_high);
                Add128(yy_sum_low, yy_sum_high, yy_z0_low, yy_z0_high, yy_sum_low, yy_sum_high);
            }

            // Add Z1 shifted by n
            if (i >= n1 && (i - n1) < 2 * n1 - 1) {
                int z1_idx = (i - n1) * 2;

                const uint64_t xx_z1_low = Z1_digitsXX[z1_idx];
                const uint64_t xx_z1_high = Z1_digitsXX[z1_idx + 1];

                const uint64_t xy_z1_low = Z1_digitsXY[z1_idx];
                const uint64_t xy_z1_high = Z1_digitsXY[z1_idx + 1];

                const uint64_t yy_z1_low = Z1_digitsYY[z1_idx];
                const uint64_t yy_z1_high = Z1_digitsYY[z1_idx + 1];

                Add128(xx_sum_low, xx_sum_high, xx_z1_low, xx_z1_high, xx_sum_low, xx_sum_high);
                Add128(xy_sum_low, xy_sum_high, xy_z1_low, xy_z1_high, xy_sum_low, xy_sum_high);
                Add128(yy_sum_low, yy_sum_high, yy_z1_low, yy_z1_high, yy_sum_low, yy_sum_high);
            }

            // Add Z2 shifted by 2*n
            if (i >= 2 * n1 && (i - 2 * n1) < 2 * n1 - 1) {
                int z2_idx = (i - 2 * n1) * 2;

                const uint64_t xx_z2_low = z2_idx < FinalZ2Size ? Z2_OutDigitsXX[z2_idx] : 0;
                const uint64_t xx_z2_high = z2_idx + 1 < FinalZ2Size ? Z2_OutDigitsXX[z2_idx + 1] : 0;

                const uint64_t xy_z2_low = z2_idx < FinalZ2Size ? Z2_OutDigitsXY[z2_idx] : 0;
                const uint64_t xy_z2_high = z2_idx + 1 < FinalZ2Size ? Z2_OutDigitsXY[z2_idx + 1] : 0;

                const uint64_t yy_z2_low = z2_idx < FinalZ2Size ? Z2_OutDigitsYY[z2_idx] : 0;
                const uint64_t yy_z2_high = z2_idx + 1 < FinalZ2Size ? Z2_OutDigitsYY[z2_idx + 1] : 0;

                Add128(xx_sum_low, xx_sum_high, xx_z2_low, xx_z2_high, xx_sum_low, xx_sum_high);
                Add128(xy_sum_low, xy_sum_high, xy_z2_low, xy_z2_high, xy_sum_low, xy_sum_high);
                Add128(yy_sum_low, yy_sum_high, yy_z2_low, yy_z2_high, yy_sum_low, yy_sum_high);
            }

            int result_idx = i * 2;

            // Store the final result
            final128XX[result_idx] = xx_sum_low;
            final128XX[result_idx + 1] = xx_sum_high;

            final128XY[result_idx] = xy_sum_low;
            final128XY[result_idx + 1] = xy_sum_high;

            final128YY[result_idx] = yy_sum_low;
            final128YY[result_idx + 1] = yy_sum_high;
        }

        if constexpr (HpShark::DebugChecksums) {
            grid.sync();

            StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Final128XX>(
                record, UseConvolutionHere, debugStates, grid, block, final128XX, total_result_digits * 2);
            StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Final128XY>(
                record, UseConvolutionHere, debugStates, grid, block, final128XY, total_result_digits * 2);
            StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Final128YY>(
                record, UseConvolutionHere, debugStates, grid, block, final128YY, total_result_digits * 2);

            EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::FinalAdd1>(
                record, debugStates, grid, block);
            EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::FinalAdd2>(
                record, debugStates, grid, block);
            EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::FinalAdd3>(
                record, debugStates, grid, block);

            EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Result_offsetXX>(
                record, debugStates, grid, block);
            EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Result_offsetXY>(
                record, debugStates, grid, block);
            EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Result_offsetYY>(
                record, debugStates, grid, block);

            EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Result_Add1>(
                record, debugStates, grid, block);
            EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Result_Add2>(
                record, debugStates, grid, block);
        }

        // Synchronize before carry propagation
        grid.sync();
    }
}

//
// static constexpr int32_t SharkFloatParams::GlobalThreadsPerBlock = /* power of 2 */;
// static constexpr int32_t SharkFloatParams::GlobalNumBlocks = /* power of 2 */;
// static constexpr int32_t SharkFloatParams::GlobalNumUint32 = SharkFloatParams::GlobalThreadsPerBlock * SharkFloatParams::GlobalNumBlocks;
// 

template<class SharkFloatParams>
static __device__ void MultiplyHelperKaratsubaV2Separates(
    const HpSharkFloat<SharkFloatParams> *SharkRestrict A,
    const HpSharkFloat<SharkFloatParams> *SharkRestrict B,
    HpSharkFloat<SharkFloatParams> *SharkRestrict OutXX,
    HpSharkFloat<SharkFloatParams> *SharkRestrict OutXY,
    HpSharkFloat<SharkFloatParams> *SharkRestrict OutYY,
    cg::grid_group &grid,
    cg::thread_block &block,
    uint64_t *SharkRestrict tempProducts) {

    extern __shared__ uint32_t shared_data[];

    constexpr auto NewN = SharkFloatParams::GlobalNumUint32;         // Total number of digits
    constexpr auto NewN1 = (NewN + 1) / 2;
    constexpr auto NewN2 = NewN - NewN1;   /* n1 is larger or same */
    constexpr auto CallIndex = 0;
    constexpr auto ExecutionBlockBase = 0;
    constexpr auto ExecutionNumBlocks = SharkFloatParams::GlobalNumBlocks;
    constexpr auto RecursionDepth = 0;

    DefineTempProductsOffsets(CallIndex);

    // Must fit inside the computed frame size (in u64 units)
    // Frame starts just after the 256-u64 per-frame header:
    constexpr auto FrameStart = TempBaseOffset + AdditionalUInt64PerFrame;
    // Frame end you already computed:
    constexpr auto FrameEnd = CarryInsEnd;
    static_assert(
        FrameEnd <= TempBaseOffset + CalculateKaratsubaFrameSize<SharkFloatParams>(),
        "Per-frame buffers overflow CalculateKaratsubaFrameSize"
        );

    // Also ensure we never intrude into the global region
    static_assert(
        TempBaseOffset >= (Checksum_offset + AdditionalGlobalChecksumSpace),
        "Per-frame region overlaps global header"
        );

    // How many NewN-sized slots did we actually consume?
    constexpr auto kSlotsUsedNewN =
        /* up to CarryInsOffset */ 79 +
        /* CarryInsEnd adds 3*NewN */ 3;

    static_assert(
        kSlotsUsedNewN <= ScratchMemoryArraysForMultiply,
        "Used more NewN slots than ScratchMemoryArraysForMultiply allows"
        );

    static_assert((Z0_offsetXX % 2) == 0, "Z0_offsetXX must be 16-byte aligned");
    static_assert((Z0_offsetXY % 2) == 0, "Z0_offsetXY must be 16-byte aligned");
    static_assert((Z0_offsetYY % 2) == 0, "Z0_offsetYY must be 16-byte aligned");


    auto *SharkRestrict debugMultiplyCounts = reinterpret_cast<DebugMultiplyCount<SharkFloatParams>*>(&tempProducts[Multiplies_offset]);
    auto *SharkRestrict debugStates = reinterpret_cast<DebugState<SharkFloatParams>*>(&tempProducts[Checksum_offset]);

    auto *SharkRestrict aDigits =
        HpShark::LoadAllInShared ?
        (shared_data) :
        const_cast<uint32_t *>(A->Digits);
    auto *SharkRestrict bDigits =
        HpShark::LoadAllInShared ?
        (aDigits + NewN + CalcAlign16Bytes32BitIndex(NewN)) :
        const_cast<uint32_t *>(B->Digits);
    auto *SharkRestrict x_diff_abs =
        HpShark::LoadAllInShared ?
        reinterpret_cast<uint32_t *>(bDigits + NewN + CalcAlign16Bytes32BitIndex(NewN)) :
        reinterpret_cast<uint32_t *>(&tempProducts[XDiff_offset]);
    auto *SharkRestrict y_diff_abs =
        HpShark::LoadAllInShared ?
        reinterpret_cast<uint32_t *>(x_diff_abs + (NewN + 1) / 2 + CalcAlign16Bytes32BitIndex((NewN + 1) / 2)) :
        reinterpret_cast<uint32_t *>(&tempProducts[YDiff_offset]);

    if constexpr (HpShark::LoadAllInShared) {
        cg::memcpy_async(block, aDigits, A->Digits, sizeof(uint32_t) * NewN);
        cg::memcpy_async(block, bDigits, B->Digits, sizeof(uint32_t) * NewN);
    }/* else {
        constexpr int AvailableShared = CalculateMultiplySharedMemorySize<SharkFloatParams>();
        memset(shared_data, 0xcd, AvailableShared);
    }*/

    if constexpr (HpShark::PrintMultiplyCounts) {
        const auto CurBlock = block.group_index().x;
        const auto CurThread = block.thread_index().x;
        debugMultiplyCounts[CurBlock * SharkFloatParams::GlobalThreadsPerBlock + CurThread].DebugMultiplyErase();
    }

    if constexpr (HpShark::DebugChecksums) {
        const auto CurBlock = block.group_index().x;
        const auto CurThread = block.thread_index().x;
        debugMultiplyCounts[CurBlock * SharkFloatParams::GlobalThreadsPerBlock + CurThread].DebugMultiplyErase();

        const RecordIt record =
            (block.thread_index().x == 0 && block.group_index().x == ExecutionBlockBase) ?
            RecordIt::Yes :
            RecordIt::No;
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Invalid>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::ADigits>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::BDigits>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::CDigits>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::DDigits>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::EDigits>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::AHalfHigh>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::AHalfLow>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::BHalfHigh>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::BHalfLow>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::XDiff>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::YDiff>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z0XX>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z0XY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z0YY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z1XX>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z1XY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z1YY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z2XX>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z2XY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z2YY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z3XX>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z3XY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z3YY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z4XX>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z4XY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z4YY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z2_Perm1>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z2_Perm2>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z2_Perm3>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z2_Perm4>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z2_Perm5>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z2_Perm6>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z1_offsetXX>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z1_offsetXY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Z1_offsetYY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Final128XX>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Final128XY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Final128YY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::FinalAdd1>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::FinalAdd2>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::FinalAdd3>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Result_offsetXX>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Result_offsetXY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Result_offsetYY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Result_Add1>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Result_Add2>(record, debugStates, grid, block);
        static_assert(static_cast<int32_t>(DebugStatePurpose::NumPurposes) == 41, "Unexpected number of purposes");
    }

    // Wait for the first batch of A to be loaded
    cg::wait(block);

    auto *SharkRestrict final128XX = &tempProducts[Convolution_offsetXX];
    auto *SharkRestrict final128XY = &tempProducts[Convolution_offsetXY];
    auto *SharkRestrict final128YY = &tempProducts[Convolution_offsetYY];

    MultiplyDigitsOnly<
        SharkFloatParams,
        RecursionDepth + 1,
        CallIndex + 1,
        NewN,
        NewN1,
        NewN2,
        ExecutionBlockBase,
        ExecutionNumBlocks,
        SharkFloatParams::GlobalNumBlocks,
        TempBase>(
            shared_data,
            A,
            B,
            aDigits,
            bDigits,
            x_diff_abs,
            y_diff_abs,
            final128XX,
            final128XY,
            final128YY,
            grid,
            block,
            tempProducts);

    grid.sync();

    // ---- Carry Propagation ----

    // Global memory for block carry-outs
    // Allocate space for grid.dim_blocks().x block carry-outs after total_result_digits
    // Note, overlaps:
    uint64_t *block_carry_outs = &tempProducts[CarryInsOffset];

    auto *SharkRestrict resultXX = &tempProducts[Result_offsetXX];
    auto *SharkRestrict resultXY = &tempProducts[Result_offsetXY];
    auto *SharkRestrict resultYY = &tempProducts[Result_offsetYY];

    if constexpr (!SharkFloatParams::DisableCarryPropagation) {

        DefineCarryDefinitions();
        constexpr bool UseParallelCarry = true;
        uint64_t *globalCarryCheck = &tempProducts[GlobalCarryOffset];

        if constexpr (UseParallelCarry) {

            // First Pass: Process convolution results to compute initial digits and local carries
            CarryPropagation<SharkFloatParams>(
                (uint64_t *)shared_data,
                grid,
                block,
                thread_start_idx,
                thread_end_idx,
                final128XX,
                final128XY,
                final128YY,
                resultXX,
                resultXY,
                resultYY,
                block_carry_outs,
                globalCarryCheck
            );

            grid.sync();
        } else {
            SerialCarryPropagation<SharkFloatParams>(
                (uint64_t *)shared_data,
                grid,
                block,
                thread_start_idx,
                thread_end_idx,
                final128XX,
                final128XY,
                final128YY,
                resultXX,
                resultXY,
                resultYY,
                nullptr,
                nullptr
            );

            grid.sync();
        }
    } else {
        grid.sync();
    }

    using DebugState = DebugState<SharkFloatParams>;
    const uint64_t *resultEntriesXX = &tempProducts[Result_offsetXX];
    const uint64_t *resultEntriesXY = &tempProducts[Result_offsetXY];
    const uint64_t *resultEntriesYY = &tempProducts[Result_offsetYY];
    const RecordIt record =
        (block.thread_index().x == 0 && block.group_index().x == ExecutionBlockBase) ?
        RecordIt::Yes :
        RecordIt::No;

    if constexpr (HpShark::DebugChecksums) {
        grid.sync();

        StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Result_offsetXX>(
            record, UseConvolution::No, debugStates, grid, block, resultEntriesXX, 2 * NewN);
        StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Result_offsetXY>(
            record, UseConvolution::No, debugStates, grid, block, resultEntriesXY, 2 * NewN);
        StoreCurrentDebugState<SharkFloatParams, RecursionDepth, CallIndex, DebugStatePurpose::Result_offsetYY>(
            record, UseConvolution::No, debugStates, grid, block, resultEntriesYY, 2 * NewN);

        grid.sync();
    }

    // ---- Finalize the Result ----
    if constexpr (!SharkFloatParams::DisableFinalConstruction) {
        // uint64_t final_carry = carryOuts_phase6[SharkFloatParams::GlobalNumBlocks - 1];

        // Initial total_result_digits is 2 * NewN
        int total_result_digits = 2 * NewN;

        // Determine the highest non-zero digit index in the full result
        int highest_nonzero_index_xx = total_result_digits - 1;
        int highest_nonzero_index_xy = total_result_digits - 1;
        int highest_nonzero_index_yy = total_result_digits - 1;

        auto HighestNonzeroIndex = [](const uint64_t *result, int &highest_nonzero_index) {
            while (highest_nonzero_index >= 0) {
                int result_idx = highest_nonzero_index;
                uint32_t digit = static_cast<uint32_t>(result[result_idx]);
                if (digit != 0) {
                    break;
                }
                highest_nonzero_index--;
            }
            };

        HighestNonzeroIndex(resultEntriesXX, highest_nonzero_index_xx);
        HighestNonzeroIndex(resultEntriesXY, highest_nonzero_index_xy);
        HighestNonzeroIndex(resultEntriesYY, highest_nonzero_index_yy);

        // Determine the number of significant digits
        const int significant_digits_xx = highest_nonzero_index_xx + 1;
        const int significant_digits_xy = highest_nonzero_index_xy + 1;
        const int significant_digits_yy = highest_nonzero_index_yy + 1;

        // Calculate the number of digits to shift to keep the most significant NewN digits
        int shift_digits_xx = significant_digits_xx - NewN;
        if (shift_digits_xx < 0) {
            shift_digits_xx = 0;  // No need to shift if we have fewer than NewN significant digits
        }

        int shift_digits_xy = significant_digits_xy - NewN;
        if (shift_digits_xy < 0) {
            shift_digits_xy = 0;  // No need to shift if we have fewer than NewN significant digits
        }

        int shift_digits_yy = significant_digits_yy - NewN;
        if (shift_digits_yy < 0) {
            shift_digits_yy = 0;  // No need to shift if we have fewer than NewN significant digits
        }

        auto ExponentAndSign = [](
            cg::thread_block &block,
            const HpSharkFloat<SharkFloatParams> *A,
            const HpSharkFloat<SharkFloatParams> *B,
            bool forcePositive,
            HpSharkFloat<SharkFloatParams> *Out,
            int shift_digits,
            int additionalFactorsOfTwo) {

                if (block.group_index().x == 0 && block.thread_index().x == 0) {
                    // Adjust the exponent based on the number of bits shifted
                    Out->Exponent = A->Exponent + B->Exponent + shift_digits * 32 + additionalFactorsOfTwo;

                    // Set the sign of the result
                    Out->SetNegative(forcePositive ? false : (A->GetNegative() ^ B->GetNegative()));
                }
            };

        constexpr auto X2_AdditionalFactorsOfTwo = 0;
        ExponentAndSign(
            block,
            A,
            A,
            true,
            OutXX,
            shift_digits_xx,
            X2_AdditionalFactorsOfTwo);

        constexpr auto XY_AdditionalFactorsOfTwo = 1;
        ExponentAndSign(
            block,
            A,
            B,
            false,
            OutXY,
            shift_digits_xy,
            XY_AdditionalFactorsOfTwo);

        constexpr auto Y2_AdditionalFactorsOfTwo = 0;
        ExponentAndSign(
            block,
            B,
            B,
            true,
            OutYY,
            shift_digits_yy,
            Y2_AdditionalFactorsOfTwo);

        auto Finalize = [](
            cg::grid_group &grid,
            cg::thread_block &block,
            const uint64_t *result,
            int highest_nonzero_index,
            int shift_digits,
            HpSharkFloat<SharkFloatParams> *Out) {

                const int tid = block.thread_index().x + block.group_index().x * block.dim_threads().x;
                const int stride = block.dim_threads().x * grid.dim_blocks().x;

                // src_idx is the starting index in tempProducts[] from which we copy
                // TODO:
                const int src_idx = shift_digits;
                const int last_src = highest_nonzero_index; // The last valid index

                // We'll do a grid-stride loop over i in [0 .. NewN)
                for (int i = tid; i < NewN; i += stride) {
                    // Corresponding source index for digit i
                    int src = src_idx + i;

                    if (src <= last_src) {
                        // Copy from tempProducts
                        Out->Digits[i] = result[src];
                    } else {
                        // Pad with zero if we've run out of digits
                        Out->Digits[i] = 0;
                    }
                }
            };

        Finalize(
            grid,
            block,
            resultEntriesXX,
            highest_nonzero_index_xx,
            shift_digits_xx,
            OutXX);

        Finalize(
            grid,
            block,
            resultEntriesXY,
            highest_nonzero_index_xy,
            shift_digits_xy,
            OutXY);

        Finalize(
            grid,
            block,
            resultEntriesYY,
            highest_nonzero_index_yy,
            shift_digits_yy,
            OutYY);
    }
}

template<class SharkFloatParams>
void PrintMaxActiveBlocks(void *kernelFn, int sharedAmountBytes) {
    std::cout << "Shared memory size: " << sharedAmountBytes << std::endl;

    int numBlocks;

    {
        // Check the maximum number of active blocks per multiprocessor
        // with the given shared memory size
        // This is useful to determine if we can fit more blocks
        // in the shared memory

        const auto err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
            &numBlocks,
            kernelFn,
            SharkFloatParams::GlobalThreadsPerBlock,
            sharedAmountBytes
        );

        if (err != hipSuccess) {
            std::cerr << "CUDA error in hipOccupancyMaxActiveBlocksPerMultiprocessor: " << hipGetErrorString(err) << std::endl;
            return;
        }

        std::cout << "Max active blocks per multiprocessor: " << numBlocks << std::endl;
    }

    {
        size_t availableSharedMemory = 0;
        const auto err = cudaOccupancyAvailableDynamicSMemPerBlock(
            &availableSharedMemory,
            kernelFn,
            numBlocks,
            SharkFloatParams::GlobalThreadsPerBlock
        );

        if (err != hipSuccess) {
            std::cerr << "CUDA error in cudaOccupancyAvailableDynamicSMemPerBlock: " << hipGetErrorString(err) << std::endl;
            return;
        }

        std::cout << "Available shared memory per block: " << availableSharedMemory << std::endl;
    }

    // Check the number of multiprocessors on the device
    int numSM;

    {
        const auto err = hipDeviceGetAttribute(
            &numSM,
            hipDeviceAttributeMultiprocessorCount,
            0
        );

        if (err != hipSuccess) {
            std::cerr << "CUDA error in hipDeviceGetAttribute: " << hipGetErrorString(err) << std::endl;
            return;
        }

        std::cout << "Number of multiprocessors: " << numSM << std::endl;
    }

    int maxConcurrentBlocks = numSM * numBlocks;

    std::cout << "Max concurrent blocks: " << maxConcurrentBlocks << std::endl;
    if (maxConcurrentBlocks < SharkFloatParams::GlobalNumBlocks) {
        std::cout << "Warning: Max concurrent blocks exceeds the number of blocks requested." << std::endl;
    }

    {
        // Check the maximum number of threads per block
        int maxThreadsPerBlock;
        const auto err = hipDeviceGetAttribute(
            &maxThreadsPerBlock,
            hipDeviceAttributeMaxThreadsPerBlock,
            0
        );

        if (err != hipSuccess) {
            std::cerr << "CUDA error in hipDeviceGetAttribute: " << hipGetErrorString(err) << std::endl;
            return;
        }

        std::cout << "Max threads per block: " << maxThreadsPerBlock << std::endl;
    }

    {
        // Check the maximum number of threads per multiprocessor
        int maxThreadsPerMultiprocessor;
        const auto err = hipDeviceGetAttribute(
            &maxThreadsPerMultiprocessor,
            hipDeviceAttributeMaxThreadsPerMultiProcessor,
            0
        );
        if (err != hipSuccess) {
            std::cerr << "CUDA error in hipDeviceGetAttribute: " << hipGetErrorString(err) << std::endl;
            return;
        }
        std::cout << "Max threads per multiprocessor: " << maxThreadsPerMultiprocessor << std::endl;
    }

    // Check if this device supports cooperative launches
    int cooperativeLaunch;

    {
        const auto err = hipDeviceGetAttribute(
            &cooperativeLaunch,
            hipDeviceAttributeCooperativeLaunch,
            0
        );

        if (err != hipSuccess) {
            std::cerr << "CUDA error in hipDeviceGetAttribute: " << hipGetErrorString(err) << std::endl;
            return;
        }

        if (cooperativeLaunch) {
            std::cout << "This device supports cooperative launches." << std::endl;
        } else {
            std::cout << "This device does not support cooperative launches." << std::endl;
        }
    }
}

// Assuming that SharkFloatParams::GlobalNumUint32 can be large and doesn't fit in shared memory
// We'll use the provided global memory buffers for large intermediates
template<class SharkFloatParams>
static __device__ void MultiplyHelperKaratsubaV2 (
    HpSharkComboResults<SharkFloatParams> *SharkRestrict combo,
    cg::grid_group &grid,
    cg::thread_block &block,
    uint64_t *SharkRestrict tempProducts) {

    MultiplyHelperKaratsubaV2Separates<SharkFloatParams>(
        &combo->A,
        &combo->B,
        &combo->ResultX2,
        &combo->Result2XY,
        &combo->ResultY2,
        grid,
        block,
        tempProducts);
}
