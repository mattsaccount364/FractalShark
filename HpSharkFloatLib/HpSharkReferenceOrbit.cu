#include "hip/hip_runtime.h"
#include "Add.cu"
#include "MultiplyNTT.cu"
#include "PeriodicityChecker.cuh"

//
// Returns true if we should continue iterating, false if we should stop (period found).
//

template <class SharkFloatParams>
__device__ [[nodiscard]] bool
ReferenceHelper(cg::grid_group &grid,
                cg::thread_block &block,
                uint64_t currentIteration,
                typename SharkFloatParams::Float *SharkRestrict cx_cast,
                typename SharkFloatParams::Float *SharkRestrict cy_cast,
                typename SharkFloatParams::Float *SharkRestrict dzdcX,
                typename SharkFloatParams::Float *SharkRestrict dzdcY,
                HpSharkReferenceResults<SharkFloatParams> *SharkRestrict reference,
                uint64_t *tempData,
                typename SharkFloatParams::ReferenceIterT *gpuReferenceIters)
{
    //
    // All threads do periodicity checking and update the period if found.
    //

    if constexpr (SharkFloatParams::Periodicity) {
        const auto shouldContinue = PeriodicityChecker(grid,
                                                       block,
                                                       currentIteration,
                                                       cx_cast,
                                                       cy_cast,
                                                       dzdcX,
                                                       dzdcY,
                                                       reference,
                                                       gpuReferenceIters);

        if (!shouldContinue) {
            return false;
        }
    }

    // Note: no synchronization needed here because periodicity checker
    // does not rely on any output before the next grid.sync inside multiply.

    //
    // Note: the multiply doesn't depend on the constants.
    // A = Z_real
    // B = Z_imaginary
    //

    MultiplyHelperNTTV2Separates<SharkFloatParams>(reference->Multiply.Roots,
                                                   &reference->Multiply.A,
                                                   &reference->Multiply.B,
                                                   &reference->Multiply.ResultX2,
                                                   &reference->Multiply.Result2XY,
                                                   &reference->Multiply.ResultY2,
                                                   grid,
                                                   block,
                                                   tempData);

    // At this point, we've calculated the intermediate results:
    // ResultX2 = Add.C_A * Add.C_A = Z_real^2
    // ResultY2 = Add.E_B * Add.E_B = Z_imaginary^2
    // ResultXY = Add.C_A * Add.E_B = Z_real * Z_imaginary
    // And just above we've multiplied ResultXY by 2.

    AddHelperSeparates<SharkFloatParams>(
        grid,
        block,
        &reference->Multiply.ResultX2,  // x^2 = Z_real^2
        &reference->Multiply.ResultY2,  // y^2 = Z_imaginary^2
        &reference->Add.C_A,            // constant C_real
        &reference->Multiply.Result2XY, // 2*x*y = 2 * Z_real * Z_imaginary
        &reference->Add.E_B,            // constant C_imaginary
        &reference->Multiply.A,         // Real result = Z_real
        &reference->Multiply.B,         // Imaginary result = Z_imaginary
        tempData);

    return true;
}

template <class SharkFloatParams>
__global__ void
HpSharkReferenceGpuKernel(HpSharkReferenceResults<SharkFloatParams> *SharkRestrict combo,
                          uint64_t *tempData)
{

    // Initialize cooperative grid group
    cg::grid_group grid = cg::this_grid();
    cg::thread_block block = cg::this_thread_block();

    // Call the AddHelper function
    constexpr auto currentIteration = 0;

    if constexpr (SharkFloatParams::Periodicity) {
        // This path is not supported: running one iteration with periodicity checking is pointless.
        // Correctness checking of all this should take place via the integrated loop version just below.
        return;
    } else {
        const auto [[maybe_unused]] shouldContinue = ReferenceHelper<SharkFloatParams>(
            grid, block, currentIteration, nullptr, nullptr, nullptr, nullptr, combo, tempData, nullptr);
    }
}

template <class SharkFloatParams>
__global__ void
HpSharkReferenceGpuLoop(HpSharkReferenceResults<SharkFloatParams> *SharkRestrict combo,
                        uint64_t numIters,
                        uint64_t *tempData,
                        typename SharkFloatParams::ReferenceIterT *gpuReferenceIters)
{

    // Initialize cooperative grid group
    cg::grid_group grid = cg::this_grid();
    cg::thread_block block = cg::this_thread_block();

    typename SharkFloatParams::Float dzdcX{1};
    typename SharkFloatParams::Float dzdcY{0};

    typename SharkFloatParams::Float cx_cast = combo->Add.C_A.ToHDRFloat<SharkFloatParams::SubType>(0);
    typename SharkFloatParams::Float cy_cast = combo->Add.E_B.ToHDRFloat<SharkFloatParams::SubType>(0);

    // if constexpr (SharkFloatParams::Periodicity) {
    // 
    //     gpuReferenceIters[0].x = {};
    //     gpuReferenceIters[0].y = {};
    // }

    for (uint64_t i = 0; i < numIters; ++i) {
        const auto shouldContinue = ReferenceHelper(
            grid, block, i, &cx_cast, &cy_cast, &dzdcX, &dzdcY, combo, tempData, gpuReferenceIters);
        if (!shouldContinue) {
            break;
        }
    }
}

template <class SharkFloatParams>
void
ComputeHpSharkReferenceGpu(void *kernelArgs[])
{

    constexpr auto sharedAmountBytes = CalculateNTTSharedMemorySize<SharkFloatParams>();

    if constexpr (SharkCustomStream) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(HpSharkReferenceGpuLoop<SharkFloatParams>),
                             hipFuncAttributeMaxDynamicSharedMemorySize,
                             sharedAmountBytes);

        PrintMaxActiveBlocks<SharkFloatParams>(HpSharkReferenceGpuLoop<SharkFloatParams>,
                                               sharedAmountBytes);
    }

    hipError_t err = hipLaunchCooperativeKernel((void *)HpSharkReferenceGpuKernel<SharkFloatParams>,
                                                  dim3(SharkFloatParams::GlobalNumBlocks),
                                                  dim3(SharkFloatParams::GlobalThreadsPerBlock),
                                                  kernelArgs,
                                                  sharedAmountBytes,
                                                  0 // Stream
    );

    auto err2 = hipGetLastError();
    if (err != hipSuccess || err2 != hipSuccess) {
        std::cerr << "CUDA error in hipLaunchCooperativeKernel: " << hipGetErrorString(err2)
                  << "err: " << err << std::endl;
    }

    hipDeviceSynchronize();

    if (err != hipSuccess) {
        std::cerr << "CUDA error in MultiplyKernelKaratsubaV2: " << hipGetErrorString(err) << std::endl;
    }
}

template <class SharkFloatParams>
void
ComputeHpSharkReferenceGpuLoop(hipStream_t &stream, void *kernelArgs[])
{

    constexpr auto sharedAmountBytes = CalculateNTTSharedMemorySize<SharkFloatParams>();

    if constexpr (SharkCustomStream) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(HpSharkReferenceGpuLoop<SharkFloatParams>),
                             hipFuncAttributeMaxDynamicSharedMemorySize,
                             sharedAmountBytes);

        PrintMaxActiveBlocks<SharkFloatParams>(HpSharkReferenceGpuLoop<SharkFloatParams>,
                                               sharedAmountBytes);
    }

    hipError_t err = hipLaunchCooperativeKernel((void *)HpSharkReferenceGpuLoop<SharkFloatParams>,
                                                  dim3(SharkFloatParams::GlobalNumBlocks),
                                                  dim3(SharkFloatParams::GlobalThreadsPerBlock),
                                                  kernelArgs,
                                                  sharedAmountBytes,
                                                  stream // Stream
    );

    auto err2 = hipGetLastError();
    if (err != hipSuccess || err2 != hipSuccess) {
        std::cerr << "CUDA error in hipLaunchCooperativeKernel: " << hipGetErrorString(err2)
                  << "err: " << err << std::endl;
    }

    hipDeviceSynchronize();

    if (err != hipSuccess) {
        std::cerr << "CUDA error in MultiplyKernelKaratsubaV2: " << hipGetErrorString(err) << std::endl;
    }
}

#define ExplicitlyInstantiate(SharkFloatParams)                                                         \
    template void ComputeHpSharkReferenceGpu<SharkFloatParams>(void *kernelArgs[]);                     \
    template void ComputeHpSharkReferenceGpuLoop<SharkFloatParams>(hipStream_t & stream,               \
                                                                   void *kernelArgs[]);

#ifdef ENABLE_REFERENCE_KERNEL
ExplicitInstantiateAll();
#endif
