#include "hip/hip_runtime.h"
#include "GPU_ReferenceIter.h"
//#include "KernelInvoke.cuh"
#include "KernelInvokeInternal.cuh"

//
// The "production" path
//
// Assumes:
// combo.Add.C_A, combo.Add.E_B, combo.Multiply.A, combo.Multiply.B are set
// C_A == Multiply.A
// E_B == Multiply.B
// combo.RadiusY is set
// combo.OutputIters is nullptr
//
// On output:
// combo.Period and combo.EscapedIteration are set
// combo.OutputIters is allocated and filled in if periodicity checking is enabled.
//   -- Free via delete[]
//

template <class SharkFloatParams>
void
InvokeHpSharkReferenceKernelProd(HpSharkReferenceResults<SharkFloatParams> &combo,
                                 mpf_t srcX,
                                 mpf_t srcY,
                                 uint64_t numIters)
{
    auto inputX = std::make_unique<HpSharkFloat<SharkFloatParams>>();
    auto inputY = std::make_unique<HpSharkFloat<SharkFloatParams>>();

    // Convert srcX and srcY to HpSharkFloat
    inputX->MpfToHpGpu(srcX, HpSharkFloat<SharkFloatParams>::DefaultMpirBits);
    inputY->MpfToHpGpu(srcY, HpSharkFloat<SharkFloatParams>::DefaultMpirBits);

    combo.Add.C_A = *inputX;
    combo.Add.E_B = *inputY;
    combo.Multiply.A = *inputX;
    combo.Multiply.B = *inputY;
    combo.Period = 0;
    combo.EscapedIteration = 0;
    combo.OutputIters = nullptr;
    assert(combo.OutputIters == nullptr);
    assert(memcmp(&combo.Add.C_A, &combo.Multiply.A, sizeof(HpSharkFloat<SharkFloatParams>)) == 0);
    assert(memcmp(&combo.Add.E_B, &combo.Multiply.B, sizeof(HpSharkFloat<SharkFloatParams>)) == 0);
    assert(combo.RadiusY.mantissa != 0); // RadiusY must be set.  Does 0 have any useful meaning here?

    InvokeHpSharkReferenceKernelPerf<SharkFloatParams>(nullptr, combo, numIters);
}

//
// This test is also something of a correctness test because
// it keeps track of the period and checks it subsequently.
//
template <class SharkFloatParams>
void
InvokeHpSharkReferenceKernelPerf(BenchmarkTimer *timer,
                                 HpSharkReferenceResults<SharkFloatParams> &combo,
                                 uint64_t numIters)
{

    typename SharkFloatParams::ReferenceIterT *gpuReferenceIters;
    hipMalloc(&gpuReferenceIters, sizeof(SharkFloatParams::ReferenceIterT) * numIters);
    if constexpr (SharkTestInitCudaMemory) {
        hipMemset(gpuReferenceIters, 0xCD, sizeof(SharkFloatParams::ReferenceIterT) * numIters);
    } else {
        hipMemset(gpuReferenceIters, 0, sizeof(SharkFloatParams::ReferenceIterT) * numIters);
    }

    // Prepare kernel arguments
    // Allocate memory for carryOuts and cumulativeCarries
    uint64_t *d_tempProducts;
    constexpr size_t BytesToAllocate =
        (AdditionalUInt64Global + CalculateNTTFrameSize<SharkFloatParams>()) * sizeof(uint64_t);
    hipMalloc(&d_tempProducts, BytesToAllocate);

    if constexpr (!SharkTestInitCudaMemory) {
        hipMemset(d_tempProducts, 0, BytesToAllocate);
    } else {
        hipMemset(d_tempProducts, 0xCD, BytesToAllocate);
    }

    HpSharkReferenceResults<SharkFloatParams> *comboGpu;
    hipMalloc(&comboGpu, sizeof(HpSharkReferenceResults<SharkFloatParams>));
    hipMemcpy(
        comboGpu, &combo, sizeof(HpSharkReferenceResults<SharkFloatParams>), hipMemcpyHostToDevice);
    assert(combo.OutputIters == nullptr); // Should not be set on input

    uint8_t byteToSet = SharkTestInitCudaMemory ? 0xCD : 0;

    // Note: we're clearing a specific set of members here, not the whole struct.
    hipMemset(&comboGpu->Add.A_X2, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Add.B_Y2, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Add.D_2X, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Add.Result1_A_B_C, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Add.Result2_D_E, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Multiply.ResultX2, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Multiply.Result2XY, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Multiply.ResultY2, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));

    // Build NTT plan + roots exactly like correctness path
    {
        SharkNTT::RootTables NTTRoots;
        SharkNTT::BuildRoots<SharkFloatParams>(
            SharkFloatParams::NTTPlan.N, SharkFloatParams::NTTPlan.stages, NTTRoots);

        CopyRootsToCuda<SharkFloatParams>(comboGpu->Multiply.Roots, NTTRoots);
    }

    void *kernelArgs[] = {
        (void *)&comboGpu, (void *)&numIters, (void *)&d_tempProducts, (void *)&gpuReferenceIters};

    hipStream_t stream = nullptr;

    if constexpr (SharkCustomStream) {
        auto res = hipStreamCreate(&stream); // Create a stream

        if (res != hipSuccess) {
            std::cerr << "CUDA error in creating stream: " << hipGetErrorString(res) << std::endl;
        }
    }

    hipDeviceProp_t prop;
    int device_id = 0;

    if constexpr (SharkCustomStream) {
        hipGetDeviceProperties(&prop, device_id);
        hipDeviceSetLimit(cudaLimitPersistingL2CacheSize,
                           prop.persistingL2CacheMaxSize); /* Set aside max possible size of L2 cache for
                                                              persisting accesses */

        auto setAccess = [&](void *ptr, size_t num_bytes) {
            hipLaunchAttributeValue stream_attribute; // Stream level attributes data structure
            stream_attribute.accessPolicyWindow.base_ptr =
                reinterpret_cast<void *>(ptr); // Global Memory data pointer
            stream_attribute.accessPolicyWindow.num_bytes =
                num_bytes; // Number of bytes for persisting accesses.
            // (Must be less than hipDeviceProp_t::accessPolicyMaxWindowSize)
            stream_attribute.accessPolicyWindow.hitRatio =
                1.0; // Hint for L2 cache hit ratio for persisting accesses in the num_bytes region
            stream_attribute.accessPolicyWindow.hitProp =
                hipAccessPropertyPersisting; // Type of access property on cache hit
            stream_attribute.accessPolicyWindow.missProp =
                hipAccessPropertyStreaming; // Type of access property on cache miss.

            // Set the attributes to a CUDA stream of type hipStream_t
            hipError_t err =
                hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute);
            if (err != hipSuccess) {
                std::cerr << "CUDA error in setting stream attribute: " << hipGetErrorString(err)
                          << std::endl;
            }
        };

        setAccess(comboGpu, sizeof(HpSharkReferenceResults<SharkFloatParams>));
        setAccess(d_tempProducts, 32 * SharkFloatParams::GlobalNumUint32 * sizeof(uint64_t));
    }

    {
        ScopedBenchmarkStopper stopper{timer};
        ComputeHpSharkReferenceGpuLoop<SharkFloatParams>(stream, kernelArgs);
    }

    hipMemcpy(
        &combo, comboGpu, sizeof(HpSharkReferenceResults<SharkFloatParams>), hipMemcpyDeviceToHost);

    // TODO Costly double-buffer, this could be improved e.g. cuda host memory allocation?
    combo.OutputIters = new typename SharkFloatParams::ReferenceIterT[numIters];
    hipMemcpy(combo.OutputIters,
               gpuReferenceIters,
               sizeof(SharkFloatParams::ReferenceIterT) * numIters,
               hipMemcpyDeviceToHost);

    // Roots were device-allocated in CopyRootsToCuda; destroy like correctness does
    SharkNTT::DestroyRoots<SharkFloatParams>(true, comboGpu->Multiply.Roots);

    hipFree(comboGpu);
    hipFree(d_tempProducts);
    hipFree(gpuReferenceIters);

    if constexpr (SharkCustomStream) {
        auto res = hipStreamDestroy(stream); // Destroy the stream

        if (res != hipSuccess) {
            std::cerr << "CUDA error in destroying stream: " << hipGetErrorString(res) << std::endl;
        }
    }
}

#if defined(ENABLE_REFERENCE_KERNEL) || defined(ENABLE_FULL_KERNEL)
#define ExplicitlyInstantiateHpSharkReference(SharkFloatParams)                                         \
    template void InvokeHpSharkReferenceKernelProd<SharkFloatParams>(                                   \
        HpSharkReferenceResults<SharkFloatParams> &, mpf_t, mpf_t, uint64_t);                           \
    template void InvokeHpSharkReferenceKernelPerf<SharkFloatParams>(                                   \
        BenchmarkTimer * timer, HpSharkReferenceResults<SharkFloatParams> & combo, uint64_t numIters);
#else
#define ExplicitlyInstantiateHpSharkReference(SharkFloatParams) ;
#endif

#define ExplicitlyInstantiate(SharkFloatParams) ExplicitlyInstantiateHpSharkReference(SharkFloatParams)

ExplicitInstantiateAll();
