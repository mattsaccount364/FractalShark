#include "hip/hip_runtime.h"
#include "MultiplyNTT.cu"

template <class SharkFloatParams>
__maxnreg__(SharkRegisterLimit) __global__
    void MultiplyKernelNTT(HpSharkComboResults<SharkFloatParams>* combo, uint64_t* tempProducts)
{

    // Initialize cooperative grid group
    cg::grid_group grid = cg::this_grid();
    cg::thread_block block = cg::this_thread_block();

    // Call the MultiplyHelper function
    // MultiplyHelper(A, B, Out, carryIns, grid, tempProducts);
    if constexpr (!SharkFloatParams::ForceNoOp) {
        MultiplyHelperNTT(combo, grid, block, tempProducts);
    } else {
        grid.sync();
    }
}

template <class SharkFloatParams>
__global__ void
__maxnreg__(SharkRegisterLimit)
    MultiplyKernelNTTTestLoop(HpSharkComboResults<SharkFloatParams>* combo,
                                      uint64_t numIters,
                                      uint64_t* tempProducts)
{ // Array to store cumulative carries

    // Initialize cooperative grid group
    cg::grid_group grid = cg::this_grid();
    cg::thread_block block = cg::this_thread_block();

    for (int i = 0; i < numIters; ++i) {
        // MultiplyHelper(A, B, Out, carryIns, grid, tempProducts);
        if constexpr (!SharkFloatParams::ForceNoOp) {
            MultiplyHelperNTT(combo, grid, block, tempProducts);
        } else {
            grid.sync();
        }
    }
}

template <class SharkFloatParams>
void
ComputeMultiplyNTTGpu(void* kernelArgs[])
{

    hipError_t err;

    constexpr auto sharedAmountBytes = CalculateNTTSharedMemorySize<SharkFloatParams>();

    if constexpr (SharkCustomStream) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(MultiplyKernelNTT<SharkFloatParams>),
                             hipFuncAttributeMaxDynamicSharedMemorySize,
                             sharedAmountBytes);

        PrintMaxActiveBlocks<SharkFloatParams>(MultiplyKernelNTT<SharkFloatParams>,
                                               sharedAmountBytes);
    }

    err = hipLaunchCooperativeKernel((void*)MultiplyKernelNTT<SharkFloatParams>,
                                      dim3(SharkFloatParams::GlobalNumBlocks),
                                      dim3(SharkFloatParams::GlobalThreadsPerBlock),
                                      kernelArgs,
                                      sharedAmountBytes, // Shared memory size
                                      0                  // Stream
    );

    auto err2 = hipGetLastError();
    if (err != hipSuccess || err2 != hipSuccess) {
        std::cerr << "CUDA error in hipLaunchCooperativeKernel: " << hipGetErrorString(err2)
                  << "err: " << err << std::endl;
    }

    hipDeviceSynchronize();

    if (err != hipSuccess) {
        std::cerr << "CUDA error in MultiplyKernelKaratsubaV2: " << hipGetErrorString(err) << std::endl;
    }
}

template <class SharkFloatParams>
void
ComputeMultiplyNTTGpuTestLoop(hipStream_t& stream, void* kernelArgs[])
{

    constexpr auto sharedAmountBytes = CalculateNTTSharedMemorySize<SharkFloatParams>();

    if constexpr (SharkCustomStream) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(MultiplyKernelNTTTestLoop<SharkFloatParams>),
                             hipFuncAttributeMaxDynamicSharedMemorySize,
                             sharedAmountBytes);

        PrintMaxActiveBlocks<SharkFloatParams>(MultiplyKernelNTTTestLoop<SharkFloatParams>,
                                               sharedAmountBytes);
    }

    hipError_t err = hipLaunchCooperativeKernel((void*)MultiplyKernelNTTTestLoop<SharkFloatParams>,
                                                  dim3(SharkFloatParams::GlobalNumBlocks),
                                                  dim3(SharkFloatParams::GlobalThreadsPerBlock),
                                                  kernelArgs,
                                                  sharedAmountBytes, // Shared memory size
                                                  stream             // Stream
    );

    hipDeviceSynchronize();

    if (err != hipSuccess) {
        std::cerr << "CUDA error in MultiplyKernelNTTTestLoop: " << hipGetErrorString(err) << std::endl;
    }
}

#define ExplicitlyInstantiate(SharkFloatParams)                                                         \
    template void ComputeMultiplyNTTGpu<SharkFloatParams>(void* kernelArgs[]);                          \
    template void ComputeMultiplyNTTGpuTestLoop<SharkFloatParams>(hipStream_t & stream,                \
                                                                  void* kernelArgs[]);

#if defined(ENABLE_MULTIPLY_NTT_KERNEL)
ExplicitInstantiateAll();
#endif
