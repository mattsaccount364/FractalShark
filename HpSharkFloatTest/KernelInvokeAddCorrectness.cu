#include "KernelInvoke.cuh"
#include "KernelInvokeInternal.cuh"

template <class SharkFloatParams>
void
InvokeAddKernelCorrectness(BenchmarkTimer &timer,
                           HpSharkAddComboResults<SharkFloatParams> &combo,
                           DebugGpuCombo *debugCombo)
{

    // Perform the calculation on the GPU
    HpSharkAddComboResults<SharkFloatParams> *comboResults;
    hipMalloc(&comboResults, sizeof(HpSharkAddComboResults<SharkFloatParams>));
    hipMemcpy(
        comboResults, &combo, sizeof(HpSharkAddComboResults<SharkFloatParams>), hipMemcpyHostToDevice);

    if constexpr (!SharkTestInitCudaMemory) {
        hipMemset(&comboResults->Result1_A_B_C, 0, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboResults->Result2_D_E, 0, sizeof(HpSharkFloat<SharkFloatParams>));
    } else {
        hipMemset(&comboResults->Result1_A_B_C, 0xCD, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboResults->Result2_D_E, 0xCD, sizeof(HpSharkFloat<SharkFloatParams>));
    }

    constexpr auto BytesToAllocate =
        (AdditionalUInt64Global + CalculateAddFrameSize<SharkFloatParams>()) * sizeof(uint64_t);
    uint64_t *g_extResult;
    hipMalloc(&g_extResult, BytesToAllocate);

    // Prepare kernel arguments
    void *kernelArgs[] = {(void *)&comboResults, (void *)&g_extResult};

    {
        ScopedBenchmarkStopper stopper{timer};
        ComputeAddGpu<SharkFloatParams>(kernelArgs);
    }

    hipMemcpy(
        &combo, comboResults, sizeof(HpSharkAddComboResults<SharkFloatParams>), hipMemcpyDeviceToHost);

    if (debugCombo != nullptr) {
        if constexpr (SharkDebugChecksums) {
            debugCombo->States.resize(SharkFloatParams::NumDebugStates);
            hipMemcpy(debugCombo->States.data(),
                       &g_extResult[AdditionalChecksumsOffset],
                       SharkFloatParams::NumDebugStates * sizeof(DebugStateRaw),
                       hipMemcpyDeviceToHost);
        }

        if constexpr (SharkPrintMultiplyCounts) {
            debugCombo->MultiplyCounts.resize(SharkFloatParams::NumDebugMultiplyCounts);
            hipMemcpy(debugCombo->MultiplyCounts.data(),
                       &g_extResult[AdditionalMultipliesOffset],
                       SharkFloatParams::NumDebugMultiplyCounts * sizeof(DebugMultiplyCountRaw),
                       hipMemcpyDeviceToHost);
        }
    }

    hipFree(g_extResult);
    hipFree(comboResults);
}

#ifdef ENABLE_ADD_KERNEL
#define ExplicitlyInstantiateAdd(SharkFloatParams)                                                      \
    template void InvokeAddKernelCorrectness<SharkFloatParams>(                                         \
        BenchmarkTimer & timer,                                                                         \
        HpSharkAddComboResults<SharkFloatParams> & combo,                                               \
        DebugGpuCombo * debugCombo);
#else
#define ExplicitlyInstantiateAdd(SharkFloatParams) ;
#endif

#define ExplicitlyInstantiate(SharkFloatParams) ExplicitlyInstantiateAdd(SharkFloatParams)

ExplicitInstantiateAll();
