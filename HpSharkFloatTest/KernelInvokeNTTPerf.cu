#include "KernelInvoke.cuh"
#include "KernelInvokeInternal.cuh"

template <class SharkFloatParams>
void
InvokeMultiplyNTTKernelPerf(BenchmarkTimer &timer,
                            HpSharkComboResults<SharkFloatParams> &combo,
                            uint64_t numIters)
{
    // --- 0) Scratch arena (global) ---------------------------------------------------------
    uint64_t *d_tempProducts = nullptr;
    constexpr size_t BytesToAllocate =
        (AdditionalUInt64Global + CalculateNTTFrameSize<SharkFloatParams>()) * sizeof(uint64_t);
    std::cout << " Allocating " << BytesToAllocate << " bytes for d_tempProducts " << std::endl;
    hipMalloc(&d_tempProducts, BytesToAllocate);

    if constexpr (!SharkTestInitCudaMemory) {
        hipMemset(d_tempProducts, 0, BytesToAllocate);
    } else {
        hipMemset(d_tempProducts, 0xCD, BytesToAllocate);
    }

    // --- 1) Stage combo struct, plan and roots on device -----------------------------------
    HpSharkComboResults<SharkFloatParams> *comboGpu = nullptr;
    hipMalloc(&comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>));
    hipMemcpy(comboGpu, &combo, sizeof(HpSharkComboResults<SharkFloatParams>), hipMemcpyHostToDevice);

    // Build NTT plan + roots exactly like correctness path
    {
        SharkNTT::RootTables NTTRoots;
        SharkNTT::BuildRoots<SharkFloatParams>(
            SharkFloatParams::NTTPlan.N, SharkFloatParams::NTTPlan.stages, NTTRoots);

        CopyRootsToCuda<SharkFloatParams>(comboGpu->Roots, NTTRoots);
    }

    // Clear result slots (matches correctness init semantics)
    {
        const uint8_t pat = SharkTestInitCudaMemory ? 0xCD : 0x00;
        hipMemset(&comboGpu->ResultX2, pat, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboGpu->Result2XY, pat, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboGpu->ResultY2, pat, sizeof(HpSharkFloat<SharkFloatParams>));
    }

    // --- 2) Stream + persisting L2 window (identical policy to correctness) ----------------
    hipStream_t stream = nullptr;

    if constexpr (SharkCustomStream) {
        auto res = hipStreamCreate(&stream);
        if (res != hipSuccess) {
            std::cerr << "CUDA error in creating stream: " << hipGetErrorString(res) << std::endl;
        }

        hipDeviceProp_t prop{};
        int device_id = 0;
        hipGetDeviceProperties(&prop, device_id);
        // Reserve as much L2 as driver allows for persisting window
        hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, prop.persistingL2CacheMaxSize);

        auto setAccess = [&](void *ptr, size_t num_bytes) {
            hipLaunchAttributeValue attr{};
            attr.accessPolicyWindow.base_ptr = ptr;
            attr.accessPolicyWindow.num_bytes = num_bytes; // must be <= accessPolicyMaxWindowSize
            attr.accessPolicyWindow.hitRatio = 1.0;        // hint
            attr.accessPolicyWindow.hitProp = hipAccessPropertyPersisting;
            attr.accessPolicyWindow.missProp = hipAccessPropertyStreaming;

            hipError_t err =
                hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &attr);
            if (err != hipSuccess) {
                std::cerr << "hipStreamSetAttribute: " << hipGetErrorString(err) << std::endl;
            }
        };

        // Keep the hot state resident
        setAccess(comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>));
        // Big scratch window (enough to cover typical working set)
        setAccess(d_tempProducts, 32ull * SharkFloatParams::GlobalNumUint32 * sizeof(uint64_t));
    }

    // --- 3) Launch (mirror correctness: test-loop entry + same arg order) ------------------
    void *kernelArgs[] = {(void *)&comboGpu, (void *)&numIters, (void *)&d_tempProducts};

    {
        ScopedBenchmarkStopper stopper{timer};
        // Use the *looping* entry so numIters lives on device (same as correctness)
        ComputeMultiplyNTTGpuTestLoop<SharkFloatParams>(stream, kernelArgs);
    }

    // --- 4) Copy results back, teardown -----------------------------------------------------
    hipMemcpy(&combo, comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>), hipMemcpyDeviceToHost);

    // Roots were device-allocated in CopyRootsToCuda; destroy like correctness does
    SharkNTT::DestroyRoots<SharkFloatParams>(true, comboGpu->Roots);

    if constexpr (SharkCustomStream) {
        hipStreamDestroy(stream);
    }

    hipFree(comboGpu);
    hipFree(d_tempProducts);
}


#ifdef ENABLE_MULTIPLY_FFT2_KERNEL
#define ExplicitlyInstantiateMultiplyNTT(SharkFloatParams)                                              \
    template void InvokeMultiplyNTTKernelPerf<SharkFloatParams>(                                        \
        BenchmarkTimer & timer, HpSharkComboResults<SharkFloatParams> & combo, uint64_t numIters);
#else
#define ExplicitlyInstantiateMultiplyNTT(SharkFloatParams) ;
#endif

#define ExplicitlyInstantiate(SharkFloatParams)                                                         \
    ExplicitlyInstantiateMultiplyNTT(SharkFloatParams)

ExplicitInstantiateAll();