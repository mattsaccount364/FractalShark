#include <hip/hip_runtime.h>

#include "HpGpu.cuh"
#include "BenchmarkTimer.h"
#include "TestTracker.h"

#include <iostream>
#include <vector>
#include <gmp.h>
#include <cstring>
#include <sstream>
#include <iomanip>
#include <cmath>
#include <algorithm>

static TestTracker Tests;

void TestConvertNumber (
    bool verbose,
    int testNum,
    const char*numberStr) {

    mpf_set_default_prec(HpGpu::DefaultMpirBits);  // Set precision for MPIR floating point

    mpf_t mpf_x;
    mpf_init(mpf_x);

    auto res = mpf_set_str(mpf_x, numberStr, 10);
    if (res == -1) {
        std::cout << "Error setting mpf_x" << std::endl;
    }

    // Print the original input values
    if (verbose) {
        std::cout << "Original input values:" << std::endl;
        std::cout << "numberStr: " << numberStr << std::endl;
        std::cout << "X: " << MpfToString(mpf_x, HpGpu::DefaultPrecBits) << std::endl;
        std::cout << "X hex: " << MpfToHexString(mpf_x, HpGpu::DefaultPrecBits) << std::endl;
    }

    // Convert the input values to HpGpu representations
    HpGpu x_num{};
    MpfToHpGpu(mpf_x, x_num, HpGpu::DefaultPrecBits);

    // Convert the HpGpu results to strings
    std::string gpu_str = x_num.ToString();

    if (verbose) {
        std::cout << "\nHighPrecisionNumber representations:" << std::endl;
        std::cout << "X: " << gpu_str << std::endl;
        std::cout << "X hex: " << x_num.ToHexString() << std::endl;
    }

    // Convert the HpGpu results to mpf_t for comparison
    mpf_t mpf_x_gpu_result;
    mpf_init(mpf_x_gpu_result);

    HpGpuToMpf(x_num, mpf_x_gpu_result);

    // Compute the differences between host and GPU results
    mpf_t mpf_diff;
    mpf_init(mpf_diff);

    mpf_sub(mpf_diff, mpf_x, mpf_x_gpu_result);

    // Take absolute delta:
    mpf_t mpf_diff_abs; 
    mpf_init(mpf_diff_abs);
    mpf_abs(mpf_diff_abs, mpf_diff);

    // Converted GPU result
    if (verbose) {
        std::cout << "\nConverted GPU result:" << std::endl;
        std::cout << "X: " << MpfToString(mpf_x_gpu_result, HpGpu::DefaultPrecBits) << std::endl;
        std::cout << "X hex: " << MpfToHexString(mpf_x_gpu_result, HpGpu::DefaultPrecBits) << std::endl;
    }

    // Print the differences
    std::cout << "\nDifference between host and GPU results:" << std::endl;
    std::cout << MpfToString(mpf_diff_abs, HpGpu::DefaultPrecBits) << std::endl;

    // If absolute delta is greater than 1e-300, the test is considered failed
    if (mpf_cmp_d(mpf_diff_abs, 1e-30) > 0) {
        Tests.MarkFailed(testNum);
    }

    // Clean up MPIR variables
    mpf_clear(mpf_x);
    mpf_clear(mpf_diff);
    mpf_clear(mpf_diff_abs);
    mpf_clear(mpf_x_gpu_result);
}

void TestConversion() {
    constexpr bool verbose = true;
    const auto set1 = 0;
    TestConvertNumber(verbose, set1 + 1, "0.0");
    TestConvertNumber(verbose, set1 + 2, "1.0");
    TestConvertNumber(verbose, set1 + 3, "2.0");
    TestConvertNumber(verbose, set1 + 4, "3.0");

    const auto set2 = 20;
    TestConvertNumber(verbose, set2 + 1, "0.1");
    TestConvertNumber(verbose, set2 + 2, "0.2");
    TestConvertNumber(verbose, set2 + 3, "0.3");
    TestConvertNumber(verbose, set2 + 4, "0.4");

    const auto set3 = 30;
    TestConvertNumber(verbose, set3 + 1, "1e-50");
    TestConvertNumber(verbose, set3 + 2, "1e-100");
    TestConvertNumber(verbose, set3 + 3, "1e-150");
    TestConvertNumber(verbose, set3 + 4, "1e-500");
    TestConvertNumber(verbose, set3 + 5, "1e-1000");
    TestConvertNumber(verbose, set3 + 6, "-1e-50");
    TestConvertNumber(verbose, set3 + 7, "-1e-100");
    TestConvertNumber(verbose, set3 + 8, "-1e-150");
    TestConvertNumber(verbose, set3 + 9, "-1e-500");

    const auto set4 = 40;
    TestConvertNumber(verbose, set4 + 1, "-1");
    TestConvertNumber(verbose, set4 + 2, "-2");
    TestConvertNumber(verbose, set4 + 3, "-3");
    TestConvertNumber(verbose, set4 + 4, "-4");

    const auto set5 = 50;
    TestConvertNumber(verbose, set5 + 1, "-0.1");
    TestConvertNumber(verbose, set5 + 2, "-0.2");
    TestConvertNumber(verbose, set5 + 3, "-0.3");
    TestConvertNumber(verbose, set5 + 4, "-0.4");

    const auto set6 = 60;
    TestConvertNumber(verbose, set6 + 1, "4294967297");
    TestConvertNumber(verbose, set6 + 2, "18446744073709551617");
    TestConvertNumber(verbose, set6 + 3, "55340232221128654849"); // 2^65 + 2^64 + 1
    TestConvertNumber(verbose, set6 + 4, "-4294967297");
    TestConvertNumber(verbose, set6 + 5, "-18446744073709551617");
    TestConvertNumber(verbose, set6 + 6, "-55340232221128654849");

    const auto set7 = 70;
    TestConvertNumber(verbose, set7 + 1, "4294967297.0000152587890625"); // 2^32 + 1 + 1/2^16
    TestConvertNumber(verbose, set7 + 2, "18446744073709551617.0000152587890625");
    TestConvertNumber(verbose, set7 + 3, "55340232221128654849.0000152587890625");
    Tests.CheckAllTestsPassed();
}