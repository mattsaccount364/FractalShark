#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "BenchmarkTimer.h"
#include "TestTracker.h"

#include "Tests.h"
#include "HpSharkFloat.cuh"
#include "Add.cuh"
#include "Multiply.cuh"
#include "HpSharkReferenceOrbit.cuh"
#include "ReferenceKaratsuba.h"
#include "DebugChecksumHost.h"

#include <iostream>
#include <vector>
#include <gmp.h>
#include <cstring>
#include <sstream>
#include <iomanip>
#include <cmath>
#include <algorithm>
#include <assert.h>

template<class SharkFloatParams>
void InvokeHpSharkReferenceKernelPerf(
    BenchmarkTimer &timer,
    HpSharkReferenceResults<SharkFloatParams> &combo,
    uint64_t numIters) {

    // Prepare kernel arguments
    // Allocate memory for carryOuts and cumulativeCarries
    uint64_t *d_tempProducts;
    constexpr auto BytesToAllocate =
        (AdditionalUInt64Global + ScratchMemoryCopies * CalculateMultiplyFrameSize<SharkFloatParams>()) * sizeof(uint64_t);
    hipError_t err = hipMalloc(&d_tempProducts, BytesToAllocate);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMalloc (d_tempProducts): " << hipGetErrorString(err) << std::endl;
        return;
    }

    if constexpr (!SharkTestInitCudaMemory) {
        err = hipMemset(d_tempProducts, 0, BytesToAllocate);
        if (err != hipSuccess) {
            std::cerr << "CUDA error in hipMemset (d_tempProducts): " << hipGetErrorString(err) << std::endl;
            hipFree(d_tempProducts);
            return;
        }
    } else {
        err = hipMemset(d_tempProducts, 0xCD, BytesToAllocate);
        if (err != hipSuccess) {
            std::cerr << "CUDA error in hipMemset (d_tempProducts): " << hipGetErrorString(err) << std::endl;
            hipFree(d_tempProducts);
            return;
        }
    }

    HpSharkReferenceResults<SharkFloatParams> *comboGpu;
    err = hipMalloc(&comboGpu, sizeof(HpSharkReferenceResults<SharkFloatParams>));
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMalloc (comboGpu): " << hipGetErrorString(err) << std::endl;
        hipFree(d_tempProducts);
        return;
    }
    err = hipMemcpy(comboGpu, &combo, sizeof(HpSharkReferenceResults<SharkFloatParams>), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMemcpy (comboGpu): " << hipGetErrorString(err) << std::endl;
        hipFree(comboGpu);
        hipFree(d_tempProducts);
        return;
    }

    uint8_t byteToSet = SharkTestInitCudaMemory ? 0xCD : 0;

    hipMemset(&comboGpu->Add.A_X2, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Add.B_Y2, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Add.D_2X, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Add.Result1_A_B_C, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Add.Result2_D_E, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Multiply.ResultX2, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Multiply.Result2XY, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Multiply.ResultY2, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));

    void *kernelArgs[] = {
        (void *)&comboGpu,
        (void *)&numIters,
        (void *)&d_tempProducts
    };

    hipStream_t stream = nullptr;

    if constexpr (SharkCustomStream) {
        auto res = hipStreamCreate(&stream); // Create a stream

        if (res != hipSuccess) {
            std::cerr << "CUDA error in creating stream: " << hipGetErrorString(res) << std::endl;
        }
    }

    hipDeviceProp_t prop;
    int device_id = 0;

    if constexpr (SharkCustomStream) {
        hipGetDeviceProperties(&prop, device_id);
        hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, prop.persistingL2CacheMaxSize); /* Set aside max possible size of L2 cache for persisting accesses */

        auto setAccess = [&](void *ptr, size_t num_bytes) {
            hipLaunchAttributeValue stream_attribute;                                         // Stream level attributes data structure
            stream_attribute.accessPolicyWindow.base_ptr = reinterpret_cast<void *>(ptr); // Global Memory data pointer
            stream_attribute.accessPolicyWindow.num_bytes = num_bytes;                    // Number of bytes for persisting accesses.
            // (Must be less than hipDeviceProp_t::accessPolicyMaxWindowSize)
            stream_attribute.accessPolicyWindow.hitRatio = 1.0;                          // Hint for L2 cache hit ratio for persisting accesses in the num_bytes region
            stream_attribute.accessPolicyWindow.hitProp = hipAccessPropertyPersisting; // Type of access property on cache hit
            stream_attribute.accessPolicyWindow.missProp = hipAccessPropertyStreaming;  // Type of access property on cache miss.

            //Set the attributes to a CUDA stream of type hipStream_t
            hipError_t err = hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute);
            if (err != hipSuccess) {
                std::cerr << "CUDA error in setting stream attribute: " << hipGetErrorString(err) << std::endl;
            }
            };

        setAccess(comboGpu, sizeof(HpSharkReferenceResults<SharkFloatParams>));
        setAccess(d_tempProducts, 32 * SharkFloatParams::GlobalNumUint32 * sizeof(uint64_t));
    }

    {
        ScopedBenchmarkStopper stopper{ timer };
        ComputeHpSharkReferenceGpuLoop<SharkFloatParams>(stream, kernelArgs);
    }

    err = hipMemcpy(&combo, comboGpu, sizeof(HpSharkReferenceResults<SharkFloatParams>), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMemcpy (device to host): " << hipGetErrorString(err) << std::endl;
    }

    err = hipFree(comboGpu);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipFree (comboGpu): " << hipGetErrorString(err) << std::endl;
    }
    err = hipFree(d_tempProducts);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipFree (d_tempProducts): " << hipGetErrorString(err) << std::endl;
    }

    if constexpr (SharkCustomStream) {
        auto res = hipStreamDestroy(stream); // Destroy the stream

        if (res != hipSuccess) {
            std::cerr << "CUDA error in destroying stream: " << hipGetErrorString(res) << std::endl;
        }
    }
}

template<class SharkFloatParams>
void InvokeMultiplyKernelPerf(
    BenchmarkTimer &timer,
    HpSharkComboResults<SharkFloatParams> &combo,
    uint64_t numIters) {

    // Prepare kernel arguments
    // Allocate memory for carryOuts and cumulativeCarries
    uint64_t *d_tempProducts;
    constexpr auto BytesToAllocate =
        (AdditionalUInt64Global + ScratchMemoryCopies * CalculateMultiplyFrameSize<SharkFloatParams>()) * sizeof(uint64_t);
    hipError_t err = hipMalloc(&d_tempProducts, BytesToAllocate);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMalloc (d_tempProducts): " << hipGetErrorString(err) << std::endl;
        return;
    }

    HpSharkComboResults<SharkFloatParams> *comboGpu;
    err = hipMalloc(&comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>));
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMalloc (comboGpu): " << hipGetErrorString(err) << std::endl;
        hipFree(d_tempProducts);
        return;
    }
    err = hipMemcpy(comboGpu, &combo, sizeof(HpSharkComboResults<SharkFloatParams>), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMemcpy (comboGpu): " << hipGetErrorString(err) << std::endl;
        hipFree(comboGpu);
        hipFree(d_tempProducts);
        return;
    }

    void *kernelArgs[] = {
        (void *)&comboGpu,
        (void *)&numIters,
        (void *)&d_tempProducts
    };

    hipStream_t stream = nullptr;

    if constexpr (SharkCustomStream) {
        err = hipStreamCreate(&stream); // Create a stream
        if (err != hipSuccess) {
            std::cerr << "CUDA error in hipStreamCreate: " << hipGetErrorString(err) << std::endl;
            hipFree(comboGpu);
            hipFree(d_tempProducts);
            return;
        }
    }

    hipDeviceProp_t prop;
    int device_id = 0;

    if constexpr (SharkCustomStream) {
        hipGetDeviceProperties(&prop, device_id);
        hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, prop.persistingL2CacheMaxSize); /* Set aside max possible size of L2 cache for persisting accesses */

        auto setAccess = [&](void *ptr, size_t num_bytes) {
            hipLaunchAttributeValue stream_attribute;                                         // Stream level attributes data structure
            stream_attribute.accessPolicyWindow.base_ptr = reinterpret_cast<void *>(ptr); // Global Memory data pointer
            stream_attribute.accessPolicyWindow.num_bytes = num_bytes;                    // Number of bytes for persisting accesses.
            // (Must be less than hipDeviceProp_t::accessPolicyMaxWindowSize)
            stream_attribute.accessPolicyWindow.hitRatio = 1.0;                          // Hint for L2 cache hit ratio for persisting accesses in the num_bytes region
            stream_attribute.accessPolicyWindow.hitProp = hipAccessPropertyPersisting; // Type of access property on cache hit
            stream_attribute.accessPolicyWindow.missProp = hipAccessPropertyStreaming;  // Type of access property on cache miss.

            //Set the attributes to a CUDA stream of type hipStream_t
            hipError_t err = hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute);
            if (err != hipSuccess) {
                std::cerr << "CUDA error in setting stream attribute: " << hipGetErrorString(err) << std::endl;
            }
            };

        setAccess(comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>));
        setAccess(d_tempProducts, 32 * SharkFloatParams::GlobalNumUint32 * sizeof(uint64_t));
    }

    {
        ScopedBenchmarkStopper stopper{ timer };
        ComputeMultiplyKaratsubaV2GpuTestLoop<SharkFloatParams>(stream, kernelArgs);
    }

    err = hipMemcpy(&combo, comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMemcpy (device to host): " << hipGetErrorString(err) << std::endl;
    }

    if constexpr (SharkCustomStream) {
        err = hipStreamDestroy(stream); // Destroy the stream
        if (err != hipSuccess) {
            std::cerr << "CUDA error in hipStreamDestroy: " << hipGetErrorString(err) << std::endl;
        }
    }

    err = hipFree(comboGpu);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipFree (comboGpu): " << hipGetErrorString(err) << std::endl;
    }
    err = hipFree(d_tempProducts);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipFree (d_tempProducts): " << hipGetErrorString(err) << std::endl;
    }
}

template<class SharkFloatParams>
void InvokeAddKernelPerf(
    BenchmarkTimer &timer,
    HpSharkAddComboResults<SharkFloatParams> &combo,
    uint64_t numIters) {

    // Perform the calculation on the GPU
    HpSharkAddComboResults<SharkFloatParams> *comboResults;
    hipError_t err = hipMalloc(&comboResults, sizeof(HpSharkAddComboResults<SharkFloatParams>));
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMalloc (comboResults): " << hipGetErrorString(err) << std::endl;
        return;
    }
    err = hipMemcpy(comboResults, &combo, sizeof(HpSharkAddComboResults<SharkFloatParams>), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMemcpy (comboResults): " << hipGetErrorString(err) << std::endl;
        hipFree(comboResults);
        return;
    }

    constexpr auto BytesToAllocate =
        (AdditionalUInt64Global + CalculateAddFrameSize<SharkFloatParams>()) * sizeof(uint64_t);
    uint64_t *g_extResult;
    err = hipMalloc(&g_extResult, BytesToAllocate);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMalloc (g_extResult): " << hipGetErrorString(err) << std::endl;
        hipFree(comboResults);
        return;
    }

    // Prepare kernel arguments
    void *kernelArgs[] = {
        (void *)&comboResults,
        (void *)&numIters,
        (void *)&g_extResult
    };

    // Launch the cooperative kernel
    {
        ScopedBenchmarkStopper stopper{ timer };
        ComputeAddGpuTestLoop<SharkFloatParams>(kernelArgs);
    }

    err = hipMemcpy(&combo, comboResults, sizeof(HpSharkAddComboResults<SharkFloatParams>), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMemcpy (device to host): " << hipGetErrorString(err) << std::endl;
    }

    // Free memory
    err = hipFree(g_extResult);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipFree (g_extResult): " << hipGetErrorString(err) << std::endl;
    }
    err = hipFree(comboResults);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipFree (comboResults): " << hipGetErrorString(err) << std::endl;
    }
}

template<class SharkFloatParams>
void InvokeHpSharkReferenceKernelCorrectness(
    BenchmarkTimer &timer,
    HpSharkReferenceResults<SharkFloatParams> &combo,
    std::vector<DebugStateRaw> *debugResults) {

    // Prepare kernel arguments
    // Allocate memory for carryOuts and cumulativeCarries

    // TODO max of add/multiply frame size
    // TODO checksum handled
    uint64_t *d_tempProducts;
    constexpr auto BytesToAllocate =
        (AdditionalUInt64Global + ScratchMemoryCopies * CalculateMultiplyFrameSize<SharkFloatParams>()) * sizeof(uint64_t);
    hipError_t err = hipMalloc(&d_tempProducts, BytesToAllocate);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMalloc (d_tempProducts): " << hipGetErrorString(err) << std::endl;
        return;
    }

    if constexpr (!SharkTestInitCudaMemory) {
        err = hipMemset(d_tempProducts, 0, BytesToAllocate);
        if (err != hipSuccess) {
            std::cerr << "CUDA error in hipMemset (d_tempProducts): " << hipGetErrorString(err) << std::endl;
            hipFree(d_tempProducts);
            return;
        }
    } else {
        err = hipMemset(d_tempProducts, 0xCD, BytesToAllocate);
        if (err != hipSuccess) {
            std::cerr << "CUDA error in hipMemset (d_tempProducts): " << hipGetErrorString(err) << std::endl;
            hipFree(d_tempProducts);
            return;
        }
    }

    HpSharkReferenceResults<SharkFloatParams> *comboGpu;
    err = hipMalloc(&comboGpu, sizeof(HpSharkReferenceResults<SharkFloatParams>));
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMalloc (comboGpu): " << hipGetErrorString(err) << std::endl;
        hipFree(d_tempProducts);
        return;
    }
    err = hipMemcpy(comboGpu, &combo, sizeof(HpSharkReferenceResults<SharkFloatParams>), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMemcpy (comboGpu): " << hipGetErrorString(err) << std::endl;
        hipFree(comboGpu);
        hipFree(d_tempProducts);
        return;
    }

    uint8_t byteToSet = SharkTestInitCudaMemory ? 0xCD : 0;

    hipMemset(&comboGpu->Add.A_X2, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Add.B_Y2, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Add.D_2X, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Add.Result1_A_B_C, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Add.Result2_D_E, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Multiply.ResultX2, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Multiply.Result2XY, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Multiply.ResultY2, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));

    void *kernelArgs[] = {
        (void *)&comboGpu,
        (void *)&d_tempProducts
    };

    {
        ScopedBenchmarkStopper stopper{ timer };
        ComputeHpSharkReferenceGpu<SharkFloatParams>(kernelArgs);
    }

    err = hipMemcpy(&combo, comboGpu, sizeof(HpSharkReferenceResults<SharkFloatParams>), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMemcpy (device to host): " << hipGetErrorString(err) << std::endl;
    }

    if (debugResults != nullptr) {
        if constexpr (SharkDebugChecksums) {
            debugResults->resize(SharkFloatParams::NumDebugStates);
            err = hipMemcpy(
                debugResults->data(),
                &d_tempProducts[AdditionalGlobalSyncSpace],
                SharkFloatParams::NumDebugStates * sizeof(DebugStateRaw),
                hipMemcpyDeviceToHost);
            if (err != hipSuccess) {
                std::cerr << "CUDA error in hipMemcpy (debug results): " << hipGetErrorString(err) << std::endl;
            }
        }
    }

    err = hipFree(comboGpu);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipFree (comboGpu): " << hipGetErrorString(err) << std::endl;
    }
    err = hipFree(d_tempProducts);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipFree (d_tempProducts): " << hipGetErrorString(err) << std::endl;
    }
}

template<class SharkFloatParams>
void InvokeMultiplyKernelCorrectness(
    BenchmarkTimer &timer,
    HpSharkComboResults<SharkFloatParams> &combo,
    std::vector<DebugStateRaw> *debugResults) {

    // Prepare kernel arguments
    // Allocate memory for carryOuts and cumulativeCarries
    uint64_t *d_tempProducts;
    constexpr auto BytesToAllocate =
        (AdditionalUInt64Global + ScratchMemoryCopies * CalculateMultiplyFrameSize<SharkFloatParams>()) * sizeof(uint64_t);
    hipError_t err = hipMalloc(&d_tempProducts, BytesToAllocate);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMalloc (d_tempProducts): " << hipGetErrorString(err) << std::endl;
        return;
    }

    if constexpr (!SharkTestInitCudaMemory) {
        err = hipMemset(d_tempProducts, 0, BytesToAllocate);
        if (err != hipSuccess) {
            std::cerr << "CUDA error in hipMemset (d_tempProducts): " << hipGetErrorString(err) << std::endl;
            hipFree(d_tempProducts);
            return;
        }
    } else {
        err = hipMemset(d_tempProducts, 0xCD, BytesToAllocate);
        if (err != hipSuccess) {
            std::cerr << "CUDA error in hipMemset (d_tempProducts): " << hipGetErrorString(err) << std::endl;
            hipFree(d_tempProducts);
            return;
        }
    }

    HpSharkComboResults<SharkFloatParams> *comboGpu;
    err = hipMalloc(&comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>));
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMalloc (comboGpu): " << hipGetErrorString(err) << std::endl;
        hipFree(d_tempProducts);
        return;
    }
    err = hipMemcpy(comboGpu, &combo, sizeof(HpSharkComboResults<SharkFloatParams>), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMemcpy (comboGpu): " << hipGetErrorString(err) << std::endl;
        hipFree(comboGpu);
        hipFree(d_tempProducts);
        return;
    }

    if constexpr (!SharkTestInitCudaMemory) {
        hipMemset(&comboGpu->ResultX2, 0, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboGpu->Result2XY, 0, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboGpu->ResultY2, 0, sizeof(HpSharkFloat<SharkFloatParams>));
    } else {
        hipMemset(&comboGpu->ResultX2, 0xCD, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboGpu->Result2XY, 0xCD, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboGpu->ResultY2, 0xCD, sizeof(HpSharkFloat<SharkFloatParams>));
    }

    void *kernelArgs[] = {
        (void *)&comboGpu,
        (void *)&d_tempProducts
    };

    {
        ScopedBenchmarkStopper stopper{ timer };
        ComputeMultiplyKaratsubaV2Gpu<SharkFloatParams>(kernelArgs);
    }

    err = hipMemcpy(&combo, comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMemcpy (device to host): " << hipGetErrorString(err) << std::endl;
    }

    if (debugResults != nullptr) {
        if constexpr (SharkDebugChecksums) {
            debugResults->resize(SharkFloatParams::NumDebugStates);
            err = hipMemcpy(
                debugResults->data(),
                &d_tempProducts[AdditionalGlobalSyncSpace],
                SharkFloatParams::NumDebugStates * sizeof(DebugStateRaw),
                hipMemcpyDeviceToHost);
            if (err != hipSuccess) {
                std::cerr << "CUDA error in hipMemcpy (debug results): " << hipGetErrorString(err) << std::endl;
            }
        }
    }

    err = hipFree(comboGpu);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipFree (comboGpu): " << hipGetErrorString(err) << std::endl;
    }
    err = hipFree(d_tempProducts);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipFree (d_tempProducts): " << hipGetErrorString(err) << std::endl;
    }
}

template<class SharkFloatParams>
void InvokeAddKernelCorrectness(
    BenchmarkTimer &timer,
    HpSharkAddComboResults<SharkFloatParams> &combo,
    std::vector<DebugStateRaw> *debugResults) {

    // Perform the calculation on the GPU
    HpSharkAddComboResults<SharkFloatParams> *comboResults;
    hipError_t err = hipMalloc(&comboResults, sizeof(HpSharkAddComboResults<SharkFloatParams>));
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMalloc (comboResults): " << hipGetErrorString(err) << std::endl;
        return;
    }
    err = hipMemcpy(comboResults, &combo, sizeof(HpSharkAddComboResults<SharkFloatParams>), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMemcpy (comboResults): " << hipGetErrorString(err) << std::endl;
        hipFree(comboResults);
        return;
    }

    if constexpr (!SharkTestInitCudaMemory) {
        hipMemset(&comboResults->Result1_A_B_C, 0, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboResults->Result2_D_E, 0, sizeof(HpSharkFloat<SharkFloatParams>));
    } else {
        hipMemset(&comboResults->Result1_A_B_C, 0xCD, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboResults->Result2_D_E, 0xCD, sizeof(HpSharkFloat<SharkFloatParams>));
    }

    constexpr auto BytesToAllocate =
        (AdditionalUInt64Global + CalculateAddFrameSize<SharkFloatParams>()) * sizeof(uint64_t);
    uint64_t *g_extResult;
    err = hipMalloc(&g_extResult, BytesToAllocate);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMalloc (g_extResult): " << hipGetErrorString(err) << std::endl;
        hipFree(comboResults);
        return;
    }

    // Prepare kernel arguments
    void *kernelArgs[] = {
        (void *)&comboResults,
        (void *)&g_extResult
    };

    {
        ScopedBenchmarkStopper stopper{ timer };
        ComputeAddGpu<SharkFloatParams>(kernelArgs);
    }

    err = hipMemcpy(&combo, comboResults, sizeof(HpSharkAddComboResults<SharkFloatParams>), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMemcpy (device to host): " << hipGetErrorString(err) << std::endl;
    }

    if (debugResults != nullptr) {
        if constexpr (SharkDebugChecksums) {
            debugResults->resize(SharkFloatParams::NumDebugStates);
            err = hipMemcpy(
                debugResults->data(),
                &g_extResult[AdditionalGlobalSyncSpace],
                SharkFloatParams::NumDebugStates * sizeof(DebugStateRaw),
                hipMemcpyDeviceToHost);
            if (err != hipSuccess) {
                std::cerr << "CUDA error in hipMemcpy (debug results): " << hipGetErrorString(err) << std::endl;
            }
        }
    }

    err = hipFree(g_extResult);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipFree (g_extResult): " << hipGetErrorString(err) << std::endl;
    }
    err = hipFree(comboResults);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipFree (comboResults): " << hipGetErrorString(err) << std::endl;
    }
}

#ifdef ENABLE_ADD_KERNEL
#define ExplicitlyInstantiateAdd(SharkFloatParams) \
    template void InvokeAddKernelPerf<SharkFloatParams>( \
        BenchmarkTimer &timer, \
        HpSharkAddComboResults<SharkFloatParams> &combo, \
        uint64_t numIters); \
    template void InvokeAddKernelCorrectness<SharkFloatParams>( \
        BenchmarkTimer &timer, \
        HpSharkAddComboResults<SharkFloatParams> &combo, \
        std::vector<DebugStateRaw> *debugResults);
#else
#define ExplicitlyInstantiateAdd(SharkFloatParams) ;
#endif

#ifdef ENABLE_MULTIPLY_KERNEL
#define ExplicitlyInstantiateMultiply(SharkFloatParams) \
    template void InvokeMultiplyKernelPerf<SharkFloatParams>( \
        BenchmarkTimer &timer, \
        HpSharkComboResults<SharkFloatParams> &combo, \
        uint64_t numIters); \
    template void InvokeMultiplyKernelCorrectness<SharkFloatParams>( \
        BenchmarkTimer &timer, \
        HpSharkComboResults<SharkFloatParams> &combo, \
        std::vector<DebugStateRaw> *debugResults);
#else
#define ExplicitlyInstantiateMultiply(SharkFloatParams) ;
#endif

#ifdef ENABLE_REFERENCE_KERNEL
#define ExplicitlyInstantiateHpSharkReference(SharkFloatParams) \
    template void InvokeHpSharkReferenceKernelPerf<SharkFloatParams>(\
        BenchmarkTimer &timer, \
        HpSharkReferenceResults<SharkFloatParams> &combo, \
        uint64_t numIters); \
    template void InvokeHpSharkReferenceKernelCorrectness<SharkFloatParams>( \
        BenchmarkTimer &timer, \
        HpSharkReferenceResults<SharkFloatParams> &combo, \
        std::vector<DebugStateRaw> *debugResults);
#else
#define ExplicitlyInstantiateHpSharkReference(SharkFloatParams) ;
#endif

#define ExplicitlyInstantiate(SharkFloatParams) \
    ExplicitlyInstantiateAdd(SharkFloatParams) \
    ExplicitlyInstantiateMultiply(SharkFloatParams) \
    ExplicitlyInstantiateHpSharkReference(SharkFloatParams)

ExplicitInstantiateAll();
