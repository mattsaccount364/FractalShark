#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "BenchmarkTimer.h"
#include "TestTracker.h"

#include "Tests.h"
#include "HpSharkFloat.cuh"
#include "Add.cuh"
#include "Multiply.cuh"
#include "ReferenceKaratsuba.h"
#include "DebugChecksumHost.h"

#include <iostream>
#include <vector>
#include <gmp.h>
#include <cstring>
#include <sstream>
#include <iomanip>
#include <cmath>
#include <algorithm>
#include <assert.h>


template<class SharkFloatParams>
void InvokeAddKernelPerf(
    BenchmarkTimer &timer,
    std::function<void(hipStream_t &, void *[])> kernel,
    HpSharkComboResults<SharkFloatParams> &combo,
    uint64_t numIters) {

    // Prepare kernel arguments
    // Allocate memory for carryOuts and cumulativeCarries
    uint64_t *d_tempProducts;
    constexpr auto BytesToAllocate =
        (AdditionalUInt64Global + ScratchMemoryCopies * CalculateFrameSize<SharkFloatParams>()) * sizeof(uint64_t);
    hipMalloc(&d_tempProducts, BytesToAllocate);

    HpSharkComboResults<SharkFloatParams> *comboGpu;
    hipMalloc(&comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>));
    hipMemcpy(comboGpu, &combo, sizeof(HpSharkComboResults<SharkFloatParams>), hipMemcpyHostToDevice);

    void *kernelArgs[] = {
        (void *)&comboGpu,
        (void *)&numIters,
        (void *)&d_tempProducts
    };

    hipStream_t stream = nullptr;

    if constexpr (SharkCustomStream) {
        hipStreamCreate(&stream); // Create a stream
    }

    hipDeviceProp_t prop;
    int device_id = 0;

    if constexpr (SharkCustomStream) {
        hipGetDeviceProperties(&prop, device_id);
        hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, prop.persistingL2CacheMaxSize); /* Set aside max possible size of L2 cache for persisting accesses */

        auto setAccess = [&](void *ptr, size_t num_bytes) {
            hipLaunchAttributeValue stream_attribute;                                         // Stream level attributes data structure
            stream_attribute.accessPolicyWindow.base_ptr = reinterpret_cast<void *>(ptr); // Global Memory data pointer
            stream_attribute.accessPolicyWindow.num_bytes = num_bytes;                    // Number of bytes for persisting accesses.
            // (Must be less than hipDeviceProp_t::accessPolicyMaxWindowSize)
            stream_attribute.accessPolicyWindow.hitRatio = 1.0;                          // Hint for L2 cache hit ratio for persisting accesses in the num_bytes region
            stream_attribute.accessPolicyWindow.hitProp = hipAccessPropertyPersisting; // Type of access property on cache hit
            stream_attribute.accessPolicyWindow.missProp = hipAccessPropertyStreaming;  // Type of access property on cache miss.

            //Set the attributes to a CUDA stream of type hipStream_t
            hipError_t err = hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute);
            if (err != hipSuccess) {
                std::cerr << "CUDA error in setting stream attribute: " << hipGetErrorString(err) << std::endl;
            }
            };

        setAccess(comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>));
        setAccess(d_tempProducts, 32 * SharkFloatParams::GlobalNumUint32 * sizeof(uint64_t));
    }

    {
        ScopedBenchmarkStopper stopper{ timer };
        kernel(stream, kernelArgs);
    }

    hipMemcpy(&combo, comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>), hipMemcpyDeviceToHost);

    if constexpr (SharkCustomStream) {
        hipStreamDestroy(stream); // Destroy the stream
    }

    hipFree(comboGpu);
    hipFree(d_tempProducts);
}

template<class SharkFloatParams>
void InvokeAddKernelPerf(
    BenchmarkTimer &timer,
    std::function<void(void *[])> kernel,
    const HpSharkFloat<SharkFloatParams> &xNum,
    const HpSharkFloat<SharkFloatParams> &yNum,
    HpSharkFloat<SharkFloatParams> &gpuResult2,
    uint64_t numIters) {

    // Perform the calculation on the GPU
    HpSharkFloat<SharkFloatParams> *xGpu;
    hipMalloc(&xGpu, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemcpy(xGpu, &xNum, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyHostToDevice);

    HpSharkFloat<SharkFloatParams> *yGpu;
    hipMalloc(&yGpu, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemcpy(yGpu, &yNum, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyHostToDevice);

    HpSharkFloat<SharkFloatParams> *internalGpuResult2;
    hipMalloc(&internalGpuResult2, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(internalGpuResult2, 0, sizeof(HpSharkFloat<SharkFloatParams>));

    uint32_t *g_extResult;
    hipMalloc(&g_extResult, (SharkFloatParams::GlobalNumUint32 + 2) * sizeof(uint32_t));

    // Prepare kernel arguments
    void *kernelArgs[] = {
        (void *)&xGpu,
        (void *)&yGpu,
        (void *)&internalGpuResult2,
        (void *)&numIters,
        (void *)&g_extResult
    };

    // Launch the cooperative kernel
    {
        ScopedBenchmarkStopper stopper{ timer };
        kernel(kernelArgs);
    }

    hipMemcpy(&gpuResult2, internalGpuResult2, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(g_extResult);
    hipFree(internalGpuResult2);
    hipFree(yGpu);
    hipFree(xGpu);
}

template<class SharkFloatParams, Operator sharkOperator>
void InvokeMultiplyKernelCorrectness(
    BenchmarkTimer &timer,
    std::function<void(void *[])> kernel,
    HpSharkComboResults<SharkFloatParams> &combo,
    std::vector<DebugStateRaw> *debugResults) {

    static constexpr bool DebugInitCudaMemory = true;

    // Prepare kernel arguments
    // Allocate memory for carryOuts and cumulativeCarries
    uint64_t *d_tempProducts;
    constexpr auto BytesToAllocate =
        (AdditionalUInt64Global + ScratchMemoryCopies * CalculateFrameSize<SharkFloatParams>()) * sizeof(uint64_t);
    hipMalloc(&d_tempProducts, BytesToAllocate);

    if constexpr (!DebugInitCudaMemory) {
        hipMemset(d_tempProducts, 0, BytesToAllocate);
    } else {
        hipMemset(d_tempProducts, 0xCD, BytesToAllocate);
    }

    HpSharkComboResults<SharkFloatParams> *comboGpu;
    hipMalloc(&comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>));
    hipMemcpy(comboGpu, &combo, sizeof(HpSharkComboResults<SharkFloatParams>), hipMemcpyHostToDevice);

    if constexpr (!DebugInitCudaMemory) {
        hipMemset(&comboGpu->ResultX2, 0, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboGpu->ResultXY, 0, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboGpu->ResultY2, 0, sizeof(HpSharkFloat<SharkFloatParams>));
    } else {
        hipMemset(&comboGpu->ResultX2, 0xCD, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboGpu->ResultXY, 0xCD, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboGpu->ResultY2, 0xCD, sizeof(HpSharkFloat<SharkFloatParams>));
    }

    void *kernelArgs[] = {
        (void *)&comboGpu,
        (void *)&d_tempProducts
    };

    {
        ScopedBenchmarkStopper stopper{ timer };
        kernel(kernelArgs);
    }

    hipMemcpy(&combo, comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>), hipMemcpyDeviceToHost);

    if (debugResults != nullptr) {
        if constexpr (SharkDebugChecksums) {
            debugResults->resize(SharkFloatParams::NumDebugStates);
            hipMemcpy(
                debugResults->data(),
                &d_tempProducts[AdditionalGlobalSyncSpace],
                SharkFloatParams::NumDebugStates * sizeof(DebugStateRaw),
                hipMemcpyDeviceToHost);
        }
    }

    hipFree(comboGpu);
    hipFree(d_tempProducts);
}

template<class SharkFloatParams, Operator sharkOperator>
void InvokeAddKernelCorrectness(
    BenchmarkTimer &timer,
    std::function<void(void *[])> kernel,
    const HpSharkFloat<SharkFloatParams> &xNum,
    const HpSharkFloat<SharkFloatParams> &yNum,
    HpSharkFloat<SharkFloatParams> &gpuResult,
    std::vector<DebugStateRaw> *debugResults) {

    // Perform the calculation on the GPU
    HpSharkFloat<SharkFloatParams> *xGpu;
    hipMalloc(&xGpu, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemcpy(xGpu, &xNum, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyHostToDevice);

    HpSharkFloat<SharkFloatParams> *yGpu;
    hipMalloc(&yGpu, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemcpy(yGpu, &yNum, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyHostToDevice);

    HpSharkFloat<SharkFloatParams> *internalGpuResult;
    hipMalloc(&internalGpuResult, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(internalGpuResult, 0, sizeof(HpSharkFloat<SharkFloatParams>));

    constexpr auto BytesToAllocate =
        (AdditionalUInt64Global + SharkFloatParams::GlobalNumUint32 * 4) * sizeof(uint32_t);
    uint32_t *g_extResult;
    hipMalloc(&g_extResult, BytesToAllocate);

    // Prepare kernel arguments
    void *kernelArgs[] = {
        (void *)&xGpu,
        (void *)&yGpu,
        (void *)&internalGpuResult,
        (void *)&g_extResult
    };

    {
        ScopedBenchmarkStopper stopper{ timer };
        ComputeAddGpu<SharkFloatParams>(kernelArgs);
    }

    hipMemcpy(&gpuResult, internalGpuResult, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyDeviceToHost);

    if (debugResults != nullptr) {
        if constexpr (SharkDebugChecksums) {
            debugResults->resize(SharkFloatParams::NumDebugStates);
            hipMemcpy(
                debugResults->data(),
                &g_extResult[AdditionalGlobalSyncSpace],
                SharkFloatParams::NumDebugStates * sizeof(DebugStateRaw),
                hipMemcpyDeviceToHost);
        }
    }

    hipFree(g_extResult);
    hipFree(internalGpuResult);
    hipFree(yGpu);
    hipFree(xGpu);
}

#define ExplicitlyInstantiate(SharkFloatParams) \
    template void InvokeAddKernelPerf<SharkFloatParams>( \
        BenchmarkTimer &timer, \
        std::function<void(hipStream_t &, void *[])> kernel, \
        HpSharkComboResults<SharkFloatParams> &combo, \
        uint64_t numIters); \
    template void InvokeAddKernelPerf<SharkFloatParams>( \
        BenchmarkTimer &timer, \
        std::function<void(void*[])> kernel, \
        const HpSharkFloat<SharkFloatParams> &xNum, \
        const HpSharkFloat<SharkFloatParams> &yNum, \
        HpSharkFloat<SharkFloatParams> &gpuResult2, \
        uint64_t numIters); \
    template void InvokeMultiplyKernelCorrectness<SharkFloatParams, Operator::MultiplyKaratsubaV2>( \
        BenchmarkTimer &timer, \
        std::function<void(void*[])> kernel, \
        HpSharkComboResults<SharkFloatParams> &combo, \
        std::vector<DebugStateRaw> *debugResults); \
    template void InvokeAddKernelCorrectness<SharkFloatParams, Operator::Add>( \
        BenchmarkTimer &timer, \
        std::function<void(void*[])> kernel, \
        const HpSharkFloat<SharkFloatParams> &xNum, \
        const HpSharkFloat<SharkFloatParams> &yNum, \
        HpSharkFloat<SharkFloatParams> &gpuResult, \
        std::vector<DebugStateRaw> *debugResults);

#ifdef SHARK_INCLUDE_KERNELS
ExplicitInstantiateAll();
#endif