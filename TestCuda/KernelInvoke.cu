#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "BenchmarkTimer.h"
#include "TestTracker.h"

#include "Tests.h"
#include "HpSharkFloat.cuh"
#include "Add.cuh"
#include "Multiply.cuh"
#include "MultiplyNTT.cuh"
#include "HpSharkReferenceOrbit.cuh"

#include <iostream>
#include <vector>
#include <gmp.h>
#include <cstring>
#include <sstream>
#include <iomanip>
#include <cmath>
#include <algorithm>
#include <assert.h>

template<class SharkFloatParams>
void InvokeHpSharkReferenceKernelPerf(
    BenchmarkTimer &timer,
    HpSharkReferenceResults<SharkFloatParams> &combo,
    uint64_t numIters) {

    // Prepare kernel arguments
    // Allocate memory for carryOuts and cumulativeCarries
    uint64_t *d_tempProducts;
    constexpr auto BytesToAllocate =
        (AdditionalUInt64Global + ScratchMemoryCopies * CalculateMultiplyFrameSize<SharkFloatParams>()) * sizeof(uint64_t);
    hipMalloc(&d_tempProducts, BytesToAllocate);

    if constexpr (!SharkTestInitCudaMemory) {
        hipMemset(d_tempProducts, 0, BytesToAllocate);
    } else {
        hipMemset(d_tempProducts, 0xCD, BytesToAllocate);
    }

    HpSharkReferenceResults<SharkFloatParams> *comboGpu;
    hipMalloc(&comboGpu, sizeof(HpSharkReferenceResults<SharkFloatParams>));
    hipMemcpy(comboGpu, &combo, sizeof(HpSharkReferenceResults<SharkFloatParams>), hipMemcpyHostToDevice);

    uint8_t byteToSet = SharkTestInitCudaMemory ? 0xCD : 0;

    hipMemset(&comboGpu->Add.A_X2, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Add.B_Y2, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Add.D_2X, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Add.Result1_A_B_C, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Add.Result2_D_E, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Multiply.ResultX2, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Multiply.Result2XY, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Multiply.ResultY2, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));

    void *kernelArgs[] = {
        (void *)&comboGpu,
        (void *)&numIters,
        (void *)&d_tempProducts
    };

    hipStream_t stream = nullptr;

    if constexpr (SharkCustomStream) {
        auto res = hipStreamCreate(&stream); // Create a stream

        if (res != hipSuccess) {
            std::cerr << "CUDA error in creating stream: " << hipGetErrorString(res) << std::endl;
        }
    }

    hipDeviceProp_t prop;
    int device_id = 0;

    if constexpr (SharkCustomStream) {
        hipGetDeviceProperties(&prop, device_id);
        hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, prop.persistingL2CacheMaxSize); /* Set aside max possible size of L2 cache for persisting accesses */

        auto setAccess = [&](void *ptr, size_t num_bytes) {
            hipLaunchAttributeValue stream_attribute;                                         // Stream level attributes data structure
            stream_attribute.accessPolicyWindow.base_ptr = reinterpret_cast<void *>(ptr); // Global Memory data pointer
            stream_attribute.accessPolicyWindow.num_bytes = num_bytes;                    // Number of bytes for persisting accesses.
            // (Must be less than hipDeviceProp_t::accessPolicyMaxWindowSize)
            stream_attribute.accessPolicyWindow.hitRatio = 1.0;                          // Hint for L2 cache hit ratio for persisting accesses in the num_bytes region
            stream_attribute.accessPolicyWindow.hitProp = hipAccessPropertyPersisting; // Type of access property on cache hit
            stream_attribute.accessPolicyWindow.missProp = hipAccessPropertyStreaming;  // Type of access property on cache miss.

            //Set the attributes to a CUDA stream of type hipStream_t
            hipError_t err = hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute);
            if (err != hipSuccess) {
                std::cerr << "CUDA error in setting stream attribute: " << hipGetErrorString(err) << std::endl;
            }
            };

        setAccess(comboGpu, sizeof(HpSharkReferenceResults<SharkFloatParams>));
        setAccess(d_tempProducts, 32 * SharkFloatParams::GlobalNumUint32 * sizeof(uint64_t));
    }

    {
        ScopedBenchmarkStopper stopper{ timer };
        ComputeHpSharkReferenceGpuLoop<SharkFloatParams>(stream, kernelArgs);
    }

    hipMemcpy(&combo, comboGpu, sizeof(HpSharkReferenceResults<SharkFloatParams>), hipMemcpyDeviceToHost);

    hipFree(comboGpu);
    hipFree(d_tempProducts);

    if constexpr (SharkCustomStream) {
        auto res = hipStreamDestroy(stream); // Destroy the stream

        if (res != hipSuccess) {
            std::cerr << "CUDA error in destroying stream: " << hipGetErrorString(res) << std::endl;
        }
    }
}

template<class SharkFloatParams>
void InvokeMultiplyKernelPerf(
    BenchmarkTimer &timer,
    HpSharkComboResults<SharkFloatParams> &combo,
    uint64_t numIters) {

    // Prepare kernel arguments
    // Allocate memory for carryOuts and cumulativeCarries
    uint64_t *d_tempProducts;
    constexpr auto BytesToAllocate =
        (AdditionalUInt64Global + ScratchMemoryCopies * CalculateMultiplyFrameSize<SharkFloatParams>()) * sizeof(uint64_t);
    hipMalloc(&d_tempProducts, BytesToAllocate);

    HpSharkComboResults<SharkFloatParams> *comboGpu;
    hipMalloc(&comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>));
    hipMemcpy(comboGpu, &combo, sizeof(HpSharkComboResults<SharkFloatParams>), hipMemcpyHostToDevice);

    void *kernelArgs[] = {
        (void *)&comboGpu,
        (void *)&numIters,
        (void *)&d_tempProducts
    };

    hipStream_t stream = nullptr;

    if constexpr (SharkCustomStream) {
        hipStreamCreate(&stream); // Create a stream
    }

    hipDeviceProp_t prop;
    int device_id = 0;

    if constexpr (SharkCustomStream) {
        hipGetDeviceProperties(&prop, device_id);
        hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, prop.persistingL2CacheMaxSize); /* Set aside max possible size of L2 cache for persisting accesses */

        auto setAccess = [&](void *ptr, size_t num_bytes) {
            hipLaunchAttributeValue stream_attribute;                                         // Stream level attributes data structure
            stream_attribute.accessPolicyWindow.base_ptr = reinterpret_cast<void *>(ptr); // Global Memory data pointer
            stream_attribute.accessPolicyWindow.num_bytes = num_bytes;                    // Number of bytes for persisting accesses.
            // (Must be less than hipDeviceProp_t::accessPolicyMaxWindowSize)
            stream_attribute.accessPolicyWindow.hitRatio = 1.0;                          // Hint for L2 cache hit ratio for persisting accesses in the num_bytes region
            stream_attribute.accessPolicyWindow.hitProp = hipAccessPropertyPersisting; // Type of access property on cache hit
            stream_attribute.accessPolicyWindow.missProp = hipAccessPropertyStreaming;  // Type of access property on cache miss.

            //Set the attributes to a CUDA stream of type hipStream_t
            hipError_t err = hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute);
            if (err != hipSuccess) {
                std::cerr << "CUDA error in setting stream attribute: " << hipGetErrorString(err) << std::endl;
            }
            };

        setAccess(comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>));
        setAccess(d_tempProducts, 32 * SharkFloatParams::GlobalNumUint32 * sizeof(uint64_t));
    }

    {
        ScopedBenchmarkStopper stopper{ timer };
        ComputeMultiplyKaratsubaV2GpuTestLoop<SharkFloatParams>(stream, kernelArgs);
    }

    hipMemcpy(&combo, comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>), hipMemcpyDeviceToHost);

    if constexpr (SharkCustomStream) {
        hipStreamDestroy(stream); // Destroy the stream
    }

    hipFree(comboGpu);
    hipFree(d_tempProducts);
}

template <class SharkFloatParams>
void
InvokeMultiplyNTTKernelPerf(BenchmarkTimer& timer,
                         HpSharkComboResults<SharkFloatParams>& combo,
                         uint64_t numIters)
{

    // Prepare kernel arguments
    // Allocate memory for carryOuts and cumulativeCarries
    uint64_t* d_tempProducts;
    constexpr auto BytesToAllocate =
        (AdditionalUInt64Global + ScratchMemoryCopies * CalculateMultiplyFrameSize<SharkFloatParams>()) *
        sizeof(uint64_t);
    hipMalloc(&d_tempProducts, BytesToAllocate);

    HpSharkComboResults<SharkFloatParams>* comboGpu;
    hipMalloc(&comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>));
    hipMemcpy(comboGpu, &combo, sizeof(HpSharkComboResults<SharkFloatParams>), hipMemcpyHostToDevice);

    void* kernelArgs[] = {(void*)&comboGpu, (void*)&numIters, (void*)&d_tempProducts};

    hipStream_t stream = nullptr;

    if constexpr (SharkCustomStream) {
        hipStreamCreate(&stream); // Create a stream
    }

    hipDeviceProp_t prop;
    int device_id = 0;

    if constexpr (SharkCustomStream) {
        hipGetDeviceProperties(&prop, device_id);
        hipDeviceSetLimit(cudaLimitPersistingL2CacheSize,
                           prop.persistingL2CacheMaxSize); /* Set aside max possible size of L2 cache for
                                                              persisting accesses */

        auto setAccess = [&](void* ptr, size_t num_bytes) {
            hipLaunchAttributeValue stream_attribute; // Stream level attributes data structure
            stream_attribute.accessPolicyWindow.base_ptr =
                reinterpret_cast<void*>(ptr); // Global Memory data pointer
            stream_attribute.accessPolicyWindow.num_bytes =
                num_bytes; // Number of bytes for persisting accesses.
            // (Must be less than hipDeviceProp_t::accessPolicyMaxWindowSize)
            stream_attribute.accessPolicyWindow.hitRatio =
                1.0; // Hint for L2 cache hit ratio for persisting accesses in the num_bytes region
            stream_attribute.accessPolicyWindow.hitProp =
                hipAccessPropertyPersisting; // Type of access property on cache hit
            stream_attribute.accessPolicyWindow.missProp =
                hipAccessPropertyStreaming; // Type of access property on cache miss.

            // Set the attributes to a CUDA stream of type hipStream_t
            hipError_t err =
                hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute);
            if (err != hipSuccess) {
                std::cerr << "CUDA error in setting stream attribute: " << hipGetErrorString(err)
                          << std::endl;
            }
        };

        setAccess(comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>));
        setAccess(d_tempProducts, 32 * SharkFloatParams::GlobalNumUint32 * sizeof(uint64_t));
    }

    {
        ScopedBenchmarkStopper stopper{timer};
        ComputeMultiplyNTTGpuTestLoop<SharkFloatParams>(stream, kernelArgs);
    }

    hipMemcpy(&combo, comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>), hipMemcpyDeviceToHost);

    if constexpr (SharkCustomStream) {
        hipStreamDestroy(stream); // Destroy the stream
    }

    hipFree(comboGpu);
    hipFree(d_tempProducts);
}

template<class SharkFloatParams>
void InvokeAddKernelPerf(
    BenchmarkTimer &timer,
    HpSharkAddComboResults<SharkFloatParams> &combo,
    uint64_t numIters) {

    // Perform the calculation on the GPU
    HpSharkAddComboResults<SharkFloatParams> *comboResults;
    hipMalloc(&comboResults, sizeof(HpSharkAddComboResults<SharkFloatParams>));
    hipMemcpy(comboResults, &combo, sizeof(HpSharkAddComboResults<SharkFloatParams>), hipMemcpyHostToDevice);

    constexpr auto BytesToAllocate =
        (AdditionalUInt64Global + CalculateAddFrameSize<SharkFloatParams>()) * sizeof(uint64_t);
    uint64_t *g_extResult;
    hipMalloc(&g_extResult, BytesToAllocate);

    // Prepare kernel arguments
    void *kernelArgs[] = {
        (void *)&comboResults,
        (void *)&numIters,
        (void *)&g_extResult
    };

    // Launch the cooperative kernel
    {
        ScopedBenchmarkStopper stopper{ timer };
        ComputeAddGpuTestLoop<SharkFloatParams>(kernelArgs);
    }

    hipMemcpy(&combo, comboResults, sizeof(HpSharkAddComboResults<SharkFloatParams>), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(g_extResult);
    hipFree(comboResults);
}

template<class SharkFloatParams>
void InvokeHpSharkReferenceKernelCorrectness(
    BenchmarkTimer &timer,
    HpSharkReferenceResults<SharkFloatParams> &combo,
    DebugGpuCombo *debugCombo) {

    // Prepare kernel arguments
    // Allocate memory for carryOuts and cumulativeCarries

    // TODO max of add/multiply frame size
    // TODO checksum handled
    uint64_t *d_tempProducts;
    constexpr auto BytesToAllocate =
        (AdditionalUInt64Global + ScratchMemoryCopies * CalculateMultiplyFrameSize<SharkFloatParams>()) * sizeof(uint64_t);
    hipMalloc(&d_tempProducts, BytesToAllocate);

    if constexpr (!SharkTestInitCudaMemory) {
        hipMemset(d_tempProducts, 0, BytesToAllocate);
    } else {
        hipMemset(d_tempProducts, 0xCD, BytesToAllocate);
    }

    HpSharkReferenceResults<SharkFloatParams> *comboGpu;
    hipMalloc(&comboGpu, sizeof(HpSharkReferenceResults<SharkFloatParams>));
    hipMemcpy(comboGpu, &combo, sizeof(HpSharkReferenceResults<SharkFloatParams>), hipMemcpyHostToDevice);

    uint8_t byteToSet = SharkTestInitCudaMemory ? 0xCD : 0;

    hipMemset(&comboGpu->Add.A_X2, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Add.B_Y2, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Add.D_2X, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Add.Result1_A_B_C, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Add.Result2_D_E, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Multiply.ResultX2, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Multiply.Result2XY, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(&comboGpu->Multiply.ResultY2, byteToSet, sizeof(HpSharkFloat<SharkFloatParams>));

    void *kernelArgs[] = {
        (void *)&comboGpu,
        (void *)&d_tempProducts
    };

    {
        ScopedBenchmarkStopper stopper{ timer };
        ComputeHpSharkReferenceGpu<SharkFloatParams>(kernelArgs);
    }

    hipMemcpy(&combo, comboGpu, sizeof(HpSharkReferenceResults<SharkFloatParams>), hipMemcpyDeviceToHost);

    if (debugCombo != nullptr) {
        if constexpr (SharkDebugChecksums) {
            debugCombo->States.resize(SharkFloatParams::NumDebugStates);
            hipMemcpy(
                debugCombo->States.data(),
                &d_tempProducts[AdditionalChecksumsOffset],
                SharkFloatParams::NumDebugStates * sizeof(DebugStateRaw),
                hipMemcpyDeviceToHost);
        }

        if constexpr (SharkPrintMultiplyCounts) {
            debugCombo->MultiplyCounts.resize(SharkFloatParams::NumDebugMultiplyCounts);
            hipMemcpy(
                debugCombo->MultiplyCounts.data(),
                &d_tempProducts[AdditionalMultipliesOffset],
                SharkFloatParams::NumDebugMultiplyCounts * sizeof(DebugMultiplyCountRaw),
                hipMemcpyDeviceToHost);
        }
    }

    hipFree(comboGpu);
    hipFree(d_tempProducts);
}

template<class SharkFloatParams>
void InvokeMultiplyKaratsubaKernelCorrectness(
    BenchmarkTimer &timer,
    HpSharkComboResults<SharkFloatParams> &combo,
    DebugGpuCombo *debugCombo) {

    // Prepare kernel arguments
    // Allocate memory for carryOuts and cumulativeCarries
    uint64_t *d_tempProducts;
    constexpr auto BytesToAllocate =
        (AdditionalUInt64Global + ScratchMemoryCopies * CalculateMultiplyFrameSize<SharkFloatParams>()) * sizeof(uint64_t);
    hipMalloc(&d_tempProducts, BytesToAllocate);

    if constexpr (!SharkTestInitCudaMemory) {
        hipMemset(d_tempProducts, 0, BytesToAllocate);
    } else {
        hipMemset(d_tempProducts, 0xCD, BytesToAllocate);
    }

    HpSharkComboResults<SharkFloatParams> *comboGpu;
    hipMalloc(&comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>));
    hipMemcpy(comboGpu, &combo, sizeof(HpSharkComboResults<SharkFloatParams>), hipMemcpyHostToDevice);

    if constexpr (!SharkTestInitCudaMemory) {
        hipMemset(&comboGpu->ResultX2, 0, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboGpu->Result2XY, 0, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboGpu->ResultY2, 0, sizeof(HpSharkFloat<SharkFloatParams>));
    } else {
        hipMemset(&comboGpu->ResultX2, 0xCD, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboGpu->Result2XY, 0xCD, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboGpu->ResultY2, 0xCD, sizeof(HpSharkFloat<SharkFloatParams>));
    }

    void *kernelArgs[] = {
        (void *)&comboGpu,
        (void *)&d_tempProducts
    };

    {
        ScopedBenchmarkStopper stopper{ timer };
        ComputeMultiplyKaratsubaV2Gpu<SharkFloatParams>(kernelArgs);
    }

    hipMemcpy(&combo, comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>), hipMemcpyDeviceToHost);

    if (debugCombo != nullptr) {
        if constexpr (SharkDebugChecksums) {
            debugCombo->States.resize(SharkFloatParams::NumDebugStates);
            hipMemcpy(
                debugCombo->States.data(),
                &d_tempProducts[AdditionalChecksumsOffset],
                SharkFloatParams::NumDebugStates * sizeof(DebugStateRaw),
                hipMemcpyDeviceToHost);
        }

        if constexpr (SharkPrintMultiplyCounts) {
            debugCombo->MultiplyCounts.resize(SharkFloatParams::NumDebugMultiplyCounts);
            hipMemcpy(
                debugCombo->MultiplyCounts.data(),
                &d_tempProducts[AdditionalMultipliesOffset],
                SharkFloatParams::NumDebugMultiplyCounts * sizeof(DebugMultiplyCountRaw),
                hipMemcpyDeviceToHost);
        }
    }

    hipFree(comboGpu);
    hipFree(d_tempProducts);
}

template <class SharkFloatParams>
void
InvokeMultiplyNTTKernelCorrectness(BenchmarkTimer& timer,
                                         HpSharkComboResults<SharkFloatParams>& combo,
                                         DebugGpuCombo* debugCombo)
{

    // Prepare kernel arguments
    // Allocate memory for carryOuts and cumulativeCarries
    uint64_t* d_tempProducts;
    constexpr auto BytesToAllocate =
        (AdditionalUInt64Global + ScratchMemoryCopies * CalculateMultiplyFrameSize<SharkFloatParams>()) *
        sizeof(uint64_t);
    hipMalloc(&d_tempProducts, BytesToAllocate);

    if constexpr (!SharkTestInitCudaMemory) {
        hipMemset(d_tempProducts, 0, BytesToAllocate);
    } else {
        hipMemset(d_tempProducts, 0xCD, BytesToAllocate);
    }

    HpSharkComboResults<SharkFloatParams>* comboGpu;
    hipMalloc(&comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>));
    hipMemcpy(comboGpu, &combo, sizeof(HpSharkComboResults<SharkFloatParams>), hipMemcpyHostToDevice);

    if constexpr (!SharkTestInitCudaMemory) {
        hipMemset(&comboGpu->ResultX2, 0, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboGpu->Result2XY, 0, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboGpu->ResultY2, 0, sizeof(HpSharkFloat<SharkFloatParams>));
    } else {
        hipMemset(&comboGpu->ResultX2, 0xCD, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboGpu->Result2XY, 0xCD, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboGpu->ResultY2, 0xCD, sizeof(HpSharkFloat<SharkFloatParams>));
    }

    void* kernelArgs[] = {(void*)&comboGpu, (void*)&d_tempProducts};

    {
        ScopedBenchmarkStopper stopper{timer};
        ComputeMultiplyNTTGpu<SharkFloatParams>(kernelArgs);
    }

    hipMemcpy(&combo, comboGpu, sizeof(HpSharkComboResults<SharkFloatParams>), hipMemcpyDeviceToHost);

    if (debugCombo != nullptr) {
        if constexpr (SharkDebugChecksums) {
            debugCombo->States.resize(SharkFloatParams::NumDebugStates);
            hipMemcpy(debugCombo->States.data(),
                       &d_tempProducts[AdditionalChecksumsOffset],
                       SharkFloatParams::NumDebugStates * sizeof(DebugStateRaw),
                       hipMemcpyDeviceToHost);
        }

        if constexpr (SharkPrintMultiplyCounts) {
            debugCombo->MultiplyCounts.resize(SharkFloatParams::NumDebugMultiplyCounts);
            hipMemcpy(debugCombo->MultiplyCounts.data(),
                       &d_tempProducts[AdditionalMultipliesOffset],
                       SharkFloatParams::NumDebugMultiplyCounts * sizeof(DebugMultiplyCountRaw),
                       hipMemcpyDeviceToHost);
        }
    }

    hipFree(comboGpu);
    hipFree(d_tempProducts);
}

template<class SharkFloatParams>
void InvokeAddKernelCorrectness(
    BenchmarkTimer &timer,
    HpSharkAddComboResults<SharkFloatParams> &combo,
    DebugGpuCombo *debugCombo) {

    // Perform the calculation on the GPU
    HpSharkAddComboResults<SharkFloatParams> *comboResults;
    hipMalloc(&comboResults, sizeof(HpSharkAddComboResults<SharkFloatParams>));
    hipMemcpy(comboResults, &combo, sizeof(HpSharkAddComboResults<SharkFloatParams>), hipMemcpyHostToDevice);

    if constexpr (!SharkTestInitCudaMemory) {
        hipMemset(&comboResults->Result1_A_B_C, 0, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboResults->Result2_D_E, 0, sizeof(HpSharkFloat<SharkFloatParams>));
    } else {
        hipMemset(&comboResults->Result1_A_B_C, 0xCD, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(&comboResults->Result2_D_E, 0xCD, sizeof(HpSharkFloat<SharkFloatParams>));
    }

    constexpr auto BytesToAllocate =
        (AdditionalUInt64Global + CalculateAddFrameSize<SharkFloatParams>()) * sizeof(uint64_t);
    uint64_t *g_extResult;
    hipMalloc(&g_extResult, BytesToAllocate);

    // Prepare kernel arguments
    void *kernelArgs[] = {
        (void *)&comboResults,
        (void *)&g_extResult
    };

    {
        ScopedBenchmarkStopper stopper{ timer };
        ComputeAddGpu<SharkFloatParams>(kernelArgs);
    }

    hipMemcpy(&combo, comboResults, sizeof(HpSharkAddComboResults<SharkFloatParams>), hipMemcpyDeviceToHost);

    if (debugCombo != nullptr) {
        if constexpr (SharkDebugChecksums) {
            debugCombo->States.resize(SharkFloatParams::NumDebugStates);
            hipMemcpy(
                debugCombo->States.data(),
                &g_extResult[AdditionalChecksumsOffset],
                SharkFloatParams::NumDebugStates * sizeof(DebugStateRaw),
                hipMemcpyDeviceToHost);
        }

        if constexpr (SharkPrintMultiplyCounts) {
            debugCombo->MultiplyCounts.resize(SharkFloatParams::NumDebugMultiplyCounts);
            hipMemcpy(
                debugCombo->MultiplyCounts.data(),
                &g_extResult[AdditionalMultipliesOffset],
                SharkFloatParams::NumDebugMultiplyCounts * sizeof(DebugMultiplyCountRaw),
                hipMemcpyDeviceToHost);
        }
    }

    hipFree(g_extResult);
    hipFree(comboResults);
}

#ifdef ENABLE_ADD_KERNEL
#define ExplicitlyInstantiateAdd(SharkFloatParams) \
    template void InvokeAddKernelPerf<SharkFloatParams>( \
        BenchmarkTimer &timer, \
        HpSharkAddComboResults<SharkFloatParams> &combo, \
        uint64_t numIters); \
    template void InvokeAddKernelCorrectness<SharkFloatParams>( \
        BenchmarkTimer &timer, \
        HpSharkAddComboResults<SharkFloatParams> &combo, \
        DebugGpuCombo *debugCombo);
#else
#define ExplicitlyInstantiateAdd(SharkFloatParams) ;
#endif

#ifdef ENABLE_MULTIPLY_KARATSUBA_KERNEL
#define ExplicitlyInstantiateMultiply(SharkFloatParams) \
    template void InvokeMultiplyKernelPerf<SharkFloatParams>( \
        BenchmarkTimer &timer, \
        HpSharkComboResults<SharkFloatParams> &combo, \
        uint64_t numIters); \
    template void InvokeMultiplyKaratsubaKernelCorrectness<SharkFloatParams>( \
        BenchmarkTimer &timer, \
        HpSharkComboResults<SharkFloatParams> &combo, \
        DebugGpuCombo *debugCombo);
#else
#define ExplicitlyInstantiateMultiply(SharkFloatParams) ;
#endif

#ifdef ENABLE_MULTIPLY_FFT2_KERNEL
#define ExplicitlyInstantiateMultiplyNTT(SharkFloatParams)                                                 \
    template void InvokeMultiplyNTTKernelPerf<SharkFloatParams>(                                           \
        BenchmarkTimer & timer, HpSharkComboResults<SharkFloatParams> & combo, uint64_t numIters);      \
    template void InvokeMultiplyNTTKernelCorrectness<SharkFloatParams>(                           \
        BenchmarkTimer & timer,                                                                         \
        HpSharkComboResults<SharkFloatParams> & combo,                                                  \
        DebugGpuCombo * debugCombo);
#else
#define ExplicitlyInstantiateMultiplyNTT(SharkFloatParams) ;
#endif


#ifdef ENABLE_REFERENCE_KERNEL
#define ExplicitlyInstantiateHpSharkReference(SharkFloatParams) \
    template void InvokeHpSharkReferenceKernelPerf<SharkFloatParams>(\
        BenchmarkTimer &timer, \
        HpSharkReferenceResults<SharkFloatParams> &combo, \
        uint64_t numIters); \
    template void InvokeHpSharkReferenceKernelCorrectness<SharkFloatParams>( \
        BenchmarkTimer &timer, \
        HpSharkReferenceResults<SharkFloatParams> &combo, \
        DebugGpuCombo *debugCombo);
#else
#define ExplicitlyInstantiateHpSharkReference(SharkFloatParams) ;
#endif

#define ExplicitlyInstantiate(SharkFloatParams) \
    ExplicitlyInstantiateAdd(SharkFloatParams) \
    ExplicitlyInstantiateMultiply(SharkFloatParams) \
    ExplicitlyInstantiateMultiplyNTT(SharkFloatParams) \
    ExplicitlyInstantiateHpSharkReference(SharkFloatParams)

ExplicitInstantiateAll();
