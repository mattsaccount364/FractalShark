#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "BenchmarkTimer.h"
#include "TestTracker.h"

#include "Tests.h"
#include "HpSharkFloat.cuh"
#include "Add.cuh"
#include "Multiply.cuh"
#include "ReferenceKaratsuba.h"
#include "DebugChecksumHost.h"

#include <iostream>
#include <vector>
#include <gmp.h>
#include <cstring>
#include <sstream>
#include <iomanip>
#include <cmath>
#include <algorithm>
#include <assert.h>


template<class SharkFloatParams>
void InvokeMultiplyKernel(
    BenchmarkTimer &timer,
    std::function<void(hipStream_t &, void *[])> kernel,
    const HpSharkFloat<SharkFloatParams> &xNum,
    const HpSharkFloat<SharkFloatParams> &yNum,
    HpSharkFloat<SharkFloatParams> &gpuResult2) {

    // Prepare kernel arguments
    // Allocate memory for carryOuts and cumulativeCarries
    uint64_t *d_tempProducts;
    constexpr auto BytesToAllocate =
        (AdditionalUInt64Global + ScratchMemoryCopies * CalculateFrameSize<SharkFloatParams>()) * sizeof(uint64_t);
    hipMalloc(&d_tempProducts, BytesToAllocate);

    // Perform the calculation on the GPU
    HpSharkFloat<SharkFloatParams> *xGpu;
    hipMalloc(&xGpu, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemcpy(xGpu, &xNum, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyHostToDevice);

    HpSharkFloat<SharkFloatParams> *yGpu;
    hipMalloc(&yGpu, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemcpy(yGpu, &yNum, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyHostToDevice);

    HpSharkFloat<SharkFloatParams> *internalGpuResult2;
    hipMalloc(&internalGpuResult2, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(internalGpuResult2, 0, sizeof(HpSharkFloat<SharkFloatParams>));

    void *kernelArgs[] = {
        (void *)&xGpu,
        (void *)&yGpu,
        (void *)&internalGpuResult2,
        (void *)&d_tempProducts
    };

    hipStream_t stream = nullptr;

    if constexpr (SharkCustomStream) {
        hipStreamCreate(&stream); // Create a stream
    }

    hipDeviceProp_t prop;
    int device_id = 0;

    if constexpr (SharkCustomStream) {
        hipGetDeviceProperties(&prop, device_id);
        hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, prop.persistingL2CacheMaxSize); /* Set aside max possible size of L2 cache for persisting accesses */

        auto setAccess = [&](void *ptr, size_t num_bytes) {
            hipLaunchAttributeValue stream_attribute;                                         // Stream level attributes data structure
            stream_attribute.accessPolicyWindow.base_ptr = reinterpret_cast<void *>(ptr); // Global Memory data pointer
            stream_attribute.accessPolicyWindow.num_bytes = num_bytes;                    // Number of bytes for persisting accesses.
            // (Must be less than hipDeviceProp_t::accessPolicyMaxWindowSize)
            stream_attribute.accessPolicyWindow.hitRatio = 1.0;                          // Hint for L2 cache hit ratio for persisting accesses in the num_bytes region
            stream_attribute.accessPolicyWindow.hitProp = hipAccessPropertyPersisting; // Type of access property on cache hit
            stream_attribute.accessPolicyWindow.missProp = hipAccessPropertyStreaming;  // Type of access property on cache miss.

            //Set the attributes to a CUDA stream of type hipStream_t
            hipError_t err = hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute);
            if (err != hipSuccess) {
                std::cerr << "CUDA error in setting stream attribute: " << hipGetErrorString(err) << std::endl;
            }
            };

        setAccess(xGpu, sizeof(HpSharkFloat<SharkFloatParams>));
        setAccess(yGpu, sizeof(HpSharkFloat<SharkFloatParams>));
        setAccess(internalGpuResult2, sizeof(HpSharkFloat<SharkFloatParams>));
        setAccess(d_tempProducts, 32 * SharkFloatParams::GlobalNumUint32 * sizeof(uint64_t));
    }

    {
        ScopedBenchmarkStopper stopper{ timer };
        kernel(stream, kernelArgs);
    }

    hipMemcpy(&gpuResult2, internalGpuResult2, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyDeviceToHost);

    if constexpr (SharkCustomStream) {
        hipStreamDestroy(stream); // Destroy the stream
    }

    hipFree(internalGpuResult2);
    hipFree(yGpu);
    hipFree(xGpu);
    hipFree(d_tempProducts);
}

template<class SharkFloatParams>
void InvokeAddKernel(
    BenchmarkTimer &timer,
    std::function<void(void *[])> kernel,
    const HpSharkFloat<SharkFloatParams> &xNum,
    const HpSharkFloat<SharkFloatParams> &yNum,
    HpSharkFloat<SharkFloatParams> &gpuResult2) {

    // Allocate memory for carryOuts and cumulativeCarries
    GlobalAddBlockData *globalBlockData;
    CarryInfo *d_carryOuts;
    uint32_t *d_cumulativeCarries;
    hipMalloc(&globalBlockData, sizeof(GlobalAddBlockData));
    hipMalloc(&d_carryOuts, (SharkFloatParams::GlobalNumBlocks + 1) * sizeof(CarryInfo));
    hipMalloc(&d_cumulativeCarries, (SharkFloatParams::GlobalNumBlocks + 1) * sizeof(uint32_t));

    // Perform the calculation on the GPU
    HpSharkFloat<SharkFloatParams> *xGpu;
    hipMalloc(&xGpu, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemcpy(xGpu, &xNum, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyHostToDevice);

    HpSharkFloat<SharkFloatParams> *yGpu;
    hipMalloc(&yGpu, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemcpy(yGpu, &yNum, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyHostToDevice);

    HpSharkFloat<SharkFloatParams> *internalGpuResult2;
    hipMalloc(&internalGpuResult2, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(internalGpuResult2, 0, sizeof(HpSharkFloat<SharkFloatParams>));

    // Prepare kernel arguments
    void *kernelArgs[] = {
        (void *)&xGpu,
        (void *)&yGpu,
        (void *)&internalGpuResult2,
        (void *)&globalBlockData,
        (void *)&d_carryOuts,
        (void *)&d_cumulativeCarries
    };

    {
        ScopedBenchmarkStopper stopper{ timer };
        kernel(kernelArgs);
    }

    // Launch the cooperative kernel

    hipMemcpy(&gpuResult2, internalGpuResult2, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyDeviceToHost);

    hipFree(internalGpuResult2);
    hipFree(yGpu);
    hipFree(xGpu);

    hipFree(globalBlockData);
    hipFree(d_carryOuts);
    hipFree(d_cumulativeCarries);
}


template<class SharkFloatParams, Operator sharkOperator>
void InvokeMultiplyKernelCorrectness(
    BenchmarkTimer &timer,
    std::function<void(void *[])> kernel,
    const HpSharkFloat<SharkFloatParams> &xNum,
    const HpSharkFloat<SharkFloatParams> &yNum,
    HpSharkFloat<SharkFloatParams> &gpuResult,
    std::vector<DebugStateRaw> *debugResults) {

    // Prepare kernel arguments
    // Allocate memory for carryOuts and cumulativeCarries
    uint64_t *d_tempProducts;
    constexpr auto BytesToAllocate =
        (AdditionalUInt64Global + ScratchMemoryCopies * CalculateFrameSize<SharkFloatParams>()) * sizeof(uint64_t);
    hipMalloc(&d_tempProducts, BytesToAllocate);

    // Perform the calculation on the GPU
    HpSharkFloat<SharkFloatParams> *xGpu;
    hipMalloc(&xGpu, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemcpy(xGpu, &xNum, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyHostToDevice);

    HpSharkFloat<SharkFloatParams> *yGpu;
    hipMalloc(&yGpu, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemcpy(yGpu, &yNum, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyHostToDevice);

    HpSharkFloat<SharkFloatParams> *internalGpuResult2;
    hipMalloc(&internalGpuResult2, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemset(internalGpuResult2, 0, sizeof(HpSharkFloat<SharkFloatParams>));

    void *kernelArgs[] = {
        (void *)&xGpu,
        (void *)&yGpu,
        (void *)&internalGpuResult2,
        (void *)&d_tempProducts
    };

    {
        ScopedBenchmarkStopper stopper{ timer };
        kernel(kernelArgs);
    }

    hipMemcpy(&gpuResult, internalGpuResult2, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyDeviceToHost);

    if (debugResults != nullptr && SharkDebug) {
        debugResults->resize(SharkFloatParams::NumDebugStates);
        hipMemcpy(
            debugResults->data(),
            &d_tempProducts[AdditionalGlobalSyncSpace],
            SharkFloatParams::NumDebugStates * sizeof(DebugStateRaw),
            hipMemcpyDeviceToHost);
    }

    hipFree(internalGpuResult2);
    hipFree(yGpu);
    hipFree(xGpu);
    hipFree(d_tempProducts);
}

template<class SharkFloatParams, Operator sharkOperator>
void InvokeAddKernelCorrectness(
    BenchmarkTimer &timer,
    std::function<void(void *[])> kernel,
    const HpSharkFloat<SharkFloatParams> &xNum,
    const HpSharkFloat<SharkFloatParams> &yNum,
    HpSharkFloat<SharkFloatParams> &gpuResult) {

    // Perform the calculation on the GPU
    HpSharkFloat<SharkFloatParams> *xGpu;
    HpSharkFloat<SharkFloatParams> *yGpu;

    hipMalloc(&xGpu, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMalloc(&yGpu, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemcpy(xGpu, &xNum, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyHostToDevice);
    hipMemcpy(yGpu, &yNum, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyHostToDevice);

    HpSharkFloat<SharkFloatParams> *internalGpuResult;
    hipMalloc(&internalGpuResult, sizeof(HpSharkFloat<SharkFloatParams>));

    // Allocate memory for carryOuts and cumulativeCarries
    GlobalAddBlockData *globalBlockData;
    CarryInfo *d_carryOuts;
    uint32_t *d_cumulativeCarries;
    hipMalloc(&globalBlockData, sizeof(GlobalAddBlockData));
    hipMalloc(&d_carryOuts, (SharkFloatParams::GlobalNumBlocks + 1) * sizeof(CarryInfo));
    hipMalloc(&d_cumulativeCarries, (SharkFloatParams::GlobalNumBlocks + 1) * sizeof(uint32_t));

    // Prepare kernel arguments
    void *kernelArgs[] = {
        (void *)&xGpu,
        (void *)&yGpu,
        (void *)&internalGpuResult,
        (void *)&globalBlockData,
        (void *)&d_carryOuts,
        (void *)&d_cumulativeCarries
    };

    {
        ScopedBenchmarkStopper stopper{ timer };
        ComputeAddGpu<SharkFloatParams>(kernelArgs);
    }

    hipFree(globalBlockData);
    hipFree(d_carryOuts);
    hipFree(d_cumulativeCarries);

    hipMemcpy(&gpuResult, internalGpuResult, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyDeviceToHost);
    hipFree(internalGpuResult);

    hipFree(yGpu);
    hipFree(xGpu);
}

#define ExplicitlyInstantiate(SharkFloatParams) \
    template void InvokeMultiplyKernel<SharkFloatParams>( \
        BenchmarkTimer &timer, \
        std::function<void(hipStream_t &, void *[])> kernel, \
        const HpSharkFloat<SharkFloatParams> &xNum, \
        const HpSharkFloat<SharkFloatParams> &yNum, \
        HpSharkFloat<SharkFloatParams> &gpuResult2); \
    template void InvokeAddKernel<SharkFloatParams>( \
        BenchmarkTimer &timer, \
        std::function<void(void*[])> kernel, \
        const HpSharkFloat<SharkFloatParams> &xNum, \
        const HpSharkFloat<SharkFloatParams> &yNum, \
        HpSharkFloat<SharkFloatParams> &gpuResult2); \
    template void InvokeMultiplyKernelCorrectness<SharkFloatParams, Operator::MultiplyKaratsubaV2>( \
        BenchmarkTimer &timer, \
        std::function<void(void*[])> kernel, \
        const HpSharkFloat<SharkFloatParams> &xNum, \
        const HpSharkFloat<SharkFloatParams> &yNum, \
        HpSharkFloat<SharkFloatParams> &gpuResult, \
        std::vector<DebugStateRaw> *debugResults); \
    template void InvokeAddKernelCorrectness<SharkFloatParams, Operator::Add>( \
        BenchmarkTimer &timer, \
        std::function<void(void*[])> kernel, \
        const HpSharkFloat<SharkFloatParams> &xNum, \
        const HpSharkFloat<SharkFloatParams> &yNum, \
        HpSharkFloat<SharkFloatParams> &gpuResult);

ExplicitInstantiateAll();