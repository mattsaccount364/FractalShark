#include "hip/hip_runtime.h"
﻿#include "Multiply.cuh"

#include <hip/hip_runtime.h>

#include "HpSharkFloat.cuh"
#include "BenchmarkTimer.h"

#include <iostream>
#include <vector>
#include <gmp.h>
#include <cstring>
#include <sstream>
#include <iomanip>
#include <cmath>
#include <algorithm>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/barrier>
namespace cg = cooperative_groups;

#ifdef _DEBUG
#define SharkForceInlineReleaseOnly
#else
#define SharkForceInlineReleaseOnly __forceinline__
#endif

template<int n1, int n2>
__device__ int CompareDigits(const uint32_t *highArray, const uint32_t *lowArray) {
    if constexpr (n1 > n2) {
        return 1;
    } else if constexpr (n2 > n1) {
        return -1;
    } else {
        // n1 == n2
        for (int i = n1 - 1; i >= 0; --i) {
            if (highArray[i] > lowArray[i]) return 1;
            if (highArray[i] < lowArray[i]) return -1;
        }
    }
    return 0;
}

template<int n1, int n2>
__device__ static void SubtractDigitsSerial(const uint32_t *a, const uint32_t *b, uint32_t *result) {
    uint64_t borrow = 0;
    for (int i = 0; i < n1; ++i) {
        uint64_t ai;
        uint64_t bi;

        ai = a[i];

        if (i >= n2) {
            bi = 0;
        } else {
            bi = b[i];
        }

        uint64_t temp = ai - bi - borrow;
        if (ai < bi + borrow) {
            borrow = 1;
            temp += ((uint64_t)1 << 32);
        } else {
            borrow = 0;
        }
        result[i] = (uint32_t)temp;
    }
}

#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

/**
 * Parallel subtraction (a - b), stored in result, using a multi-pass approach
 * to propagate borrows.
 *
 * The function attempts to subtract each digit of 'b' from 'a' in parallel,
 * then uses repeated passes (do/while) to handle newly introduced borrows
 * until no more remain or a maximum pass count is reached.
 * 
 * Corrupts contexts of x_diff_abs shared memory intentionally
 */
template<
    class SharkFloatParams,
    int n1,
    int n2,
    int ExecutionBlockBase,
    int ExecutionNumBlocks>
__device__ SharkForceInlineReleaseOnly void SubtractDigitsParallel(
    uint32_t *__restrict__ x_diff_abs,
    uint32_t *__restrict__ y_diff_abs,
    const uint32_t *__restrict__ a1,
    const uint32_t *__restrict__ b1,
    const uint32_t *__restrict__ a2,
    const uint32_t *__restrict__ b2,
    uint32_t *__restrict__ subtractionBorrows1a,
    uint32_t *__restrict__ subtractionBorrows1b,
    uint32_t *__restrict__ subtractionBorrows2a,
    uint32_t *__restrict__ subtractionBorrows2b,
    uint32_t *__restrict__ result1,
    uint32_t *__restrict__ result2,
    uint32_t *__restrict__ globalBorrowAny,
    cg::grid_group &grid,
    cg::thread_block &block
) {
    // Note: stops on this.
    auto *sharedBorrowAny = x_diff_abs;

    // Note: not ExecutionBlockBase
    if (block.group_index().x == 0 && block.thread_index().x == 0) {
        *globalBorrowAny = 0;
    }

    if (block.thread_index().x == 0) {
        *sharedBorrowAny = 0;
    }

    // Constants 
    constexpr int MaxPasses = 150;      // maximum number of multi-pass sweeps

    // We'll define a grid–stride range covering [0..n) for each pass
    // 1) global thread id
    int tid = (block.group_index().x - ExecutionBlockBase) * block.dim_threads().x + block.thread_index().x;
    // 2) stride
    int stride = block.dim_threads().x * ExecutionNumBlocks;

    // (1) First pass: naive partial difference (a[i] - b[i]) and set borrowBit
    // Instead of dividing digits among blocks, each thread does a grid–stride loop:
    for (int idx = tid; idx < n1; idx += stride) {
        uint32_t ai1;
        uint32_t bi1;
        uint32_t ai2;
        uint32_t bi2;

        if (idx >= n2) {
            ai1 = a1[idx];
            bi1 = 0;

            ai2 = a2[idx];
            bi2 = 0;
        } else {
            ai1 = a1[idx];
            bi1 = b1[idx];

            ai2 = a2[idx];
            bi2 = b2[idx];
        }

        // naive difference
        uint64_t diff1 = (uint64_t)ai1 - (uint64_t)bi1;
        uint64_t diff2 = (uint64_t)ai2 - (uint64_t)bi2;

        uint32_t borrow1 = (ai1 < bi1) ? 1u : 0u;
        uint32_t borrow2 = (ai2 < bi2) ? 1u : 0u;

        result1[idx] = static_cast<uint32_t>(diff1 & 0xFFFFFFFFu);
        subtractionBorrows1a[idx] = borrow1;

        result2[idx] = static_cast<uint32_t>(diff2 & 0xFFFFFFFFu);
        subtractionBorrows2a[idx] = borrow2;
    }

    // sync the entire grid before multi-pass fixes
    //grid.sync();

    // We'll do repeated passes to fix newly introduced borrows
    uint32_t *curBorrow1 = subtractionBorrows1a;
    uint32_t *newBorrow1 = subtractionBorrows1b;
    uint32_t *curBorrow2 = subtractionBorrows2a;
    uint32_t *newBorrow2 = subtractionBorrows2b;
    int pass = 0;
    uint32_t initialBorrowAny = 0;

    grid.sync();

    do {
        // (2) For each digit, apply the borrow from the previous digit
        for (int idx = tid; idx < n1; idx += stride) {
            uint64_t borrow_in1 = 0ULL;
            uint64_t borrow_in2 = 0ULL;
            if (idx > 0) {   // borrow_in is from digit (idx-1)
                borrow_in1 = (uint64_t)(curBorrow1[idx - 1]);
                borrow_in2 = (uint64_t)(curBorrow2[idx - 1]);
            }

            uint32_t digit1 = result1[idx];
            uint32_t digit2 = result2[idx];

            // subtract the borrow
            uint64_t sum1 = (uint64_t)digit1 - borrow_in1;
            uint64_t sum2 = (uint64_t)digit2 - borrow_in2;

            // store updated digit
            result1[idx] = static_cast<uint32_t>(sum1 & 0xFFFFFFFFULL);
            result2[idx] = static_cast<uint32_t>(sum2 & 0xFFFFFFFFULL);

            // If sum is negative => top bit is 1 => new borrow
            if (sum1 & 0x8000'0000'0000'0000ULL) {
                newBorrow1[idx] = 1;
                atomicAdd(sharedBorrowAny, 1);
            } else {
                newBorrow1[idx] = 0;
            }

            if (sum2 & 0x8000'0000'0000'0000ULL) {
                newBorrow2[idx] = 1;
                atomicAdd(sharedBorrowAny, 1);
            } else {
                newBorrow2[idx] = 0;
            }
        }

        // (a) Block-level synchronization (so all threads see final sharedBorrowAny)
        block.sync();

        // The block's thread 0 aggregates once into globalBorrowAny
        if (block.thread_index().x == 0) {
            // Add sharedBorrowAny to the global counter
            atomicAdd(globalBorrowAny, *sharedBorrowAny);

            // Reset local aggregator for the next pass
            *sharedBorrowAny = 0;
        }

        // sync before checking if any new borrows remain
        grid.sync();

        auto tempCopyGlobalBorrowAny = *globalBorrowAny;
        if (tempCopyGlobalBorrowAny == initialBorrowAny) {
            break;  // no new borrows => done
        }

        grid.sync();
        initialBorrowAny = tempCopyGlobalBorrowAny;

        // swap curBorrow, newBorrow
        uint32_t *tmp = curBorrow1;
        curBorrow1 = newBorrow1;
        newBorrow1 = tmp;

        tmp = curBorrow2;
        curBorrow2 = newBorrow2;
        newBorrow2 = tmp;

        pass++;
    } while (pass < MaxPasses);
}



// Function to perform addition with carry
__device__ SharkForceInlineReleaseOnly static void Add128(
    uint64_t a_low, uint64_t a_high,
    uint64_t b_low, uint64_t b_high,
    uint64_t &result_low, uint64_t &result_high) {

    result_low = a_low + b_low;
    uint64_t carry = (result_low < a_low) ? 1 : 0;
    result_high = a_high + b_high + carry;
}

__device__ SharkForceInlineReleaseOnly static void Subtract128(
    uint64_t a_low, uint64_t a_high,
    uint64_t b_low, uint64_t b_high,
    uint64_t &result_low, uint64_t &result_high) {

    uint64_t borrow = 0;

    // Subtract low parts
    result_low = a_low - b_low;
    borrow = (a_low < b_low) ? 1 : 0;

    // Subtract high parts with borrow
    result_high = a_high - b_high - borrow;
}

template<class SharkFloatParams>
__device__ SharkForceInlineReleaseOnly static void SerialCarryPropagation(
    uint64_t *__restrict__ shared_carries,
    cg::grid_group &grid,
    cg::thread_block &block,
    const uint3 &threadIdx,
    const uint3 &blockIdx,
    int thread_start_idx,
    int thread_end_idx,
    int Convolution_offset,
    int Result_offset,
    uint64_t *__restrict__ block_carry_outs,
    uint64_t *__restrict__ tempProducts,
    uint64_t *__restrict__ globalCarryCheck) {

    if (block.thread_index().x == 0 && block.group_index().x == 0) {
        uint64_t local_carry = 0;

        for (int idx = 0; idx < SharkFloatParams::GlobalNumUint32 * 2 + 1; ++idx) {
            int sum_low_idx = Convolution_offset + idx * 2;
            int sum_high_idx = sum_low_idx + 1;

            uint64_t sum_low = tempProducts[sum_low_idx];     // Lower 64 bits
            uint64_t sum_high = tempProducts[sum_high_idx];   // Higher 64 bits

            // Add local carry to sum_low
            bool new_sum_low_negative = false;
            uint64_t new_sum_low = sum_low + local_carry;

            // Extract one 32-bit digit from new_sum_low
            auto digit = static_cast<uint32_t>(new_sum_low & 0xFFFFFFFFULL);
            tempProducts[Result_offset + idx] = digit;

            bool local_carry_negative = ((local_carry & (1ULL << 63)) != 0);
            local_carry = 0ULL;

            if (!local_carry_negative && new_sum_low < sum_low) {
                local_carry = 1ULL << 32;
            } else if (local_carry_negative && new_sum_low > sum_low) {
                new_sum_low_negative = (new_sum_low & 0x8000'0000'0000'0000) != 0;
            }

            // Update local_carry
            if (new_sum_low_negative) {
                // Shift sum_high by 32 bits and add carry_from_low
                uint64_t upper_new_sum_low = new_sum_low >> 32;
                upper_new_sum_low |= 0xFFFF'FFFF'0000'0000;
                local_carry += upper_new_sum_low;
                local_carry += sum_high << 32;
            } else {
                local_carry += new_sum_low >> 32;
                local_carry += sum_high << 32;
            }
        }
    }
}

template<class SharkFloatParams>
__device__ SharkForceInlineReleaseOnly static void CarryPropagation (
    uint64_t *__restrict__ shared_carries,
    cg::grid_group &grid,
    cg::thread_block &block,
    const uint3 &threadIdx,
    const uint3 &blockIdx,
    int thread_start_idx,
    int thread_end_idx,
    int Convolution_offset,
    int Result_offset,
    uint64_t * __restrict__ block_carry_outs,
    uint64_t * __restrict__ tempProducts,
    uint64_t * __restrict__ globalCarryCheck) {

    // First Pass: Process convolution results to compute initial digits and local carries
    // Initialize local carry
    uint64_t local_carry = 0;

    // Constants and offsets
    constexpr int MaxPasses = 150; // Maximum number of carry propagation passes
    constexpr int total_result_digits = 2 * SharkFloatParams::GlobalNumUint32;

    uint64_t *carries_remaining_global = globalCarryCheck;

    for (int idx = thread_start_idx; idx < thread_end_idx; ++idx) {
        int sum_low_idx = Convolution_offset + idx * 2;
        int sum_high_idx = sum_low_idx + 1;

        uint64_t sum_low = tempProducts[sum_low_idx];     // Lower 64 bits
        uint64_t sum_high = tempProducts[sum_high_idx];   // Higher 64 bits

        // Add local carry to sum_low
        bool new_sum_low_negative = false;
        uint64_t new_sum_low = sum_low + local_carry;

        // Extract one 32-bit digit from new_sum_low
        auto digit = static_cast<uint32_t>(new_sum_low & 0xFFFFFFFFULL);
        tempProducts[Result_offset + idx] = digit;

        bool local_carry_negative = ((local_carry & (1ULL << 63)) != 0);
        local_carry = 0ULL;

        if (!local_carry_negative && new_sum_low < sum_low) {
            local_carry = 1ULL << 32;
        } else if (local_carry_negative && new_sum_low > sum_low) {
            new_sum_low_negative = (new_sum_low & 0x8000'0000'0000'0000) != 0;
        }

        // Update local_carry
        if (new_sum_low_negative) {
            // Shift sum_high by 32 bits and add carry_from_low
            uint64_t upper_new_sum_low = new_sum_low >> 32;
            upper_new_sum_low |= 0xFFFF'FFFF'0000'0000;
            local_carry += upper_new_sum_low;
            local_carry += sum_high << 32;
        } else {
            local_carry += new_sum_low >> 32;
            local_carry += sum_high << 32;
        }
    }

    if (block.thread_index().x == SharkFloatParams::GlobalThreadsPerBlock - 1) {
        block_carry_outs[block.group_index().x] = local_carry;
    } else {
        shared_carries[block.thread_index().x] = local_carry;
    }

    // Inter-Block Carry Propagation
    int pass = 0;

    do {
        // Synchronize all blocks
        grid.sync();

        // Zero out the global carry count for the current pass
        if (block.group_index().x == 0 && block.thread_index().x == 0) {
            *carries_remaining_global = 0;
        }

        // Get carry-in from the previous block
        local_carry = 0;
        if (block.thread_index().x == 0 && block.group_index().x > 0) {
            local_carry = block_carry_outs[block.group_index().x - 1];
        } else {
            if (block.thread_index().x > 0) {
                local_carry = shared_carries[block.thread_index().x - 1];
            }
        }

        // Each thread processes its assigned digits
        for (int idx = thread_start_idx; idx < thread_end_idx; ++idx) {
            // Read the previously stored digit
            uint32_t digit = tempProducts[Result_offset + idx];

            // Add local_carry to digit
            uint64_t sum = static_cast<uint64_t>(digit) + local_carry;

            // Update digit
            digit = static_cast<uint32_t>(sum & 0xFFFFFFFFULL);
            tempProducts[Result_offset + idx] = digit;

            local_carry = 0;

            // Check negativity of the 64-bit sum
            // If "sum" is negative, its top bit is set. 
            bool sum_is_negative = ((sum & (1ULL << 63)) != 0ULL);

            if (sum_is_negative) {
                // sign-extend the top 32 bits
                uint64_t upper_bits = (sum >> 32);
                upper_bits |= 0xFFFF'FFFF'0000'0000ULL;  // set top 32 bits to 1
                local_carry += upper_bits;               // incorporate sign-extended bits
            } else {
                // normal path: just add top 32 bits
                local_carry += (sum >> 32);
            }
        }

        shared_carries[block.thread_index().x] = local_carry;
        block.sync();

        // The block's carry-out is the carry from the last thread
        auto temp = shared_carries[block.thread_index().x];
        if (block.thread_index().x == SharkFloatParams::GlobalThreadsPerBlock - 1) {
            block_carry_outs[block.group_index().x] = temp;
        }

        if (temp != 0) {
            atomicAdd(carries_remaining_global, 1);
        }

        // Synchronize all blocks before checking if carries remain
        grid.sync();

        // If no carries remain, exit the loop
        if (*carries_remaining_global == 0) {
            break;
        }

        pass++;
    } while (pass < MaxPasses);

    // ---- Handle Final Carry-Out ----

    // Handle final carry-out
    if (block.thread_index().x == 0 && block.group_index().x == grid.dim_blocks().x - 1) {
        uint64_t final_carry = block_carry_outs[block.group_index().x];
        if (final_carry > 0) {
            // Store the final carry as an additional digit
            tempProducts[Result_offset + total_result_digits] = static_cast<uint32_t>(final_carry & 0xFFFFFFFFULL);
            // Optionally, you may need to adjust total_result_digits
        }
    }

    // Synchronize all blocks before finalization
    // grid.sync();
}

// Look for Uint64ToAllocateForMultiply and ScratchMemoryArrays
// and make sure the number of NewN arrays we're using here fits within that limit.
// The list here should go up to ScratchMemoryArrays.
#define DefineTempProductsOffsets(TempBase) \
    constexpr int n1 = NewN1; \
    constexpr int n2 = NewN2; \
    const int threadIdxGlobal = block.group_index().x * SharkFloatParams::GlobalThreadsPerBlock + block.thread_index().x; \
    constexpr int BorrowGlobalOffset = 0; \
    constexpr int Z0_offset = TempBase; \
    constexpr int Z2_offset = Z0_offset + 4 * NewN; \
    constexpr int Z1_temp_offset = Z2_offset + 4 * NewN; \
    constexpr int Z1_offset = Z1_temp_offset + 4 * NewN; \
    constexpr int Convolution_offset = Z1_offset + 4 * NewN;       /* 16 */ \
    constexpr int Result_offset = Convolution_offset + 4 * NewN;   /* 20 */ \
    constexpr int XDiff_offset = Result_offset + 2 * NewN;         /* 22 */ \
    constexpr int YDiff_offset = XDiff_offset + 1 * NewN;          /* 23 */ \
    constexpr int GlobalCarryOffset = YDiff_offset + 1 * NewN;     /* 24 */ \
    constexpr int SubtractionOffset1 = GlobalCarryOffset + 1 * NewN;   /* 25 */ \
    constexpr int SubtractionOffset2 = SubtractionOffset1 + 1 * NewN;  /* 26 */ \
    constexpr int SubtractionOffset3 = SubtractionOffset2 + 1 * NewN;  /* 27 */ \
    constexpr int SubtractionOffset4 = SubtractionOffset3 + 1 * NewN;  /* 28 */ \
    constexpr int BorrowAnyOffset = SubtractionOffset4 + 1 * NewN;     /* 29 */ \
    /* Note, overlaps: */ \
    constexpr int CarryInsOffset = TempBase;


#define DefineExtraDefinitions() \
    const auto RelativeBlockIndex = block.group_index().x - ExecutionBlockBase; \
    constexpr int total_result_digits = 2 * NewN + 1; \
    constexpr auto digits_per_block = NewN * 2 / ExecutionNumBlocks; \
    auto block_start_idx = block.group_index().x * digits_per_block; \
    auto block_end_idx = min(block_start_idx + digits_per_block, total_result_digits); \
    int digits_per_thread = (digits_per_block + block.dim_threads().x - 1) / block.dim_threads().x; \
    int thread_start_idx = block_start_idx + block.thread_index().x * digits_per_thread; \
    int thread_end_idx = min(thread_start_idx + digits_per_thread, block_end_idx);

#define DefineCarryDefinitions() \
    constexpr int total_result_digits = 2 * NewN + 1; \
    constexpr auto digits_per_block = SharkFloatParams::GlobalThreadsPerBlock * 2; \
    auto block_start_idx = block.group_index().x * digits_per_block; \
    auto block_end_idx = min(block_start_idx + digits_per_block, total_result_digits); \
    int digits_per_thread = (digits_per_block + block.dim_threads().x - 1) / block.dim_threads().x; \
    int thread_start_idx = block_start_idx + block.thread_index().x * digits_per_thread; \
    int thread_end_idx = min(thread_start_idx + digits_per_thread, block_end_idx);


// Assuming that SharkFloatParams::GlobalNumUint32 can be large and doesn't fit in shared memory
// We'll use the provided global memory buffers for large intermediates
// #define SharkRestrict __restrict__
#define SharkRestrict

template<
    class SharkFloatParams,
    int NewN,
    int NewN1,
    int NewN2,
    int ExecutionBlockBase,
    int ExecutionNumBlocks,
    int NewNumBlocks,
    int TempBase>
__device__ SharkForceInlineReleaseOnly void MultiplyDigitsOnly(
    uint32_t *SharkRestrict shared_data,
    const HpSharkFloat<SharkFloatParams> *SharkRestrict A,
    const HpSharkFloat<SharkFloatParams> *SharkRestrict B,
    const uint32_t *SharkRestrict aDigits,
    const uint32_t *SharkRestrict bDigits,
    uint32_t *SharkRestrict x_diff_abs,
    uint32_t *SharkRestrict y_diff_abs,
    uint64_t *SharkRestrict final128,
    cg::grid_group &grid,
    cg::thread_block &block,
    uint64_t *SharkRestrict tempProducts) {

    if ((ExecutionBlockBase > 0 && block.group_index().x < ExecutionBlockBase) ||
        block.group_index().x >= ExecutionBlockBase + ExecutionNumBlocks) {

        return;
    }

    DefineTempProductsOffsets(TempBase);

    const auto *a_shared = aDigits;
    const auto *b_shared = bDigits;

    auto *Z0_OutDigits = &tempProducts[Z0_offset];
    auto *Z1_temp_digits = &tempProducts[Z1_temp_offset];
    auto *Z2_OutDigits = &tempProducts[Z2_offset];

    constexpr int total_k = 2 * n1 - 1; // Total number of k values
    constexpr auto ConvolutionLimit = 9; // 3^whatevs = ConvolutionLimit
    constexpr bool UseConvolution =
        (NewNumBlocks <= std::max(SharkFloatParams::GlobalNumBlocks / ConvolutionLimit, 1) ||
        (NewNumBlocks % 3 != 0));
    constexpr bool EnableSharedDiff = false;
    constexpr bool UseParallelSubtract = true;

    // Arrays to hold the absolute differences (size n)
    auto *global_x_diff_abs = reinterpret_cast<uint32_t *>(&tempProducts[XDiff_offset]);
    auto *global_y_diff_abs = reinterpret_cast<uint32_t *>(&tempProducts[YDiff_offset]);

    // ---- Compute Differences x_diff = A1 - A0 and y_diff = B1 - B0 ----

    DefineExtraDefinitions();

    int x_diff_sign = 0; // 0 if positive, 1 if negative
    int y_diff_sign = 0; // 0 if positive, 1 if negative

    // Compute x_diff_abs and x_diff_sign
    auto *SharkRestrict subtractionBorrows = reinterpret_cast<uint32_t *>(&tempProducts[SubtractionOffset1]);
    auto *SharkRestrict subtractionBorrows2 = reinterpret_cast<uint32_t *>(&tempProducts[SubtractionOffset2]);
    auto *SharkRestrict subtractionBorrows3 = reinterpret_cast<uint32_t *>(&tempProducts[SubtractionOffset3]);
    auto *SharkRestrict subtractionBorrows4 = reinterpret_cast<uint32_t *>(&tempProducts[SubtractionOffset4]);
    auto *SharkRestrict globalBorrowAny = reinterpret_cast<uint32_t *>(&tempProducts[BorrowGlobalOffset]);

    if constexpr (!SharkFloatParams::DisableSubtraction) {
        if constexpr (UseParallelSubtract) {
            int x_compare = CompareDigits<n1, n1>(a_shared + n1, a_shared);
            int y_compare = CompareDigits<n1, n1>(b_shared + n1, b_shared);

            if (x_compare >= 0 && y_compare >= 0) {
                x_diff_sign = 0;
                y_diff_sign = 0;
                SubtractDigitsParallel<SharkFloatParams, n1, n1, ExecutionBlockBase, ExecutionNumBlocks>(
                    x_diff_abs,
                    y_diff_abs,
                    a_shared + n1,
                    a_shared,
                    b_shared + n1,
                    b_shared,
                    subtractionBorrows,
                    subtractionBorrows2,
                    subtractionBorrows3,
                    subtractionBorrows4,
                    global_x_diff_abs,
                    global_y_diff_abs,
                    globalBorrowAny,
                    grid,
                    block);
            } else if (x_compare < 0 && y_compare < 0) {
                x_diff_sign = 1;
                y_diff_sign = 1;
                SubtractDigitsParallel<SharkFloatParams, n1, n1, ExecutionBlockBase, ExecutionNumBlocks>(
                    x_diff_abs,
                    y_diff_abs,
                    a_shared,
                    a_shared + n1,
                    b_shared,
                    b_shared + n1,
                    subtractionBorrows,
                    subtractionBorrows2,
                    subtractionBorrows3,
                    subtractionBorrows4,
                    global_x_diff_abs,
                    global_y_diff_abs,
                    globalBorrowAny,
                    grid,
                    block);
            } else if (x_compare >= 0 && y_compare < 0) {
                x_diff_sign = 0;
                y_diff_sign = 1;
                SubtractDigitsParallel<SharkFloatParams, n1, n1, ExecutionBlockBase, ExecutionNumBlocks>(
                    x_diff_abs,
                    y_diff_abs,
                    a_shared + n1,
                    a_shared,
                    b_shared,
                    b_shared + n1,
                    subtractionBorrows,
                    subtractionBorrows2,
                    subtractionBorrows3,
                    subtractionBorrows4,
                    global_x_diff_abs,
                    global_y_diff_abs,
                    globalBorrowAny,
                    grid,
                    block);
            } else {
                x_diff_sign = 1;
                y_diff_sign = 0;
                SubtractDigitsParallel<SharkFloatParams, n1, n1, ExecutionBlockBase, ExecutionNumBlocks>(
                    x_diff_abs,
                    y_diff_abs,
                    a_shared,
                    a_shared + n1,
                    b_shared + n1,
                    b_shared,
                    subtractionBorrows,
                    subtractionBorrows2,
                    subtractionBorrows3,
                    subtractionBorrows4,
                    global_x_diff_abs,
                    global_y_diff_abs,
                    globalBorrowAny,
                    grid,
                    block);
            }
        } else {
            if (block.thread_index().x == 0 && block.group_index().x == ExecutionBlockBase) {
                int x_compare = CompareDigits<n1, n1>(a_shared + n1, a_shared);

                if (x_compare >= 0) {
                    x_diff_sign = 0;
                    SubtractDigitsSerial<n1, n1>(a_shared + n1, a_shared, global_x_diff_abs); // x_diff = A1 - A0
                } else {
                    x_diff_sign = 1;
                    SubtractDigitsSerial<n1, n1>(a_shared, a_shared + n1, global_x_diff_abs); // x_diff = A0 - A1
                }

                // Compute y_diff_abs and y_diff_sign
                int y_compare = CompareDigits<n1, n1>(b_shared + n1, b_shared);
                if (y_compare >= 0) {
                    y_diff_sign = 0;
                    SubtractDigitsSerial<n1, n1>(b_shared + n1, b_shared, global_y_diff_abs); // y_diff = B1 - B0
                } else {
                    y_diff_sign = 1;
                    SubtractDigitsSerial<n1, n1>(b_shared, b_shared + n1, global_y_diff_abs); // y_diff = B0 - B1
                }
            }
        }
    }

    // Determine the sign of Z1_temp
    int z1_sign = x_diff_sign ^ y_diff_sign;

    if constexpr (UseConvolution) {
        // Replace A and B in shared memory with their absolute differences
        if constexpr (EnableSharedDiff) {
            cg::memcpy_async(block, const_cast<uint32_t *>(x_diff_abs), global_x_diff_abs, sizeof(uint32_t) * n1);
            cg::memcpy_async(block, const_cast<uint32_t *>(y_diff_abs), global_y_diff_abs, sizeof(uint32_t) * n1);
        }

        const int tid = RelativeBlockIndex * block.dim_threads().x + block.thread_index().x;
        const int stride = block.dim_threads().x * ExecutionNumBlocks;

        if constexpr (EnableSharedDiff) {
            // Wait for the first batch of A to be loaded
            cg::wait(block);
        }

        // A single loop that covers 2*total_k elements
        for (int idx = tid; idx < 3 * total_k; idx += stride) {
            // Check if idx < total_k => handle Z0, else handle Z2
            if (idx < total_k) {
                // Z0 partial sums
                int k = idx;
                uint64_t sum_low = 0ULL, sum_high = 0ULL;

                int i_start = max(0, k - (n1 - 1));
                int i_end = min(k, n1 - 1);
                for (int i = i_start; i <= i_end; i++) {
                    uint64_t a = a_shared[i]; //A_shared[i];         // A0[i]

                    uint64_t b;
                    
                    //if (k - i < n1) { // TODO
                        b = b_shared[k - i]; // B_shared[k - i];     // B0[k - i]
                    //} else {
                    //    b = 0;
                    //}

                    uint64_t product = a * b;

                    // Add product to sum
                    sum_low += product;
                    if (sum_low < product) {
                        sum_high += 1;
                    }
                }

                // store sum_low, sum_high in Z0_OutDigits
                int out_idx = k * 2;
                Z0_OutDigits[out_idx] = sum_low;
                Z0_OutDigits[out_idx + 1] = sum_high;
            } else if (idx < 2 * total_k) {
                // Z2 partial sums
                int k = idx - total_k; // shift to [0..total_k-1]
                uint64_t sum_low = 0ULL, sum_high = 0ULL;

                int i_start = max(0, k - (n1 - 1));
                int i_end = min(k, n1 - 1);
                for (int i = i_start; i <= i_end; i++) {
                    uint64_t a = a_shared[i + n1]; // A_shared[i];         // A1[i]
                    uint64_t b;
                    
                    //if (k - i + n1 < n1) { // TODO
                        b = b_shared[k - i + n1]; // B_shared[k - i];     // B1[k - i]
                    //} else {
                    //    b = 0;
                    //}

                    uint64_t product = a * b;

                    // Add product to sum
                    sum_low += product;
                    if (sum_low < product) {
                        sum_high += 1;
                    }
                }

                // store sum_low, sum_high in Z2_OutDigits
                int out_idx = k * 2;
                Z2_OutDigits[out_idx] = sum_low;
                Z2_OutDigits[out_idx + 1] = sum_high;
            } else {
                int k = idx - 2 * total_k; // shift to [0..total_k-1]
                uint64_t sum_low = 0;
                uint64_t sum_high = 0;

                int i_start = max(0, k - (n1 - 1));
                int i_end = min(k, n1 - 1);

                for (int i = i_start; i <= i_end; ++i) {
                    uint64_t a = EnableSharedDiff ? x_diff_abs[i] : global_x_diff_abs[i];
                    uint64_t b;
                    
                    //if (k - i < n1) {
                        b = EnableSharedDiff ? y_diff_abs[k - i] : global_y_diff_abs[k - i];
                    //} else {
                    //    b = 0;
                    //}

                    uint64_t product = a * b;

                    // Accumulate the product
                    sum_low += product;
                    if (sum_low < product) {
                        sum_high += 1;
                    }
                }

                // Store sum_low and sum_high in tempProducts
                int out_idx = k * 2;
                Z1_temp_digits[out_idx] = sum_low;
                Z1_temp_digits[out_idx + 1] = sum_high;
            }
        }
    } else {
        constexpr auto NumBlocksRatio = ConvolutionLimit * SharkFloatParams::GlobalNumBlocks / NewNumBlocks;

        constexpr auto NewTempBase1 =
            TempBase + ScratchMemoryArrays * SharkFloatParams::GlobalNumUint32 * NumBlocksRatio;

        constexpr auto SubNewN = NewN / 2;
        constexpr auto SubNewNRoundUp = (NewN + 1) / 2;
        constexpr auto SubNewN1 = (SubNewN + 1) / 2;
        //constexpr auto SubNewN2 = SubNewNRoundUp - SubNewN1;
        constexpr auto SubNewN2 = SubNewN1;

        MultiplyDigitsOnly<
            SharkFloatParams,
            SubNewN,
            SubNewN1,
            SubNewN1,
            ExecutionBlockBase,
            ExecutionNumBlocks / 3,
            NewNumBlocks / 3,
            NewTempBase1>(
            shared_data,
            A,
            B,
            a_shared,
            b_shared,
            x_diff_abs,
            y_diff_abs,
            Z0_OutDigits,
            grid,
            block,
            tempProducts);

        constexpr auto NewTempBase2 =
            TempBase + ScratchMemoryArrays * SharkFloatParams::GlobalNumUint32 * (NumBlocksRatio * 2);
        // constexpr auto SubRemainingNewN = NewN - SubNewN;
        constexpr auto SubRemainingNewN = SubNewN;
        MultiplyDigitsOnly<
            SharkFloatParams,
            SubRemainingNewN,
            SubNewN1,
            SubNewN1,
            ExecutionBlockBase + ExecutionNumBlocks / 3,
            ExecutionNumBlocks / 3,
            NewNumBlocks / 3,
            NewTempBase2>(
            shared_data,
            A,
            B,
            a_shared + n1,
            b_shared + n1,
            x_diff_abs,
            y_diff_abs,
            Z2_OutDigits,
            grid,
            block,
            tempProducts);

        //grid.sync();

        {
            constexpr auto NewExecutionBlockBase = ExecutionBlockBase + 2 * ExecutionNumBlocks / 3;
            constexpr auto NewExecutionNumBlocks = ExecutionNumBlocks / 3;

            const bool ExecuteAtAll =
                !((NewExecutionBlockBase > 0 && block.group_index().x < NewExecutionBlockBase) ||
                    block.group_index().x >= NewExecutionBlockBase + NewExecutionNumBlocks);
            constexpr bool EnableMoreParallelism = true;
            constexpr auto NewTempBase3 =
                TempBase + ScratchMemoryArrays * SharkFloatParams::GlobalNumUint32 * (NumBlocksRatio * 3);

            // Replace A and B in shared memory with their absolute differences
            if constexpr (EnableSharedDiff) {
                cg::memcpy_async(block,
                    const_cast<uint32_t *>(a_shared),
                    global_x_diff_abs,
                    sizeof(uint32_t) * n1);
                cg::memcpy_async(block,
                    const_cast<uint32_t *>(b_shared),
                    global_y_diff_abs,
                    sizeof(uint32_t) * n1);
                cg::wait(block);
            }

            if (EnableMoreParallelism && ExecuteAtAll) {
                MultiplyDigitsOnly<
                    SharkFloatParams,
                    SubNewN,
                    SubNewN1,
                    SubNewN1,
                    NewExecutionBlockBase,
                    NewExecutionNumBlocks,
                    NewNumBlocks / 3,
                    NewTempBase3>(
                        shared_data,
                        A,
                        B,
                        EnableSharedDiff ? a_shared : global_x_diff_abs,
                        EnableSharedDiff ? b_shared : global_y_diff_abs,
                        x_diff_abs,
                        y_diff_abs,
                        Z1_temp_digits,
                        grid,
                        block,
                        tempProducts);
            }

            if constexpr (!EnableMoreParallelism) {
                MultiplyDigitsOnly<
                    SharkFloatParams,
                    SubNewN,
                    SubNewN1,
                    SubNewN1,
                    ExecutionBlockBase,
                    ExecutionNumBlocks,
                    NewNumBlocks / 3,
                    NewTempBase3>(
                        shared_data,
                        A,
                        B,
                        EnableSharedDiff ? a_shared : global_x_diff_abs,
                        EnableSharedDiff ? b_shared : global_y_diff_abs,
                        x_diff_abs,
                        y_diff_abs,
                        Z1_temp_digits,
                        grid,
                        block,
                        tempProducts);
            }

            if constexpr (EnableSharedDiff) {
                cg::memcpy_async(block,
                    const_cast<uint32_t *>(a_shared),
                    A->Digits,
                    sizeof(uint32_t) * SharkFloatParams::GlobalNumUint32);
                cg::memcpy_async(block,
                    const_cast<uint32_t *>(b_shared),
                    B->Digits,
                    sizeof(uint32_t) * SharkFloatParams::GlobalNumUint32);
                cg::wait(block);
            }
        }
    }

    grid.sync();

    auto *Z1_digits = &tempProducts[Z1_offset];

    if constexpr (!SharkFloatParams::DisableAllAdditions) {

        // After computing Z1_temp (Z1'), we now form Z1 directly:
        // If z1_sign == 0: Z1 = Z2 + Z0 - Z1_temp
        // If z1_sign == 1: Z1 = Z2 + Z0 + Z1_temp

        const int tid = RelativeBlockIndex * block.dim_threads().x + block.thread_index().x;
        const int stride = block.dim_threads().x * ExecutionNumBlocks;

        for (int i = tid; i < total_k; i += stride) {
            // Retrieve Z0
            int z0_idx = i * 2;
            uint64_t z0_low = Z0_OutDigits[z0_idx];
            uint64_t z0_high = Z0_OutDigits[z0_idx + 1];

            // Retrieve Z2
            int z2_idx = i * 2;
            uint64_t z2_low = Z2_OutDigits[z2_idx];
            uint64_t z2_high = Z2_OutDigits[z2_idx + 1];

            // Retrieve Z1_temp (Z1')
            int z1_temp_idx = i * 2;
            uint64_t z1_temp_low = Z1_temp_digits[z1_temp_idx];
            uint64_t z1_temp_high = Z1_temp_digits[z1_temp_idx + 1];

            // Combine Z2 + Z0 first
            uint64_t temp_low, temp_high;
            Add128(z2_low, z2_high, z0_low, z0_high, temp_low, temp_high);

            uint64_t z1_low, z1_high;
            if (z1_sign == 0) {
                // same sign: Z1 = (Z2 + Z0) - Z1_temp
                Subtract128(temp_low, temp_high, z1_temp_low, z1_temp_high, z1_low, z1_high);
            } else {
                // opposite signs: Z1 = (Z2 + Z0) + Z1_temp
                Add128(temp_low, temp_high, z1_temp_low, z1_temp_high, z1_low, z1_high);
            }

            // Store fully formed Z1
            int z1_idx = i * 2;
            Z1_digits[z1_idx] = z1_low;
            Z1_digits[z1_idx + 1] = z1_high;
        }

        // Synchronize before final combination
        grid.sync();

        // Now the final combination is just:
        // final = Z0 + (Z1 << (32*n)) + (Z2 << (64*n))
        for (int i = tid; i < total_result_digits; i += stride) {
            uint64_t sum_low = 0;
            uint64_t sum_high = 0;

            // Add Z0
            if (i < 2 * n1 - 1) {
                int z0_idx = i * 2;
                uint64_t z0_low = Z0_OutDigits[z0_idx];
                uint64_t z0_high = Z0_OutDigits[z0_idx + 1];
                Add128(sum_low, sum_high, z0_low, z0_high, sum_low, sum_high);
            }

            // Add Z1 shifted by n
            if (i >= n1 && (i - n1) < 2 * n1 - 1) {
                int z1_idx = (i - n1) * 2;
                uint64_t z1_low = Z1_digits[z1_idx];
                uint64_t z1_high = Z1_digits[z1_idx + 1];
                Add128(sum_low, sum_high, z1_low, z1_high, sum_low, sum_high);
            }

            // Add Z2 shifted by 2*n
            if (i >= 2 * n1 && (i - 2 * n1) < 2 * n1 - 1) {
                int z2_idx = (i - 2 * n1) * 2;
                uint64_t z2_low = Z2_OutDigits[z2_idx];
                uint64_t z2_high = Z2_OutDigits[z2_idx + 1];
                Add128(sum_low, sum_high, z2_low, z2_high, sum_low, sum_high);
            }

            int result_idx = i * 2;
            final128[result_idx] = sum_low;
            final128[result_idx + 1] = sum_high;
        }

        // Synchronize before carry propagation
        grid.sync();
    }
}

//
// static constexpr int32_t SharkFloatParams::GlobalThreadsPerBlock = /* power of 2 */;
// static constexpr int32_t SharkFloatParams::GlobalNumBlocks = /* power of 2 */;
// static constexpr int32_t SharkFloatParams::GlobalNumUint32 = SharkFloatParams::GlobalThreadsPerBlock * SharkFloatParams::GlobalNumBlocks;
// 

// Assuming that SharkFloatParams::GlobalNumUint32 can be large and doesn't fit in shared memory
// We'll use the provided global memory buffers for large intermediates
template<class SharkFloatParams>
__device__ void MultiplyHelperKaratsubaV2(
    const HpSharkFloat<SharkFloatParams> *__restrict__ A,
    const HpSharkFloat<SharkFloatParams> *__restrict__ B,
    HpSharkFloat<SharkFloatParams> *__restrict__ Out,
    cg::grid_group &grid,
    cg::thread_block &block,
    uint64_t *__restrict__ tempProducts) {

    constexpr int N = SharkFloatParams::GlobalNumUint32;         // Total number of digits
    constexpr int NewN = N;
    extern __shared__ uint32_t shared_data[];


    // Define a shared global space for a bit of synchronization data.
    // Note using 128-bytes for perf
    constexpr auto EndOfSharedGlobalMemory = 128;
    constexpr auto TempBase = EndOfSharedGlobalMemory;

    constexpr auto NewN1 = (NewN + 1) / 2;
    //constexpr auto NewN2 = NewN - NewN / 2;   /* n1 is larger or same */
    constexpr auto NewN2 = NewN1;
    DefineTempProductsOffsets(TempBase);

    auto *SharkRestrict a_shared = shared_data;
    auto *SharkRestrict b_shared = a_shared + NewN;
    auto *SharkRestrict x_diff_abs = b_shared + NewN;
    auto *SharkRestrict y_diff_abs = x_diff_abs + NewN / 2;

    cg::memcpy_async(block, a_shared, A->Digits, sizeof(uint32_t) * NewN);
    cg::memcpy_async(block, b_shared, B->Digits, sizeof(uint32_t) * NewN);

    // Wait for the first batch of A to be loaded
    cg::wait(block);

    constexpr auto ExecutionBlockBase = 0;
    constexpr auto ExecutionNumBlocks = SharkFloatParams::GlobalNumBlocks;

    auto *final128 = &tempProducts[Convolution_offset];
    MultiplyDigitsOnly<SharkFloatParams,
        N,
        n1,
        n1,
        ExecutionBlockBase,
        ExecutionNumBlocks,
        SharkFloatParams::GlobalNumBlocks,
        TempBase>(
        shared_data,
        A,
        B,
        a_shared,
        b_shared,
        x_diff_abs,
        y_diff_abs,
        final128,
        grid,
        block,
        tempProducts);

    grid.sync();

    // ---- Carry Propagation ----

    // Global memory for block carry-outs
    // Allocate space for grid.dim_blocks().x block carry-outs after total_result_digits
    uint64_t *block_carry_outs = &tempProducts[CarryInsOffset];
    auto *resultDigits = &tempProducts[Result_offset];

    if constexpr (!SharkFloatParams::DisableCarryPropagation) {

        DefineCarryDefinitions();

        constexpr bool UseParallelCarry = true;
        uint64_t *globalCarryCheck = &tempProducts[GlobalCarryOffset];

        if constexpr (UseParallelCarry) {

            // First Pass: Process convolution results to compute initial digits and local carries
            CarryPropagation<SharkFloatParams>(
                (uint64_t *)shared_data,
                grid,
                block,
                block.thread_index(),
                block.group_index(),
                thread_start_idx,
                thread_end_idx,
                Convolution_offset,
                Result_offset,
                block_carry_outs,
                tempProducts,
                globalCarryCheck
            );
        } else {
            SerialCarryPropagation<SharkFloatParams>(
                (uint64_t *)shared_data,
                grid,
                block,
                block.thread_index(),
                block.group_index(),
                thread_start_idx,
                thread_end_idx,
                Convolution_offset,
                Result_offset,
                block_carry_outs,
                tempProducts,
                globalCarryCheck
            );

            grid.sync();
        }
    } else {
        grid.sync();
    }

    // ---- Finalize the Result ----
    if constexpr (!SharkFloatParams::DisableFinalConstruction) {
        // uint64_t final_carry = carryOuts_phase6[SharkFloatParams::GlobalNumBlocks - 1];

        // Initial total_result_digits is 2 * N
        int total_result_digits = 2 * N;

        // Determine the highest non-zero digit index in the full result
        int highest_nonzero_index = total_result_digits - 1;

        while (highest_nonzero_index >= 0) {
            int result_idx = Result_offset + highest_nonzero_index;
            uint32_t digit = static_cast<uint32_t>(tempProducts[result_idx]);
            if (digit != 0) {
                break;
            }

            highest_nonzero_index--;
        }

        // Determine the number of significant digits
        int significant_digits = highest_nonzero_index + 1;
        // Calculate the number of digits to shift to keep the most significant N digits
        int shift_digits = significant_digits - N;
        if (shift_digits < 0) {
            shift_digits = 0;  // No need to shift if we have fewer than N significant digits
        }

        if (block.group_index().x == 0 && block.thread_index().x == 0) {
            // Adjust the exponent based on the number of bits shifted
            Out->Exponent = A->Exponent + B->Exponent + shift_digits * 32;

            // Set the sign of the result
            Out->IsNegative = A->IsNegative ^ B->IsNegative;
        }

        int tid = block.thread_index().x + block.group_index().x * block.dim_threads().x;
        int stride = block.dim_threads().x * grid.dim_blocks().x;

        // src_idx is the starting index in tempProducts[] from which we copy
        int src_idx = Result_offset + shift_digits;
        int last_src = Result_offset + highest_nonzero_index; // The last valid index

        // We'll do a grid-stride loop over i in [0 .. N)
        for (int i = tid; i < N; i += stride) {
            // Corresponding source index for digit i
            int src = src_idx + i;

            if (src <= last_src) {
                // Copy from tempProducts
                Out->Digits[i] = tempProducts[src];
            } else {
                // Pad with zero if we've run out of digits
                Out->Digits[i] = 0;
            }
        }
    }
}

template<class SharkFloatParams>
__global__ void MultiplyKernelKaratsubaV2(
    const HpSharkFloat<SharkFloatParams> *A,
    const HpSharkFloat<SharkFloatParams> *B,
    HpSharkFloat<SharkFloatParams> *Out,
    uint64_t *tempProducts) {

    // Initialize cooperative grid group
    cg::grid_group grid = cg::this_grid();
    cg::thread_block block = cg::this_thread_block();

    // Call the MultiplyHelper function
    //MultiplyHelper(A, B, Out, carryIns, grid, tempProducts);
    MultiplyHelperKaratsubaV2(A, B, Out, grid, block, tempProducts);
}

template<class SharkFloatParams>
__global__ void MultiplyKernelKaratsubaV2TestLoop(
    HpSharkFloat<SharkFloatParams> *A,
    HpSharkFloat<SharkFloatParams> *B,
    HpSharkFloat<SharkFloatParams> *Out,
    uint64_t *tempProducts) { // Array to store cumulative carries

    // Initialize cooperative grid group
    cg::grid_group grid = cg::this_grid();
    cg::thread_block block = cg::this_thread_block();

    for (int i = 0; i < TestIterCount; ++i) {
        // MultiplyHelper(A, B, Out, carryIns, grid, tempProducts);
        if constexpr (!SharkFloatParams::ForceNoOp) {
            MultiplyHelperKaratsubaV2(A, B, Out, grid, block, tempProducts);
        } else {
            grid.sync();
        }
    }
}

template<class SharkFloatParams>
void PrintMaxActiveBlocks(int sharedAmountBytes) {
    std::cout << "Shared memory size: " << sharedAmountBytes << std::endl;

    int numBlocks;
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocks,
        MultiplyKernelKaratsubaV2<SharkFloatParams>,
        SharkFloatParams::GlobalThreadsPerBlock,
        sharedAmountBytes
    );

    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipOccupancyMaxActiveBlocksPerMultiprocessor: " << hipGetErrorString(err) << std::endl;
        return;
    }

    std::cout << "Max active blocks: " << numBlocks << std::endl;
}

template<class SharkFloatParams>
void ComputeMultiplyKaratsubaV2Gpu(void *kernelArgs[]) {

    hipError_t err;

    constexpr int N = SharkFloatParams::GlobalNumUint32;
    constexpr auto n = (N + 1) / 2;              // Half of N
    constexpr auto sharedAmountBytes = UseSharedMemory ? (2 * N + 2 * n) * sizeof(uint32_t) : 0;

    if constexpr (UseCustomStream) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            MultiplyKernelKaratsubaV2<SharkFloatParams>),
            hipFuncAttributeMaxDynamicSharedMemorySize,
            sharedAmountBytes);

        PrintMaxActiveBlocks<SharkFloatParams>(sharedAmountBytes);
    }

    err = hipLaunchCooperativeKernel(
        (void *)MultiplyKernelKaratsubaV2<SharkFloatParams>,
        dim3(SharkFloatParams::GlobalNumBlocks),
        dim3(SharkFloatParams::GlobalThreadsPerBlock),
        kernelArgs,
        sharedAmountBytes, // Shared memory size
        0 // Stream
    );

    hipDeviceSynchronize();

    if (err != hipSuccess) {
        std::cerr << "CUDA error in MultiplyKernelKaratsubaV2: " << hipGetErrorString(err) << std::endl;
    }
}

template<class SharkFloatParams>
void ComputeMultiplyKaratsubaV2GpuTestLoop(hipStream_t &stream, void *kernelArgs[]) {

    constexpr int N = SharkFloatParams::GlobalNumUint32;
    constexpr auto n = (N + 1) / 2;              // Half of N
    constexpr auto sharedAmountBytes = UseSharedMemory ? (2 * N + 2 * n) * sizeof(uint32_t) : 0;

    if constexpr (UseCustomStream) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            MultiplyKernelKaratsubaV2TestLoop<SharkFloatParams>),
            hipFuncAttributeMaxDynamicSharedMemorySize,
            sharedAmountBytes);

        PrintMaxActiveBlocks<SharkFloatParams>(sharedAmountBytes);
    }

    hipError_t err = hipLaunchCooperativeKernel(
        (void *)MultiplyKernelKaratsubaV2TestLoop<SharkFloatParams>,
        dim3(SharkFloatParams::GlobalNumBlocks),
        dim3(SharkFloatParams::GlobalThreadsPerBlock),
        kernelArgs,
        sharedAmountBytes, // Shared memory size
        stream // Stream
    );

    hipDeviceSynchronize();

    if (err != hipSuccess) {
        std::cerr << "CUDA error in MultiplyKernelKaratsubaTestLoop: " << hipGetErrorString(err) << std::endl;
    }
}

#define ExplicitlyInstantiate(SharkFloatParams) \
    template void ComputeMultiplyKaratsubaV2Gpu<SharkFloatParams>(void *kernelArgs[]); \
    template void ComputeMultiplyKaratsubaV2GpuTestLoop<SharkFloatParams>(hipStream_t &stream, void *kernelArgs[]);

ExplicitInstantiateAll();