#include "hip/hip_runtime.h"
﻿#include "Multiply.cuh"

#include <hip/hip_runtime.h>

#include "HpSharkFloat.cuh"
#include "BenchmarkTimer.h"
#include "DebugChecksum.cuh"

#include <iostream>
#include <vector>
#include <gmp.h>
#include <cstring>
#include <sstream>
#include <iomanip>
#include <cmath>
#include <algorithm>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/barrier>


namespace cg = cooperative_groups;

#ifdef _DEBUG
#define SharkForceInlineReleaseOnly
#else
// #define SharkForceInlineReleaseOnly __forceinline__
#define SharkForceInlineReleaseOnly
#endif

template<int n1, int n2>
__device__ int CompareDigits(const uint32_t *highArray, const uint32_t *lowArray) {
    // The biggest possible “digit index” is one less
    // than the max of the two sizes.
    int maxLen = std::max(n1, n2);

    // Compare top-down, from maxLen-1 down to 0
    for (int i = maxLen - 1; i >= 0; --i) {
        // Treat out-of-range as zero
        uint32_t a_val = (i < n1) ? highArray[i] : 0u;
        uint32_t b_val = (i < n2) ? lowArray[i] : 0u;

        if (a_val > b_val) {
            return 1;  // A is bigger
        } else if (a_val < b_val) {
            return -1; // B is bigger
        }
    }
    return 0;
}

template<int n1, int n2>
__device__ static void SubtractDigitsSerial(const uint32_t *a, const uint32_t *b, uint32_t *result) {
    uint64_t borrow = 0;
    for (int i = 0; i < n1; ++i) {
        uint64_t ai;
        uint64_t bi;

        ai = a[i];

        if (i >= n2) {
            bi = 0;
        } else {
            bi = b[i];
        }

        uint64_t temp = ai - bi - borrow;
        if (ai < bi + borrow) {
            borrow = 1;
            temp += ((uint64_t)1 << 32);
        } else {
            borrow = 0;
        }
        result[i] = (uint32_t)temp;
    }
}

#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

/**
 * Parallel subtraction (a1 - b1) and (a2 - b2), stored in global_(x|y)_diff_abs,
 * using a multi-pass approach to propagate borrows.
 *
 * The function attempts to subtract each digit of 'b' from 'a' in parallel,
 * then uses repeated passes (do/while) to handle newly introduced borrows
 * until no more remain or a maximum pass count is reached.
 * 
 * Corrupts x_diff_abs shared memory intentionally
 */
template<
    class SharkFloatParams,
    int a1n,
    int b1n,
    int a2n,
    int b2n,
    int ExecutionBlockBase,
    int ExecutionNumBlocks>
__device__ SharkForceInlineReleaseOnly void SubtractDigitsParallel(
    uint32_t *__restrict__ x_diff_abs,
    uint32_t *__restrict__ y_diff_abs,
    const uint32_t *__restrict__ a1,
    const uint32_t *__restrict__ b1,
    const uint32_t *__restrict__ a2,
    const uint32_t *__restrict__ b2,
    uint32_t *__restrict__ subtractionBorrows1a,
    uint32_t *__restrict__ subtractionBorrows1b,
    uint32_t *__restrict__ subtractionBorrows2a,
    uint32_t *__restrict__ subtractionBorrows2b,
    uint32_t *__restrict__ global_x_diff_abs,
    uint32_t *__restrict__ global_y_diff_abs,
    uint32_t *__restrict__ globalBorrowAny,
    cg::grid_group &grid,
    cg::thread_block &block
) {
    // Note: stops on this.
    auto *sharedBorrowAny = x_diff_abs;

    // Note: not ExecutionBlockBase
    if (block.group_index().x == 0 && block.thread_index().x == 0) {
        *globalBorrowAny = 0;
    }

    if (block.thread_index().x == 0) {
        *sharedBorrowAny = 0;
    }

    // Constants 
    constexpr int MaxPasses = 5000;     // maximum number of multi-pass sweeps

    // We'll define a grid–stride range covering [0..n) for each pass
    // 1) global thread id
    int tid = (block.group_index().x - ExecutionBlockBase) * block.dim_threads().x + block.thread_index().x;
    // 2) stride
    int stride = block.dim_threads().x * ExecutionNumBlocks;

    constexpr auto n1max = std::max(a1n, b1n);
    constexpr auto n2max = std::max(a2n, b2n);
    constexpr auto nmax = std::max(n1max, n2max);

    // (1) First pass: naive partial difference (a[i] - b[i]) and set borrowBit
    // Instead of dividing digits among blocks, each thread does a grid–stride loop:
    for (int idx = tid; idx < nmax; idx += stride) {
        uint32_t ai1;
        uint32_t bi1;
        uint32_t ai2;
        uint32_t bi2;

        // Fill in with 0s if idx is out of bounds
        if (idx < a1n) {
            ai1 = a1[idx];
        } else {
            ai1 = 0;
        }

        if (idx < a2n) {
            ai2 = a2[idx];
        } else {
            ai2 = 0;
        }

        if (idx < b1n) {
            bi1 = b1[idx];
        } else {
            bi1 = 0;
        }

        if (idx < b2n) {
            bi2 = b2[idx];
        } else {
            bi2 = 0;
        }

        // naive difference
        uint64_t diff1 = (uint64_t)ai1 - (uint64_t)bi1;
        uint64_t diff2 = (uint64_t)ai2 - (uint64_t)bi2;

        uint32_t borrow1 = (ai1 < bi1) ? 1u : 0u;
        uint32_t borrow2 = (ai2 < bi2) ? 1u : 0u;

        global_x_diff_abs[idx] = static_cast<uint32_t>(diff1 & 0xFFFFFFFFu);
        subtractionBorrows1a[idx] = borrow1;

        global_y_diff_abs[idx] = static_cast<uint32_t>(diff2 & 0xFFFFFFFFu);
        subtractionBorrows2a[idx] = borrow2;
    }

    // sync the entire grid before multi-pass fixes
    //grid.sync();

    // We'll do repeated passes to fix newly introduced borrows
    uint32_t *curBorrow1 = subtractionBorrows1a;
    uint32_t *newBorrow1 = subtractionBorrows1b;
    uint32_t *curBorrow2 = subtractionBorrows2a;
    uint32_t *newBorrow2 = subtractionBorrows2b;
    int pass = 0;
    uint32_t initialBorrowAny = 0;

    grid.sync();

    do {
        // (2) For each digit, apply the borrow from the previous digit
        for (int idx = tid; idx < nmax; idx += stride) {
            uint64_t borrow_in1 = 0ULL;
            uint64_t borrow_in2 = 0ULL;
            if (idx > 0) {   // borrow_in is from digit (idx-1)
                borrow_in1 = (uint64_t)(curBorrow1[idx - 1]);
                borrow_in2 = (uint64_t)(curBorrow2[idx - 1]);
            }

            uint32_t digit1 = global_x_diff_abs[idx];
            uint32_t digit2 = global_y_diff_abs[idx];

            // subtract the borrow
            uint64_t sum1 = (uint64_t)digit1 - borrow_in1;
            uint64_t sum2 = (uint64_t)digit2 - borrow_in2;

            // store updated digit
            global_x_diff_abs[idx] = static_cast<uint32_t>(sum1 & 0xFFFFFFFFULL);
            global_y_diff_abs[idx] = static_cast<uint32_t>(sum2 & 0xFFFFFFFFULL);

            // If sum is negative => top bit is 1 => new borrow
            if (sum1 & 0x8000'0000'0000'0000ULL) {
                newBorrow1[idx] = 1;
                atomicAdd(sharedBorrowAny, 1);
            } else {
                newBorrow1[idx] = 0;
            }

            if (sum2 & 0x8000'0000'0000'0000ULL) {
                newBorrow2[idx] = 1;
                atomicAdd(sharedBorrowAny, 1);
            } else {
                newBorrow2[idx] = 0;
            }
        }

        // (a) Block-level synchronization (so all threads see final sharedBorrowAny)
        block.sync();

        // The block's thread 0 aggregates once into globalBorrowAny
        if (block.thread_index().x == 0) {
            // Add sharedBorrowAny to the global counter
            atomicAdd(globalBorrowAny, *sharedBorrowAny);

            // Reset local aggregator for the next pass
            *sharedBorrowAny = 0;
        }

        // sync before checking if any new borrows remain
        grid.sync();

        auto tempCopyGlobalBorrowAny = *globalBorrowAny;
        if (tempCopyGlobalBorrowAny == initialBorrowAny) {
            break;  // no new borrows => done
        }

        grid.sync();
        initialBorrowAny = tempCopyGlobalBorrowAny;

        // swap curBorrow, newBorrow
        uint32_t *tmp = curBorrow1;
        curBorrow1 = newBorrow1;
        newBorrow1 = tmp;

        tmp = curBorrow2;
        curBorrow2 = newBorrow2;
        newBorrow2 = tmp;

        pass++;
    } while (pass < MaxPasses);

    if constexpr (SharkDebug) {
        if (pass == MaxPasses && block.group_index().x == 0) {
            // This will deadlock the kernel because this problem is hard to diagnose
            grid.sync();
        }
    }
}



// Function to perform addition with carry
__device__ SharkForceInlineReleaseOnly static void Add128(
    uint64_t a_low, uint64_t a_high,
    uint64_t b_low, uint64_t b_high,
    uint64_t &result_low, uint64_t &result_high) {

    result_low = a_low + b_low;
    uint64_t carry = (result_low < a_low) ? 1 : 0;
    result_high = a_high + b_high + carry;
}

__device__ SharkForceInlineReleaseOnly static void Subtract128(
    uint64_t a_low, uint64_t a_high,
    uint64_t b_low, uint64_t b_high,
    uint64_t &result_low, uint64_t &result_high) {

    uint64_t borrow = 0;

    // Subtract low parts
    result_low = a_low - b_low;
    borrow = (a_low < b_low) ? 1 : 0;

    // Subtract high parts with borrow
    result_high = a_high - b_high - borrow;
}

template<class SharkFloatParams>
__device__ SharkForceInlineReleaseOnly static void SerialCarryPropagation(
    uint64_t *__restrict__ shared_carries,
    cg::grid_group &grid,
    cg::thread_block &block,
    const uint3 &threadIdx,
    const uint3 &blockIdx,
    int thread_start_idx,
    int thread_end_idx,
    int Convolution_offset,
    int Result_offset,
    uint64_t *__restrict__ block_carry_outs,
    uint64_t *__restrict__ tempProducts,
    uint64_t *__restrict__ globalCarryCheck) {

    if (block.thread_index().x == 0 && block.group_index().x == 0) {
        uint64_t local_carry = 0;

        for (int idx = 0; idx < SharkFloatParams::GlobalNumUint32 * 2 + 1; ++idx) {
            int sum_low_idx = Convolution_offset + idx * 2;
            int sum_high_idx = sum_low_idx + 1;

            uint64_t sum_low = tempProducts[sum_low_idx];     // Lower 64 bits
            uint64_t sum_high = tempProducts[sum_high_idx];   // Higher 64 bits

            // Add local carry to sum_low
            bool new_sum_low_negative = false;
            uint64_t new_sum_low = sum_low + local_carry;

            // Extract one 32-bit digit from new_sum_low
            auto digit = static_cast<uint32_t>(new_sum_low & 0xFFFFFFFFULL);
            tempProducts[Result_offset + idx] = digit;

            bool local_carry_negative = ((local_carry & (1ULL << 63)) != 0);
            local_carry = 0ULL;

            if (!local_carry_negative && new_sum_low < sum_low) {
                local_carry = 1ULL << 32;
            } else if (local_carry_negative && new_sum_low > sum_low) {
                new_sum_low_negative = (new_sum_low & 0x8000'0000'0000'0000) != 0;
            }

            // Update local_carry
            if (new_sum_low_negative) {
                // Shift sum_high by 32 bits and add carry_from_low
                uint64_t upper_new_sum_low = new_sum_low >> 32;
                upper_new_sum_low |= 0xFFFF'FFFF'0000'0000;
                local_carry += upper_new_sum_low;
                local_carry += sum_high << 32;
            } else {
                local_carry += new_sum_low >> 32;
                local_carry += sum_high << 32;
            }
        }
    }
}

template<class SharkFloatParams>
__device__ SharkForceInlineReleaseOnly static void CarryPropagation (
    uint64_t *__restrict__ shared_carries,
    cg::grid_group &grid,
    cg::thread_block &block,
    const uint3 &threadIdx,
    const uint3 &blockIdx,
    int thread_start_idx,
    int thread_end_idx,
    int Convolution_offset,
    int Result_offset,
    uint64_t * __restrict__ block_carry_outs,
    uint64_t * __restrict__ tempProducts,
    uint64_t * __restrict__ globalCarryCheck) {

    // First Pass: Process convolution results to compute initial digits and local carries
    // Initialize local carry
    uint64_t local_carry = 0;

    // Constants and offsets
    constexpr int MaxPasses = 150; // Maximum number of carry propagation passes
    constexpr int total_result_digits = 2 * SharkFloatParams::GlobalNumUint32;

    uint64_t *carries_remaining_global = globalCarryCheck;

    for (int idx = thread_start_idx; idx < thread_end_idx; ++idx) {
        int sum_low_idx = Convolution_offset + idx * 2;
        int sum_high_idx = sum_low_idx + 1;

        uint64_t sum_low = tempProducts[sum_low_idx];     // Lower 64 bits
        uint64_t sum_high = tempProducts[sum_high_idx];   // Higher 64 bits

        // Add local carry to sum_low
        bool new_sum_low_negative = false;
        uint64_t new_sum_low = sum_low + local_carry;

        // Extract one 32-bit digit from new_sum_low
        auto digit = static_cast<uint32_t>(new_sum_low & 0xFFFFFFFFULL);
        tempProducts[Result_offset + idx] = digit;

        bool local_carry_negative = ((local_carry & (1ULL << 63)) != 0);
        local_carry = 0ULL;

        if (!local_carry_negative && new_sum_low < sum_low) {
            local_carry = 1ULL << 32;
        } else if (local_carry_negative && new_sum_low > sum_low) {
            new_sum_low_negative = (new_sum_low & 0x8000'0000'0000'0000) != 0;
        }

        // Update local_carry
        if (new_sum_low_negative) {
            // Shift sum_high by 32 bits and add carry_from_low
            uint64_t upper_new_sum_low = new_sum_low >> 32;
            upper_new_sum_low |= 0xFFFF'FFFF'0000'0000;
            local_carry += upper_new_sum_low;
            local_carry += sum_high << 32;
        } else {
            local_carry += new_sum_low >> 32;
            local_carry += sum_high << 32;
        }
    }

    if (block.thread_index().x == SharkFloatParams::GlobalThreadsPerBlock - 1) {
        block_carry_outs[block.group_index().x] = local_carry;
    } else {
        shared_carries[block.thread_index().x] = local_carry;
    }

    // Inter-Block Carry Propagation
    int pass = 0;

    do {
        // Synchronize all blocks
        grid.sync();

        // Zero out the global carry count for the current pass
        if (block.group_index().x == 0 && block.thread_index().x == 0) {
            *carries_remaining_global = 0;
        }

        // Get carry-in from the previous block
        local_carry = 0;
        if (block.thread_index().x == 0 && block.group_index().x > 0) {
            local_carry = block_carry_outs[block.group_index().x - 1];
        } else {
            if (block.thread_index().x > 0) {
                local_carry = shared_carries[block.thread_index().x - 1];
            }
        }

        // Each thread processes its assigned digits
        for (int idx = thread_start_idx; idx < thread_end_idx; ++idx) {
            // Read the previously stored digit
            uint32_t digit = tempProducts[Result_offset + idx];

            // Add local_carry to digit
            uint64_t sum = static_cast<uint64_t>(digit) + local_carry;

            // Update digit
            digit = static_cast<uint32_t>(sum & 0xFFFFFFFFULL);
            tempProducts[Result_offset + idx] = digit;

            local_carry = 0;

            // Check negativity of the 64-bit sum
            // If "sum" is negative, its top bit is set. 
            bool sum_is_negative = ((sum & (1ULL << 63)) != 0ULL);

            if (sum_is_negative) {
                // sign-extend the top 32 bits
                uint64_t upper_bits = (sum >> 32);
                upper_bits |= 0xFFFF'FFFF'0000'0000ULL;  // set top 32 bits to 1
                local_carry += upper_bits;               // incorporate sign-extended bits
            } else {
                // normal path: just add top 32 bits
                local_carry += (sum >> 32);
            }
        }

        shared_carries[block.thread_index().x] = local_carry;
        block.sync();

        // The block's carry-out is the carry from the last thread
        auto temp = shared_carries[block.thread_index().x];
        if (block.thread_index().x == SharkFloatParams::GlobalThreadsPerBlock - 1) {
            block_carry_outs[block.group_index().x] = temp;
        }

        if (temp != 0) {
            atomicAdd(carries_remaining_global, 1);
        }

        // Synchronize all blocks before checking if carries remain
        grid.sync();

        // If no carries remain, exit the loop
        if (*carries_remaining_global == 0) {
            break;
        }

        pass++;
    } while (pass < MaxPasses);

    // ---- Handle Final Carry-Out ----

    // Handle final carry-out
    if (block.thread_index().x == 0 && block.group_index().x == grid.dim_blocks().x - 1) {
        uint64_t final_carry = block_carry_outs[block.group_index().x];
        if (final_carry > 0) {
            // Store the final carry as an additional digit
            tempProducts[Result_offset + total_result_digits] = static_cast<uint32_t>(final_carry & 0xFFFFFFFFULL);
            // Optionally, you may need to adjust total_result_digits
        }
    }

    // Synchronize all blocks before finalization
    // grid.sync();
}

// Look for CalculateFrameSize and ScratchMemoryArrays
// and make sure the number of NewN arrays we're using here fits within that limit.
// The list here should go up to ScratchMemoryArrays.
static_assert(AdditionalUInt64PerFrame == 256, "See below");
#define DefineTempProductsOffsets(TempBase, CallIndex) \
    const int threadIdxGlobal = block.group_index().x * SharkFloatParams::GlobalThreadsPerBlock + block.thread_index().x; \
    constexpr int TestMultiplier = 1; \
    constexpr auto CallOffset = CallIndex * CalculateFrameSize<SharkFloatParams>(); \
    constexpr auto TempBaseOffset = TempBase + CallOffset; \
    constexpr auto BorrowGlobalOffset = 0; \
    constexpr auto Checksum_offset = AdditionalGlobalSyncSpace; \
    auto *debugTrackerArray = reinterpret_cast<DebugState<SharkFloatParams>*>(&tempProducts[Checksum_offset]); \
    constexpr auto Z0_offset = TempBaseOffset + AdditionalUInt64PerFrame; \
    constexpr auto Z2_offset = Z0_offset + 4 * NewN * TestMultiplier; \
    constexpr auto Z1_temp_offset = Z2_offset + 4 * NewN * TestMultiplier; \
    constexpr auto Z1_offset = Z1_temp_offset + 4 * NewN * TestMultiplier; \
    constexpr auto Convolution_offset = Z1_offset + 4 * NewN * TestMultiplier;       /* 17 */ \
    constexpr auto Result_offset = Convolution_offset + 4 * NewN * TestMultiplier;   /* 21 */ \
    constexpr auto XDiff_offset = Result_offset + 2 * NewN * TestMultiplier;         /* 23 */ \
    constexpr auto YDiff_offset = XDiff_offset + 1 * NewN * TestMultiplier;          /* 24 */ \
    constexpr auto GlobalCarryOffset = YDiff_offset + 1 * NewN * TestMultiplier;     /* 25 */ \
    constexpr auto SubtractionOffset1 = GlobalCarryOffset + 1 * NewN * TestMultiplier;   /* 26 */ \
    constexpr auto SubtractionOffset2 = SubtractionOffset1 + 1 * NewN * TestMultiplier;  /* 27 */ \
    constexpr auto SubtractionOffset3 = SubtractionOffset2 + 1 * NewN * TestMultiplier;  /* 28 */ \
    constexpr auto SubtractionOffset4 = SubtractionOffset3 + 1 * NewN * TestMultiplier;  /* 29 */


#define DefineExtraDefinitions() \
    const auto RelativeBlockIndex = block.group_index().x - ExecutionBlockBase; \
    constexpr int total_result_digits = 2 * NewN; \
    constexpr auto digits_per_block = NewN * 2 / ExecutionNumBlocks; \
    auto block_start_idx = block.group_index().x * digits_per_block; \
    auto block_end_idx = min(block_start_idx + digits_per_block, total_result_digits); \
    int digits_per_thread = (digits_per_block + block.dim_threads().x - 1) / block.dim_threads().x; \
    int thread_start_idx = block_start_idx + block.thread_index().x * digits_per_thread; \
    int thread_end_idx = min(thread_start_idx + digits_per_thread, block_end_idx);

#define DefineCarryDefinitions() \
    constexpr int total_result_digits = 2 * NewN; \
    constexpr auto digits_per_block = SharkFloatParams::GlobalThreadsPerBlock * 2; \
    auto block_start_idx = block.group_index().x * digits_per_block; \
    auto block_end_idx = min(block_start_idx + digits_per_block, total_result_digits); \
    int digits_per_thread = (digits_per_block + block.dim_threads().x - 1) / block.dim_threads().x; \
    int thread_start_idx = block_start_idx + block.thread_index().x * digits_per_thread; \
    int thread_end_idx = min(thread_start_idx + digits_per_thread, block_end_idx);

template<
    class SharkFloatParams,
    int CallIndex,
    DebugStatePurpose Purpose>
__device__ SharkForceInlineReleaseOnly void
EraseCurrentDebugState(
    bool record,
    DebugState<SharkFloatParams> *debugTrackerArray,
    cooperative_groups::grid_group &grid,
    cooperative_groups::thread_block &block) {

    constexpr auto maxPurposes = static_cast<int>(DebugStatePurpose::NumPurposes);
    constexpr auto curPurpose = static_cast<int>(Purpose);
    debugTrackerArray[CallIndex * maxPurposes + curPurpose].Erase(
        record, grid, block, Purpose, CallIndex);
}

template<
    class SharkFloatParams,
    int CallIndex,
    DebugStatePurpose Purpose,
    typename ArrayType>
__device__ SharkForceInlineReleaseOnly void
StoreCurrentDebugState (
    bool record,
    DebugState<SharkFloatParams> *debugTrackerArray,
    cooperative_groups::grid_group &grid,
    cooperative_groups::thread_block &block,
    const ArrayType *arrayToChecksum,
    size_t arraySize)
{
    constexpr auto maxPurposes = static_cast<int>(DebugStatePurpose::NumPurposes);
    constexpr auto curPurpose = static_cast<int>(Purpose);
    debugTrackerArray[CallIndex * maxPurposes + curPurpose].Reset(
        record, grid, block, arrayToChecksum, arraySize, Purpose, CallIndex);
}

// Assuming that SharkFloatParams::GlobalNumUint32 can be large and doesn't fit in shared memory
// We'll use the provided global memory buffers for large intermediates
// #define SharkRestrict __restrict__
#define SharkRestrict

template<
    class SharkFloatParams,
    int RecursionDepth,
    int CallIndex,
    int NewN,
    int n1,
    int n2,
    int ExecutionBlockBase,
    int ExecutionNumBlocks,
    int NewNumBlocks,
    int TempBase>
__device__ SharkForceInlineReleaseOnly void MultiplyDigitsOnly(
    uint32_t *SharkRestrict shared_data,
    const HpSharkFloat<SharkFloatParams> *SharkRestrict A,
    const HpSharkFloat<SharkFloatParams> *SharkRestrict B,
    const uint32_t *SharkRestrict aDigits,
    const uint32_t *SharkRestrict bDigits,
    uint32_t *SharkRestrict x_diff_abs,
    uint32_t *SharkRestrict y_diff_abs,
    uint64_t *SharkRestrict final128,
    cg::grid_group &grid,
    cg::thread_block &block,
    uint64_t *SharkRestrict tempProducts) {

    if ((ExecutionBlockBase > 0 && block.group_index().x < ExecutionBlockBase) ||
        block.group_index().x >= ExecutionBlockBase + ExecutionNumBlocks) {

        return;
    }

    DefineTempProductsOffsets(TempBase, CallIndex);
    constexpr auto MaxHalfN = std::max(n1, n2);
    constexpr int total_k = MaxHalfN * 2 - 1; // Total number of k values
    constexpr bool UseConvolution =
        (NewNumBlocks <= std::max(SharkFloatParams::GlobalNumBlocks / SharkFloatParams::ConvolutionLimit, 1) ||
        (NewNumBlocks % 3 != 0));
    constexpr bool EnableSharedDiff = true; // TODO
    constexpr bool UseParallelSubtract = true;

    using DebugState = DebugState<SharkFloatParams>;

    const bool record = block.thread_index().x == 0 && block.group_index().x == ExecutionBlockBase;

    if constexpr (DebugChecksums) {
        grid.sync();

        EraseCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::Invalid>(
            record, debugTrackerArray, grid, block);
        StoreCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::ADigits, uint32_t>(
            record, debugTrackerArray, grid, block, aDigits, NewN);
        StoreCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::BDigits, uint32_t>(
            record, debugTrackerArray, grid, block, bDigits, NewN);

        grid.sync();
    }

    auto *Z0_OutDigits = &tempProducts[Z0_offset];
    auto *Z1_temp_digits = &tempProducts[Z1_temp_offset];
    auto *Z2_OutDigits = &tempProducts[Z2_offset];

    // Arrays to hold the absolute differences (size n)
    auto *global_x_diff_abs = reinterpret_cast<uint32_t *>(&tempProducts[XDiff_offset]);
    auto *global_y_diff_abs = reinterpret_cast<uint32_t *>(&tempProducts[YDiff_offset]);

    // ---- Compute Differences x_diff = A1 - A0 and y_diff = B1 - B0 ----

    DefineExtraDefinitions();

    int x_diff_sign = 0; // 0 if positive, 1 if negative
    int y_diff_sign = 0; // 0 if positive, 1 if negative

    // Compute x_diff_abs and x_diff_sign
    auto *SharkRestrict subtractionBorrows = reinterpret_cast<uint32_t *>(&tempProducts[SubtractionOffset1]);
    auto *SharkRestrict subtractionBorrows2 = reinterpret_cast<uint32_t *>(&tempProducts[SubtractionOffset2]);
    auto *SharkRestrict subtractionBorrows3 = reinterpret_cast<uint32_t *>(&tempProducts[SubtractionOffset3]);
    auto *SharkRestrict subtractionBorrows4 = reinterpret_cast<uint32_t *>(&tempProducts[SubtractionOffset4]);
    auto *SharkRestrict globalBorrowAny = reinterpret_cast<uint32_t *>(&tempProducts[BorrowGlobalOffset]);

    const auto SharkRestrict *a_high = aDigits + n1;
    const auto SharkRestrict *b_high = bDigits + n1;
    const auto SharkRestrict *a_low = aDigits;
    const auto SharkRestrict *b_low = bDigits;

    if constexpr (DebugChecksums) {
        grid.sync();

        StoreCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::AHalfHigh>(
            record, debugTrackerArray, grid, block, a_high, n2);
        StoreCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::AHalfLow>(
            record, debugTrackerArray, grid, block, a_low, n1);
        StoreCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::BHalfHigh>(
            record, debugTrackerArray, grid, block, b_high, n2);
        StoreCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::BHalfLow>(
            record, debugTrackerArray, grid, block, b_low, n1);

        grid.sync();
    }

    if constexpr (!SharkFloatParams::DisableSubtraction) {
        if constexpr (UseParallelSubtract) {
            int x_compare = CompareDigits<n2, n1>(a_high, a_low);
            int y_compare = CompareDigits<n2, n1>(b_high, b_low);

            if (x_compare >= 0 && y_compare >= 0) {
                x_diff_sign = 0;
                y_diff_sign = 0;
                SubtractDigitsParallel<
                    SharkFloatParams,
                    n2,
                    n1,
                    n2,
                    n1,
                    ExecutionBlockBase,
                    ExecutionNumBlocks>(
                        x_diff_abs,
                        y_diff_abs,
                        a_high,
                        a_low,
                        b_high,
                        b_low,
                        subtractionBorrows,
                        subtractionBorrows2,
                        subtractionBorrows3,
                        subtractionBorrows4,
                        global_x_diff_abs,
                        global_y_diff_abs,
                        globalBorrowAny,
                        grid,
                        block);
            } else if (x_compare < 0 && y_compare < 0) {
                x_diff_sign = 1;
                y_diff_sign = 1;
                SubtractDigitsParallel<
                    SharkFloatParams,
                    n1,
                    n2,
                    n1,
                    n2,
                    ExecutionBlockBase,
                    ExecutionNumBlocks>(
                        x_diff_abs,
                        y_diff_abs,
                        a_low,
                        a_high,
                        b_low,
                        b_high,
                        subtractionBorrows,
                        subtractionBorrows2,
                        subtractionBorrows3,
                        subtractionBorrows4,
                        global_x_diff_abs,
                        global_y_diff_abs,
                        globalBorrowAny,
                        grid,
                        block);
            } else if (x_compare >= 0 && y_compare < 0) {
                x_diff_sign = 0;
                y_diff_sign = 1;
                SubtractDigitsParallel<
                    SharkFloatParams,
                    n2,
                    n1,
                    n1,
                    n2,
                    ExecutionBlockBase,
                    ExecutionNumBlocks>(
                        x_diff_abs,
                        y_diff_abs,
                        a_high,
                        a_low,
                        b_low,
                        b_high,
                        subtractionBorrows,
                        subtractionBorrows2,
                        subtractionBorrows3,
                        subtractionBorrows4,
                        global_x_diff_abs,
                        global_y_diff_abs,
                        globalBorrowAny,
                        grid,
                        block);
            } else {
                x_diff_sign = 1;
                y_diff_sign = 0;
                SubtractDigitsParallel<
                    SharkFloatParams,
                    n1,
                    n2,
                    n2,
                    n1,
                    ExecutionBlockBase,
                    ExecutionNumBlocks>(
                        x_diff_abs,
                        y_diff_abs,
                        a_low,
                        a_high,
                        b_high,
                        b_low,
                        subtractionBorrows,
                        subtractionBorrows2,
                        subtractionBorrows3,
                        subtractionBorrows4,
                        global_x_diff_abs,
                        global_y_diff_abs,
                        globalBorrowAny,
                        grid,
                        block);
            }
        } else {
            if (block.thread_index().x == 0 && block.group_index().x == ExecutionBlockBase) {
                int x_compare = CompareDigits<n1, n2>(a_high, a_low);

                if (x_compare >= 0) {
                    x_diff_sign = 0;
                    SubtractDigitsSerial<n2, n1>(a_high, a_low, global_x_diff_abs); // x_diff = A1 - A0
                } else {
                    x_diff_sign = 1;
                    SubtractDigitsSerial<n1, n2>(a_low, a_high, global_x_diff_abs); // x_diff = A0 - A1
                }

                // Compute y_diff_abs and y_diff_sign
                int y_compare = CompareDigits<n1, n2>(b_high, b_low);
                if (y_compare >= 0) {
                    y_diff_sign = 0;
                    SubtractDigitsSerial<n2, n1>(b_high, b_low, global_y_diff_abs); // y_diff = B1 - B0
                } else {
                    y_diff_sign = 1;
                    SubtractDigitsSerial<n1, n2>(b_low, b_high, global_y_diff_abs); // y_diff = B0 - B1
                }
            }
        }
    }


    if constexpr (DebugChecksums) {
        grid.sync();
    
        StoreCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::XDiff>(
            record, debugTrackerArray, grid, block, global_x_diff_abs, MaxHalfN);
        StoreCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::YDiff>(
            record, debugTrackerArray, grid, block, global_y_diff_abs, MaxHalfN);

        grid.sync();
    }

    constexpr auto NumBlocksRatio =
        SharkFloatParams::ConvolutionLimitPow *
        SharkFloatParams::GlobalNumBlocks /
        NewNumBlocks;

    constexpr auto SubNewNRoundUp = (NewN + 1) / 2;
    constexpr auto SubNewN2a = SubNewNRoundUp / 2;
    constexpr auto SubNewN1a = SubNewNRoundUp - SubNewN2a;   /* n1 is larger or same */

    constexpr auto SubRemainingNewN = NewN - SubNewNRoundUp;
    constexpr auto SubNewN2b = SubRemainingNewN / 2;
    constexpr auto SubNewN1b = SubRemainingNewN - SubNewN2b;   /* n1 is larger or same */

    // Determine the sign of Z1_temp
    int z1_sign = x_diff_sign ^ y_diff_sign;

    if constexpr (UseConvolution) {
        // Replace A and B in shared memory with their absolute differences
        if constexpr (EnableSharedDiff) {
            cg::memcpy_async(block, const_cast<uint32_t *>(x_diff_abs), global_x_diff_abs, sizeof(uint32_t) * MaxHalfN);
            cg::memcpy_async(block, const_cast<uint32_t *>(y_diff_abs), global_y_diff_abs, sizeof(uint32_t) * MaxHalfN);
        }

        const int tid = RelativeBlockIndex * block.dim_threads().x + block.thread_index().x;
        const int stride = block.dim_threads().x * ExecutionNumBlocks;

        if constexpr (EnableSharedDiff) {
            // Wait for the first batch of A to be loaded
            cg::wait(block);
        }

        // A single loop that covers 2*total_k elements
        for (int idx = tid; idx < 3 * total_k; idx += stride) {
            
            // Check if idx < total_k => handle Z0, else handle Z2
            if (idx < total_k) {
                // Z0 partial sums
                int k = idx;
                uint64_t sum_low = 0ULL, sum_high = 0ULL;

                int i_start = (k < n1) ? 0 : (k - (n1 - 1));
                int i_end = (k < n1) ? k : (n1 - 1);

                for (int i = i_start; i <= i_end; i++) {
                    uint64_t a;
                    uint64_t b;

                    a = aDigits[i]; // A_shared[i];         // A0[i]
                    b = bDigits[k - i]; // B_shared[k - i];     // B0[k - i]

                    uint64_t product = a * b;

                    // Add product to sum
                    sum_low += product;
                    if (sum_low < product) {
                        sum_high += 1;
                    }
                }

                // store sum_low, sum_high in Z0_OutDigits
                int out_idx = k * 2;
                Z0_OutDigits[out_idx] = sum_low;
                Z0_OutDigits[out_idx + 1] = sum_high;
            } else if (idx < 2 * total_k) {
                // Z2 partial sums
                int k = idx - total_k; // shift to [0..total_k-1]
                uint64_t sum_low = 0ULL, sum_high = 0ULL;

                int i_start = (k < n2) ? 0 : (k - (n2 - 1));
                int i_end = (k < n2) ? k : (n2 - 1);

                for (int i = i_start; i <= i_end; i++) {
                    uint64_t a;
                    uint64_t b;

                    a = aDigits[i + n1]; // A_shared[i];         // A1[i]
                    b = bDigits[k - i + n1]; // B_shared[k - i];     // B1[k - i]

                    uint64_t product = a * b;

                    // Add product to sum
                    sum_low += product;
                    if (sum_low < product) {
                        sum_high += 1;
                    }
                }

                // store sum_low, sum_high in Z2_OutDigits
                int out_idx = k * 2;
                Z2_OutDigits[out_idx] = sum_low;
                Z2_OutDigits[out_idx + 1] = sum_high;
            } else {
                int k = idx - 2 * total_k; // shift to [0..total_k-1]
                uint64_t sum_low = 0;
                uint64_t sum_high = 0;

                int i_start = (k < MaxHalfN) ? 0 : (k - (MaxHalfN - 1));
                int i_end = (k < MaxHalfN) ? k : (MaxHalfN - 1);

                for (int i = i_start; i <= i_end; ++i) {
                    uint64_t a;
                    uint64_t b;

                    a = EnableSharedDiff ? x_diff_abs[i] : global_x_diff_abs[i];
                    b = EnableSharedDiff ? y_diff_abs[k - i] : global_y_diff_abs[k - i];

                    uint64_t product = a * b;

                    // Accumulate the product
                    sum_low += product;
                    if (sum_low < product) {
                        sum_high += 1;
                    }
                }

                // Store sum_low and sum_high in tempProducts
                int out_idx = k * 2;
                Z1_temp_digits[out_idx] = sum_low;
                Z1_temp_digits[out_idx + 1] = sum_high;
            }
        }

        if constexpr (DebugChecksums) {
            grid.sync();

            StoreCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::Z0>(
                record, debugTrackerArray, grid, block, Z0_OutDigits, total_k * 2);
            StoreCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::Z2>(
                record, debugTrackerArray, grid, block, Z2_OutDigits, total_k * 2);
            StoreCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::Z1_offset>(
                record, debugTrackerArray, grid, block, Z1_temp_digits, total_k * 2);

            grid.sync();
        }

    } else {
        static_assert(RecursionDepth <= 5, "Unexpected recursion depth");

        MultiplyDigitsOnly<
            SharkFloatParams,
            RecursionDepth + 1,
            CallIndex * 3 - 1,
            SubNewNRoundUp,
            SubNewN1a,
            SubNewN2a,
            ExecutionBlockBase,
            ExecutionNumBlocks / 3,
            NewNumBlocks / 3,
            TempBase>(
            shared_data,
            A,
            B,
            aDigits,
            bDigits,
            x_diff_abs,
            y_diff_abs,
            Z0_OutDigits,
            grid,
            block,
            tempProducts);

        MultiplyDigitsOnly<
            SharkFloatParams,
            RecursionDepth + 1,
            CallIndex * 3,
            SubRemainingNewN,
            SubNewN1b,
            SubNewN2b,
            ExecutionBlockBase + ExecutionNumBlocks / 3,
            ExecutionNumBlocks / 3,
            NewNumBlocks / 3,
            TempBase>(
            shared_data,
            A,
            B,
            aDigits + n1,
            bDigits + n1,
            x_diff_abs,
            y_diff_abs,
            Z2_OutDigits,
            grid,
            block,
            tempProducts);

        //grid.sync();

        {
            constexpr auto NewExecutionBlockBase = ExecutionBlockBase + 2 * ExecutionNumBlocks / 3;
            constexpr auto NewExecutionNumBlocks = ExecutionNumBlocks / 3;

            const bool ExecuteAtAll =
                !((NewExecutionBlockBase > 0 && block.group_index().x < NewExecutionBlockBase) ||
                    block.group_index().x >= NewExecutionBlockBase + NewExecutionNumBlocks);
            constexpr auto MaxSubNewN = std::max(SubNewN1a, SubNewN2a);

            if (ExecuteAtAll) {
                // Replace A and B in shared memory with their absolute differences
                if constexpr (EnableSharedDiff) {
                    cg::memcpy_async(block,
                        const_cast<uint32_t *>(aDigits),
                        global_x_diff_abs,
                        sizeof(uint32_t) * MaxHalfN);
                    cg::memcpy_async(block,
                        const_cast<uint32_t *>(bDigits),
                        global_y_diff_abs,
                        sizeof(uint32_t) * MaxHalfN);
                    cg::wait(block);
                }

                MultiplyDigitsOnly<
                    SharkFloatParams,
                    RecursionDepth + 1,
                    CallIndex * 3 + 1,
                    SubNewNRoundUp,
                    SubNewN1a,
                    SubNewN2a,
                    NewExecutionBlockBase,
                    NewExecutionNumBlocks,
                    NewNumBlocks / 3,
                    TempBase>(
                        shared_data,
                        A,
                        B,
                        EnableSharedDiff ? aDigits : global_x_diff_abs,
                        EnableSharedDiff ? bDigits : global_y_diff_abs,
                        x_diff_abs,
                        y_diff_abs,
                        Z1_temp_digits,
                        grid,
                        block,
                        tempProducts);

                if constexpr (EnableSharedDiff) {
                    cg::memcpy_async(block,
                        const_cast<uint32_t *>(aDigits),
                        A->Digits,
                        sizeof(uint32_t) * SharkFloatParams::GlobalNumUint32);
                    cg::memcpy_async(block,
                        const_cast<uint32_t *>(bDigits),
                        B->Digits,
                        sizeof(uint32_t) * SharkFloatParams::GlobalNumUint32);
                    cg::wait(block);
                }
            }

            if constexpr (DebugChecksums) {
                grid.sync();

                StoreCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::Z0>(
                    record, debugTrackerArray, grid, block, Z0_OutDigits, SubNewNRoundUp * 2 * 2);
                StoreCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::Z2>(
                    record, debugTrackerArray, grid, block, Z2_OutDigits, SubRemainingNewN * 2 * 2);
                StoreCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::Z1_offset>(
                    record, debugTrackerArray, grid, block, Z1_temp_digits, SubNewNRoundUp * 2 * 2);

                grid.sync();
            }
        }
    }

    grid.sync();

    auto *Z1_digits = &tempProducts[Z1_offset];

    if constexpr (!SharkFloatParams::DisableAllAdditions) {

        // After computing Z1_temp (Z1'), we now form Z1 directly:
        // If z1_sign == 0: Z1 = Z2 + Z0 - Z1_temp
        // If z1_sign == 1: Z1 = Z2 + Z0 + Z1_temp

        const int tid = RelativeBlockIndex * block.dim_threads().x + block.thread_index().x;
        const int stride = block.dim_threads().x * ExecutionNumBlocks;

        for (int i = tid; i < total_k; i += stride) {
            // Retrieve Z0
            int z0_idx = i * 2;
            uint64_t z0_low = Z0_OutDigits[z0_idx];
            uint64_t z0_high = Z0_OutDigits[z0_idx + 1];

            // Retrieve Z2
            int z2_idx = i * 2;
            uint64_t z2_low = Z2_OutDigits[z2_idx];
            uint64_t z2_high = Z2_OutDigits[z2_idx + 1];

            // Retrieve Z1_temp (Z1')
            int z1_temp_idx = i * 2;
            uint64_t z1_temp_low = Z1_temp_digits[z1_temp_idx];
            uint64_t z1_temp_high = Z1_temp_digits[z1_temp_idx + 1];

            // Combine Z2 + Z0 first
            uint64_t temp_low, temp_high;
            Add128(z2_low, z2_high, z0_low, z0_high, temp_low, temp_high);

            uint64_t z1_low, z1_high;
            if (z1_sign == 0) {
                // same sign: Z1 = (Z2 + Z0) - Z1_temp
                Subtract128(temp_low, temp_high, z1_temp_low, z1_temp_high, z1_low, z1_high);
            } else {
                // opposite signs: Z1 = (Z2 + Z0) + Z1_temp
                Add128(temp_low, temp_high, z1_temp_low, z1_temp_high, z1_low, z1_high);
            }

            // Store fully formed Z1
            int z1_idx = i * 2;
            Z1_digits[z1_idx] = z1_low;
            Z1_digits[z1_idx + 1] = z1_high;
        }

        if constexpr (DebugChecksums) {
            grid.sync();

            StoreCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::Z1>(
                record, debugTrackerArray, grid, block, Z1_digits, total_k * 2);
        }

        // Synchronize before final combination
        grid.sync();

        // Now the final combination is just:
        // final = Z0 + (Z1 << (32*n)) + (Z2 << (64*n))
        for (int i = tid; i < total_result_digits; i += stride) {
            uint64_t sum_low = 0;
            uint64_t sum_high = 0;

            // Add Z0
            if (i < 2 * n1 - 1) {
                int z0_idx = i * 2;
                uint64_t z0_low = Z0_OutDigits[z0_idx];
                uint64_t z0_high = Z0_OutDigits[z0_idx + 1];
                Add128(sum_low, sum_high, z0_low, z0_high, sum_low, sum_high);
            }

            // Add Z1 shifted by n
            if (i >= n1 && (i - n1) < 2 * n1 - 1) {
                int z1_idx = (i - n1) * 2;
                uint64_t z1_low = Z1_digits[z1_idx];
                uint64_t z1_high = Z1_digits[z1_idx + 1];
                Add128(sum_low, sum_high, z1_low, z1_high, sum_low, sum_high);
            }

            // Add Z2 shifted by 2*n
            if (i >= 2 * n1 && (i - 2 * n1) < 2 * n1 - 1) {
                int z2_idx = (i - 2 * n1) * 2;
                uint64_t z2_low = Z2_OutDigits[z2_idx];
                uint64_t z2_high = Z2_OutDigits[z2_idx + 1];
                Add128(sum_low, sum_high, z2_low, z2_high, sum_low, sum_high);
            }

            int result_idx = i * 2;
            final128[result_idx] = sum_low;
            final128[result_idx + 1] = sum_high;
        }

        if constexpr (DebugChecksums) {
            grid.sync();

            StoreCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::Final128>(
                record, debugTrackerArray, grid, block, final128, total_result_digits * 2);
            EraseCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::Result_offset>(
                record, debugTrackerArray, grid, block);
        }

        // Synchronize before carry propagation
        grid.sync();
    }
}

//
// static constexpr int32_t SharkFloatParams::GlobalThreadsPerBlock = /* power of 2 */;
// static constexpr int32_t SharkFloatParams::GlobalNumBlocks = /* power of 2 */;
// static constexpr int32_t SharkFloatParams::GlobalNumUint32 = SharkFloatParams::GlobalThreadsPerBlock * SharkFloatParams::GlobalNumBlocks;
// 

// Assuming that SharkFloatParams::GlobalNumUint32 can be large and doesn't fit in shared memory
// We'll use the provided global memory buffers for large intermediates
template<class SharkFloatParams>
__device__ void MultiplyHelperKaratsubaV2 (
    const HpSharkFloat<SharkFloatParams> *__restrict__ A,
    const HpSharkFloat<SharkFloatParams> *__restrict__ B,
    HpSharkFloat<SharkFloatParams> *__restrict__ Out,
    cg::grid_group &grid,
    cg::thread_block &block,
    uint64_t *__restrict__ tempProducts) {

    extern __shared__ uint32_t shared_data[];

    constexpr auto NewN = SharkFloatParams::GlobalNumUint32;         // Total number of digits
    constexpr auto NewN1 = (NewN + 1) / 2;
    constexpr auto NewN2 = NewN - NewN1;   /* n1 is larger or same */
    constexpr auto TempBase = AdditionalUInt64Global;
    constexpr auto CallIndex = 0;
    constexpr auto CarryInsOffset = TempBase;
    constexpr auto ExecutionBlockBase = 0;
    constexpr auto ExecutionNumBlocks = SharkFloatParams::GlobalNumBlocks;
    constexpr auto RecursionDepth = 1;
    DefineTempProductsOffsets(TempBase, CallIndex);

    auto *SharkRestrict aDigits = shared_data;
    auto *SharkRestrict bDigits = aDigits + NewN;
    auto *SharkRestrict x_diff_abs = bDigits + NewN;
    auto *SharkRestrict y_diff_abs = x_diff_abs + (NewN + 1) / 2;

    cg::memcpy_async(block, aDigits, A->Digits, sizeof(uint32_t) * NewN);
    cg::memcpy_async(block, bDigits, B->Digits, sizeof(uint32_t) * NewN);

    if constexpr (DebugChecksums) {
        const bool record = block.thread_index().x == 0 && block.group_index().x == ExecutionBlockBase;
        EraseCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::Invalid>(record, debugTrackerArray, grid, block);
        EraseCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::ADigits>(record, debugTrackerArray, grid, block);
        EraseCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::BDigits>(record, debugTrackerArray, grid, block);
        EraseCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::AHalfHigh>(record, debugTrackerArray, grid, block);
        EraseCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::AHalfLow>(record, debugTrackerArray, grid, block);
        EraseCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::BHalfHigh>(record, debugTrackerArray, grid, block);
        EraseCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::BHalfLow>(record, debugTrackerArray, grid, block);
        EraseCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::XDiff>(record, debugTrackerArray, grid, block);
        EraseCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::YDiff>(record, debugTrackerArray, grid, block);
        EraseCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::Z0>(record, debugTrackerArray, grid, block);
        EraseCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::Z1>(record, debugTrackerArray, grid, block);
        EraseCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::Z2>(record, debugTrackerArray, grid, block);
        EraseCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::Z1_offset>(record, debugTrackerArray, grid, block);
        EraseCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::Final128>(record, debugTrackerArray, grid, block);
        EraseCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::Result_offset>(record, debugTrackerArray, grid, block);
        static_assert(static_cast<int>(DebugStatePurpose::NumPurposes) == 15, "Unexpected number of purposes");
    }

    // Wait for the first batch of A to be loaded
    cg::wait(block);

    auto *final128 = &tempProducts[Convolution_offset];
    MultiplyDigitsOnly<
        SharkFloatParams,
        RecursionDepth,
        CallIndex + 1,
        NewN,
        NewN1,
        NewN2,
        ExecutionBlockBase,
        ExecutionNumBlocks,
        SharkFloatParams::GlobalNumBlocks,
        TempBase>(
        shared_data,
        A,
        B,
        aDigits,
        bDigits,
        x_diff_abs,
        y_diff_abs,
        final128,
        grid,
        block,
        tempProducts);

    grid.sync();

    // ---- Carry Propagation ----

    // Global memory for block carry-outs
    // Allocate space for grid.dim_blocks().x block carry-outs after total_result_digits
    // Note, overlaps:
    uint64_t *block_carry_outs = &tempProducts[CarryInsOffset];

    if constexpr (!SharkFloatParams::DisableCarryPropagation) {

        DefineCarryDefinitions();

        constexpr bool UseParallelCarry = true;
        uint64_t *globalCarryCheck = &tempProducts[GlobalCarryOffset];

        if constexpr (UseParallelCarry) {

            // First Pass: Process convolution results to compute initial digits and local carries
            CarryPropagation<SharkFloatParams>(
                (uint64_t *)shared_data,
                grid,
                block,
                block.thread_index(),
                block.group_index(),
                thread_start_idx,
                thread_end_idx,
                Convolution_offset,
                Result_offset,
                block_carry_outs,
                tempProducts,
                globalCarryCheck
            );
        } else {
            SerialCarryPropagation<SharkFloatParams>(
                (uint64_t *)shared_data,
                grid,
                block,
                block.thread_index(),
                block.group_index(),
                thread_start_idx,
                thread_end_idx,
                Convolution_offset,
                Result_offset,
                block_carry_outs,
                tempProducts,
                globalCarryCheck
            );

            grid.sync();
        }
    } else {
        grid.sync();
    }

    using DebugState = DebugState<SharkFloatParams>;
    const uint64_t *resultEntries = &tempProducts[Result_offset];
    const bool record = block.thread_index().x == 0 && block.group_index().x == ExecutionBlockBase;

    if constexpr (DebugChecksums) {
        grid.sync();

        StoreCurrentDebugState<SharkFloatParams, CallIndex, DebugStatePurpose::Result_offset>(
            record, debugTrackerArray, grid, block, resultEntries, 2 * NewN);

        grid.sync();
    }

    // ---- Finalize the Result ----
    if constexpr (!SharkFloatParams::DisableFinalConstruction) {
        // uint64_t final_carry = carryOuts_phase6[SharkFloatParams::GlobalNumBlocks - 1];

        // Initial total_result_digits is 2 * NewN
        int total_result_digits = 2 * NewN;

        // Determine the highest non-zero digit index in the full result
        int highest_nonzero_index = total_result_digits - 1;

        while (highest_nonzero_index >= 0) {
            int result_idx = Result_offset + highest_nonzero_index;
            uint32_t digit = static_cast<uint32_t>(tempProducts[result_idx]);
            if (digit != 0) {
                break;
            }

            highest_nonzero_index--;
        }

        // Determine the number of significant digits
        int significant_digits = highest_nonzero_index + 1;
        // Calculate the number of digits to shift to keep the most significant NewN digits
        int shift_digits = significant_digits - NewN;
        if (shift_digits < 0) {
            shift_digits = 0;  // No need to shift if we have fewer than NewN significant digits
        }

        if (block.group_index().x == 0 && block.thread_index().x == 0) {
            // Adjust the exponent based on the number of bits shifted
            Out->Exponent = A->Exponent + B->Exponent + shift_digits * 32;

            // Set the sign of the result
            Out->IsNegative = A->IsNegative ^ B->IsNegative;
        }

        int tid = block.thread_index().x + block.group_index().x * block.dim_threads().x;
        int stride = block.dim_threads().x * grid.dim_blocks().x;

        // src_idx is the starting index in tempProducts[] from which we copy
        int src_idx = Result_offset + shift_digits;
        int last_src = Result_offset + highest_nonzero_index; // The last valid index

        // We'll do a grid-stride loop over i in [0 .. NewN)
        for (int i = tid; i < NewN; i += stride) {
            // Corresponding source index for digit i
            int src = src_idx + i;

            if (src <= last_src) {
                // Copy from tempProducts
                Out->Digits[i] = tempProducts[src];
            } else {
                // Pad with zero if we've run out of digits
                Out->Digits[i] = 0;
            }
        }
    }
}

template<class SharkFloatParams>
__global__ void MultiplyKernelKaratsubaV2(
    const HpSharkFloat<SharkFloatParams> *A,
    const HpSharkFloat<SharkFloatParams> *B,
    HpSharkFloat<SharkFloatParams> *Out,
    uint64_t *tempProducts) {

    // Initialize cooperative grid group
    cg::grid_group grid = cg::this_grid();
    cg::thread_block block = cg::this_thread_block();

    // Call the MultiplyHelper function
    //MultiplyHelper(A, B, Out, carryIns, grid, tempProducts);
    MultiplyHelperKaratsubaV2(A, B, Out, grid, block, tempProducts);
}

template<class SharkFloatParams>
__global__ void MultiplyKernelKaratsubaV2TestLoop(
    HpSharkFloat<SharkFloatParams> *A,
    HpSharkFloat<SharkFloatParams> *B,
    HpSharkFloat<SharkFloatParams> *Out,
    uint64_t *tempProducts) { // Array to store cumulative carries

    // Initialize cooperative grid group
    cg::grid_group grid = cg::this_grid();
    cg::thread_block block = cg::this_thread_block();

    for (int i = 0; i < SharkTestIterCount; ++i) {
        // MultiplyHelper(A, B, Out, carryIns, grid, tempProducts);
        if constexpr (!SharkFloatParams::ForceNoOp) {
            MultiplyHelperKaratsubaV2(A, B, Out, grid, block, tempProducts);
        } else {
            grid.sync();
        }
    }
}

template<class SharkFloatParams>
void PrintMaxActiveBlocks(int sharedAmountBytes) {
    std::cout << "Shared memory size: " << sharedAmountBytes << std::endl;

    int numBlocks;
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocks,
        MultiplyKernelKaratsubaV2<SharkFloatParams>,
        SharkFloatParams::GlobalThreadsPerBlock,
        sharedAmountBytes
    );

    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipOccupancyMaxActiveBlocksPerMultiprocessor: " << hipGetErrorString(err) << std::endl;
        return;
    }

    std::cout << "Max active blocks: " << numBlocks << std::endl;
}

template<class SharkFloatParams>
void ComputeMultiplyKaratsubaV2Gpu(void *kernelArgs[]) {

    hipError_t err;

    constexpr int NewN = SharkFloatParams::GlobalNumUint32;
    constexpr auto n = (NewN + 1) / 2;              // Half of NewN
    constexpr auto sharedAmountBytes = UseSharedMemory ? (2 * NewN + 2 * n) * sizeof(uint32_t) : 0;

    if constexpr (SharkCustomStream) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            MultiplyKernelKaratsubaV2<SharkFloatParams>),
            hipFuncAttributeMaxDynamicSharedMemorySize,
            sharedAmountBytes);

        PrintMaxActiveBlocks<SharkFloatParams>(sharedAmountBytes);
    }

    err = hipLaunchCooperativeKernel(
        (void *)MultiplyKernelKaratsubaV2<SharkFloatParams>,
        dim3(SharkFloatParams::GlobalNumBlocks),
        dim3(SharkFloatParams::GlobalThreadsPerBlock),
        kernelArgs,
        sharedAmountBytes, // Shared memory size
        0 // Stream
    );

    hipDeviceSynchronize();

    if (err != hipSuccess) {
        std::cerr << "CUDA error in MultiplyKernelKaratsubaV2: " << hipGetErrorString(err) << std::endl;
    }
}

template<class SharkFloatParams>
void ComputeMultiplyKaratsubaV2GpuTestLoop(hipStream_t &stream, void *kernelArgs[]) {

    constexpr int NewN = SharkFloatParams::GlobalNumUint32;
    constexpr auto n = (NewN + 1) / 2;              // Half of NewN
    constexpr auto sharedAmountBytes = UseSharedMemory ? (2 * NewN + 2 * n) * sizeof(uint32_t) : 0;

    if constexpr (SharkCustomStream) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            MultiplyKernelKaratsubaV2TestLoop<SharkFloatParams>),
            hipFuncAttributeMaxDynamicSharedMemorySize,
            sharedAmountBytes);

        PrintMaxActiveBlocks<SharkFloatParams>(sharedAmountBytes);
    }

    hipError_t err = hipLaunchCooperativeKernel(
        (void *)MultiplyKernelKaratsubaV2TestLoop<SharkFloatParams>,
        dim3(SharkFloatParams::GlobalNumBlocks),
        dim3(SharkFloatParams::GlobalThreadsPerBlock),
        kernelArgs,
        sharedAmountBytes, // Shared memory size
        stream // Stream
    );

    hipDeviceSynchronize();

    if (err != hipSuccess) {
        std::cerr << "CUDA error in MultiplyKernelKaratsubaTestLoop: " << hipGetErrorString(err) << std::endl;
    }
}

#define ExplicitlyInstantiate(SharkFloatParams) \
    template void ComputeMultiplyKaratsubaV2Gpu<SharkFloatParams>(void *kernelArgs[]); \
    template void ComputeMultiplyKaratsubaV2GpuTestLoop<SharkFloatParams>(hipStream_t &stream, void *kernelArgs[]);

ExplicitInstantiateAll();