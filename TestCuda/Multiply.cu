#include "hip/hip_runtime.h"
#include "Multiply.cuh"

#include <hip/hip_runtime.h>

#include "HpGpu.cuh"
#include "BenchmarkTimer.h"

#include <iostream>
#include <vector>
#include <gmp.h>
#include <cstring>
#include <sstream>
#include <iomanip>
#include <cmath>
#include <algorithm>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/barrier>
namespace cg = cooperative_groups;


// Structs for carry handling (similar to addition)
struct GlobalMulBlockData {
    // Define any necessary global data for multiplication
    // Placeholder: can be expanded as needed
};

struct PartialSum {
    uint64_t sum;
};

#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;


//
// static constexpr int32_t ThreadsPerBlock = /* power of 2 */;
// static constexpr int32_t NumBlocks = /* power of 2 */;
// static constexpr int32_t HpGpu::NumUint32 = ThreadsPerBlock * NumBlocks;
// 

// Device function to perform high-precision multiplication
__device__ void MultiplyHelper(
    const HpGpu * __restrict__ A,
    const HpGpu * __restrict__ B,
    HpGpu *__restrict__ Out,
    uint64_t * __restrict__ carryOuts_phase3, // Array to store carry-out from Phase 3
    uint64_t * __restrict__ carryOuts_phase6, // Array to store carry-out from Phase 6
    uint64_t * __restrict__ carryIns,          // Array to store carry-in for each block
    cg::grid_group grid,
    uint64_t * __restrict__ tempProducts      // Temporary buffer to store intermediate products
) {
    // Calculate the thread's unique index
    const int threadIdxGlobal = blockIdx.x * blockDim.x + threadIdx.x;

    const int threadIdxGlobalMin = blockIdx.x * blockDim.x;
    const int threadIdxGlobalMax = threadIdxGlobalMin + blockDim.x - 1;

    const int lowDigitIdxMin = threadIdxGlobalMin * 2;
    const int lowDigitIdxMax = threadIdxGlobalMax * 2;

    const int highDigitIdxMin = lowDigitIdxMin + 1;
    const int highDigitIdxMax = lowDigitIdxMax + 1;

    // Each thread handles two digits: low and high
    const int lowDigitIdx = threadIdxGlobal * 2;
    const int highDigitIdx = lowDigitIdx + 1;

    // Ensure indices do not exceed the temporary buffer size
    if (lowDigitIdx >= 2 * HpGpu::NumUint32) return;

    // Initialize temporary products to zero
    tempProducts[lowDigitIdx] = 0;
    if (highDigitIdx < 2 * HpGpu::NumUint32) {
        tempProducts[highDigitIdx] = 0;
    }

    static constexpr int32_t BATCH_SIZE_A = BatchSize;
    static constexpr int32_t BATCH_SIZE_B = BatchSize;

    // Compute k_min and k_max
    const int k_min = 2 * blockIdx.x * blockDim.x;
    const int k_max = min(2 * (blockIdx.x + 1) * blockDim.x - 1, 2 * HpGpu::NumUint32 - 1);

    // Compute j_min_block and j_max_block
    const int j_min_block = max(0, k_min - (HpGpu::NumUint32 - 1));
    const int j_max_block = min(k_max, HpGpu::NumUint32 - 1);

    const int a_shared_size_required = j_max_block - j_min_block + 1;

    // Shared memory for A and B with double buffering
    __shared__ __align__(16) uint32_t A_shared[2][BATCH_SIZE_A];
    __shared__ __align__(16) uint32_t B_shared[2][BATCH_SIZE_B];

    const int numBatches_A = (a_shared_size_required + BATCH_SIZE_A - 1) / BATCH_SIZE_A;
    const int numBatches_B = (HpGpu::NumUint32 + BATCH_SIZE_B - 1) / BATCH_SIZE_B;

    uint32_t * __restrict__ tempBufferA = nullptr;
    uint32_t * __restrict__ currentBufferA = A_shared[0];
    uint32_t * __restrict__ nextBufferA = A_shared[1];

    uint32_t * __restrict__ tempBufferB = nullptr;
    uint32_t * __restrict__ currentBufferB = B_shared[0];
    uint32_t * __restrict__ nextBufferB = B_shared[1];

    cg::thread_block block = cg::this_thread_block();

    // Start loading the first batch of A asynchronously
    const int batchStartA = j_min_block;
    const int elementsToCopyA = min(BATCH_SIZE_A, a_shared_size_required);

    cg::memcpy_async(block, &currentBufferA[0], &A->Digits[batchStartA], sizeof(uint32_t) * elementsToCopyA);

    // Wait for the first batch of A to be loaded
    cg::wait(block);

    uint64_t lowDigitIdxSum = 0;
    uint64_t highDigitIdxSum = 0;

    // Loop over batches of A
    for (int32_t batchA = 0; batchA < numBatches_A; ++batchA) {
        block.sync();

        const int batchStartA = j_min_block + batchA * BATCH_SIZE_A;
        const int batchEndA = batchStartA + elementsToCopyA - 1;

        // Start loading the next batch of A asynchronously if not the last batch
        if (batchA + 1 < numBatches_A) {
            const int nextBatchStartA = j_min_block + (batchA + 1) * BATCH_SIZE_A;
            const int nextElementsToCopyA = min(BATCH_SIZE_A, a_shared_size_required - (batchA + 1) * BATCH_SIZE_A);

            cg::memcpy_async(block, &nextBufferA[0], &A->Digits[nextBatchStartA], sizeof(uint32_t) * nextElementsToCopyA);
        }

        const int bIndex_min_low = lowDigitIdxMin - batchEndA;
        const int bIndex_max_low = lowDigitIdxMax - batchStartA;

        const int bIndex_min_high = highDigitIdxMin - batchEndA;
        const int bIndex_max_high = highDigitIdxMax - batchStartA;

        const int bIndex_min = max(0, min(bIndex_min_low, bIndex_min_high));
        const int bIndex_max = min(HpGpu::NumUint32 - 1, max(bIndex_max_low, bIndex_max_high));

        const int batchB_start = bIndex_min / BATCH_SIZE_B;
        // const int batchB_end = bIndex_max / BATCH_SIZE_B;

        int batchStartB = batchB_start * BATCH_SIZE_B;
        {
            const int elementsToCopyB = min(BATCH_SIZE_B, HpGpu::NumUint32 - batchStartB);
            cg::memcpy_async(block, &currentBufferB[0], &B->Digits[batchStartB], sizeof(uint32_t) * elementsToCopyB);
        }

        // Loop over batches of B
        for (int batchB = batchB_start; batchB < numBatches_B; ++batchB) {
            //block.sync();

            const int elementsToCopyB = min(BATCH_SIZE_B, HpGpu::NumUint32 - batchStartB);
            const int batchEndB = batchStartB + elementsToCopyB - 1;

            // Start loading the next batch of B asynchronously if not the last batch
            if (batchB + 1 < numBatches_B) {
                int nextBatchStartB = (batchB + 1) * BATCH_SIZE_B;
                int nextElementsToCopyB = min(BATCH_SIZE_B, HpGpu::NumUint32 - nextBatchStartB);

                cg::memcpy_async(block, &nextBufferB[0], &B->Digits[nextBatchStartB], sizeof(uint32_t) * nextElementsToCopyB);
                cg::wait_prior<1>(block);
            } else {
                cg::wait(block);
            }

            // Compute partial products for lowDigitIdx
            {
                uint64_t sumLow = 0;
                uint64_t sumHigh = 0;

                // Calculate the valid ranges of j for lowDigitIdx and highDigitIdx
                int j_min_low = max(batchStartA, max(j_min_block, lowDigitIdx - batchEndB));
                int j_max_low = min(batchEndA, min(j_max_block, lowDigitIdx - batchStartB));

                int j_min_high = max(batchStartA, max(j_min_block, highDigitIdx - batchEndB));
                int j_max_high = min(batchEndA, min(j_max_block, highDigitIdx - batchStartB));

                // Combined range
                int j_min = min(j_min_low, j_min_high);
                int j_max = max(j_max_low, j_max_high);

                // Iterate over the combined range
                for (int j = j_min; j <= j_max; ++j) {
                    int aSharedIndex = j - batchStartA;
                    uint32_t aValue = currentBufferA[aSharedIndex];

                    // Compute for lowDigitIdx
                    if (j >= j_min_low && j <= j_max_low) {
                        int bIndexLow = lowDigitIdx - j;
                        int bSharedIndexLow = bIndexLow - batchStartB;
                        uint32_t bValueLow = currentBufferB[bSharedIndexLow];

                        sumLow += static_cast<uint64_t>(aValue) * static_cast<uint64_t>(bValueLow);
                    }

                    // Compute for highDigitIdx
                    if (highDigitIdx < 2 * HpGpu::NumUint32 && j >= j_min_high && j <= j_max_high) {
                        int bIndexHigh = highDigitIdx - j;
                        int bSharedIndexHigh = bIndexHigh - batchStartB;
                        uint32_t bValueHigh = currentBufferB[bSharedIndexHigh];

                        sumHigh += static_cast<uint64_t>(aValue) * static_cast<uint64_t>(bValueHigh);
                    }
                }
                lowDigitIdxSum += sumLow;
                highDigitIdxSum += sumHigh;
            }

            // Switch buffers for double buffering of B
            tempBufferB = currentBufferB;
            currentBufferB = nextBufferB;
            nextBufferB = tempBufferB;

            batchStartB += BATCH_SIZE_B;
        }

        // Switch buffers for double buffering of A
        tempBufferA = currentBufferA;
        currentBufferA = nextBufferA;
        nextBufferA = tempBufferA;

        // Wait for the next batch of A to be loaded
        if (batchA + 1 < numBatches_A) {
            cg::wait(block);
        }
        //block.sync();
    }

    // Phase 2: Perform initial carry propagation for two digits per thread
    uint64_t lowValue = lowDigitIdxSum;
    uint32_t digitLow = static_cast<uint32_t>(lowValue & 0xFFFFFFFF);
    uint32_t carryLow = static_cast<uint32_t>(lowValue >> 32);

    uint64_t highValue = highDigitIdxSum;
    uint32_t digitHigh = static_cast<uint32_t>(highValue & 0xFFFFFFFF);
    uint32_t carryHigh = static_cast<uint32_t>(highValue >> 32);

    // Apply carry from low to high digit
    uint64_t highSum = static_cast<uint64_t>(digitHigh) + carryLow;
    digitHigh = static_cast<uint32_t>(highSum & 0xFFFFFFFF);
    carryHigh += static_cast<uint32_t>(highSum >> 32);

    // Store the digits back into tempProducts
    tempProducts[lowDigitIdx] = digitLow;
    tempProducts[highDigitIdx] = static_cast<uint64_t>(digitHigh);

    // Phase 3: Sequentially propagate carries within the block
    __shared__ uint32_t carryOutsShared[ThreadsPerBlock];
    carryOutsShared[threadIdx.x] = carryHigh; // Each thread's carryOut from high digit
    //grid.sync();

    // Sequential carry propagation within the block
    for (int i = 1; i < ThreadsPerBlock; ++i) {
        if (threadIdx.x == i) {
            uint32_t carryIn = carryOutsShared[i - 1];
            carryOutsShared[i - 1] = 0; // Reset carryOut for this thread
            if (carryIn) {
                // Add carryIn to this thread's low digit
                uint64_t currentLow = tempProducts[lowDigitIdx];
                uint64_t newLowSum = currentLow + carryIn;
                tempProducts[lowDigitIdx] = newLowSum;
                uint32_t newCarry = static_cast<uint32_t>(newLowSum >> 32);

                // Add carryIn to this thread's high digit
                uint64_t currentHigh = tempProducts[highDigitIdx];
                uint64_t newHighSum = currentHigh + newCarry;
                tempProducts[highDigitIdx] = newHighSum;
                newCarry = static_cast<uint32_t>(newHighSum >> 32);
                carryOutsShared[i] += newCarry;
            }
        }
        block.sync();
    }

    // After propagation, the last thread's carryOut is the block's carryOut
    if (threadIdx.x == ThreadsPerBlock - 1) {
        carryOuts_phase3[blockIdx.x] = carryOutsShared[ThreadsPerBlock - 1];
    }
    //grid.sync(); // Ensure all blocks have computed carryIns

    //// Phase 4: Compute carry-ins using prefix sum on carryOuts_phase3
    //if (blockIdx.x == 0 && threadIdx.x == 0) {
    //    carryIns[0] = 0; // First block has no carry-in
    //    for (int i = 1; i < NumBlocks; ++i) {
    //        carryIns[i] = carryIns[i - 1] + carryOuts_phase3[i - 1];
    //    }
    //}
    //grid.sync(); // Ensure all blocks have computed carryIns


    // Phase 5: Apply carry-ins to each block's output digits
    //if (lowDigitIdx < HpGpu::NumUint32) {
    //    uint64_t sumLow = static_cast<uint64_t>(tempProducts[lowDigitIdx] & 0xFFFFFFFF) + carryOuts_phase3[blockIdx.x];
    //    Out->Digits[lowDigitIdx] = static_cast<uint32_t>(sumLow & 0xFFFFFFFF);
    //    //uint32_t newCarryLow = static_cast<uint32_t>(sumLow >> 32);
    //    //carryOutsShared[threadIdx.x * 2] = newCarryLow;
    //} else {
    //    //carryOutsShared[threadIdx.x * 2] = 0;
    //}

    //if (highDigitIdx < HpGpu::NumUint32) {
    //    uint64_t sumHigh = static_cast<uint64_t>(tempProducts[highDigitIdx] & 0xFFFFFFFF) + carryOutsShared[threadIdx.x * 2];
    //    Out->Digits[highDigitIdx] = static_cast<uint32_t>(sumHigh & 0xFFFFFFFF);
    //    //uint32_t newCarryHigh = static_cast<uint32_t>(sumHigh >> 32);
    //    //carryOutsShared[threadIdx.x * 2 + 1] = newCarryHigh;
    //} else {
    //    //carryOutsShared[threadIdx.x * 2 + 1] = 0;
    //}
    ////block.sync();

    // Phase 6: Record any new carry-outs generated by carry-ins
    //if (threadIdx.x == 0) {
    //    uint64_t blockCarryOut = 0;
    //    for (int i = 0; i < ThreadsPerBlock * 2; ++i) {
    //        blockCarryOut += carryOutsShared[i];
    //    }
    //    carryOuts_phase6[blockIdx.x] = blockCarryOut;
    //}
    //block.sync();

    // FIX ME: carryOuts_phase6 is unused.

    // Synchronize all blocks before handling final carry-outs
    grid.sync();

    // Corrected Phase 7: Handle significant digits beyond the mantissa
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        // Step 1: Find the highest non-zero index in tempProducts
        int highestNonZeroIndex = -1;
        for (int i = 2 * HpGpu::NumUint32 - 1; i >= 0; --i) {
            if (tempProducts[i] != 0) {
                highestNonZeroIndex = i;
                break;
            }
        }

        // Step 2: Calculate the total number of significant digits
        int totalResultDigits = highestNonZeroIndex + 1;

        // Step 3: Determine the number of shifts needed
        int shifts = totalResultDigits - HpGpu::NumUint32;
        if (shifts < 0) {
            shifts = 0; // No shift needed if result fits within the mantissa
        }

        // Step 4: Shift the mantissa to the right by 'shifts' digits
        for (int i = 0; i < HpGpu::NumUint32; ++i) {
            int srcIndex = i + shifts;
            if (srcIndex < 2 * HpGpu::NumUint32) {
                Out->Digits[i] = static_cast<uint32_t>(tempProducts[srcIndex] & 0xFFFFFFFF);
            } else {
                Out->Digits[i] = 0; // Pad with zeros if beyond tempProducts
            }
        }

        // Step 5: Adjust the exponent accordingly
        Out->Exponent += shifts * 32;

        // Step 6: Perform carry propagation within the mantissa
        uint64_t carry = 0;
        for (int i = 0; i < HpGpu::NumUint32; ++i) {
            uint64_t sum = static_cast<uint64_t>(Out->Digits[i]) + carry;
            Out->Digits[i] = static_cast<uint32_t>(sum & 0xFFFFFFFF);
            carry = sum >> 32;
        }

        // Step 7: Handle any remaining carry
        if (carry > 0) {
            // Shift the mantissa to the right by one more digit
            for (int i = 0; i < HpGpu::NumUint32 - 1; ++i) {
                Out->Digits[i] = Out->Digits[i + 1];
            }
            Out->Digits[HpGpu::NumUint32 - 1] = static_cast<uint32_t>(carry & 0xFFFFFFFF);
            // Adjust the exponent
            Out->Exponent += 32;
        }

        // Step 8: Set the sign and adjust the exponent
        Out->IsNegative = A->IsNegative != B->IsNegative;
        Out->Exponent += A->Exponent + B->Exponent;
    }
    block.sync();

    //// Phase 8: Initialize result properties (only block 0's thread 0 does this)
    //if (blockIdx.x == 0 && threadIdx.x == 0) {
    //    // Determine the sign of the result
    //    Out->IsNegative = A->IsNegative != B->IsNegative;
    //    // Calculate the initial exponent of the result
    //    Out->Exponent = A->Exponent + B->Exponent;
    //    // Note: Any additional exponent adjustments have been handled in Phase 7
    //}
}

__global__ void MultiplyKernel(
    const HpGpu *A,
    const HpGpu *B,
    HpGpu *Out,
    uint64_t *carryOuts_phase3,
    uint64_t *carryOuts_phase6,
    uint64_t *carryIns,
    uint64_t *tempProducts) {

    // Initialize cooperative grid group
    cg::grid_group grid = cg::this_grid();

    // Call the MultiplyHelper function
    MultiplyHelper(A, B, Out, carryOuts_phase3, carryOuts_phase6, carryIns, grid, tempProducts);
}

__global__ void MultiplyKernelTestLoop(
    HpGpu *A,
    HpGpu *B,
    HpGpu *Out,
    uint64_t *carryOuts_phase3,
    uint64_t *carryOuts_phase6,
    uint64_t *carryIns,
    uint64_t *tempProducts) { // Array to store cumulative carries

    // Initialize cooperative grid group
    cg::grid_group grid = cg::this_grid();

    for (int i = 0; i < NUM_ITER; ++i) {
        MultiplyHelper(A, B, Out, carryOuts_phase3, carryOuts_phase6, carryIns, grid, tempProducts);
    }
}


void ComputeMultiplyGpu(void *kernelArgs[]) {

    hipError_t err = hipLaunchCooperativeKernel(
        (void *)MultiplyKernel,
        dim3(NumBlocks),
        dim3(ThreadsPerBlock),
        kernelArgs,
        0, // Shared memory size
        0 // Stream
    );

    hipDeviceSynchronize();
}

void ComputeMultiplyGpuTestLoop(void *kernelArgs[]) {

    hipError_t err = hipLaunchCooperativeKernel(
        (void *)MultiplyKernelTestLoop,
        dim3(NumBlocks),
        dim3(ThreadsPerBlock),
        kernelArgs,
        0, // Shared memory size
        0 // Stream
    );

    hipDeviceSynchronize();
}

