#include "hip/hip_runtime.h"
#include "Multiply.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/barrier>

#include "HpSharkFloat.cuh"

#include <iostream>

namespace cg = cooperative_groups;

/////////////////////////////////////////////////////////////////////////////////////////
// MultiplyHelperKaratsuba


__device__ static void multiply_uint64(
    uint64_t a, uint64_t b,
    uint64_t &low, uint64_t &high) {
    low = a * b;
    high = __umul64hi(a, b);
}

// Function to perform addition with carry
__device__ static void addWithCarry(
    uint64_t a_low, uint64_t a_high,
    uint64_t b_low, uint64_t b_high,
    uint64_t &result_low, uint64_t &result_high) {

    result_low = a_low + b_low;
    uint64_t carry = (result_low < a_low) ? 1 : 0;
    result_high = a_high + b_high + carry;
}

// Function to perform subtraction with borrow
__device__ static void subtractWithBorrow(
    uint64_t a_low, uint64_t a_high,
    uint64_t b_low, uint64_t b_high,
    uint64_t &result_low, uint64_t &result_high) {

    bool borrow_low = a_low < b_low;
    result_low = a_low - b_low;
    uint64_t borrow_high = borrow_low ? 1 : 0;
    result_high = a_high - b_high - borrow_high;
}

template<class SharkFloatParams>
__device__ static void CarryPropagation(
    uint64_t *__restrict__ shared_carries,
    cg::grid_group &grid,
    cg::thread_block &block,
    const uint3 &threadIdx,
    const uint3 &blockIdx,
    const uint3 &blockDim,
    const uint3 &gridDim,
    int thread_start_idx,
    int thread_end_idx,
    int Convolution_offset,
    int Result_offset,
    uint64_t *__restrict__ block_carry_outs,
    uint64_t *__restrict__ tempProducts,
    uint64_t *__restrict__ globalCarryCheck) {

    // First Pass: Process convolution results to compute initial digits and local carries
    // Initialize local carry
    uint64_t local_carry = 0;

    // Constants and offsets
    constexpr int MaxPasses = 10; // Maximum number of carry propagation passes
    constexpr int total_result_digits = 2 * SharkFloatParams::NumUint32;

    uint64_t *carries_remaining_global = globalCarryCheck;

    for (int idx = thread_start_idx; idx < thread_end_idx; ++idx) {
        int sum_low_idx = Convolution_offset + idx * 2;
        int sum_high_idx = sum_low_idx + 1;

        uint64_t sum_low = tempProducts[sum_low_idx];     // Lower 64 bits
        uint64_t sum_high = tempProducts[sum_high_idx];   // Higher 64 bits

        // Add local carry to sum_low
        bool new_sum_low_negative = false;
        uint64_t new_sum_low = sum_low + local_carry;

        // Extract one 32-bit digit from new_sum_low
        auto digit = static_cast<uint32_t>(new_sum_low & 0xFFFFFFFFULL);
        tempProducts[Result_offset + idx] = digit;

        bool local_carry_negative = ((local_carry & (1ULL << 63)) != 0);
        local_carry = 0ULL;

        if (!local_carry_negative && new_sum_low < sum_low) {
            local_carry = 1ULL << 32;
        } else if (local_carry_negative && new_sum_low > sum_low) {
            new_sum_low_negative = (new_sum_low & 0x8000'0000'0000'0000) != 0;
        }

        // Update local_carry
        if (new_sum_low_negative) {
            // Shift sum_high by 32 bits and add carry_from_low
            uint64_t upper_new_sum_low = new_sum_low >> 32;
            upper_new_sum_low |= 0xFFFF'FFFF'0000'0000;
            local_carry += upper_new_sum_low;
            local_carry += sum_high << 32;
        } else {
            local_carry += new_sum_low >> 32;
            local_carry += sum_high << 32;
        }
    }

    if (threadIdx.x == SharkFloatParams::ThreadsPerBlock - 1) {
        block_carry_outs[blockIdx.x] = local_carry;
    } else {
        shared_carries[threadIdx.x] = local_carry;
    }

    // Synchronize all blocks
    grid.sync();

    // Inter-Block Carry Propagation
    int pass = 0;

    do {
        // Zero out the global carry count for the current pass
        if (blockIdx.x == 0 && threadIdx.x == 0) {
            *carries_remaining_global = 0;
        }

        // Get carry-in from the previous block
        local_carry = 0;
        if (threadIdx.x == 0 && blockIdx.x > 0) {
            local_carry = block_carry_outs[blockIdx.x - 1];
        } else {
            if (threadIdx.x > 0) {
                local_carry = shared_carries[threadIdx.x - 1];
            }
        }

        // Each thread processes its assigned digits
        bool local_carry_negative = false;
        for (int idx = thread_start_idx; idx < thread_end_idx; ++idx) {
            // Read the previously stored digit
            uint32_t digit = tempProducts[Result_offset + idx];

            // Add local_carry to digit
            uint64_t sum = static_cast<uint64_t>(digit) + local_carry;
            if (local_carry_negative) {
                // Clear high order 32 bits of sum:
                sum &= 0x0000'0000'FFFF'FFFF;
            }

            // Update digit
            digit = static_cast<uint32_t>(sum & 0xFFFFFFFFULL);
            tempProducts[Result_offset + idx] = digit;

            // Compute new local_carry for next digit
            local_carry_negative = ((local_carry & (1ULL << 63)) != 0);
            local_carry = sum >> 32;
        }

        shared_carries[threadIdx.x] = local_carry;
        block.sync();

        // The block's carry-out is the carry from the last thread
        auto temp = shared_carries[threadIdx.x];
        if (threadIdx.x == SharkFloatParams::ThreadsPerBlock - 1) {
            block_carry_outs[blockIdx.x] = temp;
        }

        if (temp != 0) {
            atomicAdd(carries_remaining_global, 1);
        }

        // Synchronize all blocks before checking if carries remain
        grid.sync();

        // If no carries remain, exit the loop
        if (*carries_remaining_global == 0) {
            break;
        }

        grid.sync();
        pass++;
    } while (pass < MaxPasses);

    // ---- Handle Final Carry-Out ----

    // Synchronize all blocks
    grid.sync();

    // Handle final carry-out
    if (threadIdx.x == 0 && blockIdx.x == gridDim.x - 1) {
        uint64_t final_carry = block_carry_outs[blockIdx.x];
        if (final_carry > 0) {
            // Store the final carry as an additional digit
            tempProducts[Result_offset + total_result_digits] = static_cast<uint32_t>(final_carry & 0xFFFFFFFFULL);
            // Optionally, you may need to adjust total_result_digits
        }
    }

    // Synchronize all blocks before finalization
    grid.sync();
}

// Assuming that SharkFloatParams::NumUint32 can be large and doesn't fit in shared memory
// We'll use the provided global memory buffers for large intermediates
template<class SharkFloatParams>
__device__ void MultiplyHelperKaratsubaV1(
    const HpSharkFloat<SharkFloatParams> *__restrict__ A,
    const HpSharkFloat<SharkFloatParams> *__restrict__ B,
    HpSharkFloat<SharkFloatParams> *__restrict__ Out,
    cg::grid_group grid,
    uint64_t *__restrict__ tempProducts) {

    cg::thread_block block = cg::this_thread_block();

    const int threadIdxGlobal = blockIdx.x * SharkFloatParams::ThreadsPerBlock + threadIdx.x;

    constexpr int total_threads = SharkFloatParams::ThreadsPerBlock * SharkFloatParams::NumBlocks;
    constexpr int N = SharkFloatParams::NumUint32;         // Total number of digits
    constexpr int n = (N + 1) / 2;              // Half of N

    // Constants for tempProducts offsets
    constexpr int Z0_offset = 0;
    constexpr int Z2_offset = Z0_offset + 4 * N;
    constexpr int Z1_temp_offset = Z2_offset + 4 * N;
    constexpr int Z1_offset = Z1_temp_offset + 4 * N;
    constexpr int Convolution_offset = Z1_offset + 4 * N;
    constexpr int Result_offset = Convolution_offset + 4 * N;
    constexpr int GlobalCarryOffset = Result_offset + 1 * N;
    constexpr int CarryInsOffset = GlobalCarryOffset + 1 * N;

    //// Shared memory allocation
    //__shared__ uint64_t A_shared[n];
    //__shared__ uint64_t B_shared[n];

    //// Load segments of A and B into shared memory
    //for (int i = threadIdx.x; i < n; i += SharkFloatParams::ThreadsPerBlock) {
    //    A_shared[i] = (i < N) ? A->Digits[i] : 0;  // A0
    //    B_shared[i] = (i < N) ? B->Digits[i] : 0;  // B0
    //}

    // Synchronize before starting convolutions
    //block.sync();

    // Common variables for convolution loops
    constexpr int total_k = 2 * n - 1; // Total number of k values

    int k_start = (threadIdxGlobal * total_k) / total_threads;
    int k_end = ((threadIdxGlobal + 1) * total_k) / total_threads;

    // ---- Convolution for Z0 = A0 * B0 ----
    for (int k = k_start; k < k_end; ++k) {
        uint64_t sum_low = 0;
        uint64_t sum_high = 0;

        int i_start = max(0, k - (n - 1));
        int i_end = min(k, n - 1);

        for (int i = i_start; i <= i_end; ++i) {
            uint64_t a = A->Digits[i]; //A_shared[i];         // A0[i]
            uint64_t b = B->Digits[k - i]; //B_shared[k - i];     // B0[k - i]

            uint64_t product = a * b;

            // Add product to sum
            sum_low += product;
            if (sum_low < product) {
                sum_high += 1;
            }
        }

        // Store sum_low and sum_high in tempProducts
        int idx = Z0_offset + k * 2;
        tempProducts[idx] = sum_low;
        tempProducts[idx + 1] = sum_high;
    }

    // Synchronize before next convolution
    //block.sync();

    //// Load A1 and B1 into shared memory
    //for (int i = threadIdx.x; i < n; i += SharkFloatParams::ThreadsPerBlock) {
    //    int index = i + n;
    //    A_shared[i] = (index < N) ? A->Digits[index] : 0;    // A1
    //    B_shared[i] = (index < N) ? B->Digits[index] : 0;    // B1
    //}

    //block.sync();

    // ---- Convolution for Z2 = A1 * B1 ----
    for (int k = k_start; k < k_end; ++k) {
        uint64_t sum_low = 0;
        uint64_t sum_high = 0;

        int i_start = max(0, k - (n - 1));
        int i_end = min(k, n - 1);

        for (int i = i_start; i <= i_end; ++i) {
            uint64_t a = A->Digits[i + n]; // A_shared[i];         // A1[i]
            uint64_t b = B->Digits[k - i + n]; // B_shared[k - i];     // B1[k - i]

            uint64_t product = a * b;

            // Add product to sum
            sum_low += product;
            if (sum_low < product) {
                sum_high += 1;
            }
        }

        // Store sum_low and sum_high in tempProducts
        int idx = Z2_offset + k * 2;
        tempProducts[idx] = sum_low;
        tempProducts[idx + 1] = sum_high;
    }

    //// Synchronize before next convolution
    //block.sync();

    //// Compute (A0 + A1) and (B0 + B1) and store in shared memory
    //for (int i = threadIdx.x; i < n; i += SharkFloatParams::ThreadsPerBlock) {
    //    uint64_t A0 = (i < N) ? A->Digits[i] : 0;
    //    uint64_t A1 = (i + n < N) ? A->Digits[i + n] : 0;
    //    A_shared[i] = A0 + A1;               // (A0 + A1)

    //    uint64_t B0 = (i < N) ? B->Digits[i] : 0;
    //    uint64_t B1 = (i + n < N) ? B->Digits[i + n] : 0;
    //    B_shared[i] = B0 + B1;               // (B0 + B1)
    //}

    //block.sync();

    // ---- Convolution for Z1_temp = (A0 + A1) * (B0 + B1) ----
    for (int k = k_start; k < k_end; ++k) {
        uint64_t sum_low = 0;
        uint64_t sum_high = 0;

        int i_start = max(0, k - (n - 1));
        int i_end = min(k, n - 1);

        for (int i = i_start; i <= i_end; ++i) {
            // uint64_t a = A_shared[i];         // (A0 + A1)[i]
            // uint64_t b = B_shared[k - i];     // (B0 + B1)[k - i]

            uint64_t A0 = A->Digits[i];
            uint64_t A1 = A->Digits[i + n];
            uint64_t B0 = B->Digits[k - i];
            uint64_t B1 = B->Digits[k - i + n];
            auto a = A0 + A1;
            auto b = B0 + B1;

            // Compute full 128-bit product
            uint64_t prod_low, prod_high;
            multiply_uint64(a, b, prod_low, prod_high);

            // Accumulate the product
            addWithCarry(sum_low, sum_high, prod_low, prod_high, sum_low, sum_high);
        }

        // Store sum_low and sum_high in tempProducts
        int idx = Z1_temp_offset + k * 2;
        tempProducts[idx] = sum_low;
        tempProducts[idx + 1] = sum_high;
    }

    // Synchronize before subtraction
    grid.sync();

    // ---- Compute Z1 = Z1_temp - Z0 - Z2 ----
    for (int k = k_start; k < k_end; ++k) {
        // Retrieve Z1_temp
        int z1_temp_idx = Z1_temp_offset + k * 2;
        uint64_t z1_temp_low = tempProducts[z1_temp_idx];
        uint64_t z1_temp_high = tempProducts[z1_temp_idx + 1];

        // Retrieve Z0
        int z0_idx = Z0_offset + k * 2;
        uint64_t z0_low = tempProducts[z0_idx];
        uint64_t z0_high = tempProducts[z0_idx + 1];

        // Retrieve Z2
        int z2_idx = Z2_offset + k * 2;
        uint64_t z2_low = tempProducts[z2_idx];
        uint64_t z2_high = tempProducts[z2_idx + 1];

        // Compute z0 + z2
        uint64_t z0z2_low, z0z2_high;
        addWithCarry(z0_low, z0_high, z2_low, z2_high, z0z2_low, z0z2_high);

        // Compute Z1 = Z1_temp - (Z0 + Z2)
        uint64_t z1_low, z1_high;
        subtractWithBorrow(z1_temp_low, z1_temp_high, z0z2_low, z0z2_high, z1_low, z1_high);

        // Store z1_low and z1_high in tempProducts
        int z1_idx = Z1_offset + k * 2;
        tempProducts[z1_idx] = z1_low;
        tempProducts[z1_idx + 1] = z1_high;
    }

    // Synchronize before combining results
    grid.sync();

    // ---- Combine Z0, Z1, Z2 into the final result ----
    constexpr int total_result_digits = 2 * N;
    int idx_start = (threadIdxGlobal * total_result_digits) / total_threads;
    int idx_end = ((threadIdxGlobal + 1) * total_result_digits) / total_threads;

    for (int idx = idx_start; idx < idx_end; ++idx) {
        uint64_t sum_low = 0;
        uint64_t sum_high = 0;

        // Add Z0 component
        if (idx < 2 * n - 1) {
            int z0_idx = Z0_offset + idx * 2;
            uint64_t z0_low = tempProducts[z0_idx];
            uint64_t z0_high = tempProducts[z0_idx + 1];
            addWithCarry(sum_low, sum_high, z0_low, z0_high, sum_low, sum_high);
        }

        // Add Z1 component shifted by n digits
        if (idx >= n && (idx - n) < 2 * n - 1) {
            int z1_idx = Z1_offset + (idx - n) * 2;
            uint64_t z1_low = tempProducts[z1_idx];
            uint64_t z1_high = tempProducts[z1_idx + 1];
            addWithCarry(sum_low, sum_high, z1_low, z1_high, sum_low, sum_high);
        }

        // Add Z2 component shifted by 2n digits
        if (idx >= 2 * n && (idx - 2 * n) < 2 * n - 1) {
            int z2_idx = Z2_offset + (idx - 2 * n) * 2;
            uint64_t z2_low = tempProducts[z2_idx];
            uint64_t z2_high = tempProducts[z2_idx + 1];
            addWithCarry(sum_low, sum_high, z2_low, z2_high, sum_low, sum_high);
        }

        // Store sum_low and sum_high in tempProducts
        int result_idx = Convolution_offset + idx * 2;
        tempProducts[result_idx] = sum_low;
        tempProducts[result_idx + 1] = sum_high;
    }

    // Synchronize before carry propagation
    block.sync();

    // ---- Carry Propagation ----

    //if (blockIdx.x == 0 && threadIdx.x == 0) {
    //    // Only one thread performs the carry propagation
    //    uint64_t carry = 0;
    //    int total_result_digits = 2 * N;

    //    for (int idx = 0; idx < total_result_digits; ++idx) {
    //        int result_idx = Convolution_offset + idx * 2;
    //        uint64_t sum_low = tempProducts[result_idx];        // Lower 64 bits
    //        uint64_t sum_high = tempProducts[result_idx + 1];   // Higher 64 bits

    //        // Add carry to sum_low
    //        uint64_t new_sum_low = sum_low + carry;
    //        uint64_t carry_from_low = (new_sum_low < sum_low) ? 1 : 0;

    //        // Add carry_from_low to sum_high
    //        uint64_t new_sum_high = (sum_high << 32) + carry_from_low;

    //        // Extract digit (lower 32 bits of new_sum_low)
    //        uint32_t digit = static_cast<uint32_t>(new_sum_low & 0xFFFFFFFFULL);

    //        // Compute carry for the next digit
    //        carry = new_sum_high + (new_sum_low >> 32);

    //        // Store the digit
    //        tempProducts[Result_offset + idx] = digit;
    //    }

    //    // Handle final carry
    //    if (carry > 0) {
    //        tempProducts[Result_offset + total_result_digits] = static_cast<uint32_t>(carry & 0xFFFFFFFFULL);
    //        total_result_digits += 1;
    //    }
    //}

    // Initialize variables

    // Global memory for block carry-outs
    // Allocate space for gridDim.x block carry-outs after total_result_digits
    uint64_t *block_carry_outs = &tempProducts[CarryInsOffset];

    constexpr auto digits_per_block = SharkFloatParams::ThreadsPerBlock * 2;
    auto block_start_idx = blockIdx.x * digits_per_block;
    auto block_end_idx = min(block_start_idx + digits_per_block, total_result_digits);

    int digits_per_thread = (digits_per_block + blockDim.x - 1) / blockDim.x;

    int thread_start_idx = block_start_idx + threadIdx.x * digits_per_thread;
    int thread_end_idx = min(thread_start_idx + digits_per_thread, block_end_idx);

    //So the idea is we process the chunk of digits that has digits interleaved with carries.
    //    This logic should be similar to the global carry propagation but done in parallel on
    //    each block.
    //After that step is done, we should have reduced the number of digits we care about
    //    because weve propagated all the intermediate junk produced above during convolution.
    //    so we end up with 2 * SharkFloatParams::NumBlocks * SharkFloatParams::ThreadsPerBlock digits.
    //At that point we do inter-block carry propagation, which is iterative.

    __shared__ uint64_t shared_carries[SharkFloatParams::ThreadsPerBlock];


    if constexpr (!SharkFloatParams::DisableCarryPropagation) {

        uint64_t *globalCarryCheck = &tempProducts[GlobalCarryOffset];

        CarryPropagation<SharkFloatParams>(
            shared_carries,
            grid,
            block,
            threadIdx,
            blockIdx,
            blockDim,
            gridDim,
            thread_start_idx,
            thread_end_idx,
            Convolution_offset,
            Result_offset,
            block_carry_outs,
            tempProducts,
            globalCarryCheck
        );
    } else {
        grid.sync();
    }

    // ---- Finalize the Result ----

    // ---- Handle Any Remaining Final Carry ----

    // Only one thread handles the final carry propagation
    if constexpr (!SharkFloatParams::DisableFinalConstruction) {
        if (blockIdx.x == 0 && threadIdx.x == 0) {
            // uint64_t final_carry = carryOuts_phase6[SharkFloatParams::NumBlocks - 1];

            // Initial total_result_digits is 2 * N
            int total_result_digits = 2 * N;

            // Handle the final carry-out from the most significant digit
            //if (final_carry > 0) {
            //    // Append the final carry as a new digit at the end (most significant digit)
            //    tempProducts[Result_offset + total_result_digits] = static_cast<uint32_t>(final_carry & 0xFFFFFFFFULL);
            //    total_result_digits += 1;
            //}

            // Determine the highest non-zero digit index in the full result
            int highest_nonzero_index = total_result_digits - 1;

            while (highest_nonzero_index >= 0) {
                int result_idx = Result_offset + highest_nonzero_index;
                uint32_t digit = static_cast<uint32_t>(tempProducts[result_idx]);
                if (digit != 0) {
                    break;
                }

                highest_nonzero_index--;
            }

            // Determine the number of significant digits
            int significant_digits = highest_nonzero_index + 1;
            // Calculate the number of digits to shift to keep the most significant N digits
            int shift_digits = significant_digits - N;
            if (shift_digits < 0) {
                shift_digits = 0;  // No need to shift if we have fewer than N significant digits
            }

            // Adjust the exponent based on the number of bits shifted
            Out->Exponent = A->Exponent + B->Exponent + shift_digits * 32;

            // Copy the least significant N digits to Out->Digits
            int src_idx = Result_offset + shift_digits;
            for (int i = 0; i < N; ++i, ++src_idx) {
                if (src_idx <= Result_offset + highest_nonzero_index) {
                    Out->Digits[i] = tempProducts[src_idx];
                } else {
                    // If we've run out of digits, pad with zeros
                    Out->Digits[i] = 0;
                }
            }

            // Set the sign of the result
            Out->IsNegative = A->IsNegative ^ B->IsNegative;
        }
    }
}


template<class SharkFloatParams>
__global__ void MultiplyKernelKaratsubaV1(
    const HpSharkFloat<SharkFloatParams> *A,
    const HpSharkFloat<SharkFloatParams> *B,
    HpSharkFloat<SharkFloatParams> *Out,
    uint64_t *tempProducts) {

    // Initialize cooperative grid group
    cg::grid_group grid = cg::this_grid();

    // Call the MultiplyHelper function
    //MultiplyHelper(A, B, Out, grid, tempProducts);
    MultiplyHelperKaratsubaV1(A, B, Out, grid, tempProducts);
}

template<class SharkFloatParams>
__global__ void MultiplyKernelKaratsubaV1TestLoop(
    HpSharkFloat<SharkFloatParams> *A,
    HpSharkFloat<SharkFloatParams> *B,
    HpSharkFloat<SharkFloatParams> *Out,
    uint64_t *tempProducts) { // Array to store cumulative carries

    // Initialize cooperative grid group
    cg::grid_group grid = cg::this_grid();

    for (int i = 0; i < TestIterCount; ++i) {
        // MultiplyHelper(A, B, Out, grid, tempProducts);
        MultiplyHelperKaratsubaV1(A, B, Out, grid, tempProducts);
    }
}

template<class SharkFloatParams>
void ComputeMultiplyKaratsubaV1Gpu(void *kernelArgs[]) {

    hipError_t err = hipLaunchCooperativeKernel(
        (void *)MultiplyKernelKaratsubaV1<SharkFloatParams>,
        dim3(SharkFloatParams::NumBlocks),
        dim3(SharkFloatParams::ThreadsPerBlock),
        kernelArgs,
        0, // Shared memory size
        0 // Stream
    );

    hipDeviceSynchronize();

    if (err != hipSuccess) {
        std::cerr << "CUDA error in MultiplyKernelKaratsubaV1: " << hipGetErrorString(err) << std::endl;
    }
}

template<class SharkFloatParams>
void ComputeMultiplyKaratsubaV1GpuTestLoop(hipStream_t &stream, void *kernelArgs[]) {

    hipError_t err = hipLaunchCooperativeKernel(
        (void *)MultiplyKernelKaratsubaV1TestLoop<SharkFloatParams>,
        dim3(SharkFloatParams::NumBlocks),
        dim3(SharkFloatParams::ThreadsPerBlock),
        kernelArgs,
        0, // Shared memory size
        stream // Stream
    );

    hipDeviceSynchronize();

    if (err != hipSuccess) {
        std::cerr << "CUDA error in MultiplyKernelKaratsubaTestLoop: " << hipGetErrorString(err) << std::endl;
    }
}

#define ExplicitlyInstantiate(SharkFloatParams) \
    template void ComputeMultiplyKaratsubaV1Gpu<SharkFloatParams>(void *kernelArgs[]); \
    template void ComputeMultiplyKaratsubaV1GpuTestLoop<SharkFloatParams>(hipStream_t &stream, void *kernelArgs[]);

ExplicitlyInstantiate(Test4x4SharkParams);
ExplicitlyInstantiate(Test4x2SharkParams);
ExplicitlyInstantiate(Test8x1SharkParams);
ExplicitlyInstantiate(Test8x8SharkParams);

ExplicitlyInstantiate(Test128x64SharkParams);
ExplicitlyInstantiate(Test64x64SharkParams);
ExplicitlyInstantiate(Test32x64SharkParams);
ExplicitlyInstantiate(Test16x64SharkParams);

ExplicitlyInstantiate(Test128x32SharkParams);
ExplicitlyInstantiate(Test128x16SharkParams);
ExplicitlyInstantiate(Test128x8SharkParams);
ExplicitlyInstantiate(Test128x4SharkParams);