#include "hip/hip_runtime.h"
#include "MultiplyNTT.cuh"

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "BenchmarkTimer.h"
#include "DebugChecksum.cuh"
#include "HpSharkFloat.cuh"

#include <algorithm>
#include <cmath>
#include <cstring>
#include <gmp.h>
#include <iomanip>
#include <iostream>
#include <sstream>
#include <vector>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/barrier>

namespace cg = cooperative_groups;


template <class SharkFloatParams, DebugStatePurpose Purpose>
__device__ SharkForceInlineReleaseOnly static void
EraseCurrentDebugState(RecordIt record,
                       DebugState<SharkFloatParams>* debugStates,
                       cooperative_groups::grid_group& grid,
                       cooperative_groups::thread_block& block)
{
    constexpr auto RecursionDepth = 0;
    constexpr auto CallIndex = 0;
    constexpr auto maxPurposes = static_cast<int>(DebugStatePurpose::NumPurposes);
    constexpr auto curPurpose = static_cast<int>(Purpose);
    debugStates[curPurpose].Erase(
        record, grid, block, Purpose, RecursionDepth, CallIndex);
}

template <class SharkFloatParams>
static __device__ void
MultiplyHelperNTTV2Separates(const HpSharkFloat<SharkFloatParams>* SharkRestrict A,
                                   const HpSharkFloat<SharkFloatParams>* SharkRestrict B,
                                   HpSharkFloat<SharkFloatParams>* SharkRestrict OutXX,
                                   HpSharkFloat<SharkFloatParams>* SharkRestrict OutXY,
                                   HpSharkFloat<SharkFloatParams>* SharkRestrict OutYY,
                                   cg::grid_group& grid,
                                   cg::thread_block& block,
                                   uint64_t* SharkRestrict tempProducts)
{

    extern __shared__ uint32_t shared_data[];

    constexpr auto ExecutionBlockBase = 0;
    constexpr auto ExecutionNumBlocks = SharkFloatParams::GlobalNumBlocks;

    // TODO: indexes
    auto* SharkRestrict debugMultiplyCounts =
        reinterpret_cast<DebugMultiplyCount<SharkFloatParams>*>(&tempProducts[0]);
    auto* SharkRestrict debugStates =
        reinterpret_cast<DebugState<SharkFloatParams>*>(&tempProducts[0]);

    if constexpr (SharkPrintMultiplyCounts) {
        const auto CurBlock = block.group_index().x;
        const auto CurThread = block.thread_index().x;
        debugMultiplyCounts[CurBlock * SharkFloatParams::GlobalThreadsPerBlock + CurThread]
            .DebugMultiplyErase();
    }

    if constexpr (SharkDebugChecksums) {
        const auto CurBlock = block.group_index().x;
        const auto CurThread = block.thread_index().x;
        debugMultiplyCounts[CurBlock * SharkFloatParams::GlobalThreadsPerBlock + CurThread]
            .DebugMultiplyErase();

        const RecordIt record =
            (block.thread_index().x == 0 && block.group_index().x == ExecutionBlockBase) ? RecordIt::Yes
                                                                                         : RecordIt::No;
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::Invalid>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::ADigits>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::BDigits>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::CDigits>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::DDigits>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::EDigits>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams,
                               DebugStatePurpose::AHalfHigh>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::AHalfLow>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams,
                               DebugStatePurpose::BHalfHigh>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::BHalfLow>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::XDiff>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::YDiff>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::Z0XX>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::Z0XY>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::Z0YY>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::Z1XX>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::Z1XY>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::Z1YY>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::Z2XX>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::Z2XY>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::Z2YY>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::Z2_Perm1>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::Z2_Perm2>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::Z2_Perm3>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::Z2_Perm4>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::Z2_Perm5>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams, DebugStatePurpose::Z2_Perm6>(
            record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams,
                               DebugStatePurpose::Z1_offsetXX>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams,
                               DebugStatePurpose::Z1_offsetXY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams,
                               DebugStatePurpose::Z1_offsetYY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams,
                               DebugStatePurpose::Final128XX>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams,
                               DebugStatePurpose::Final128XY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams,
                               DebugStatePurpose::Final128YY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams,
                               DebugStatePurpose::FinalAdd1>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams,
                               DebugStatePurpose::FinalAdd2>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams,
                               DebugStatePurpose::FinalAdd3>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams,
                               DebugStatePurpose::Result_offsetXX>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams,
                               DebugStatePurpose::Result_offsetXY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams,
                               DebugStatePurpose::Result_offsetYY>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams,
                               DebugStatePurpose::Result_Add1>(record, debugStates, grid, block);
        EraseCurrentDebugState<SharkFloatParams,
                               DebugStatePurpose::Result_Add2>(record, debugStates, grid, block);
        static_assert(static_cast<int32_t>(DebugStatePurpose::NumPurposes) == 41,
                      "Unexpected number of purposes");
    }
}

template <class SharkFloatParams>
void
PrintMaxActiveBlocks(void* kernelFn, int sharedAmountBytes)
{
    std::cout << "Shared memory size: " << sharedAmountBytes << std::endl;

    int numBlocks;

    {
        // Check the maximum number of active blocks per multiprocessor
        // with the given shared memory size
        // This is useful to determine if we can fit more blocks
        // in the shared memory

        const auto err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
            &numBlocks, kernelFn, SharkFloatParams::GlobalThreadsPerBlock, sharedAmountBytes);

        if (err != hipSuccess) {
            std::cerr << "CUDA error in hipOccupancyMaxActiveBlocksPerMultiprocessor: "
                      << hipGetErrorString(err) << std::endl;
            return;
        }

        std::cout << "Max active blocks per multiprocessor: " << numBlocks << std::endl;
    }

    {
        size_t availableSharedMemory = 0;
        const auto err = cudaOccupancyAvailableDynamicSMemPerBlock(
            &availableSharedMemory, kernelFn, numBlocks, SharkFloatParams::GlobalThreadsPerBlock);

        if (err != hipSuccess) {
            std::cerr << "CUDA error in cudaOccupancyAvailableDynamicSMemPerBlock: "
                      << hipGetErrorString(err) << std::endl;
            return;
        }

        std::cout << "Available shared memory per block: " << availableSharedMemory << std::endl;
    }

    // Check the number of multiprocessors on the device
    int numSM;

    {
        const auto err = hipDeviceGetAttribute(&numSM, hipDeviceAttributeMultiprocessorCount, 0);

        if (err != hipSuccess) {
            std::cerr << "CUDA error in hipDeviceGetAttribute: " << hipGetErrorString(err)
                      << std::endl;
            return;
        }

        std::cout << "Number of multiprocessors: " << numSM << std::endl;
    }

    int maxConcurrentBlocks = numSM * numBlocks;

    std::cout << "Max concurrent blocks: " << maxConcurrentBlocks << std::endl;
    if (maxConcurrentBlocks < SharkFloatParams::GlobalNumBlocks) {
        std::cout << "Warning: Max concurrent blocks exceeds the number of blocks requested."
                  << std::endl;
    }

    {
        // Check the maximum number of threads per block
        int maxThreadsPerBlock;
        const auto err = hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0);

        if (err != hipSuccess) {
            std::cerr << "CUDA error in hipDeviceGetAttribute: " << hipGetErrorString(err)
                      << std::endl;
            return;
        }

        std::cout << "Max threads per block: " << maxThreadsPerBlock << std::endl;
    }

    {
        // Check the maximum number of threads per multiprocessor
        int maxThreadsPerMultiprocessor;
        const auto err = hipDeviceGetAttribute(
            &maxThreadsPerMultiprocessor, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);
        if (err != hipSuccess) {
            std::cerr << "CUDA error in hipDeviceGetAttribute: " << hipGetErrorString(err)
                      << std::endl;
            return;
        }
        std::cout << "Max threads per multiprocessor: " << maxThreadsPerMultiprocessor << std::endl;
    }

    // Check if this device supports cooperative launches
    int cooperativeLaunch;

    {
        const auto err = hipDeviceGetAttribute(&cooperativeLaunch, hipDeviceAttributeCooperativeLaunch, 0);

        if (err != hipSuccess) {
            std::cerr << "CUDA error in hipDeviceGetAttribute: " << hipGetErrorString(err)
                      << std::endl;
            return;
        }

        if (cooperativeLaunch) {
            std::cout << "This device supports cooperative launches." << std::endl;
        } else {
            std::cout << "This device does not support cooperative launches." << std::endl;
        }
    }
}

// Assuming that SharkFloatParams::GlobalNumUint32 can be large and doesn't fit in shared memory
// We'll use the provided global memory buffers for large intermediates
template <class SharkFloatParams>
static __device__ void
MultiplyHelperNTT(HpSharkComboResults<SharkFloatParams>* SharkRestrict combo,
                          cg::grid_group& grid,
                          cg::thread_block& block,
                          uint64_t* SharkRestrict tempProducts)
{

    MultiplyHelperNTTV2Separates<SharkFloatParams>(&combo->A,
                                                         &combo->B,
                                                         &combo->ResultX2,
                                                         &combo->Result2XY,
                                                         &combo->ResultY2,
                                                         grid,
                                                         block,
                                                         tempProducts);
}
