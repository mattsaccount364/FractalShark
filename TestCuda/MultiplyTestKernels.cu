#include "hip/hip_runtime.h"
#include "Multiply.cu"

template<class SharkFloatParams>
__maxnreg__(SharkRegisterLimit)
__global__ void MultiplyKernelKaratsubaV2(
    HpSharkComboResults<SharkFloatParams> *combo,
    uint64_t *tempProducts) {

    // Initialize cooperative grid group
    cg::grid_group grid = cg::this_grid();
    cg::thread_block block = cg::this_thread_block();

    // Call the MultiplyHelper function
    //MultiplyHelper(A, B, Out, carryIns, grid, tempProducts);
    if constexpr (!SharkFloatParams::ForceNoOp) {
        MultiplyHelperKaratsubaV2(combo, grid, block, tempProducts);
    } else {
        grid.sync();
    }
}

template<class SharkFloatParams>
__global__ void
__maxnreg__(SharkRegisterLimit)
MultiplyKernelKaratsubaV2TestLoop(
    HpSharkComboResults<SharkFloatParams> *combo,
    uint64_t numIters,
    uint64_t *tempProducts) { // Array to store cumulative carries

    // Initialize cooperative grid group
    cg::grid_group grid = cg::this_grid();
    cg::thread_block block = cg::this_thread_block();

    for (int i = 0; i < numIters; ++i) {
        // MultiplyHelper(A, B, Out, carryIns, grid, tempProducts);
        if constexpr (!SharkFloatParams::ForceNoOp) {
            MultiplyHelperKaratsubaV2(combo, grid, block, tempProducts);
        } else {
            grid.sync();
        }
    }
}

template<class SharkFloatParams>
void ComputeMultiplyKaratsubaV2Gpu(void *kernelArgs[]) {

    hipError_t err;

    constexpr auto sharedAmountBytes = CalculateMultiplySharedMemorySize<SharkFloatParams>();

    if constexpr (SharkCustomStream) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            MultiplyKernelKaratsubaV2<SharkFloatParams>),
            hipFuncAttributeMaxDynamicSharedMemorySize,
            sharedAmountBytes);

        PrintMaxActiveBlocks<SharkFloatParams>(
            MultiplyKernelKaratsubaV2<SharkFloatParams>,
            sharedAmountBytes);
    }

    err = hipLaunchCooperativeKernel(
        (void *)MultiplyKernelKaratsubaV2<SharkFloatParams>,
        dim3(SharkFloatParams::GlobalNumBlocks),
        dim3(SharkFloatParams::GlobalThreadsPerBlock),
        kernelArgs,
        sharedAmountBytes, // Shared memory size
        0 // Stream
    );

    auto err2 = hipGetLastError();
    if (err != hipSuccess || err2 != hipSuccess) {
        std::cerr << "CUDA error in hipLaunchCooperativeKernel: " << hipGetErrorString(err2) <<
            "err: " << err << std::endl;
    }

    hipDeviceSynchronize();

    if (err != hipSuccess) {
        std::cerr << "CUDA error in MultiplyKernelKaratsubaV2: " << hipGetErrorString(err) << std::endl;
    }
}

template<class SharkFloatParams>
void ComputeMultiplyKaratsubaV2GpuTestLoop(hipStream_t &stream, void *kernelArgs[]) {

    constexpr auto sharedAmountBytes = CalculateMultiplySharedMemorySize<SharkFloatParams>();

    if constexpr (SharkCustomStream) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            MultiplyKernelKaratsubaV2TestLoop<SharkFloatParams>),
            hipFuncAttributeMaxDynamicSharedMemorySize,
            sharedAmountBytes);

        PrintMaxActiveBlocks<SharkFloatParams>(
            MultiplyKernelKaratsubaV2TestLoop<SharkFloatParams>,
            sharedAmountBytes);
    }

    hipError_t err = hipLaunchCooperativeKernel(
        (void *)MultiplyKernelKaratsubaV2TestLoop<SharkFloatParams>,
        dim3(SharkFloatParams::GlobalNumBlocks),
        dim3(SharkFloatParams::GlobalThreadsPerBlock),
        kernelArgs,
        sharedAmountBytes, // Shared memory size
        stream // Stream
    );

    hipDeviceSynchronize();

    if (err != hipSuccess) {
        std::cerr << "CUDA error in MultiplyKernelKaratsubaTestLoop: " << hipGetErrorString(err) << std::endl;
    }
}

#define ExplicitlyInstantiate(SharkFloatParams) \
    template void ComputeMultiplyKaratsubaV2Gpu<SharkFloatParams>(void *kernelArgs[]); \
    template void ComputeMultiplyKaratsubaV2GpuTestLoop<SharkFloatParams>(hipStream_t &stream, void *kernelArgs[]);

#ifdef SHARK_INCLUDE_KERNELS
ExplicitInstantiateAll();
#endif