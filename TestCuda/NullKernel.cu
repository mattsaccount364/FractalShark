#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "BenchmarkTimer.h"

#include <iostream>

__global__ void trivial_kernel(int *output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    output[idx] = idx;
}

void TestNullKernel() {
    const int N = 256;
    const auto NumIterations = 100000;
    int *d_output;
    hipError_t err = hipMalloc(&d_output, N * sizeof(int));
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipMalloc: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // Warm-up kernel to mitigate startup overhead
    trivial_kernel << <1, N >> > (d_output);
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipDeviceSynchronize (warm-up): " << hipGetErrorString(err) << std::endl;
        hipFree(d_output);
        return;
    }

    BenchmarkTimer timer;
    {
        ScopedBenchmarkStopper stopper(timer);

        for (int i = 0; i < NumIterations; ++i) {

            // Launch the trivial kernel
            trivial_kernel << <1, N >> > (d_output);

            // Ensure the kernel has completed
            err = hipDeviceSynchronize();
            if (err != hipSuccess) {
                std::cerr << "CUDA error in hipDeviceSynchronize (iteration " << i << "): " << hipGetErrorString(err) << std::endl;
                break;
            }
        }
    }

    uint64_t elapsed_ms = timer.GetDeltaInMs();
    std::cout << "Null kernel elapsed time: " << elapsed_ms << " ms" << std::endl;

    err = hipFree(d_output);
    if (err != hipSuccess) {
        std::cerr << "CUDA error in hipFree: " << hipGetErrorString(err) << std::endl;
    }
}
