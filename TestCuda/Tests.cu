#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "HpSharkFloat.cuh"
#include "BenchmarkTimer.h"
#include "TestTracker.h"

#include "Tests.cuh"
#include "Add.cuh"
#include "Multiply.cuh"

#include <iostream>
#include <vector>
#include <gmp.h>
#include <cstring>
#include <sstream>
#include <iomanip>
#include <cmath>
#include <algorithm>
#include <assert.h>

static TestTracker Tests;

template<class SharkFloatParams, Operator sharkOperator>
void DiffAgainstHost(
    int testNum,
    const mpf_t mpfHostResult,
    const HpSharkFloat<SharkFloatParams> &gpuResult) {

    if (Verbose) {
        std::cout << "\nGPU result: " << std::endl;
        std::cout << gpuResult.ToString() << std::endl;
        std::cout << gpuResult.ToHexString() << std::endl;
    }

    // Convert the HpSharkFloat<SharkFloatParams> results to mpf_t for comparison
    mpf_t mpfXGpuResult;
    mpf_init(mpfXGpuResult);

    HpGpuToMpf(gpuResult, mpfXGpuResult);

    // Compute the differences between host and GPU results
    mpf_t mpfDiff;
    mpf_init(mpfDiff);

    mpf_sub(mpfDiff, mpfHostResult, mpfXGpuResult);

    // Take absolute delta:
    mpf_t mpfDiffAbs;
    mpf_init(mpfDiffAbs);
    mpf_abs(mpfDiffAbs, mpfDiff);

    // Converted GPU result
    if (Verbose) {
        std::cout << "\nConverted GPU result:" << std::endl;
        std::cout << MpfToString<SharkFloatParams>(mpfXGpuResult, HpSharkFloat<SharkFloatParams>::DefaultPrecBits) << std::endl;

        // Print the differences
        std::cout << "\nDifference between host and GPU results:" << std::endl;
        std::cout << MpfToString<SharkFloatParams>(mpfDiffAbs, LowPrec) << std::endl;
    }

    // Check if the host result is zero to avoid division by zero
    mp_bitcnt_t gpuPrecBits = HpSharkFloat<SharkFloatParams>::DefaultPrecBits;
    mp_bitcnt_t margin = sizeof(uint32_t) * 8 * 2;
    mp_bitcnt_t totalPrecBits = (gpuPrecBits > margin) ? (gpuPrecBits - margin) : 1;
    mpf_t acceptableError;

    if (mpf_cmp_ui(mpfHostResult, 0) != 0) {
        // Host result is non-zero

        // Compute relative error
        mpf_t relativeError;
        mpf_init(relativeError);
        mpf_sub(relativeError, mpfHostResult, mpfXGpuResult);
        mpf_div(relativeError, relativeError, mpfHostResult);
        mpf_abs(relativeError, relativeError);

        // Compute machine epsilon: epsilon = 2^(-totalPrecBits)
        mpf_t epsilon;
        mpf_init2(epsilon, totalPrecBits);
        mpf_set_ui(epsilon, 1);
        mpf_div_2exp(epsilon, epsilon, totalPrecBits);

        // Compute acceptable error: acceptableError = epsilon * abs(hostResult)
        mpf_init(acceptableError);
        mpf_mul(acceptableError, epsilon, mpfHostResult);
        mpf_abs(acceptableError, acceptableError);

        // Compare absolute error with acceptable threshold
        auto relativeErrorStr = MpfToString<SharkFloatParams>(relativeError, LowPrec);
        auto epsilonStr = MpfToString<SharkFloatParams>(epsilon, LowPrec);
        if (mpf_cmp(relativeError, epsilon) <= 0) {
            if (Verbose) {
                std::cout << "\nThe relative error is within acceptable bounds." << std::endl;
                std::cout << "Relative error: " << epsilonStr << std::endl;
            }
        } else {
            std::cerr << "\nError: The relative error exceeds acceptable bounds." << std::endl;
            std::cout << "Relative error: " << relativeErrorStr << std::endl;
            Tests.MarkFailed(testNum, relativeErrorStr, epsilonStr);
        }

        // Clean up
        mpf_clear(relativeError);
        mpf_clear(epsilon);
        mpf_clear(acceptableError);
    } else {
        // Host result is zero

        // For zero host result, use an absolute error threshold
        mpf_init2(acceptableError, totalPrecBits);
        mpf_set_ui(acceptableError, 1);
        mpf_div_2exp(acceptableError, acceptableError, totalPrecBits);

        auto mpfDiffAbsStr = MpfToString<SharkFloatParams>(mpfDiffAbs, LowPrec);
        auto absoluteErrorStr = MpfToString<SharkFloatParams>(acceptableError, LowPrec);

        if (mpf_cmp(mpfDiffAbs, acceptableError) <= 0) {
            if (Verbose) {
                std::cout << "\nThe absolute error is within acceptable bounds." << std::endl;
            }
        } else {
            std::cerr << "\nError: The absolute error exceeds acceptable bounds." << std::endl;
            Tests.MarkFailed(testNum, mpfDiffAbsStr, absoluteErrorStr);
        }

        mpf_clear(acceptableError);
    }

    mpf_clear(mpfDiff);
    mpf_clear(mpfDiffAbs);
    mpf_clear(mpfXGpuResult);
}

template<class SharkFloatParams, Operator sharkOperator>
void TestAddTwoNumbersPerf(
    int testNum,
    const char *num1,
    const char *num2,
    const mpf_t mpfX,
    const mpf_t mpfY) {

    // Print the original input values
    if (Verbose) {
        std::cout << "Original input values:" << std::endl;
        std::cout << "num1: " << num1 << std::endl;
        std::cout << "X: " << MpfToString<SharkFloatParams>(mpfX, HpSharkFloat<SharkFloatParams>::DefaultPrecBits) << std::endl;
        std::cout << "num2: " << num2 << std::endl;
        std::cout << "Y: " << MpfToString<SharkFloatParams>(mpfY, HpSharkFloat<SharkFloatParams>::DefaultPrecBits) << std::endl;
    }

    std::unique_ptr<HpSharkFloat<SharkFloatParams>> xNum = std::make_unique<HpSharkFloat<SharkFloatParams>>();
    std::unique_ptr<HpSharkFloat<SharkFloatParams>> yNum = std::make_unique<HpSharkFloat<SharkFloatParams>>();
    std::unique_ptr<HpSharkFloat<SharkFloatParams>> resultNum = std::make_unique<HpSharkFloat<SharkFloatParams>>();
    MpfToHpGpu(mpfX, *xNum, HpSharkFloat<SharkFloatParams>::DefaultPrecBits);
    MpfToHpGpu(mpfY, *yNum, HpSharkFloat<SharkFloatParams>::DefaultPrecBits);
    if (Verbose) {
        std::cout << "\nConverted HpSharkFloat<SharkFloatParams> representations:" << std::endl;
        std::cout << "X: " << xNum->ToString() << std::endl;
        std::cout << "Y: " << yNum->ToString() << std::endl;
    }

    // Perform the calculation on the host using MPIR
    mpf_t mpfHostResult;
    mpf_init(mpfHostResult);

    {
        BenchmarkTimer hostTimer;
        ScopedBenchmarkStopper hostStopper{ hostTimer };

        for (int i = 0; i < TestIterCount; ++i) {
            if constexpr (sharkOperator == Operator::Add) {
                mpf_add(mpfHostResult, mpfX, mpfY);
            } else if constexpr (sharkOperator == Operator::Multiply) {
                mpf_mul(mpfHostResult, mpfX, mpfY);
            }
        }

        hostTimer.StopTimer();

        std::cout << "Host iter time: " << hostTimer.GetDeltaInMs() << " ms" << std::endl;
    }

    std::unique_ptr<HpSharkFloat<SharkFloatParams>> gpuResult2 = std::make_unique<HpSharkFloat<SharkFloatParams>>();

    {
        // Perform the calculation on the GPU
        HpSharkFloat<SharkFloatParams> *xGpu;
        hipMalloc(&xGpu, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemcpy(xGpu, xNum.get(), sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyHostToDevice);

        HpSharkFloat<SharkFloatParams> *yGpu;
        hipMalloc(&yGpu, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemcpy(yGpu, yNum.get(), sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyHostToDevice);

        HpSharkFloat<SharkFloatParams> *internalGpuResult2;
        hipMalloc(&internalGpuResult2, sizeof(HpSharkFloat<SharkFloatParams>));
        hipMemset(internalGpuResult2, 0, sizeof(HpSharkFloat<SharkFloatParams>));

        BenchmarkTimer timer;
        ScopedBenchmarkStopper stopper{ timer };

        if constexpr (sharkOperator == Operator::Add) {
            // Allocate memory for carryOuts and cumulativeCarries
            GlobalAddBlockData *globalBlockData;
            CarryInfo *d_carryOuts;
            uint32_t *d_cumulativeCarries;
            hipMalloc(&globalBlockData, sizeof(GlobalAddBlockData));
            hipMalloc(&d_carryOuts, (SharkFloatParams::NumBlocks + 1) * sizeof(CarryInfo));
            hipMalloc(&d_cumulativeCarries, (SharkFloatParams::NumBlocks + 1) * sizeof(uint32_t));

            // Prepare kernel arguments
            void *kernelArgs[] = {
                (void *)&xGpu,
                (void *)&yGpu,
                (void *)&internalGpuResult2,
                (void *)&globalBlockData,
                (void *)&d_carryOuts,
                (void *)&d_cumulativeCarries
            };

            ComputeAddGpuTestLoop<SharkFloatParams>(kernelArgs);

            // Launch the cooperative kernel

            hipMemcpy(gpuResult2.get(), internalGpuResult2, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyDeviceToHost);

            hipFree(globalBlockData);
            hipFree(d_carryOuts);
            hipFree(d_cumulativeCarries);
        } else if constexpr (sharkOperator == Operator::Multiply) {
            // Prepare kernel arguments
            // Allocate memory for carryOuts and cumulativeCarries
            uint64_t *d_carry1;
            uint64_t *d_carry2;
            uint64_t *d_carry3;
            uint64_t *d_tempProducts;
            hipMalloc(&d_carry1,        2 * SharkFloatParams::NumBlocks * sizeof(uint64_t));
            hipMalloc(&d_carry2,        2 * SharkFloatParams::NumBlocks * sizeof(uint64_t));
            hipMalloc(&d_carry3,        2 * SharkFloatParams::NumBlocks * sizeof(uint64_t));
            hipMalloc(&d_tempProducts, 32 * SharkFloatParams::NumUint32 * sizeof(uint64_t));

            void *kernelArgs[] = {
                (void *)&xGpu,
                (void *)&yGpu,
                (void *)&internalGpuResult2,
                (void *)&d_carry1,
                (void *)&d_carry2,
                (void *)&d_carry3,
                (void *)&d_tempProducts
            };

            ComputeMultiplyGpuTestLoop<SharkFloatParams>(kernelArgs);

            hipFree(d_carry1);
            hipFree(d_carry2);
            hipFree(d_carry3);
            hipFree(d_tempProducts);

            hipMemcpy(gpuResult2.get(), internalGpuResult2, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyDeviceToHost);
        }

        timer.StopTimer();
        Tests.AddTime(testNum, timer.GetDeltaInMs());

        std::cout << "GPU iter time: " << timer.GetDeltaInMs() << " ms" << std::endl;

        hipFree(internalGpuResult2);
        hipFree(xGpu);
    }

    DiffAgainstHost<SharkFloatParams, sharkOperator>(testNum, mpfHostResult, *gpuResult2);

    // Clean up MPIR variables
    mpf_clear(mpfHostResult);
}

template<class SharkFloatParams, Operator sharkOperator>
void TestAddTwoNumbersPerf(
    int testNum,
    const char *num1,
    const char *num2) {

    mpf_set_default_prec(HpSharkFloat<SharkFloatParams>::DefaultMpirBits);  // Set precision for MPIR floating point

    mpf_t mpfX;
    mpf_t mpfY;
    mpf_init(mpfX);
    mpf_init(mpfY);

    auto res = mpf_set_str(mpfX, num1, 10);
    if (res == -1) {
        std::cout << "Error setting mpfX" << std::endl;
    }

    res = mpf_set_str(mpfY, num2, 10);
    if (res == -1) {
        std::cout << "Error setting mpfY" << std::endl;
    }

    TestAddTwoNumbersPerf<SharkFloatParams, sharkOperator>(testNum, num1, num2, mpfX, mpfY);

    mpf_clear(mpfX);
    mpf_clear(mpfY);
}

template<class SharkFloatParams, Operator sharkOperator>
void TestBinOperatorTwoNumbers(
    int testNum,
    const HpSharkFloat<SharkFloatParams> &xNum,
    const HpSharkFloat<SharkFloatParams> &yNum,
    const mpf_t &mpfX,
    const mpf_t &mpfY) {

    if (Verbose) {
        std::cout << "\nConverted HpSharkFloat<SharkFloatParams> representations:" << std::endl;
        std::cout << "X: " << xNum.ToString() << std::endl;
        std::cout << "X hex: " << xNum.ToHexString() << std::endl;
        std::cout << "Y: " << yNum.ToString() << std::endl;
        std::cout << "Y hex: " << yNum.ToHexString() << std::endl;
    }

    // Perform the calculation on the GPU
    HpSharkFloat<SharkFloatParams> *xGpu;
    HpSharkFloat<SharkFloatParams> *yGpu;

    hipMalloc(&xGpu, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMalloc(&yGpu, sizeof(HpSharkFloat<SharkFloatParams>));
    hipMemcpy(xGpu, &xNum, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyHostToDevice);
    hipMemcpy(yGpu, &yNum, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyHostToDevice);

    {
        // Perform the calculation on the host using MPIR
        HpSharkFloat<SharkFloatParams> gpuResult{};
        mpf_t mpfHostResult;
        mpf_init(mpfHostResult);

        if constexpr (sharkOperator == Operator::Add) {
            mpf_add(mpfHostResult, mpfX, mpfY);
        } else if constexpr (sharkOperator == Operator::Multiply) {
            mpf_mul(mpfHostResult, mpfX, mpfY);
        }

        // Print host result
        if (Verbose) {
            std::cout << "\nHost result:" << std::endl;
            std::cout << "Host result: " << MpfToString<SharkFloatParams>(mpfHostResult, HpSharkFloat<SharkFloatParams>::DefaultPrecBits) << std::endl;
            std::cout << "Host hex: " << std::endl;
            std::cout << "" << MpfToHexString(mpfHostResult) << std::endl;
        }

        HpSharkFloat<SharkFloatParams> *internalGpuResult;
        hipMalloc(&internalGpuResult, sizeof(HpSharkFloat<SharkFloatParams>));

        BenchmarkTimer timer;
        ScopedBenchmarkStopper stopper{ timer };

        if constexpr (sharkOperator == Operator::Add) {
            // Allocate memory for carryOuts and cumulativeCarries
            GlobalAddBlockData *globalBlockData;
            CarryInfo *d_carryOuts;
            uint32_t *d_cumulativeCarries;
            hipMalloc(&globalBlockData, sizeof(GlobalAddBlockData));
            hipMalloc(&d_carryOuts, (SharkFloatParams::NumBlocks + 1) * sizeof(CarryInfo));
            hipMalloc(&d_cumulativeCarries, (SharkFloatParams::NumBlocks + 1) * sizeof(uint32_t));

            // Prepare kernel arguments
            void *kernelArgs[] = {
                (void *)&xGpu,
                (void *)&yGpu,
                (void *)&internalGpuResult,
                (void *)&globalBlockData,
                (void *)&d_carryOuts,
                (void *)&d_cumulativeCarries
            };

            ComputeAddGpu<SharkFloatParams>(kernelArgs);

            hipFree(globalBlockData);
            hipFree(d_carryOuts);
            hipFree(d_cumulativeCarries);
        } else if constexpr (sharkOperator == Operator::Multiply) {
            // Prepare kernel arguments
            // Allocate memory for carryOuts and cumulativeCarries
            uint64_t *d_carry1;
            uint64_t *d_carry2;
            uint64_t *d_carry3;
            uint64_t *d_tempProducts;
            hipMalloc(&d_carry1,        2 * SharkFloatParams::NumUint32 * sizeof(uint64_t));
            hipMalloc(&d_carry2,        2 * SharkFloatParams::NumUint32 * sizeof(uint64_t));
            hipMalloc(&d_carry3,        2 * SharkFloatParams::NumUint32 * sizeof(uint64_t));
            hipMalloc(&d_tempProducts, 32 * SharkFloatParams::NumUint32 * sizeof(uint64_t));

            void *kernelArgs[] = {
                (void *)&xGpu,
                (void *)&yGpu,
                (void *)&internalGpuResult,
                (void *)&d_carry1,
                (void *)&d_carry2,
                (void *)&d_carry3,
                (void *)&d_tempProducts
            };

            ComputeMultiplyGpu<SharkFloatParams>(kernelArgs);

            hipFree(d_carry1);
            hipFree(d_carry2);
            hipFree(d_carry3);
            hipFree(d_tempProducts);
        }

        hipMemcpy(&gpuResult, internalGpuResult, sizeof(HpSharkFloat<SharkFloatParams>), hipMemcpyDeviceToHost);

        timer.StopTimer();
        Tests.AddTime(testNum, timer.GetDeltaInMs());

        if (Verbose) {
            std::cout << "GPU single time: " << timer.GetDeltaInMs() << " ms" << std::endl;
        }

        hipFree(internalGpuResult);

        DiffAgainstHost<SharkFloatParams, sharkOperator>(testNum, mpfHostResult, gpuResult);

        // Clean up MPIR variables
        mpf_clear(mpfHostResult);
    }

    hipFree(xGpu);
    hipFree(yGpu);
}

template<class SharkFloatParams, Operator sharkOperator>
void TestBinOperatorTwoNumbers(
    int testNum,
    const char *num1,
    const char *num2,
    const mpf_t &mpfX,
    const mpf_t &mpfY) {

    // Print the original input values
    if (Verbose) {
        std::cout << "Original input strings:" << std::endl;
        std::cout << "num1: " << num1 << std::endl;
        std::cout << "num2: " << num2 << std::endl;
        std::cout << "MpfX: " << MpfToString<SharkFloatParams>(mpfX, HpSharkFloat<SharkFloatParams>::DefaultPrecBits) << std::endl;
        std::cout << "MpfY: " << MpfToString<SharkFloatParams>(mpfY, HpSharkFloat<SharkFloatParams>::DefaultPrecBits) << std::endl;
    }

    // Convert the input values to HpSharkFloat<SharkFloatParams> representations
    std::unique_ptr<HpSharkFloat<SharkFloatParams>> xNum = std::make_unique<HpSharkFloat<SharkFloatParams>>();
    std::unique_ptr<HpSharkFloat<SharkFloatParams>> yNum = std::make_unique<HpSharkFloat<SharkFloatParams>>();
    MpfToHpGpu(mpfX, *xNum, HpSharkFloat<SharkFloatParams>::DefaultPrecBits);
    MpfToHpGpu(mpfY, *yNum, HpSharkFloat<SharkFloatParams>::DefaultPrecBits);

    TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(testNum, *xNum, *yNum, mpfX, mpfY);
}

template<class SharkFloatParams, Operator sharkOperator>
void TestBinOperatorTwoNumbers(
    int testNum,
    const char *num1,
    const char *num2) {

    std::cout << std::endl;
    std::cout << std::endl;
    std::cout << "Test " << testNum << std::endl;

    mpf_set_default_prec(HpSharkFloat<SharkFloatParams>::DefaultMpirBits);  // Set precision for MPIR floating point

    mpf_t mpfX, mpfY;
    mpf_init(mpfX);
    mpf_init(mpfY);

    auto res = mpf_set_str(mpfX, num1, 10);
    if (res == -1) {
        std::cout << "Error setting mpfX" << std::endl;
    }

    res = mpf_set_str(mpfY, num2, 10);
    if (res == -1) {
        std::cout << "Error setting mpfY" << std::endl;
    }

    TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(testNum, num1, num2, mpfX, mpfY);

    mpf_clear(mpfX);
    mpf_clear(mpfY);
}

template<class SharkFloatParams, Operator sharkOperator>
void TestAddSpecialNumbers(int testNum, std::vector<uint32_t> &digits1, std::vector<uint32_t> &digits2) {
    mpf_t x, y;
    mpf_init(x);
    mpf_init(y);

    std::cout << std::endl;
    std::cout << std::endl;
    std::cout << "Test " << testNum << std::endl;

    auto strLargeX = Uint32ToMpf<SharkFloatParams>(digits1.data(), SharkFloatParams::NumUint32 / 2, x);
    auto strLargeY = Uint32ToMpf<SharkFloatParams>(digits2.data(), SharkFloatParams::NumUint32 / 2, y);
    TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(testNum, strLargeX.c_str(), strLargeY.c_str(), x, y);

    mpf_clear(x);
    mpf_clear(y);
}

template<class SharkFloatParams, Operator sharkOperator>
void TestAddSpecialNumbers(
    int testNum,
    const HpSharkFloat<SharkFloatParams> &xNum,
    const HpSharkFloat<SharkFloatParams> &yNum) {

    mpf_t mpfX;
    mpf_t mpfY;
    mpf_init(mpfX);
    mpf_init(mpfY);
    HpGpuToMpf(xNum, mpfX);
    HpGpuToMpf(yNum, mpfY);

    TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(testNum, xNum, yNum, mpfX, mpfY);
}

template<class SharkFloatParams, Operator sharkOperator>
void TestAddSpecialNumbers1(int testNum) {
    std::vector<uint32_t> testData;
    for (size_t i = 0; i < SharkFloatParams::NumUint32; ++i) {
        testData.push_back(0);
    }

    assert(testData.size() == SharkFloatParams::NumUint32);
    testData[testData.size() - 1] = 0x80000000;

    TestAddSpecialNumbers<SharkFloatParams, sharkOperator>(testNum, testData, testData);
}

template<class SharkFloatParams, Operator sharkOperator>
void TestAddSpecialNumbers2(int testNum) {
    std::vector<uint32_t> testData;
    for (size_t i = 0; i < SharkFloatParams::NumUint32; ++i) {
        testData.push_back(0);
    }

    assert(testData.size() == SharkFloatParams::NumUint32);
    testData[testData.size() - 1] = 0xC0000000;

    TestAddSpecialNumbers<SharkFloatParams, sharkOperator>(testNum, testData, testData);
}

template<class SharkFloatParams, Operator sharkOperator>
void TestAddSpecialNumbers3(int testNum) {
    std::vector<uint32_t> testData;
    for (size_t i = 0; i < SharkFloatParams::NumUint32; ++i) {
        testData.push_back(0);
    }

    assert(testData.size() == SharkFloatParams::NumUint32);
    testData[testData.size() - 1] = 0xFFFFFFFF;

    TestAddSpecialNumbers<SharkFloatParams, sharkOperator>(testNum, testData, testData);
}

template<class SharkFloatParams, Operator sharkOperator>
void TestAddSpecialNumbersHelper(
    int testNum,
    bool isNegative1,
    std::vector<uint32_t> testData1,
    bool isNegative2,
    std::vector<uint32_t> testData2) {

    std::cout << std::endl;
    std::cout << std::endl;
    std::cout << "Test " << testNum << std::endl;

    std::vector<uint32_t> testData1Copy;
    testData1Copy = testData1;
    testData1Copy.resize(SharkFloatParams::NumUint32);

    std::vector<uint32_t> testData2Copy;
    testData2Copy = testData2;
    testData2Copy.resize(SharkFloatParams::NumUint32);

    std::unique_ptr<HpSharkFloat<SharkFloatParams>> xNum{ std::make_unique<HpSharkFloat<SharkFloatParams>>(testData1Copy.data(), 0, isNegative1) };
    std::unique_ptr<HpSharkFloat<SharkFloatParams>> yNum{ std::make_unique<HpSharkFloat<SharkFloatParams>>(testData2Copy.data(), 0, isNegative2) };

    TestAddSpecialNumbers<SharkFloatParams, sharkOperator>(testNum, *xNum, *yNum);
}

template<class SharkFloatParams, Operator sharkOperator>
void TestAddSpecialNumbers4(int testNum) {
    TestAddSpecialNumbersHelper<SharkFloatParams, sharkOperator>(
        testNum,
        true,
        std::vector<uint32_t>{ 0xF26D37FC, 0xA96025CE, 0xB03FC716, 0x1DF7182B, 0xCCBD69BD, 0x40C0F80C, 0xFAA0222E, 0xD1FDA456 },
        true,
        std::vector<uint32_t>{ 0x8BBCDF3, 0x4C3E7ACB, 0x6691A71D, 0xDFE03842, 0x3FADCA11, 0x4058BC9E, 0xF30FD7DE, 0xAA6CA582 });
}

template<class SharkFloatParams, Operator sharkOperator>
void TestAddSpecialNumbers5(int testNum) {
    TestAddSpecialNumbersHelper<SharkFloatParams, sharkOperator>(
        testNum,
        false,
        std::vector<uint32_t>{ 0, 0, 0, 0, 0, 0, 0, 0xFFFFFFFF },
        true,
        std::vector<uint32_t>{ 0, 0, 0, 0, 0, 0, 0, 0xFFFFFFFF });
}

template<class SharkFloatParams, Operator sharkOperator>
void TestAddSpecialNumbers6(int testNum) {
    TestAddSpecialNumbersHelper<SharkFloatParams, sharkOperator>(
        testNum,
        true,
        std::vector<uint32_t>{ 0xFFFFFFFF, 0xFFFFFFFF },
        true,
        std::vector<uint32_t>{ 0xFFFFFFFF, 0xFFFFFFFF });
}

template<class SharkFloatParams, Operator sharkOperator>
void TestAddSpecialNumbers7(int testNum) {
    TestAddSpecialNumbersHelper<SharkFloatParams, sharkOperator>(
        testNum,
        true,
        std::vector<uint32_t>{ 0, 0xFFFFFFFF, 0xFFFFFFFF },
        true,
        std::vector<uint32_t>{ 0, 0xFFFFFFFF, 0xFFFFFFFF });
}

template<class SharkFloatParams, Operator sharkOperator>
void TestAddSpecialNumbers8(int testNum) {

    TestAddSpecialNumbersHelper<SharkFloatParams, sharkOperator>(
        testNum,
        true,
        std::vector<uint32_t>{ 0, 0, 0xFFFFFFFF, 0xFFFFFFFF },
        true,
        std::vector<uint32_t>{ 0, 0, 0xFFFFFFFF, 0xFFFFFFFF });
}

template<class SharkFloatParams, Operator sharkOperator>
void TestAddSpecialNumbers9(int testNum) {

    TestAddSpecialNumbersHelper<SharkFloatParams, sharkOperator>(
        testNum,
        true,
        std::vector<uint32_t>{ 0, 0, 0, 0xFFFFFFFF, 0xFFFFFFFF },
        true,
        std::vector<uint32_t>{ 0, 0, 0, 0xFFFFFFFF, 0xFFFFFFFF });
}


template<class SharkFloatParams, Operator sharkOperator>
bool TestAllBinaryOp(int testBase) {
    constexpr bool includeSet1 = true;
    constexpr bool includeSet2 = true;
    constexpr bool includeSet3 = true;
    constexpr bool includeSet4 = true;
    constexpr bool includeSet5 = true;
    constexpr bool includeSet6 = true;
    constexpr bool includeSet10 = true;

    // 200s is multiply
    // 400s is add
    
    if constexpr (includeSet1) {
        const auto set = testBase + 10;
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 1, "1", "2");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 2, "4294967295", "1");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 3, "4294967296", "1");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 4, "4294967295", "4294967296");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 5, "4294967296", "-1");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 6, "18446744073709551615", "1");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 7, "0", "0.1");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 8, "0.1", "0");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 9, "0", "0");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 10, "0.1", "0.1");
    }

    if constexpr (includeSet2) {
        const auto set = testBase + 30;
        TestAddSpecialNumbers1<SharkFloatParams, sharkOperator>(set + 1);
        TestAddSpecialNumbers2<SharkFloatParams, sharkOperator>(set + 2);
        TestAddSpecialNumbers3<SharkFloatParams, sharkOperator>(set + 3);
        TestAddSpecialNumbers4<SharkFloatParams, sharkOperator>(set + 4);
        TestAddSpecialNumbers5<SharkFloatParams, sharkOperator>(set + 5);
        TestAddSpecialNumbers6<SharkFloatParams, sharkOperator>(set + 6);
        TestAddSpecialNumbers7<SharkFloatParams, sharkOperator>(set + 7);
        TestAddSpecialNumbers8<SharkFloatParams, sharkOperator>(set + 8);
        TestAddSpecialNumbers9<SharkFloatParams, sharkOperator>(set + 9);
    }

    if constexpr (includeSet3) {
        const auto set = testBase + 40;
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 1, "2", "0.1");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 2, "0.2", "0.1");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 3, "0.5", "1.2");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 4, "0.6", "1.3");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 5, "0.7", "1.4");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 6, "0.1", "1.99999999999999999999999999999");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 7, "0.123124561464451654461", "1.2395123123127298375982735");
    }

    if constexpr (includeSet4) {
        const auto set = testBase + 50;
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 1, "-0.5", "1.2");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 2, "-0.6", "1.3");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 3, "-0.7", "1.4");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 4, "-0.1", "1.99999999999999999999999999999");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 5, "-0.123124561464451654461", "1.2395123123127298375982735");
    }

    if constexpr (includeSet5) {
        const auto set = testBase + 60;
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 1, "-0.5", "-1.2");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 2, "-0.6", "-1.3");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 3, "-0.7", "-1.4");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 4, "-0.1", "-1.99999999999999999999999999999");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 5, "-0.123124561464451654461", "-1.2395123123127298375982735");
    }

    if constexpr (includeSet6) {
        const auto set = testBase + 70;
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 1, "0.5265542653452654526545625456254565446654545645649789871322131213156435546435", "-1.263468375787958774985473345435632415334245268476928454653443234164658776634854746584532186639173047328910730217803271839216");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 2, "0.2999999999965542653452654526545625456254565446654545645649789871322131213156435546435", "-1.263468375787958774985473345435632415334245268476928454653443234164658776634854746584532186639173047328910730217803271839216");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 3, "0.1265542653452654526545625456254565446654545645649789871322131213156435546435", "-1.2634683757879587749854733454356324153342452684769284546534432341646587766348547465845321866391730473289107302178039999999999999271839216");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 4, "0.0265542653452654526545625456254565446654545645649789871322131213156435546435", "-1.263468375787958774985473345435632415334245268476928454653443234164658776634854746584532186639173047328910730217803271839216");
        TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set + 5, "0.00000000000000000265542653452654526545625456254565446654545645649789871322131213156435546435", "-1.263468375787958774985473345435632415334245268476928454653443234164658776634854746584532186639173047328910730217803271839216");
    }

    if constexpr (includeSet10) {
        const auto set10 = testBase + 100;
        for (auto i = 0; i < 100; i++) {
            std::unique_ptr<HpSharkFloat<SharkFloatParams>> x = std::make_unique<HpSharkFloat<SharkFloatParams>>();
            std::unique_ptr<HpSharkFloat<SharkFloatParams>> y = std::make_unique<HpSharkFloat<SharkFloatParams>>();

            x->GenerateRandomNumber();
            y->GenerateRandomNumber();

            if (Verbose) {
                std::cout << "x.Exponent: " << x->Exponent << ", neg: " << x->IsNegative << std::endl;
                std::cout << "y.Exponent: " << y->Exponent << ", neg: " << y->IsNegative << std::endl;
            }
            const std::string x_str = x->ToString();
            const std::string y_str = y->ToString();
            TestBinOperatorTwoNumbers<SharkFloatParams, sharkOperator>(set10 + i, x_str.c_str(), y_str.c_str());
        }
    }

    return Tests.CheckAllTestsPassed();
}

template<class SharkFloatParams, Operator sharkOperator>
bool TestBinaryOperatorPerf(int testBase) {
    TestAddTwoNumbersPerf<SharkFloatParams, sharkOperator>(testBase + 1, ".1", ".1");
    return Tests.CheckAllTestsPassed();
}

// Explicitly instantiate TestAllBinaryOp
#define ExplicitlyInstantiate(SharkFloatParams) \
    template bool TestAllBinaryOp<SharkFloatParams, Operator::Add>(int testBase); \
    template bool TestAllBinaryOp<SharkFloatParams, Operator::Multiply>(int testBase); \
    template bool TestBinaryOperatorPerf<SharkFloatParams, Operator::Add>(int testBase); \
    template bool TestBinaryOperatorPerf<SharkFloatParams, Operator::Multiply>(int testBase);


ExplicitlyInstantiate(Test4x4SharkParams);
ExplicitlyInstantiate(Test4x2SharkParams);
ExplicitlyInstantiate(Test8x1SharkParams);
ExplicitlyInstantiate(Test128x64SharkParams);