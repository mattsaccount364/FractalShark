#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "HpGpu.cuh"
#include "BenchmarkTimer.h"
#include "TestTracker.h"

#include "Tests.cuh"
#include "Add.cuh"
#include "Multiply.cuh"

#include <iostream>
#include <vector>
#include <gmp.h>
#include <cstring>
#include <sstream>
#include <iomanip>
#include <cmath>
#include <algorithm>
#include <assert.h>

static TestTracker Tests;

template<Operator sharkOperator>
void DiffAgainstHost(
    int testNum,
    const mpf_t mpfHostResult,
    const HpGpu &gpuResult) {

    if (Verbose) {
        std::cout << "\nGPU result: " << std::endl;
        std::cout << gpuResult.ToString() << std::endl;
        std::cout << gpuResult.ToHexString() << std::endl;
    }

    // Convert the HpGpu results to mpf_t for comparison
    mpf_t mpfXGpuResult;
    mpf_init(mpfXGpuResult);

    HpGpuToMpf(gpuResult, mpfXGpuResult);

    // Compute the differences between host and GPU results
    mpf_t mpfDiff;
    mpf_init(mpfDiff);

    mpf_sub(mpfDiff, mpfHostResult, mpfXGpuResult);

    // Take absolute delta:
    mpf_t mpfDiffAbs;
    mpf_init(mpfDiffAbs);
    mpf_abs(mpfDiffAbs, mpfDiff);

    // Converted GPU result
    if (Verbose) {
        std::cout << "\nConverted GPU result:" << std::endl;
        std::cout << MpfToString(mpfXGpuResult, HpGpu::DefaultPrecBits) << std::endl;

        // Print the differences
        std::cout << "\nDifference between host and GPU results:" << std::endl;
        std::cout << MpfToString(mpfDiffAbs, LowPrec) << std::endl;
    }

    // Check if the host result is zero to avoid division by zero
    mp_bitcnt_t gpuPrecBits = HpGpu::DefaultPrecBits;
    mp_bitcnt_t margin = sizeof(uint32_t) * 8 * 2;
    mp_bitcnt_t totalPrecBits = (gpuPrecBits > margin) ? (gpuPrecBits - margin) : 1;
    mpf_t acceptableError;

    if (mpf_cmp_ui(mpfHostResult, 0) != 0) {
        // Host result is non-zero

        // Compute relative error
        mpf_t relativeError;
        mpf_init(relativeError);
        mpf_sub(relativeError, mpfHostResult, mpfXGpuResult);
        mpf_div(relativeError, relativeError, mpfHostResult);
        mpf_abs(relativeError, relativeError);

        // Compute machine epsilon: epsilon = 2^(-totalPrecBits)
        mpf_t epsilon;
        mpf_init2(epsilon, totalPrecBits);
        mpf_set_ui(epsilon, 1);
        mpf_div_2exp(epsilon, epsilon, totalPrecBits);

        // Compute acceptable error: acceptableError = epsilon * abs(hostResult)
        mpf_init(acceptableError);
        mpf_mul(acceptableError, epsilon, mpfHostResult);
        mpf_abs(acceptableError, acceptableError);

        // Compare absolute error with acceptable threshold
        if (mpf_cmp(relativeError, epsilon) <= 0) {
            if (Verbose) {
                std::cout << "\nThe relative error is within acceptable bounds." << std::endl;
                std::cout << "Relative error: " << MpfToString(relativeError, LowPrec) << std::endl;
            }
        } else {
            std::cerr << "\nError: The relative error exceeds acceptable bounds." << std::endl;
            std::cout << "Relative error: " << MpfToString(relativeError, LowPrec) << std::endl;
            Tests.MarkFailed(testNum, relativeError, epsilon);
        }

        // Clean up
        mpf_clear(relativeError);
        mpf_clear(epsilon);
        mpf_clear(acceptableError);
    } else {
        // Host result is zero

        // For zero host result, use an absolute error threshold
        mpf_init2(acceptableError, totalPrecBits);
        mpf_set_ui(acceptableError, 1);
        mpf_div_2exp(acceptableError, acceptableError, totalPrecBits);

        if (mpf_cmp(mpfDiffAbs, acceptableError) <= 0) {
            if (Verbose) {
                std::cout << "\nThe absolute error is within acceptable bounds." << std::endl;
            }
        } else {
            std::cerr << "\nError: The absolute error exceeds acceptable bounds." << std::endl;
            Tests.MarkFailed(testNum, mpfDiffAbs, acceptableError);
        }

        mpf_clear(acceptableError);
    }

    mpf_clear(mpfDiff);
    mpf_clear(mpfDiffAbs);
    mpf_clear(mpfXGpuResult);
}

template<Operator sharkOperator>
void TestAddTwoNumbersPerf(
    int testNum,
    const char *num1,
    const char *num2,
    const mpf_t mpfX,
    const mpf_t mpfY) {

    // Print the original input values
    if (Verbose) {
        std::cout << "Original input values:" << std::endl;
        std::cout << "num1: " << num1 << std::endl;
        std::cout << "X: " << MpfToString(mpfX, HpGpu::DefaultPrecBits) << std::endl;
        std::cout << "num2: " << num2 << std::endl;
        std::cout << "Y: " << MpfToString(mpfY, HpGpu::DefaultPrecBits) << std::endl;
    }

    std::unique_ptr<HpGpu> xNum = std::make_unique<HpGpu>();
    std::unique_ptr<HpGpu> yNum = std::make_unique<HpGpu>();
    std::unique_ptr<HpGpu> resultNum = std::make_unique<HpGpu>();
    MpfToHpGpu(mpfX, *xNum, HpGpu::DefaultPrecBits);
    MpfToHpGpu(mpfY, *yNum, HpGpu::DefaultPrecBits);
    if (Verbose) {
        std::cout << "\nConverted HpGpu representations:" << std::endl;
        std::cout << "X: " << xNum->ToString() << std::endl;
        std::cout << "Y: " << yNum->ToString() << std::endl;
    }

    // Perform the calculation on the host using MPIR
    mpf_t mpfHostResult;
    mpf_init(mpfHostResult);

    {
        BenchmarkTimer hostTimer;
        ScopedBenchmarkStopper hostStopper{ hostTimer };

        for (int i = 0; i < NUM_ITER; ++i) {
            if constexpr (sharkOperator == Operator::Add) {
                mpf_add(mpfHostResult, mpfX, mpfY);
            } else if constexpr (sharkOperator == Operator::Multiply) {
                mpf_mul(mpfHostResult, mpfX, mpfY);
            }
        }

        hostTimer.StopTimer();

        std::cout << "Host iter time: " << hostTimer.GetDeltaInMs() << " ms" << std::endl;
    }

    std::unique_ptr<HpGpu> gpuResult2 = std::make_unique<HpGpu>();

    {
        // Perform the calculation on the GPU
        HpGpu *xGpu;
        hipMalloc(&xGpu, sizeof(HpGpu));
        hipMemcpy(xGpu, xNum.get(), sizeof(HpGpu), hipMemcpyHostToDevice);

        HpGpu *yGpu;
        hipMalloc(&yGpu, sizeof(HpGpu));
        hipMemcpy(yGpu, yNum.get(), sizeof(HpGpu), hipMemcpyHostToDevice);

        HpGpu *internalGpuResult2;
        hipMalloc(&internalGpuResult2, sizeof(HpGpu));
        hipMemset(internalGpuResult2, 0, sizeof(HpGpu));

        BenchmarkTimer timer;
        ScopedBenchmarkStopper stopper{ timer };

        if constexpr (sharkOperator == Operator::Add) {
            // Allocate memory for carryOuts and cumulativeCarries
            GlobalAddBlockData *globalBlockData;
            CarryInfo *d_carryOuts;
            uint32_t *d_cumulativeCarries;
            hipMalloc(&globalBlockData, sizeof(GlobalAddBlockData));
            hipMalloc(&d_carryOuts, (NumBlocks + 1) * sizeof(CarryInfo));
            hipMalloc(&d_cumulativeCarries, (NumBlocks + 1) * sizeof(uint32_t));

            // Prepare kernel arguments
            void *kernelArgs[] = {
                (void *)&xGpu,
                (void *)&yGpu,
                (void *)&internalGpuResult2,
                (void *)&globalBlockData,
                (void *)&d_carryOuts,
                (void *)&d_cumulativeCarries
            };

            ComputeAddGpuTestLoop(kernelArgs);

            // Launch the cooperative kernel

            hipMemcpy(gpuResult2.get(), internalGpuResult2, sizeof(HpGpu), hipMemcpyDeviceToHost);

            hipFree(globalBlockData);
            hipFree(d_carryOuts);
            hipFree(d_cumulativeCarries);
        } else if constexpr (sharkOperator == Operator::Multiply) {
            // Prepare kernel arguments
            // Allocate memory for carryOuts and cumulativeCarries
            uint64_t *d_carry1;
            uint64_t *d_carry2;
            uint64_t *d_carry3;
            uint64_t *d_tempProducts;
            hipMalloc(&d_carry1, (NumBlocks + 1) * sizeof(uint64_t));
            hipMalloc(&d_carry2, (NumBlocks + 1) * sizeof(uint64_t));
            hipMalloc(&d_carry3, (NumBlocks + 1) * sizeof(uint64_t));
            hipMalloc(&d_tempProducts, 32 * HpGpu::NumUint32 * sizeof(uint64_t));

            void *kernelArgs[] = {
                (void *)&xGpu,
                (void *)&yGpu,
                (void *)&internalGpuResult2,
                (void *)&d_carry1,
                (void *)&d_carry2,
                (void *)&d_carry3,
                (void *)&d_tempProducts
            };

            ComputeMultiplyGpuTestLoop(kernelArgs);

            hipFree(d_carry1);
            hipFree(d_carry2);
            hipFree(d_carry3);
            hipFree(d_tempProducts);

            hipMemcpy(gpuResult2.get(), internalGpuResult2, sizeof(HpGpu), hipMemcpyDeviceToHost);
        }

        timer.StopTimer();
        Tests.AddTime(testNum, timer.GetDeltaInMs());

        std::cout << "GPU iter time: " << timer.GetDeltaInMs() << " ms" << std::endl;

        hipFree(internalGpuResult2);
        hipFree(xGpu);
    }

    DiffAgainstHost<sharkOperator>(testNum, mpfHostResult, *gpuResult2);

    // Clean up MPIR variables
    mpf_clear(mpfHostResult);
}

template<Operator sharkOperator>
void TestAddTwoNumbersPerf(
    int testNum,
    const char *num1,
    const char *num2) {

    mpf_set_default_prec(HpGpu::DefaultMpirBits);  // Set precision for MPIR floating point

    mpf_t mpfX;
    mpf_t mpfY;
    mpf_init(mpfX);
    mpf_init(mpfY);

    auto res = mpf_set_str(mpfX, num1, 10);
    if (res == -1) {
        std::cout << "Error setting mpfX" << std::endl;
    }

    res = mpf_set_str(mpfY, num2, 10);
    if (res == -1) {
        std::cout << "Error setting mpfY" << std::endl;
    }

    TestAddTwoNumbersPerf<sharkOperator>(testNum, num1, num2, mpfX, mpfY);

    mpf_clear(mpfX);
    mpf_clear(mpfY);
}

template<Operator sharkOperator>
void TestBinOperatorTwoNumbers(
    int testNum,
    const HpGpu &xNum,
    const HpGpu &yNum,
    const mpf_t &mpfX,
    const mpf_t &mpfY) {

    if (Verbose) {
        std::cout << "\nConverted HpGpu representations:" << std::endl;
        std::cout << "X: " << xNum.ToString() << std::endl;
        std::cout << "X hex: " << xNum.ToHexString() << std::endl;
        std::cout << "Y: " << yNum.ToString() << std::endl;
        std::cout << "Y hex: " << yNum.ToHexString() << std::endl;
    }

    // Perform the calculation on the GPU
    HpGpu *xGpu;
    HpGpu *yGpu;

    hipMalloc(&xGpu, sizeof(HpGpu));
    hipMalloc(&yGpu, sizeof(HpGpu));
    hipMemcpy(xGpu, &xNum, sizeof(HpGpu), hipMemcpyHostToDevice);
    hipMemcpy(yGpu, &yNum, sizeof(HpGpu), hipMemcpyHostToDevice);

    {
        // Perform the calculation on the host using MPIR
        HpGpu gpuResult{};
        mpf_t mpfHostResult;
        mpf_init(mpfHostResult);

        if constexpr (sharkOperator == Operator::Add) {
            mpf_add(mpfHostResult, mpfX, mpfY);
        } else if constexpr (sharkOperator == Operator::Multiply) {
            mpf_mul(mpfHostResult, mpfX, mpfY);
        }

        // Print host result
        if (Verbose) {
            std::cout << "\nHost result:" << std::endl;
            std::cout << "Host result: " << MpfToString(mpfHostResult, HpGpu::DefaultPrecBits) << std::endl;
            std::cout << "Host hex: " << MpfToHexString(mpfHostResult) << std::endl;
        }

        HpGpu *internalGpuResult;
        hipMalloc(&internalGpuResult, sizeof(HpGpu));

        BenchmarkTimer timer;
        ScopedBenchmarkStopper stopper{ timer };

        if constexpr (sharkOperator == Operator::Add) {
            // Allocate memory for carryOuts and cumulativeCarries
            GlobalAddBlockData *globalBlockData;
            CarryInfo *d_carryOuts;
            uint32_t *d_cumulativeCarries;
            hipMalloc(&globalBlockData, sizeof(GlobalAddBlockData));
            hipMalloc(&d_carryOuts, (NumBlocks + 1) * sizeof(CarryInfo));
            hipMalloc(&d_cumulativeCarries, (NumBlocks + 1) * sizeof(uint32_t));

            // Prepare kernel arguments
            void *kernelArgs[] = {
                (void *)&xGpu,
                (void *)&yGpu,
                (void *)&internalGpuResult,
                (void *)&globalBlockData,
                (void *)&d_carryOuts,
                (void *)&d_cumulativeCarries
            };

            ComputeAddGpu(kernelArgs);

            hipFree(globalBlockData);
            hipFree(d_carryOuts);
            hipFree(d_cumulativeCarries);
        } else if constexpr (sharkOperator == Operator::Multiply) {
            // Prepare kernel arguments
            // Allocate memory for carryOuts and cumulativeCarries
            uint64_t *d_carry1;
            uint64_t *d_carry2;
            uint64_t *d_carry3;
            uint64_t *d_tempProducts;
            hipMalloc(&d_carry1, (NumBlocks + 1) * sizeof(uint64_t));
            hipMalloc(&d_carry2, (NumBlocks + 1) * sizeof(uint64_t));
            hipMalloc(&d_carry3, (NumBlocks + 1) * sizeof(uint64_t));
            hipMalloc(&d_tempProducts, 32 * HpGpu::NumUint32 * sizeof(uint64_t));

            void *kernelArgs[] = {
                (void *)&xGpu,
                (void *)&yGpu,
                (void *)&internalGpuResult,
                (void *)&d_carry1,
                (void *)&d_carry2,
                (void *)&d_carry3,
                (void *)&d_tempProducts
            };

            ComputeMultiplyGpu(kernelArgs);

            hipFree(d_carry1);
            hipFree(d_carry2);
            hipFree(d_carry3);
            hipFree(d_tempProducts);
        }

        hipMemcpy(&gpuResult, internalGpuResult, sizeof(HpGpu), hipMemcpyDeviceToHost);

        timer.StopTimer();
        Tests.AddTime(testNum, timer.GetDeltaInMs());

        if (Verbose) {
            std::cout << "GPU single time: " << timer.GetDeltaInMs() << " ms" << std::endl;
        }

        hipFree(internalGpuResult);

        DiffAgainstHost<sharkOperator>(testNum, mpfHostResult, gpuResult);

        // Clean up MPIR variables
        mpf_clear(mpfHostResult);
    }

    hipFree(xGpu);
    hipFree(yGpu);
}

template<Operator sharkOperator>
void TestBinOperatorTwoNumbers(
    int testNum,
    const char *num1,
    const char *num2,
    const mpf_t &mpfX,
    const mpf_t &mpfY) {

    // Print the original input values
    if (Verbose) {
        std::cout << "Original input strings:" << std::endl;
        std::cout << "num1: " << num1 << std::endl;
        std::cout << "num2: " << num2 << std::endl;
        std::cout << "MpfX: " << MpfToString(mpfX, HpGpu::DefaultPrecBits) << std::endl;
        std::cout << "MpfY: " << MpfToString(mpfY, HpGpu::DefaultPrecBits) << std::endl;
    }

    // Convert the input values to HpGpu representations
    std::unique_ptr<HpGpu> xNum = std::make_unique<HpGpu>();
    std::unique_ptr<HpGpu> yNum = std::make_unique<HpGpu>();
    MpfToHpGpu(mpfX, *xNum, HpGpu::DefaultPrecBits);
    MpfToHpGpu(mpfY, *yNum, HpGpu::DefaultPrecBits);

    TestBinOperatorTwoNumbers<sharkOperator>(testNum, *xNum, *yNum, mpfX, mpfY);
}

template<Operator sharkOperator>
void TestBinOperatorTwoNumbers(
    int testNum,
    const char *num1,
    const char *num2) {

    std::cout << std::endl;
    std::cout << std::endl;
    std::cout << "Test " << testNum << std::endl;

    mpf_set_default_prec(HpGpu::DefaultMpirBits);  // Set precision for MPIR floating point

    mpf_t mpfX, mpfY;
    mpf_init(mpfX);
    mpf_init(mpfY);

    auto res = mpf_set_str(mpfX, num1, 10);
    if (res == -1) {
        std::cout << "Error setting mpfX" << std::endl;
    }

    res = mpf_set_str(mpfY, num2, 10);
    if (res == -1) {
        std::cout << "Error setting mpfY" << std::endl;
    }

    TestBinOperatorTwoNumbers<sharkOperator>(testNum, num1, num2, mpfX, mpfY);

    mpf_clear(mpfX);
    mpf_clear(mpfY);
}

template<Operator sharkOperator>
void TestAddSpecialNumbers(int testNum, std::vector<uint32_t> &digits1, std::vector<uint32_t> &digits2) {
    mpf_t x, y;
    mpf_init(x);
    mpf_init(y);

    std::cout << std::endl;
    std::cout << std::endl;
    std::cout << "Test " << testNum << std::endl;

    auto strLargeX = Uint32ToMpf(digits1.data(), HpGpu::NumUint32 / 2, x);
    auto strLargeY = Uint32ToMpf(digits2.data(), HpGpu::NumUint32 / 2, y);
    TestBinOperatorTwoNumbers<sharkOperator>(testNum, strLargeX.c_str(), strLargeY.c_str(), x, y);

    mpf_clear(x);
    mpf_clear(y);
}

template<Operator sharkOperator>
void TestAddSpecialNumbers(
    int testNum,
    const HpGpu &xNum,
    const HpGpu &yNum) {

    mpf_t mpfX;
    mpf_t mpfY;
    mpf_init(mpfX);
    mpf_init(mpfY);
    HpGpuToMpf(xNum, mpfX);
    HpGpuToMpf(yNum, mpfY);

    TestBinOperatorTwoNumbers<sharkOperator>(testNum, xNum, yNum, mpfX, mpfY);
}

template<Operator sharkOperator>
void TestAddSpecialNumbers1(int testNum) {
    std::vector<uint32_t> testData;
    for (size_t i = 0; i < HpGpu::NumUint32; ++i) {
        testData.push_back(0);
    }

    assert(testData.size() == HpGpu::NumUint32);
    testData[testData.size() - 1] = 0x80000000;

    TestAddSpecialNumbers<sharkOperator>(testNum, testData, testData);
}

template<Operator sharkOperator>
void TestAddSpecialNumbers2(int testNum) {
    std::vector<uint32_t> testData;
    for (size_t i = 0; i < HpGpu::NumUint32; ++i) {
        testData.push_back(0);
    }

    assert(testData.size() == HpGpu::NumUint32);
    testData[testData.size() - 1] = 0xC0000000;

    TestAddSpecialNumbers<sharkOperator>(testNum, testData, testData);
}

template<Operator sharkOperator>
void TestAddSpecialNumbers3(int testNum) {
    std::vector<uint32_t> testData;
    for (size_t i = 0; i < HpGpu::NumUint32; ++i) {
        testData.push_back(0);
    }

    assert(testData.size() == HpGpu::NumUint32);
    testData[testData.size() - 1] = 0xFFFFFFFF;

    TestAddSpecialNumbers<sharkOperator>(testNum, testData, testData);
}

template<Operator sharkOperator>
void TestAddSpecialNumbersHelper(
    int testNum,
    bool isNegative1,
    std::vector<uint32_t> testData1,
    bool isNegative2,
    std::vector<uint32_t> testData2) {

    std::cout << std::endl;
    std::cout << std::endl;
    std::cout << "Test " << testNum << std::endl;

    std::vector<uint32_t> testData1Copy;
    testData1Copy = testData1;
    testData1Copy.resize(HpGpu::NumUint32);

    std::vector<uint32_t> testData2Copy;
    testData2Copy = testData2;
    testData2Copy.resize(HpGpu::NumUint32);

    std::unique_ptr<HpGpu> xNum{ std::make_unique<HpGpu>(testData1Copy.data(), 0, isNegative1) };
    std::unique_ptr<HpGpu> yNum{ std::make_unique<HpGpu>(testData2Copy.data(), 0, isNegative2) };

    TestAddSpecialNumbers<sharkOperator>(testNum, *xNum, *yNum);
}

template<Operator sharkOperator>
void TestAddSpecialNumbers4(int testNum) {
    TestAddSpecialNumbersHelper<sharkOperator>(
        testNum,
        true,
        std::vector<uint32_t>{ 0xF26D37FC, 0xA96025CE, 0xB03FC716, 0x1DF7182B, 0xCCBD69BD, 0x40C0F80C, 0xFAA0222E, 0xD1FDA456 },
        true,
        std::vector<uint32_t>{ 0x8BBCDF3, 0x4C3E7ACB, 0x6691A71D, 0xDFE03842, 0x3FADCA11, 0x4058BC9E, 0xF30FD7DE, 0xAA6CA582 });
}

template<Operator sharkOperator>
void TestAddSpecialNumbers5(int testNum) {
    TestAddSpecialNumbersHelper<sharkOperator>(
        testNum,
        false,
        std::vector<uint32_t>{ 0, 0, 0, 0, 0, 0, 0, 0xFFFFFFFF },
        true,
        std::vector<uint32_t>{ 0, 0, 0, 0, 0, 0, 0, 0xFFFFFFFF });
}

template<Operator sharkOperator>
void TestAddSpecialNumbers6(int testNum) {
    TestAddSpecialNumbersHelper<sharkOperator>(
        testNum,
        true,
        std::vector<uint32_t>{ 0xFFFFFFFF, 0xFFFFFFFF },
        true,
        std::vector<uint32_t>{ 0xFFFFFFFF, 0xFFFFFFFF });
}

template<Operator sharkOperator>
void TestAddSpecialNumbers7(int testNum) {
    TestAddSpecialNumbersHelper<sharkOperator>(
        testNum,
        true,
        std::vector<uint32_t>{ 0, 0xFFFFFFFF, 0xFFFFFFFF },
        true,
        std::vector<uint32_t>{ 0, 0xFFFFFFFF, 0xFFFFFFFF });
}

template<Operator sharkOperator>
void TestAddSpecialNumbers8(int testNum) {

    TestAddSpecialNumbersHelper<sharkOperator>(
        testNum,
        true,
        std::vector<uint32_t>{ 0, 0, 0xFFFFFFFF, 0xFFFFFFFF },
        true,
        std::vector<uint32_t>{ 0, 0, 0xFFFFFFFF, 0xFFFFFFFF });
}

template<Operator sharkOperator>
void TestAddSpecialNumbers9(int testNum) {

    TestAddSpecialNumbersHelper<sharkOperator>(
        testNum,
        true,
        std::vector<uint32_t>{ 0, 0, 0, 0xFFFFFFFF, 0xFFFFFFFF },
        true,
        std::vector<uint32_t>{ 0, 0, 0, 0xFFFFFFFF, 0xFFFFFFFF });
}


template<Operator sharkOperator>
bool TestAllBinaryOp(int testBase) {
    constexpr bool includeSet1 = true;
    constexpr bool includeSet2 = true;
    constexpr bool includeSet3 = true;
    constexpr bool includeSet4 = true;
    constexpr bool includeSet5 = true;
    constexpr bool includeSet6 = true;
    constexpr bool includeSet10 = true;

    // 200s is multiply
    // 400s is add
    
    if constexpr (includeSet1) {
        const auto set = testBase + 10;
        TestBinOperatorTwoNumbers<sharkOperator>(set + 1, "1", "2");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 2, "4294967295", "1");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 3, "4294967296", "1");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 4, "4294967295", "4294967296");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 5, "4294967296", "-1");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 6, "18446744073709551615", "1");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 7, "0", "0.1");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 8, "0.1", "0");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 9, "0", "0");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 10, "0.1", "0.1");
    }

    if constexpr (includeSet2) {
        const auto set = testBase + 30;
        TestAddSpecialNumbers1<sharkOperator>(set + 1);
        TestAddSpecialNumbers2<sharkOperator>(set + 2);
        TestAddSpecialNumbers3<sharkOperator>(set + 3);
        TestAddSpecialNumbers4<sharkOperator>(set + 4);
        TestAddSpecialNumbers5<sharkOperator>(set + 5);
        TestAddSpecialNumbers6<sharkOperator>(set + 6);
        TestAddSpecialNumbers7<sharkOperator>(set + 7);
        TestAddSpecialNumbers8<sharkOperator>(set + 8);
        TestAddSpecialNumbers9<sharkOperator>(set + 9);
    }

    if constexpr (includeSet3) {
        const auto set = testBase + 40;
        TestBinOperatorTwoNumbers<sharkOperator>(set + 1, "2", "0.1");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 2, "0.2", "0.1");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 3, "0.5", "1.2");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 4, "0.6", "1.3");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 5, "0.7", "1.4");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 6, "0.1", "1.99999999999999999999999999999");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 7, "0.123124561464451654461", "1.2395123123127298375982735");
    }

    if constexpr (includeSet4) {
        const auto set = testBase + 50;
        TestBinOperatorTwoNumbers<sharkOperator>(set + 1, "-0.5", "1.2");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 2, "-0.6", "1.3");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 3, "-0.7", "1.4");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 4, "-0.1", "1.99999999999999999999999999999");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 5, "-0.123124561464451654461", "1.2395123123127298375982735");
    }

    if constexpr (includeSet5) {
        const auto set = testBase + 60;
        TestBinOperatorTwoNumbers<sharkOperator>(set + 1, "-0.5", "-1.2");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 2, "-0.6", "-1.3");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 3, "-0.7", "-1.4");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 4, "-0.1", "-1.99999999999999999999999999999");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 5, "-0.123124561464451654461", "-1.2395123123127298375982735");
    }

    if constexpr (includeSet6) {
        const auto set = testBase + 70;
        TestBinOperatorTwoNumbers<sharkOperator>(set + 1, "0.5265542653452654526545625456254565446654545645649789871322131213156435546435", "-1.263468375787958774985473345435632415334245268476928454653443234164658776634854746584532186639173047328910730217803271839216");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 2, "0.2999999999965542653452654526545625456254565446654545645649789871322131213156435546435", "-1.263468375787958774985473345435632415334245268476928454653443234164658776634854746584532186639173047328910730217803271839216");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 3, "0.1265542653452654526545625456254565446654545645649789871322131213156435546435", "-1.2634683757879587749854733454356324153342452684769284546534432341646587766348547465845321866391730473289107302178039999999999999271839216");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 4, "0.0265542653452654526545625456254565446654545645649789871322131213156435546435", "-1.263468375787958774985473345435632415334245268476928454653443234164658776634854746584532186639173047328910730217803271839216");
        TestBinOperatorTwoNumbers<sharkOperator>(set + 5, "0.00000000000000000265542653452654526545625456254565446654545645649789871322131213156435546435", "-1.263468375787958774985473345435632415334245268476928454653443234164658776634854746584532186639173047328910730217803271839216");
    }

    if constexpr (includeSet10) {
        const auto set10 = testBase + 100;
        for (auto i = 0; i < 100; i++) {
            std::unique_ptr<HpGpu> x = std::make_unique<HpGpu>();
            std::unique_ptr<HpGpu> y = std::make_unique<HpGpu>();

            x->GenerateRandomNumber();
            y->GenerateRandomNumber();

            if (Verbose) {
                std::cout << "x.Exponent: " << x->Exponent << ", neg: " << x->IsNegative << std::endl;
                std::cout << "y.Exponent: " << y->Exponent << ", neg: " << y->IsNegative << std::endl;
            }
            const std::string x_str = x->ToString();
            const std::string y_str = y->ToString();
            TestBinOperatorTwoNumbers<sharkOperator>(set10 + i, x_str.c_str(), y_str.c_str());
        }
    }

    return Tests.CheckAllTestsPassed();
}

template<Operator sharkOperator>
bool TestBinaryOperatorPerf(int testBase) {
    TestAddTwoNumbersPerf<sharkOperator>(testBase + 1, ".1", ".1");
    return Tests.CheckAllTestsPassed();
}

// Explicitly instantiate TestBinaryOperatorPerf
template bool TestBinaryOperatorPerf<Operator::Add>(int testBase);
template bool TestBinaryOperatorPerf<Operator::Multiply>(int testBase);

// Explicitly instantiate TestAllBinaryOp
template bool TestAllBinaryOp<Operator::Add>(int testBase);
template bool TestAllBinaryOp<Operator::Multiply>(int testBase);