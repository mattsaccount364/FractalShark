#include "hip/hip_runtime.h"
// TODO: 2x32 perturb is busted, do git diff
// Re-run  profile on current default view
#include <stdio.h>
#include <iostream>

#include "render_gpu.h"
#include "dbldbl.cuh"
#include "dblflt.cuh"
#include "../QuadDouble/gqd_basic.cuh"
#include "../QuadFloat/gqf_basic.cuh"

#include "GPU_BLAS.h"

#include "HDRFloatComplex.h"
#include "BLA.h"
#include "HDRFloat.h"

#include "GPU_LAReference.h"

#include "GPU_LAInfoDeep.h"
#include "LAReference.h"

#include <type_traits>
//#include <cuda/pipeline>
//#include <cuda_pipeline.h>

#ifdef __HIPCC__
__device__ __constant__ double twoPowExpDataDbl[2048];
__device__ __constant__ float twoPowExpDataFlt[256];

#ifdef __CUDA_ARCH__
__device__ void InitStatics()
{
    //LN2 = ::log(2);
    //LN2_REC = 1.0 / LN2;

    twoPowExpDbl = twoPowExpDataDbl;
    twoPowExpFlt = twoPowExpDataFlt;

    static constexpr int MaxDoubleExponent = 1023;
    static constexpr int MinDoubleExponent = -1022;

    static constexpr int MaxFloatExponent = 127;
    static constexpr int MinFloatExponent = -126;

    //twoPowExp.resize(MaxDoubleExponent - MinDoubleExponent + 1);
    for (int i = MinDoubleExponent; i <= MaxDoubleExponent; i++) {
        double d = scalbn(1.0, i);
        int index = i - MinDoubleExponent;
        twoPowExpDbl[index] = d;
    }

    for (int i = MinFloatExponent; i <= MaxFloatExponent; i++) {
        float f = scalbn(1.0, i);
        int index = i - MinFloatExponent;
        twoPowExpFlt[index] = f;
    }
}
#endif
#endif

////////////////////////////////////////////////////////////////////////////////////////
// Bilinear approximation
////////////////////////////////////////////////////////////////////////////////////////

template<class T>
CUDA_CRAP constexpr BLA<T>::BLA(T r2, T RealA, T ImagA, T RealB, T ImagB, int l)
    : Ax(RealA),
      Ay(ImagA),
      Bx(RealB),
      By(ImagB),
      r2(r2),
      l(l) {
    //HdrReduce(this->Ax);
    //HdrReduce(this->Ay);
    //HdrReduce(this->Bx);
    //HdrReduce(this->By);
    //HdrReduce(this->r2);
}

template<class T>
CUDA_CRAP void BLA<T>::getValue(
    T &RealDeltaSubN,
    T &ImagDeltaSubN,
    const T &RealDeltaSub0,
    const T &ImagDeltaSub0
) const {

    //T zxn = Ax * zx - Ay * zy + Bx * cx - By * cy;
    //T zyn = Ax * zy + Ay * zx + Bx * cy + By * cx;
    T NewRealValue = Ax * RealDeltaSubN - Ay * ImagDeltaSubN + Bx * RealDeltaSub0 - By * ImagDeltaSub0;
    T NewImagValue = Ax * ImagDeltaSubN + Ay * RealDeltaSubN + Bx * ImagDeltaSub0 + By * RealDeltaSub0;
    RealDeltaSubN = NewRealValue;
    //HdrReduce(RealDeltaSubN);

    ImagDeltaSubN = NewImagValue;
    //HdrReduce(ImagDeltaSubN);
}

template<class T>
CUDA_CRAP T BLA<T>::hypotA() const {
    auto ret = HdrSqrt<T>(Ax * Ax + Ay * Ay);
    HdrReduce(ret);
    return ret;
}

template<class T>
CUDA_CRAP T BLA<T>::hypotB() const {
    auto ret = HdrSqrt<T>(Bx * Bx + By * By);
    HdrReduce(ret);
    return ret;
}

template<class T>
CUDA_CRAP BLA<T> BLA<T>::getGenericStep(
    T r2,
    T RealA,
    T ImagA,
    T RealB,
    T ImagB,
    int l
) {
    return BLA(r2, RealA, ImagA, RealB, ImagB, l);
}

// A = y.A * x.A
template<class T>
CUDA_CRAP void BLA<T>::getNewA(const BLA &x, const BLA &y, T &RealValue, T &ImagValue) {
    RealValue = y.Ax * x.Ax - y.Ay * x.Ay;
    HdrReduce(RealValue);

    ImagValue = y.Ax * x.Ay + y.Ay * x.Ax;
    HdrReduce(ImagValue);
}

// B = y.A * x.B + y.B
template<class T>
CUDA_CRAP void BLA<T>::getNewB(const BLA &x, const BLA &y, T& RealValue, T& ImagValue) {
    T xBx = x.Bx;
    T xBy = x.By;
    RealValue = y.Ax * xBx - y.Ay * xBy + y.Bx;
    HdrReduce(RealValue);

    ImagValue = y.Ax * xBy + y.Ay * xBx + y.By;
    HdrReduce(ImagValue);
}

template<class T>
CUDA_CRAP int BLA<T>::getL() const {
    return l;
}

template<class T>
CUDA_CRAP T BLA<T>::getR2() const {
    return r2;
}

template<class T>
CUDA_CRAP const T *BLA<T>::getR2Addr() const {
    return &r2;
}

template class BLA<float>;
template class BLA<double>;
template class BLA<HDRFloat<float>>;
template class BLA<HDRFloat<double>>;

////////////////////////////////////////////////////////////////////////////////////////
// Bilinear approximation.  GPU copy.
////////////////////////////////////////////////////////////////////////////////////////

template<class T, class GPUBLA_TYPE, int32_t LM2>
GPU_BLAS<T, GPUBLA_TYPE, LM2>::GPU_BLAS(const std::vector<std::vector<GPUBLA_TYPE>>& B)
    : m_B(nullptr),
      m_Err(),
      m_Owned(true) {

    GPUBLA_TYPE** tempB;
    m_Err = hipMallocManaged(&tempB, m_NumLevels * sizeof(GPUBLA_TYPE*), hipMemAttachGlobal);
    if (m_Err != hipSuccess) {
        return;
    }

    m_B = tempB;
    hipMemset(m_B, 0, m_NumLevels * sizeof(GPUBLA_TYPE*));

    size_t total = 0;

    for (size_t i = 0; i < B.size(); i++) {
        total += sizeof(GPUBLA_TYPE) * B[i].size();
    }

    m_Err = hipMalloc(&m_BMem, total);
    if (m_Err != hipSuccess) {
        return;
    }

    size_t curTotal = 0;
    for (size_t i = 0; i < B.size(); i++) {
        m_B[i] = &m_BMem[curTotal];
        curTotal += B[i].size();

        hipMemcpy(m_B[i],
            B[i].data(),
            sizeof(GPUBLA_TYPE) * B[i].size(),
            hipMemcpyDefault);
    }
}

template<class T, class GPUBLA_TYPE, int32_t LM2>
GPU_BLAS<T, GPUBLA_TYPE, LM2>::~GPU_BLAS() {
    if (m_Owned) {
        if (m_BMem != nullptr) {
            hipFree(m_BMem);
            m_BMem = nullptr;
        }

        if (m_B != nullptr) {
            hipFree(m_B);
            m_B = nullptr;
        }
    }
}

template<class T, class GPUBLA_TYPE, int32_t LM2>
GPU_BLAS<T, GPUBLA_TYPE, LM2>::GPU_BLAS(const GPU_BLAS& other) : m_Owned(false) {
    if (this == &other) {
        return;
    }

    m_BMem = other.m_BMem;
    m_B = other.m_B;
    //for (size_t i = 0; i < m_NumLevels; i++) {
    //    m_B[i] = other.m_B[i];
    //}
}

template<class T, class GPUBLA_TYPE, int32_t LM2>
uint32_t GPU_BLAS<T, GPUBLA_TYPE, LM2>::CheckValid() const {
    return m_Err;
}

#ifdef __CUDA_ARCH__
template<class T, class GPUBLA_TYPE, int32_t LM2>
CUDA_CRAP const GPUBLA_TYPE* GPU_BLAS<T, GPUBLA_TYPE, LM2>::LookupBackwards(
    const GPUBLA_TYPE* __restrict__ *altB,
    //const GPUBLA_TYPE* __restrict__ nullBla,
    /*T* curBR2,*/
    size_t m,
    T z2) const {

    const int32_t k = (int32_t)m - 1;

    //// Option A:
    //const GPUBLA_TYPE* __restrict__ tempB = nullptr;
    //const float v = (float)(k & -k);
    //const uint32_t bits = *reinterpret_cast<const uint32_t * __restrict__>(&v);
    //const uint32_t zeros = (bits >> 23) - 0x7f;
    //uint32_t ix = k >> zeros;

    //// Option B: pretty similar results:
    //const GPUBLA_TYPE* __restrict__ tempB = nullptr;
    //uint32_t zeros;
    //uint32_t ix;
    //int r;           // result goes here
    //static constexpr int MultiplyDeBruijnBitPosition[32] =
    //{
    //  0, 1, 28, 2, 29, 14, 24, 3, 30, 22, 20, 15, 25, 17, 4, 8,
    //  31, 27, 13, 23, 21, 19, 16, 7, 26, 12, 18, 6, 11, 5, 10, 9
    //};
    //zeros = MultiplyDeBruijnBitPosition[((uint32_t)((k & -k) * 0x077CB531U)) >> 27];
    //ix = k >> zeros;

    //// Option C:
    // Get position of low-order 1 bit, subtract 1.
    //const GPUBLA_TYPE* __restrict__ tempB = nullptr;
    //const uint32_t zeros = __ffs(k) - 1;
    //uint32_t ix = k >> zeros;

    // Option D:
    // Reverse bit order, count high order zeros.
    const GPUBLA_TYPE* __restrict__ tempB = nullptr;
    const uint32_t zeros = __clz(__brev(k));
    uint32_t ix = k >> zeros;

    const int32_t startLevel =
        (LM2 == 0) ? 0 : (((zeros < LM2) ? zeros : LM2));

    //for (int32_t level = startLevel; level >= m_FirstLevel; --level) {
    //    __pipeline_memcpy_async(
    //        &curBR2[level],
    //        altB[level][ix].getR2Addr(),
    //        sizeof(T),
    //        0);
    //    ix = ix << 1;
    //}
    //__pipeline_commit();

    //ix = ixcopy;

    //__pipeline_wait_prior(0);

    for (int32_t level = startLevel; level >= m_FirstLevel; --level) {
        if (z2 < (tempB = &altB[level][ix])->getR2()) {
        //if (z2 < curBR2[level]) {
            return tempB;
        }
        ix = ix << 1;
    }
    return nullptr;
    //return nullBla;

    //GPUBLA_TYPE* __restrict__ tempB = nullptr;
    //uint32_t zeros;
    //uint32_t ix;
    //float v = (float)(k & -k);
    //uint32_t bits = *reinterpret_cast<const uint32_t * __restrict__>(&v);
    //zeros = (bits >> 23) - 0x7f;
    //ix = k >> zeros;
    //int32_t startLevel = ((zeros <= m_LM2) ? zeros : m_LM2);
    //ix = ix << (startLevel - m_FirstLevel);
    //for (int32_t level = m_FirstLevel; level <= startLevel; level++) {
    //    tempB = (z2 < m_B[level][ix].getR2()) ? &m_B[level][ix] : tempB;
    //    ix = ix >> 1;
    //}
    //return tempB;
}
#endif

#define LargeSwitch \
        switch (blas->m_LM2) {                                         \
        case  0: result = Run.template operator()<0> (); break;     \
        case  1: result = Run.template operator()<1> (); break;     \
        case  2: result = Run.template operator()<2> (); break;     \
        case  3: result = Run.template operator()<3> (); break;     \
        case  4: result = Run.template operator()<4> (); break;     \
        case  5: result = Run.template operator()<5> (); break;     \
        case  6: result = Run.template operator()<6> (); break;     \
        case  7: result = Run.template operator()<7> (); break;     \
        case  8: result = Run.template operator()<8> (); break;     \
        case  9: result = Run.template operator()<9> (); break;     \
        case 10: result = Run.template operator()<10> (); break;    \
        case 11: result = Run.template operator()<11> (); break;    \
        case 12: result = Run.template operator()<12> (); break;    \
        case 13: result = Run.template operator()<13> (); break;    \
        case 14: result = Run.template operator()<14> (); break;    \
        case 15: result = Run.template operator()<15> (); break;    \
        case 16: result = Run.template operator()<16> (); break;    \
        case 17: result = Run.template operator()<17> (); break;    \
        case 18: result = Run.template operator()<18> (); break;    \
        case 19: result = Run.template operator()<19> (); break;    \
        case 20: result = Run.template operator()<20> (); break;    \
        case 21: result = Run.template operator()<21> (); break;    \
        case 22: result = Run.template operator()<22> (); break;    \
        case 23: result = Run.template operator()<23> (); break;    \
        case 24: result = Run.template operator()<24> (); break;    \
        case 25: result = Run.template operator()<25> (); break;    \
        case 26: result = Run.template operator()<26> (); break;    \
        case 27: result = Run.template operator()<27> (); break;    \
        case 28: result = Run.template operator()<28> (); break;    \
        case 29: result = Run.template operator()<29> (); break;    \
        case 30: result = Run.template operator()<30> (); break;    \
        case 31: result = Run.template operator()<31> (); break;    \
        default: break;                                                \
        }

////////////////////////////////////////////////////////////////////////////////////////
// Perturbation results
////////////////////////////////////////////////////////////////////////////////////////

static_assert(sizeof(MattReferenceSingleIter<float>) == 16, "Float");
static_assert(sizeof(MattReferenceSingleIter<double>) == 24, "Double");
static_assert(sizeof(MattReferenceSingleIter<dblflt>) == 24, "Dblflt");

//char(*__kaboom1)[sizeof(MattReferenceSingleIter<float>)] = 1;
//char(*__kaboom2)[sizeof(MattReferenceSingleIter<double>)] = 1;
//char(*__kaboom3)[sizeof(MattReferenceSingleIter<dblflt>)] = 1;

template<typename Type>
struct MattPerturbSingleResults {
    MattReferenceSingleIter<Type>* __restrict__ iters;
    size_t size;
    bool own;
    hipError_t err;
    size_t PeriodMaybeZero;

    MattPerturbSingleResults(
        size_t sz,
        size_t PeriodMaybeZero,
        MattReferenceSingleIter<Type> *in_iters)
        : size(sz),
        PeriodMaybeZero(PeriodMaybeZero),
        iters(nullptr),
        own(true),
        err(hipSuccess) {

        static_assert(sizeof(MattDblflt) == sizeof(dblflt), "No");

        MattReferenceSingleIter<Type>* tempIters;
        err = hipMalloc(&tempIters, size * sizeof(MattReferenceSingleIter<Type>));
        if (err != hipSuccess) {
            size = 0;
            return;
        }

        iters = tempIters;
        hipMemcpy(iters, in_iters, size * sizeof(MattReferenceSingleIter<Type>), hipMemcpyDefault);

        //err = hipMemAdvise(iters,
        //    size * sizeof(MattReferenceSingleIter<Type>),
        //    hipMemAdviseSetReadMostly,
        //    0);
        //if (err != hipSuccess) {
        //    size = 0;
        //    return;
        //}
    }

    // funny semantics, copy doesn't own the pointers.
    MattPerturbSingleResults(const MattPerturbSingleResults& other) {
        if (this == &other) {
            return;
        }

        iters = other.iters;
        size = other.size;
        PeriodMaybeZero = other.PeriodMaybeZero;
        own = false;
    }

    uint32_t CheckValid() const {
        return err;
    }

    __device__ HDRFloatComplex<float> GetComplex(size_t index) const {
        
        //auto temp = *(float4*)(&iters[index].x);
        //const auto &re = *(HDRFloat<float>*)&temp.x;
        //const auto &im = *(HDRFloat<float>*)&temp.z;
        //static_assert(sizeof(iters[index].x) == 8, "Misaligned");
        //static_assert(sizeof(iters[index].y) == 8, "Misaligned");
        //static_assert(sizeof(HDRFloat<float>) == 8, "Misaligned");
        //static_assert(sizeof(float4) == 16, "Misaligned float4");
        //return HDRFloatComplex<float>(re, im);
        return HDRFloatComplex<float>(iters[index].x, iters[index].y);
    }

    MattPerturbSingleResults(MattPerturbSingleResults&& other) = delete;
    MattPerturbSingleResults &operator=(const MattPerturbSingleResults& other) = delete;
    MattPerturbSingleResults &operator=(MattPerturbSingleResults&& other) = delete;

    ~MattPerturbSingleResults() {
        if (own) {
            if (iters != nullptr) {
                hipFree(iters);
            }
        }
    }
};


//////////////////////////////////////////////////////////////////////////////
// GPU_LAReference
//////////////////////////////////////////////////////////////////////////////

__host__
GPU_LAReference::GPU_LAReference(const LAReference& other) :
    UseAT{other.UseAT},
    AT{other.AT},
    LAStageCount{other.LAStageCount},
    isValid{other.isValid},
    m_Err{},
    m_Owned(true),
    LAs{},
    LAStages{} {

    GPU_LAInfoDeep<float>* tempLAs;
    LAStageInfo* tempLAStages;

    m_Err = hipMallocManaged(&tempLAs, other.LAs.size() * sizeof(GPU_LAInfoDeep<float>), hipMemAttachGlobal);
    if (m_Err != hipSuccess) {
        return;
    }

    LAs = tempLAs;

    m_Err = hipMallocManaged(&tempLAStages, other.LAStages.size() * sizeof(LAStageInfo), hipMemAttachGlobal);
    if (m_Err != hipSuccess) {
        return;
    }

    LAStages = tempLAStages;

    for (size_t i = 0; i < other.LAs.size(); i++) {
        LAs[i] = other.LAs[i];
    }

    for (size_t i = 0; i < other.LAStages.size(); i++) {
        LAStages[i] = other.LAStages[i];
    }
}

GPU_LAReference::~GPU_LAReference() {
    if (m_Owned) {
        if (LAs != nullptr) {
            hipFree(LAs);
            LAs = nullptr;
        }

        if (LAStages != nullptr) {
            hipFree(LAStages);
            LAStages = nullptr;
        }
    }
}

__host__
GPU_LAReference::GPU_LAReference(const GPU_LAReference& other) : m_Owned(false) {
    this->UseAT = other.UseAT;
    this->AT = other.AT;
    this->LAStageCount = other.LAStageCount;
    this->isValid = other.isValid;
    this->m_Err = hipSuccess;
    this->LAs = other.LAs;
    this->LAStages = other.LAStages;
}


//////////////////////////////////////////////////////////////////////////////

__global__
void mandel_4x_float(uint32_t* iter_matrix,
    int width,
    int height,
    GQF::gqf_real cx,
    GQF::gqf_real cy,
    GQF::gqf_real dx,
    GQF::gqf_real dy,
    uint32_t n_iterations)
{
    using namespace GQF;
    int X = blockIdx.x * blockDim.x + threadIdx.x;
    int Y = blockIdx.y * blockDim.y + threadIdx.y;

    if (X >= width || Y >= height)
        return;

    size_t idx = width * (height - Y - 1) + X;

    // Approach 2
    //// For reference
    ////{
    ////    x = 0;
    ////    y = 0;
    ////    float zrsqr = x * x;
    ////    float zisqr = y * y;
    ////    while (zrsqr + zisqr <= 4.0 && iter < n_iterations)
    ////    {
    ////        y = x * y;
    ////        y += y; // Multiply by two
    ////        y += y0;
    ////        x = zrsqr - zisqr + x0;
    ////        zrsqr = x * x;
    ////        zisqr = y * y;
    ////        iter++;
    ////    }
    ////}

    int iter = 0;
    gqf_real x = make_qf(0.0f, 0.0f, 0.0f, 0.0f);
    gqf_real y = make_qf(0.0f, 0.0f, 0.0f, 0.0f);

    gqf_real y0;
    gqf_real Y_QF = make_qf(Y, 0.0f, 0.0f, 0.0f);
    y0 = cy + dy * Y_QF;

    gqf_real x0;
    gqf_real X_QF = make_qf(X, 0.0f, 0.0f, 0.0f);
    x0 = cx + dx * X_QF;

    gqf_real four;
    four = make_qf(4.0f, 0.0f, 0.0f, 0.0f);

    gqf_real zrsqr = sqr(x);
    gqf_real zisqr = sqr(y);
    while (zrsqr + zisqr <= four && iter < n_iterations)
    {
        y = x * y;
        y = mul_pwr2(y, 2.0f); // Multiply by two
        y = y + y0;
        x = zrsqr - zisqr + x0;
        zrsqr = sqr(x);
        zisqr = sqr(y);
        iter++;
    }

    iter_matrix[idx] = iter;
}

__global__
void mandel_4x_double(uint32_t* iter_matrix,
    int width,
    int height,
    GQD::gqd_real cx,
    GQD::gqd_real cy,
    GQD::gqd_real dx,
    GQD::gqd_real dy,
    uint32_t n_iterations)
{
    using namespace GQD;
    int X = blockIdx.x * blockDim.x + threadIdx.x;
    int Y = blockIdx.y * blockDim.y + threadIdx.y;

    if (X >= width || Y >= height)
        return;

    size_t idx = width * (height - Y - 1) + X;

    // Approach 2
    //// For reference
    ////{
    ////    x = 0;
    ////    y = 0;
    ////    float zrsqr = x * x;
    ////    float zisqr = y * y;
    ////    while (zrsqr + zisqr <= 4.0 && iter < n_iterations)
    ////    {
    ////        y = x * y;
    ////        y += y; // Multiply by two
    ////        y += y0;
    ////        x = zrsqr - zisqr + x0;
    ////        zrsqr = x * x;
    ////        zisqr = y * y;
    ////        iter++;
    ////    }
    ////}

    int iter = 0;
    gqd_real x = make_qd(0, 0, 0, 0);
    gqd_real y = make_qd(0, 0, 0, 0);
    gqd_real y0;
    y0 = cy + dy * Y;

    gqd_real x0;
    x0 = cx + dx * X;

    gqd_real zrsqr = x * x;
    gqd_real zisqr = y * y;
    while (zrsqr + zisqr <= 4.0 && iter < n_iterations)
    {
        y = x * y;
        y = y * 2.0; // Multiply by two
        y = y + y0;
        x = zrsqr - zisqr + x0;
        zrsqr = x * x;
        zisqr = y * y;
        iter++;
    }

    iter_matrix[idx] = iter;
}


/*
* 
* Nvidia double double library (128-bit precision)

We have released a library that contains code for negation, addition, subtraction, multiplication, division, and square root of double-double operands using a simple C-style interface.

Developers whose applications require precision beyond double precision will likely find this helpful, as double-double offers almost twice the precision of double precision.

It is available in the CUDA Registered Developer Page. The tar file also contains a simple example ( solution of a quadratic equation with different precisions)

$ ./example_dd

Solving quadratic equation with a = 1 b = -100000 c = 1

Using double precision (std. quadratic formula):
x1 = 9.99999999900e+04 ax1**2+bx1+c = 0.00000000000e+00
x2 = 1.00000033854e-05 ax2**2+bx2+c =-3.38435755864e-07

Using double-double (std. quadratic formula):
x1 = 9.99999999900e+04 ax1**2+bx1+c = 0.00000000000e+00
x2 = 1.00000000010e-05 ax2**2+bx2+c = 0.00000000000e+00

Using double precision (more robust formula):
x1 = 9.99999999900e+04 ax1**2+bx1+c = 0.00000000000e+00
x2 = 1.00000000010e-05 ax2**2+bx2+c = 0.00000000000e+00

*/

__global__
void mandel_2x_double(uint32_t* iter_matrix,
    int width,
    int height,
    dbldbl cx,
    dbldbl cy,
    dbldbl dx,
    dbldbl dy,
    uint32_t n_iterations)
{
    int X = blockIdx.x * blockDim.x + threadIdx.x;
    int Y = blockIdx.y * blockDim.y + threadIdx.y;

    if (X >= width || Y >= height)
        return;

    size_t idx = width * (height - Y - 1) + X;

    //// Approach 1
    // TODO need to take dbldbl as parameters to this
    // convert boost high precision to dbldbl?

    dbldbl cx2 = add_double_to_dbldbl(cx.y, cx.x);
    dbldbl cy2 = add_double_to_dbldbl(cy.y, cy.x);
    
    dbldbl dx2 = add_double_to_dbldbl(dx.y, dx.x);
    dbldbl dy2 = add_double_to_dbldbl(dy.y, dy.x);

    dbldbl X2 = add_double_to_dbldbl(X, 0);
    dbldbl Y2 = add_double_to_dbldbl(Y, 0);

    dbldbl x0;
    x0 = add_dbldbl(cx2, mul_dbldbl(dx2, X2));

    dbldbl y0 = add_dbldbl(cy2, mul_dbldbl(dy2, Y2));

    dbldbl x = add_double_to_dbldbl(0,0);
    dbldbl y = add_double_to_dbldbl(0, 0);

    int iter = 0;
    dbldbl xtemp;
    const dbldbl two = add_double_to_dbldbl(2.0, 0);

    dbldbl zrsqr = sqr_dbldbl(x);
    dbldbl zisqr = sqr_dbldbl(y);

    while (zrsqr.y + zisqr.y < 4.0 && iter < n_iterations)
    {
        xtemp = add_dbldbl(sub_dbldbl(zrsqr, zisqr), x0);
        y = add_dbldbl(mul_dbldbl(two, mul_dbldbl(x, y)), y0);
        x = xtemp;
        zrsqr = sqr_dbldbl(x);
        zisqr = sqr_dbldbl(y);
        iter++;
    }

    iter_matrix[idx] = iter;

    // // Approach 2
    // // For reference
    //{
    //    x = 0;
    //    y = 0;
    //    float zrsqr = x * x;
    //    float zisqr = y * y;
    //    while (zrsqr + zisqr <= 4.0 && iter < n_iterations)
    //    {
    //        y = x * y;
    //        y += y; // Multiply by two
    //        y += y0;
    //        x = zrsqr - zisqr + x0;
    //        zrsqr = x * x;
    //        zisqr = y * y;
    //        iter++;
    //    }
    //}

    //dbldbl cx2 = add_double_to_dbldbl(cx.y, cx.x);
    //dbldbl cy2 = add_double_to_dbldbl(cy.y, cy.x);

    //dbldbl dx2 = add_double_to_dbldbl(dx.y, dx.x);
    //dbldbl dy2 = add_double_to_dbldbl(dy.y, dy.x);

    //dbldbl X2 = add_double_to_dbldbl(X, 0);
    //dbldbl Y2 = add_double_to_dbldbl(Y, 0);

    //dbldbl x0;
    //x0 = add_dbldbl(cx2, mul_dbldbl(dx2, X2));

    //dbldbl y0;
    //y0 = add_dbldbl(cy2, mul_dbldbl(dy2, Y2));

    //dbldbl x = add_double_to_dbldbl(0, 0);
    //dbldbl y = add_double_to_dbldbl(0, 0);

    //dbldbl two;
    //two = add_double_to_dbldbl(2.0, 0);

    //int iter = 0;
    //dbldbl zrsqr = mul_dbldbl(x, x);
    //dbldbl zisqr = mul_dbldbl(y, y);

    //while (get_dbldbl_head(add_dbldbl(zrsqr, zisqr)) < 4.0 && iter < n_iterations)
    //{
    //    y = mul_dbldbl(x, y);
    //    y = shiftleft_dbldbl(y);
    //    y = add_dbldbl(y, y0);
    //    x = sub_dbldbl(zrsqr, zisqr);
    //    x = add_dbldbl(x, x0);
    //    zrsqr = sqr_dbldbl(x);
    //    zisqr = sqr_dbldbl(y);
    //    iter++;
    //}

    //iter_matrix[idx] = iter;
}

template<int iteration_precision>
__global__
void mandel_1x_double(uint32_t* iter_matrix,
                 int width,
                 int height,
                 double cx,
                 double cy,
                 double dx,
                 double dy,
                 uint32_t n_iterations)
{
    int X = blockIdx.x * blockDim.x + threadIdx.x;
    int Y = blockIdx.y * blockDim.y + threadIdx.y;

    if (X >= width || Y >= height)
        return;

    size_t idx = width * (height - Y - 1) + X;

    double x0 = cx + dx * X;
    double y0 = cy + dy * Y;

    double x = 0.0;
    double y = 0.0;

    n_iterations -= iteration_precision - 1;

    int iter = 0;
    double xtemp, xtemp2;
    double ytemp;

    auto MANDEL_1X_DOUBLE = [&]() {
        ytemp = __fma_rd(-y, y, x0);
        xtemp = __fma_rd(x, x, ytemp);
        xtemp2 = 2.0 * x;
        y = __fma_rd(xtemp2, y, y0);
        x = xtemp;
    };

    while (x * x + y * y < 4.0 && iter < n_iterations)
    {
        //xtemp = x * x - y * y + x0;
        //y = 2.0 * x * y + y0;
        //x = xtemp;
        //iter++;

        if (iteration_precision == 1) {
            MANDEL_1X_DOUBLE();
            iter++;
        }
        else if (iteration_precision == 2) {
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            iter += 2;
        }
        else if (iteration_precision == 4) {
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            iter += 4;
        }
        else if (iteration_precision == 8) {
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            iter += 8;
        }
        else if (iteration_precision == 16) {
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            MANDEL_1X_DOUBLE();
            iter += 16;
        }
    }

    iter_matrix[idx] = iter;
}

__global__
void mandel_1x_double_perturb(uint32_t* iter_matrix,
    MattPerturbSingleResults<double> PerturbDouble,
    int width,
    int height,
    double cx,
    double cy,
    double dx,
    double dy,
    double centerX,
    double centerY,
    uint32_t n_iterations)
{
    int X = blockIdx.x * blockDim.x + threadIdx.x;
    int Y = blockIdx.y * blockDim.y + threadIdx.y;

    if (X >= width || Y >= height)
        return;

    //size_t idx = width * (height - Y - 1) + X;
    size_t idx = width * Y + X;

    if (iter_matrix[idx] != 0) {
        return;
    }

    size_t iter = 0;
    size_t RefIteration = 0;
    double DeltaReal = dx * X - centerX;
    double DeltaImaginary = -dy * Y - centerY;

    double DeltaSub0X = DeltaReal;
    double DeltaSub0Y = DeltaImaginary;
    double DeltaSubNX = 0;
    double DeltaSubNY = 0;
    size_t MaxRefIteration = PerturbDouble.size - 1;

    while (iter < n_iterations) {
        MattReferenceSingleIter<double> *CurIter = &PerturbDouble.iters[RefIteration];

        const double DeltaSubNXOrig = DeltaSubNX;
        const double DeltaSubNYOrig = DeltaSubNY;

        DeltaSubNX =
            DeltaSubNXOrig * (CurIter->x * 2 + DeltaSubNXOrig) -
            DeltaSubNYOrig * (CurIter->y * 2 + DeltaSubNYOrig) +
            DeltaSub0X;
        DeltaSubNY =
            DeltaSubNXOrig * (CurIter->y * 2 + DeltaSubNYOrig) +
            DeltaSubNYOrig * (CurIter->x * 2 + DeltaSubNXOrig) +
            DeltaSub0Y;

        ++RefIteration;
        CurIter = &PerturbDouble.iters[RefIteration];

        const double tempZX = CurIter->x + DeltaSubNX;
        const double tempZY = CurIter->y + DeltaSubNY;
        const double zn_size = tempZX * tempZX + tempZY * tempZY;
        const double normDeltaSubN = DeltaSubNX * DeltaSubNX + DeltaSubNY * DeltaSubNY;

        if (zn_size > 256) {
            break;
        }

        if (zn_size < normDeltaSubN ||
            RefIteration == MaxRefIteration) {
            DeltaSubNX = tempZX;
            DeltaSubNY = tempZY;
            RefIteration = 0;
        }

        ++iter;
    }

    iter_matrix[idx] = (uint32_t)iter;
}

__global__
void mandel_1xHDR_InitStatics()
{
    if (blockIdx.x == 0 &&
        threadIdx.x == 0 &&
        blockIdx.y == 0 &&
        threadIdx.y == 0) {
        InitStatics();
    }
}

template<class HDRFloatType>
struct SharedMemStruct {
    using GPUBLA_TYPE = BLA<HDRFloatType>;
    const GPUBLA_TYPE* __restrict__ altB[32];
    //GPUBLA_TYPE nullBla;
    //struct {
    //    //HDRFloatType curBR2[16];
    //    //MattReferenceSingleIter<HDRFloatType> CurResult;
    //    //HDRFloatType NextX1;
    //    //HDRFloatType NextY1;
    //} PerThread[NB_THREADS_W][NB_THREADS_H];
};

template<int32_t LM2>
__global__
void mandel_1x_double_perturb_bla(uint32_t* iter_matrix,
    MattPerturbSingleResults<double> PerturbDouble,
    GPU_BLAS<double, BLA<double>, LM2> doubleBlas,
    int width,
    int height,
    double cx,
    double cy,
    double dx,
    double dy,
    double centerX,
    double centerY,
    uint32_t n_iterations)
{
    int X = blockIdx.x * blockDim.x + threadIdx.x;
    int Y = blockIdx.y * blockDim.y + threadIdx.y;

    if (X >= width || Y >= height)
        return;

    //size_t idx = width * (height - Y - 1) + X;
    size_t idx = width * Y + X;

    if (iter_matrix[idx] != 0) {
        return;
    }

    using GPUBLA_TYPE = BLA<double>;
    char __shared__ SharedMem[sizeof(SharedMemStruct<double>)];
    auto* shared =
        reinterpret_cast<SharedMemStruct<double>*>(SharedMem);

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        GPUBLA_TYPE** elts = doubleBlas.GetB();

        for (size_t i = 0; i < doubleBlas.m_NumLevels; i++) {
            shared->altB[i] = elts[i];
        }
    }

    __syncthreads();

    size_t iter = 0;
    size_t RefIteration = 0;
    double DeltaReal = dx * X - centerX;
    double DeltaImaginary = -dy * Y - centerY;

    double DeltaSub0X = DeltaReal;
    double DeltaSub0Y = DeltaImaginary;
    double DeltaSubNX = 0;
    double DeltaSubNY = 0;
    double DeltaNormSquared = 0;

    while (iter < n_iterations) {
        const BLA<double>* b = nullptr;
        while ((b = doubleBlas.LookupBackwards(shared->altB, RefIteration, DeltaNormSquared)) != nullptr) {
            int l = b->getL();

            // TODO this first RefIteration + l check bugs me
            if (RefIteration + l >= PerturbDouble.size) {
                break;
            }

            if (iter + l >= n_iterations) {
                break;
            }

            iter += l;
            RefIteration += l;

            b->getValue(DeltaSubNX, DeltaSubNY, DeltaSub0X, DeltaSub0Y);

            const double tempZX = PerturbDouble.iters[RefIteration].x + DeltaSubNX;
            const double tempZY = PerturbDouble.iters[RefIteration].y + DeltaSubNY;
            const double normSquared = tempZX * tempZX + tempZY * tempZY;
            DeltaNormSquared = DeltaSubNX * DeltaSubNX + DeltaSubNY * DeltaSubNY;

            if (normSquared > 256) {
                break;
            }

            if (normSquared < DeltaNormSquared ||
                RefIteration >= PerturbDouble.size - 1) {
                DeltaSubNX = tempZX;
                DeltaSubNY = tempZY;
                DeltaNormSquared = normSquared;
                RefIteration = 0;
            }
        }

        if (iter >= n_iterations) {
            break;
        }

        const double DeltaSubNXOrig = DeltaSubNX;
        const double DeltaSubNYOrig = DeltaSubNY;

        DeltaSubNX = DeltaSubNXOrig * (PerturbDouble.iters[RefIteration].x * 2 + DeltaSubNXOrig) -
            DeltaSubNYOrig * (PerturbDouble.iters[RefIteration].y * 2 + DeltaSubNYOrig) +
            DeltaSub0X;
        DeltaSubNY = DeltaSubNXOrig * (PerturbDouble.iters[RefIteration].y * 2 + DeltaSubNYOrig) +
            DeltaSubNYOrig * (PerturbDouble.iters[RefIteration].x * 2 + DeltaSubNXOrig) +
            DeltaSub0Y;

        ++RefIteration;

        const double tempZX = PerturbDouble.iters[RefIteration].x + DeltaSubNX;
        const double tempZY = PerturbDouble.iters[RefIteration].y + DeltaSubNY;
        const double normSquared = tempZX * tempZX + tempZY * tempZY;
        DeltaNormSquared = DeltaSubNX * DeltaSubNX + DeltaSubNY * DeltaSubNY;

        if (normSquared > 256) {
            break;
        }

        if (normSquared < DeltaNormSquared ||
            RefIteration >= PerturbDouble.size - 1) {
            DeltaSubNX = tempZX;
            DeltaSubNY = tempZY;
            DeltaNormSquared = normSquared;
            RefIteration = 0;
        }

        ++iter;
    }

    iter_matrix[idx] = (uint32_t)iter;
}

//#define DEVICE_STATIC_INTRINSIC_QUALIFIERS  static __device__ __forceinline__

//#if (defined(_MSC_VER) && defined(_WIN64)) || defined(__LP64__)
//#define PXL_GLOBAL_PTR   "l"
//#else
//#define PXL_GLOBAL_PTR   "r"
//#endif

//DEVICE_STATIC_INTRINSIC_QUALIFIERS void __prefetch_global_l1(const void* const ptr)
//{
//    asm("prefetch.global.L1 [%0];" : : PXL_GLOBAL_PTR(ptr));
//}
//
//DEVICE_STATIC_INTRINSIC_QUALIFIERS void __prefetch_global_uniform(const void* const ptr)
//{
//    asm("prefetchu.L1 [%0];" : : PXL_GLOBAL_PTR(ptr));
//}
//
//DEVICE_STATIC_INTRINSIC_QUALIFIERS void __prefetch_global_l2(const void* const ptr)
//{
//    asm("prefetch.global.L2 [%0];" : : PXL_GLOBAL_PTR(ptr));
//}

template<class HDRFloatType, int32_t LM2>
__global__
void
//__launch_bounds__(NB_THREADS_W * NB_THREADS_H, 2)
mandel_1xHDR_float_perturb_bla(uint32_t* iter_matrix,
    MattPerturbSingleResults<HDRFloatType> Perturb,
    GPU_BLAS<HDRFloatType, BLA<HDRFloatType>, LM2> blas,
    int width,
    int height,
    const HDRFloatType cx,
    const HDRFloatType cy,
    const HDRFloatType dx,
    const HDRFloatType dy,
    const HDRFloatType centerX,
    const HDRFloatType centerY,
    uint32_t n_iterations)
{
    const int X = blockIdx.x * blockDim.x + threadIdx.x;
    const int Y = blockIdx.y * blockDim.y + threadIdx.y;

    if (X >= width || Y >= height)
        return;

    //size_t idx = width * (height - Y - 1) + X;
    size_t idx = width * Y + X;

    if (iter_matrix[idx] != 0) { 
        return;
    }

    using GPUBLA_TYPE = BLA<HDRFloatType>;
    char __shared__ SharedMem[sizeof(SharedMemStruct<HDRFloatType>)];
    auto* shared =
        reinterpret_cast<SharedMemStruct<HDRFloatType>*>(SharedMem);

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        GPUBLA_TYPE**elts = blas.GetB();

        for (size_t i = 0; i < blas.m_NumLevels; i++) {
            shared->altB[i] = elts[i];
        }

        //const GPUBLA_TYPE nullBla{
        //    HDRFloatType(),
        //    HDRFloatType(1.0f),
        //    HDRFloatType(),
        //    HDRFloatType(),
        //    HDRFloatType(),
        //    0 };
        //shared->nullBla = nullBla;
    }

    __syncthreads();

    size_t iter = 0;
    size_t RefIteration = 0;
    const HDRFloatType DeltaReal = dx * X - centerX;
    const HDRFloatType DeltaImaginary = -dy * Y - centerY;

    const HDRFloatType DeltaSub0X = DeltaReal;
    const HDRFloatType DeltaSub0Y = DeltaImaginary;
    HDRFloatType DeltaSubNX = HDRFloatType(0);
    HDRFloatType DeltaSubNY = HDRFloatType(0);
    HDRFloatType DeltaNormSquared = HDRFloatType(0);
    const HDRFloatType TwoFiftySix = HDRFloatType(256);
    const HDRFloatType Two = HDRFloatType(2);

    while (iter < n_iterations) {
        //auto* next1X = &shared->PerThread[threadIdx.x][threadIdx.y].NextX1;
        //auto* next1Y = &shared->PerThread[threadIdx.x][threadIdx.y].NextY1;
        //__pipeline_memcpy_async(
        //    next1X,
        //    &Perturb.iters[RefIteration + 1].x,
        //    sizeof(Perturb.iters[RefIteration + 1].x),
        //    0);
        //__pipeline_memcpy_async(
        //    next1Y,
        //    &Perturb.iters[RefIteration + 1].y,
        //    sizeof(Perturb.iters[RefIteration + 1].y),
        //    0);
        //__pipeline_commit();

        const HDRFloatType DeltaSubNXOrig = DeltaSubNX;
        const HDRFloatType DeltaSubNYOrig = DeltaSubNY;

        //__prefetch_global_l2(&Perturb.iters[RefIteration + 1].x);
        //__prefetch_global_l2(&Perturb.iters[RefIteration + 1].y);

        const auto tempMulX2 = Perturb.iters[RefIteration].x * Two;
        const auto tempMulY2 = Perturb.iters[RefIteration].y * Two;

        ++RefIteration;

        //if (RefIteration >= Perturb.size) {
        //    // TODO this first RefIteration + l check bugs me
        //    iter = 255;
        //    break;
        //}

        const auto tempSum1 = (tempMulY2 + DeltaSubNYOrig);
        const auto tempSum2 = (tempMulX2 + DeltaSubNXOrig);

        //DeltaSubNX = DeltaSubNXOrig * tempSum2 -
        //    DeltaSubNYOrig * tempSum1 +
        //    DeltaSub0X;
        //HdrReduce(DeltaSubNX);
        DeltaSubNX = HDRFloatType::custom_perturb1<false>(
            DeltaSubNXOrig,
            tempSum2,
            DeltaSubNYOrig,
            tempSum1,
            DeltaSub0X);

        DeltaSubNY = HDRFloatType::custom_perturb1<true>(
            DeltaSubNXOrig,
            tempSum1,
            DeltaSubNYOrig,
            tempSum2,
            DeltaSub0Y);

        //DeltaSubNY = DeltaSubNXOrig * tempSum1 +
        //    DeltaSubNYOrig * tempSum2 +
        //    DeltaSub0Y;
        //HdrReduce(DeltaSubNY);

        //__pipeline_wait_prior(0);

        const auto tempVal1X = Perturb.iters[RefIteration].x;
        const auto tempVal1Y = Perturb.iters[RefIteration].y;

        const HDRFloatType tempZX = tempVal1X + DeltaSubNX;
        const HDRFloatType tempZY = tempVal1Y + DeltaSubNY;
        HDRFloatType normSquared = tempZX * tempZX + tempZY * tempZY;
        HdrReduce(normSquared);

        if (normSquared <= TwoFiftySix && iter < n_iterations) {
            DeltaNormSquared = DeltaSubNX * DeltaSubNX + DeltaSubNY * DeltaSubNY;
            HdrReduce(DeltaNormSquared);

            if (normSquared < DeltaNormSquared ||
                RefIteration >= Perturb.size - 1) {
                DeltaSubNX = tempZX;
                DeltaSubNY = tempZY;
                DeltaNormSquared = normSquared;
                RefIteration = 0;
            }

            ++iter;
        }
        else {
            break;
        }

        const BLA<HDRFloatType>* b = nullptr;

        for (;;) {
            b = blas.LookupBackwards(
                shared->altB,
                /*shared->PerThread[threadIdx.x][threadIdx.y].curBR2,*/
                //&shared->nullBla,
                RefIteration,
                DeltaNormSquared);
            if (b == nullptr) {
                break;
            }

            const int l = b->getL();

            // TODO this first RefIteration + l check bugs me
            const bool res1 = (RefIteration + l >= Perturb.size);
            const bool res2 = (iter + l >= n_iterations);
            const bool res3 = (RefIteration + l < Perturb.size - 1);
            //const bool res4 = l == 0; // nullBla
            const bool res12 = (/*res4 || */res1 || res2) == false;
            if (res12 && res3) {
                iter += l;
                RefIteration += l;

                b->getValue(DeltaSubNX, DeltaSubNY, DeltaSub0X, DeltaSub0Y);

                DeltaNormSquared = DeltaSubNX * DeltaSubNX + DeltaSubNY * DeltaSubNY;
                HdrReduce(DeltaNormSquared);
                continue;
            }
            else if (res12 && !res3) {
                iter += l;
                RefIteration += l;

                b->getValue(DeltaSubNX, DeltaSubNY, DeltaSub0X, DeltaSub0Y);

                //__pipeline_memcpy_async(
                //    &shared->PerThread[threadIdx.x][threadIdx.y].CurResult.x,
                //    &Perturb.iters[RefIteration].x,
                //    sizeof(Perturb.iters[RefIteration].x),
                //    0);
                //__pipeline_memcpy_async(
                //    &shared->PerThread[threadIdx.x][threadIdx.y].CurResult.y,
                //    &Perturb.iters[RefIteration].y,
                //    sizeof(Perturb.iters[RefIteration].y),
                //    0);
                //__pipeline_commit();
                //__pipeline_wait_prior(0);

                //HDRFloatType tempZX = shared->PerThread[threadIdx.x][threadIdx.y].CurResult.x + DeltaSubNX;
                //HDRFloatType tempZY = shared->PerThread[threadIdx.x][threadIdx.y].CurResult.y + DeltaSubNY;
                HDRFloatType tempZX = Perturb.iters[RefIteration].x + DeltaSubNX;
                HDRFloatType tempZY = Perturb.iters[RefIteration].y + DeltaSubNY;

                DeltaSubNX = tempZX;
                DeltaSubNY = tempZY;

                DeltaNormSquared = tempZX.square_mutable() + tempZY.square_mutable();
                HdrReduce(DeltaNormSquared);
                RefIteration = 0;
                break;
            }
            else {
                break;
            }
        }
    }

    iter_matrix[idx] = (uint32_t)iter;
}

template<class HDRFloatType>
__global__
void
//__launch_bounds__(NB_THREADS_W * NB_THREADS_H, 2)
mandel_1xHDR_float_perturb_lav2(uint32_t* iter_matrix,
    MattPerturbSingleResults<HDRFloat<float>> Perturb,
    GPU_LAReference LaReference, // "copy"
    int width,
    int height,
    const HDRFloatType cx,
    const HDRFloatType cy,
    const HDRFloatType dx,
    const HDRFloatType dy,
    const HDRFloatType centerX,
    const HDRFloatType centerY,
    uint32_t n_iterations)
{
    const int X = blockIdx.x * blockDim.x + threadIdx.x;
    const int Y = blockIdx.y * blockDim.y + threadIdx.y;

    if (X >= width || Y >= height)
        return;

    int32_t idx = width * Y + X;
    if (iter_matrix[idx] != 0) {
        return;
    }

    int32_t iter = 0;
    int32_t RefIteration = 0;
    const HDRFloatType DeltaReal = dx * X - centerX;
    const HDRFloatType DeltaImaginary = -dy * Y - centerY;

    const HDRFloatType DeltaSub0X = DeltaReal;
    const HDRFloatType DeltaSub0Y = DeltaImaginary;
    HDRFloatType DeltaSubNX = HDRFloatType(0);
    HDRFloatType DeltaSubNY = HDRFloatType(0);
    HDRFloatType DeltaNormSquared = HDRFloatType(0);
    const HDRFloatType TwoFiftySix = HDRFloatType(256);
    const HDRFloatType Two = HDRFloatType(2);

    using SubType = float;
    using TComplex = HDRFloatComplex<float>;

    ////////////
    int32_t BLA2SkippedIterations;

    BLA2SkippedIterations = 0;
    
    TComplex DeltaSub0;
    TComplex DeltaSubN;

    DeltaSub0 = { DeltaReal, DeltaImaginary };
    DeltaSubN = { 0, 0 };

    if (LaReference.isValid && LaReference.UseAT && LaReference.AT.isValid(DeltaSub0)) {
        ATResult res;
        LaReference.AT.PerformAT(n_iterations, DeltaSub0, res);
        BLA2SkippedIterations = res.bla_iterations;
        DeltaSubN = res.dz;
    }

    int32_t MaxRefIteration = Perturb.size - 1;

    iter = BLA2SkippedIterations;

    TComplex complex0{ DeltaReal, DeltaImaginary };

    if (iter != 0 && RefIteration < MaxRefIteration) {
        complex0 = Perturb.GetComplex(RefIteration).plus_mutable(DeltaSubN);
    }
    else if (iter != 0 && Perturb.PeriodMaybeZero != 0) {
        RefIteration = RefIteration % Perturb.PeriodMaybeZero;
        complex0 = Perturb.GetComplex(RefIteration).plus_mutable(DeltaSubN);
    }

    int32_t CurrentLAStage = LaReference.isValid ? LaReference.LAStageCount : 0;

    while (CurrentLAStage > 0) {
        CurrentLAStage--;

        int32_t LAIndex = LaReference.getLAIndex(CurrentLAStage);

        if (LaReference.isLAStageInvalid(LAIndex, DeltaSub0)) {
            continue;
        }

        int32_t MacroItCount = LaReference.getMacroItCount(CurrentLAStage);
        int32_t j = RefIteration;

        while (iter < n_iterations) {
            GPU_LAstep las = LaReference.getLA(LAIndex, DeltaSubN, j, iter, n_iterations);

            if (las.unusable) {
                RefIteration = las.nextStageLAindex;
                break;
            }

            iter += las.step;
            DeltaSubN = las.Evaluate(DeltaSub0);
            complex0 = las.getZ(DeltaSubN);
            j++;

            auto lhs = complex0.chebychevNorm();
            HdrReduce(lhs);
            auto rhs = DeltaSubN.chebychevNorm();
            HdrReduce(rhs);

            if (lhs < rhs || j >= MacroItCount) {
                DeltaSubN = complex0;
                j = 0;
            }

            //HdrReduce(DeltaSubN); // maybe don't need
        }

        if (iter >= n_iterations) {
            break;
        }
    }

    HDRFloatType normSquared{};

    DeltaSubNX = DeltaSubN.getRe();
    DeltaSubNY = DeltaSubN.getIm();

    for (;;) {
        const HDRFloatType DeltaSubNXOrig = DeltaSubNX;
        const HDRFloatType DeltaSubNYOrig = DeltaSubNY;

        const auto tempMulX2 = Perturb.iters[RefIteration].x * Two;
        const auto tempMulY2 = Perturb.iters[RefIteration].y * Two;

        ++RefIteration;

        const auto tempSum1 = (tempMulY2 + DeltaSubNYOrig);
        const auto tempSum2 = (tempMulX2 + DeltaSubNXOrig);

        DeltaSubNX = HDRFloatType::custom_perturb1<false>(
            DeltaSubNXOrig,
            tempSum2,
            DeltaSubNYOrig,
            tempSum1,
            DeltaSub0X);

        DeltaSubNY = HDRFloatType::custom_perturb1<true>(
            DeltaSubNXOrig,
            tempSum1,
            DeltaSubNYOrig,
            tempSum2,
            DeltaSub0Y);

        const auto tempVal1X = Perturb.iters[RefIteration].x;
        const auto tempVal1Y = Perturb.iters[RefIteration].y;

        const HDRFloatType tempZX = tempVal1X + DeltaSubNX;
        const HDRFloatType tempZY = tempVal1Y + DeltaSubNY;
        HDRFloatType normSquared = tempZX * tempZX + tempZY * tempZY;
        HdrReduce(normSquared);

        if (normSquared <= TwoFiftySix && iter < n_iterations) {
            DeltaNormSquared = DeltaSubNX * DeltaSubNX + DeltaSubNY * DeltaSubNY;
            HdrReduce(DeltaNormSquared);

            if (normSquared < DeltaNormSquared ||
                RefIteration >= Perturb.size - 1) {
                DeltaSubNX = tempZX;
                DeltaSubNY = tempZY;
                DeltaNormSquared = normSquared;
                RefIteration = 0;
            }

            ++iter;
        }
        else {
            break;
        }
    }

    iter_matrix[idx] = (uint32_t)iter;
}

template<int iteration_precision>
__global__
void mandel_2x_float(uint32_t* iter_matrix,
    int width,
    int height,
    dblflt cx,
    dblflt cy,
    dblflt dx,
    dblflt dy,
    uint32_t n_iterations)
{
    int X = blockIdx.x * blockDim.x + threadIdx.x;
    int Y = blockIdx.y * blockDim.y + threadIdx.y;

    if (X >= width || Y >= height)
        return;

    size_t idx = width * (height - Y - 1) + X;

    ////// Approach 1
    //// TODO need to take dblflt as parameters to this
    //// convert boost high precision to dblflt?

    //dblflt cx2 = add_float_to_dblflt(cx.y, cx.x);
    //dblflt cy2 = add_float_to_dblflt(cy.y, cy.x);

    //dblflt dx2 = add_float_to_dblflt(dx.y, dx.x);
    //dblflt dy2 = add_float_to_dblflt(dy.y, dy.x);

    //dblflt X2 = add_float_to_dblflt(X, 0);
    //dblflt Y2 = add_float_to_dblflt(Y, 0);

    //dblflt x0;
    //x0 = add_dblflt(cx2, mul_dblflt(dx2, X2));

    //dblflt y0;
    //y0 = add_dblflt(cy2, mul_dblflt(dy2, Y2));

    //dblflt x = add_float_to_dblflt(0, 0);
    //dblflt y = add_float_to_dblflt(0, 0);

    //int iter = 0;
    //dblflt xtemp;
    //dblflt two;
    //two = add_float_to_dblflt(2.0, 0);

    //dblflt xtemp2, xtemp3;
    //dblflt ytemp2, ytemp3;
    //// while (x * x + y * y < 4.0 && iter < n_iterations)
    //while (get_dblflt_head(add_dblflt(mul_dblflt(x, x), mul_dblflt(y, y))) < 4.0 && iter < n_iterations)
    //{
    //    //xtemp = x * x - y * y + x0;
    //    //y = 2.0 * x * y + y0;
    //    //x = xtemp;
    //    //iter++;

    //    //xtemp = add_dblflt(sub_dblflt(mul_dblflt(x, x), mul_dblflt(y, y)), x0);
    //    //y = add_dblflt(mul_dblflt(two, mul_dblflt(x, y)), y0);
    //    //x = xtemp;
    //    //iter++;

    //    xtemp2 = mul_dblflt(x, x);
    //    ytemp2 = mul_dblflt(y, y);
    //    ytemp3 = sub_dblflt(xtemp2, ytemp2);
    //    xtemp = add_dblflt(ytemp3, x0);
    //    y = add_dblflt(mul_dblflt(two, mul_dblflt(x, y)), y0);
    //    x = xtemp;
    //    iter++;
    //}


    // Approach 2
    // // For reference
    //{
    //    x = 0;
    //    y = 0;
    //    float zrsqr = x * x;
    //    float zisqr = y * y;
    //    while (zrsqr + zisqr <= 4.0 && iter < n_iterations)
    //    {
    //        y = x * y;
    //        y += y; // Multiply by two
    //        y += y0;
    //        x = zrsqr - zisqr + x0;
    //        zrsqr = x * x;
    //        zisqr = y * y;
    //        iter++;
    //    }
    //}

    dblflt cx2 = add_float_to_dblflt(cx.y, cx.x);
    dblflt cy2 = add_float_to_dblflt(cy.y, cy.x);

    dblflt dx2 = add_float_to_dblflt(dx.y, dx.x);
    dblflt dy2 = add_float_to_dblflt(dy.y, dy.x);

    dblflt X2 = add_float_to_dblflt(X, 0);
    dblflt Y2 = add_float_to_dblflt(Y, 0);

    dblflt x0;
    x0 = add_dblflt(cx2, mul_dblflt(dx2, X2));

    dblflt y0;
    y0 = add_dblflt(cy2, mul_dblflt(dy2, Y2));

    dblflt x = {};
    dblflt y = {};

    int iter = 0;
    dblflt zrsqr = {};
    dblflt zisqr = {};

    auto MANDEL_2X_FLOAT = [&]() {
        y = mul_dblflt2x(x, y);
        y = add_dblflt(y, y0);
        x = sub_dblflt(zrsqr, zisqr);
        x = add_dblflt(x, x0);
        zrsqr = sqr_dblflt(x);
        zisqr = sqr_dblflt(y);
    };

    while (zrsqr.y + zisqr.y < 4.0f && iter < n_iterations)
    {
        if (iteration_precision == 1) {
            MANDEL_2X_FLOAT();
            iter++;
        }
        else if (iteration_precision == 2) {
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            iter += 2;
        }
        else if (iteration_precision == 4) {
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            iter += 4;
        }
        else if (iteration_precision == 8) {
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            iter += 8;
        }
        else if (iteration_precision == 16) {
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            MANDEL_2X_FLOAT();
            iter += 16;
        }
    }

    iter_matrix[idx] = iter;
}

__global__
void mandel_2x_float_perturb_setup(MattPerturbSingleResults<dblflt> PerturbDblFlt)
{
    if (blockIdx.x != 0 || blockIdx.y != 0 || threadIdx.x != 0 || threadIdx.y != 0)
        return;

    for (size_t i = 0; i < PerturbDblFlt.size; i++) {
        PerturbDblFlt.iters[i].x = add_float_to_dblflt(PerturbDblFlt.iters[i].x.y, PerturbDblFlt.iters[i].x.x);
        PerturbDblFlt.iters[i].y = add_float_to_dblflt(PerturbDblFlt.iters[i].y.y, PerturbDblFlt.iters[i].y.x);
    }
}

__global__
void mandel_2x_float_perturb(uint32_t* iter_matrix,
    MattPerturbSingleResults<dblflt> PerturbDblFlt,
    int width,
    int height,
    dblflt cx,
    dblflt cy,
    dblflt dx,
    dblflt dy,
    dblflt centerX,
    dblflt centerY,
    uint32_t n_iterations)
{

    //int X = blockIdx.x * blockDim.x + threadIdx.x;
    //int Y = blockIdx.y * blockDim.y + threadIdx.y;

    //if (X >= width || Y >= height)
    //    return;

    ////size_t idx = width * (height - Y - 1) + X;
    //size_t idx = width * Y + X;

    //if (iter_matrix[idx] != 0) {
    //    return;
    //}

    //size_t iter = 0;
    //size_t RefIteration = 0;
    //double DeltaReal = dx * X - centerX;
    //double DeltaImaginary = -dy * Y - centerY;

    //double DeltaSub0X = DeltaReal;
    //double DeltaSub0Y = DeltaImaginary;
    //double DeltaSubNX = 0;
    //double DeltaSubNY = 0;

    //while (iter < n_iterations) {
    //    const double DeltaSubNXOrig = DeltaSubNX;
    //    const double DeltaSubNYOrig = DeltaSubNY;

    //    DeltaSubNX =
    //        DeltaSubNXOrig * (results_x2[RefIteration] + DeltaSubNXOrig) -
    //        DeltaSubNYOrig * (results_y2[RefIteration] + DeltaSubNYOrig) +
    //        DeltaSub0X;
    //    DeltaSubNY =
    //        DeltaSubNXOrig * (results_y2[RefIteration] + DeltaSubNYOrig) +
    //        DeltaSubNYOrig * (results_x2[RefIteration] + DeltaSubNXOrig) +
    //        DeltaSub0Y;

    //    ++RefIteration;

    //    const double tempZX = results_x[RefIteration] + DeltaSubNX;
    //    const double tempZY = results_y[RefIteration] + DeltaSubNY;
    //    const double zn_size = tempZX * tempZX + tempZY * tempZY;
    //    const double normDeltaSubN = DeltaSubNX * DeltaSubNX + DeltaSubNY * DeltaSubNY;

    //    if (zn_size > 256) {
    //        break;
    //    }

    //    if (zn_size < normDeltaSubN ||
    //        RefIteration == sz - 1) {
    //        DeltaSubNX = tempZX;
    //        DeltaSubNY = tempZY;
    //        RefIteration = 0;
    //    }

    //    ++iter;
    //}

    //iter_matrix[idx] = (uint32_t)iter;

    int X = blockIdx.x * blockDim.x + threadIdx.x;
    int Y = blockIdx.y * blockDim.y + threadIdx.y;

    if (X >= width || Y >= height)
        return;

    //size_t idx = width * (height - Y - 1) + X;
    size_t idx = width * Y + X;

    if (iter_matrix[idx] != 0) {
        return;
    }

    int iter = 0;
    size_t RefIteration = 0;

    dblflt Two = add_float_to_dblflt(0, 2);

    dblflt X2 = add_float_to_dblflt(X, 0);
    dblflt Y2 = add_float_to_dblflt(Y, 0);
    dblflt MinusY2 = add_float_to_dblflt(-Y, 0);

    dblflt DeltaReal = sub_dblflt(mul_dblflt(dx, X2), centerX);
    dblflt DeltaImaginary = sub_dblflt(mul_dblflt(dy, MinusY2), centerY);

    dblflt DeltaSub0X = DeltaReal;
    dblflt DeltaSub0Y = DeltaImaginary;
    dblflt DeltaSubNX, DeltaSubNY;

    size_t MaxRefIteration = PerturbDblFlt.size - 1;

    DeltaSubNX = add_float_to_dblflt(0, 0);
    DeltaSubNY = add_float_to_dblflt(0, 0);

    while (iter < n_iterations) {
        MattReferenceSingleIter<dblflt>* CurIter = &PerturbDblFlt.iters[RefIteration];

        const dblflt DeltaSubNXOrig = DeltaSubNX;
        const dblflt DeltaSubNYOrig = DeltaSubNY;

        const dblflt tempX = mul_dblflt(CurIter->x, Two);
        const dblflt tempY = mul_dblflt(CurIter->y, Two);

        const dblflt tempTermX1 = add_dblflt(tempX, DeltaSubNXOrig);
        const dblflt tempTermX2 = add_dblflt(tempY, DeltaSubNYOrig);

        DeltaSubNX =
            sub_dblflt(
                mul_dblflt(DeltaSubNXOrig, tempTermX1),
                mul_dblflt(DeltaSubNYOrig, tempTermX2)
            );
        DeltaSubNX = add_dblflt(DeltaSubNX, DeltaSub0X);

        DeltaSubNY =
            add_dblflt(
                mul_dblflt(DeltaSubNXOrig, tempTermX2),
                mul_dblflt(DeltaSubNYOrig, tempTermX1)
            );
        DeltaSubNY = add_dblflt(DeltaSubNY, DeltaSub0Y);

        ++RefIteration;
        CurIter = &PerturbDblFlt.iters[RefIteration];

        const dblflt tempZX = add_dblflt(CurIter->x, DeltaSubNX);
        const dblflt tempZY = add_dblflt(CurIter->y, DeltaSubNY);
        const dblflt zn_size = add_dblflt(sqr_dblflt(tempZX), sqr_dblflt(tempZY));
        const dblflt normDeltaSubN = add_dblflt(sqr_dblflt(DeltaSubNX), sqr_dblflt(DeltaSubNY));

        if (zn_size.y > 256) {
            break;
        }

        if (zn_size.y < normDeltaSubN.y ||
            RefIteration == MaxRefIteration) {
            DeltaSubNX = tempZX;
            DeltaSubNY = tempZY;
            RefIteration = 0;
        }

        ++iter;
    }

    iter_matrix[idx] = iter;
}

template<int iteration_precision>
__global__
void mandel_1x_float(uint32_t* iter_matrix,
    int width,
    int height,
    float cx,
    float cy,
    float dx,
    float dy,
    uint32_t n_iterations)
{
    int X = blockIdx.x * blockDim.x + threadIdx.x;
    int Y = blockIdx.y * blockDim.y + threadIdx.y;

    if (X >= width || Y >= height)
        return;

    size_t idx = width * (height - Y - 1) + X;

    float x0 = cx + dx * X;
    float y0 = cy + dy * Y;

    float x = 0.0;
    float y = 0.0;

    n_iterations -= iteration_precision - 1;

    int iter = 0;

    //{
    //    x = 0;
    //    y = 0;
    //    float zrsqr = x * x;
    //    float zisqr = y * y;
    //    while (zrsqr + zisqr <= 4.0 && iter < n_iterations)
    //    {
    //        y = x * y;
    //        y += y; // Multiply by two
    //        y += y0;
    //        x = zrsqr - zisqr + x0;
    //        zrsqr = x * x;
    //        zisqr = y * y;
    //        iter++;
    //    }
    //}

    float xtemp, xtemp2;
    float ytemp;

    auto MANDEL_1X_FLOAT = [&]() {
        ytemp = __fmaf_rd(-y, y, x0);
        xtemp = __fmaf_rd(x, x, ytemp);
        xtemp2 = 2.0f * x;
        y = __fmaf_rd(xtemp2, y, y0);
        x = xtemp;
    };

    while (x * x + y * y < 4.0 && iter < n_iterations)
    {
        //xtemp = x * x - y * y + x0;
        //y = 2.0 * x * y + y0;
        //x = xtemp;
        //iter++;

        if (iteration_precision == 1) {
            MANDEL_1X_FLOAT();
            iter++;
        }
        else if (iteration_precision == 2) {
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            iter += 2;
        }
        else if (iteration_precision == 4) {
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            iter += 4;
        }
        else if (iteration_precision == 8) {
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            iter += 8;
        }
        else if (iteration_precision == 16) {
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            MANDEL_1X_FLOAT();
            iter += 16;
        }
    }

    iter_matrix[idx] = iter;
}

template<bool Periodic>
__global__
void mandel_1x_float_perturb(uint32_t* iter_matrix,
    MattPerturbSingleResults<float> PerturbFloat,
    int width,
    int height,
    float cx,
    float cy,
    float dx,
    float dy,
    float centerX,
    float centerY,
    uint32_t n_iterations)
{
    int X = blockIdx.x * blockDim.x + threadIdx.x;
    int Y = blockIdx.y * blockDim.y + threadIdx.y;

    if (X >= width || Y >= height)
        return;

    //size_t idx = width * (height - Y - 1) + X;
    size_t idx = width * Y + X;

    if (iter_matrix[idx] != 0) {
        return;
    }

    size_t iter = 0;
    size_t RefIteration = 0;
    float DeltaReal = dx * X - centerX;
    float DeltaImaginary = -dy * Y - centerY;

    float DeltaSub0X = DeltaReal;
    float DeltaSub0Y = DeltaImaginary;
    float DeltaSubNX = 0;
    float DeltaSubNY = 0;
    size_t MaxRefIteration = PerturbFloat.size - 1;

    //float dzdcX = max(max(x.dzdc), 1.0f);
    float scalingFactor = 1.0f / (max(max(abs(dx), abs(dy)), 1.0f));
    //float scalingFactor = 1.0f;
    float dzdcX = scalingFactor;
    float dzdcY = float(0.0);

    float maxRadius = max(abs(dx), abs(dy));
    float maxRadiusSq = maxRadius * maxRadius;

    while (iter < n_iterations) {
        const MattReferenceSingleIter<float> *curIter = &PerturbFloat.iters[RefIteration];

        const float DeltaSubNXOrig = DeltaSubNX;
        const float DeltaSubNYOrig = DeltaSubNY;

        const float tempSubX = curIter->x * 2 + DeltaSubNXOrig;
        const float tempSubY = curIter->y * 2 + DeltaSubNYOrig;

        ++RefIteration;
        curIter = &PerturbFloat.iters[RefIteration];

        DeltaSubNX =
            DeltaSubNXOrig * tempSubX -
            DeltaSubNYOrig * tempSubY +
            DeltaSub0X;
        DeltaSubNY =
            DeltaSubNXOrig * tempSubY +
            DeltaSubNYOrig * tempSubX +
            DeltaSub0Y;

        const float tempZX = curIter->x + DeltaSubNX;
        const float tempZY = curIter->y + DeltaSubNY;
        const float zn_size = tempZX * tempZX + tempZY * tempZY;
        const float normDeltaSubN = DeltaSubNX * DeltaSubNX + DeltaSubNY * DeltaSubNY;

        if (zn_size > 256) {
            break;
        }

        // Just finds the interesting Misiurewicz points.  Breaks so they're colored differently
        if constexpr (Periodic) {
            auto n3 = maxRadiusSq * (dzdcX * dzdcX + dzdcY * dzdcY);
            if (zn_size >= n3) {
                // dzdc = dzdc * 2.0 * z + ScalingFactor;
                // dzdc = dzdc * 2.0 * tempZ + ScalingFactor;
                // dzdc = (dzdcX + dzdcY * i) * 2.0 * (tempZX + tempZY * i) + ScalingFactor;
                // dzdc = (dzdcX * 2.0 + dzdcY * i * 2.0) * (tempZX + tempZY * i) + ScalingFactor;
                // dzdc = (dzdcX * 2.0) * tempZX +
                //        (dzdcX * 2.0) * (tempZY * i) +
                //        (dzdcY * i * 2.0) * tempZX +
                //        (dzdcY * i * 2.0) * tempZY * i
                //
                // dzdcX = (dzdcX * 2.0) * tempZX -
                //         (dzdcY * 2.0) * tempZY
                // dzdcY = (dzdcX * 2.0) * (tempZY) +
                //         (dzdcY * 2.0) * tempZX
                auto dzdcXOrig = dzdcX;
                dzdcX = 2.0f * tempZX * dzdcX - 2.0f * tempZY * dzdcY + scalingFactor;
                dzdcY = 2.0f * tempZY * dzdcXOrig + 2.0f * tempZX * dzdcY;
            }
            else {
                //iter = n_iterations;
                break;
            }
        }

        if (zn_size < normDeltaSubN ||
            RefIteration == MaxRefIteration) {
            DeltaSubNX = tempZX;
            DeltaSubNY = tempZY;
            RefIteration = 0;
        }

        ++iter;
    }

    iter_matrix[idx] = (uint32_t)iter;
}

template<class T>
__global__
void mandel_1x_float_perturb_scaled(uint32_t* iter_matrix,
    MattPerturbSingleResults<float> PerturbFloat,
    MattPerturbSingleResults<T> PerturbDouble,
    int width,
    int height,
    T cx,
    T cy,
    T dx,
    T dy,
    T centerX,
    T centerY,
    uint32_t n_iterations)
{
    int X = blockIdx.x * blockDim.x + threadIdx.x;
    int Y = blockIdx.y * blockDim.y + threadIdx.y;
    const float LARGE_MANTISSA = 1e30;

    if (X >= width || Y >= height)
        return;

    size_t idx = width * Y + X;

    if (iter_matrix[idx] != 0) {
        return;
    }

    size_t iter = 0;
    size_t RefIteration = 0;
    T DeltaReal = dx * X - centerX;
    HdrReduce(DeltaReal);

    T DeltaImaginary = -dy * Y - centerY;
    HdrReduce(DeltaImaginary);

    // DeltaSubNWX = 2 * DeltaSubNWX * PerturbFloat.x[RefIteration] - 2 * DeltaSubNWY * PerturbFloat.y[RefIteration] +
    //               S * DeltaSubNWX * DeltaSubNWX - S * DeltaSubNWY * DeltaSubNWY +
    //               dX
    // DeltaSubNWY = 2 * DeltaSubNWX * PerturbFloat.y[RefIteration] + 2 * DeltaSubNWY * PerturbFloat.x[RefIteration] +
    //               2 * S * DeltaSubNWX * DeltaSubNWY +
    //               dY
    // 
    // wrn = (2 * Xr + wr * s) * wr - (2 * Xi + wi * s) * wi + ur;
    //     = 2 * Xr * wr + wr * wr * s - 2 * Xi * wi - wi * wi * s + ur;
    // win = 2 * ((Xr + wr * s) * wi + Xi * wr) + ui;
    //     = 2 * (Xr * wi + wr * s * wi + Xi * wr) + ui;
    //     = 2 * Xr * wi + 2 * wr * s * wi + 2 * Xi * wr + ui;

    T S = HdrSqrt(DeltaReal * DeltaReal + DeltaImaginary * DeltaImaginary);
    HdrReduce(S);

    //double S = 1;
    float DeltaSub0DX = (float)(DeltaReal / S);
    float DeltaSub0DY = (float)(DeltaImaginary / S);
    float DeltaSubNWX = 0;
    float DeltaSubNWY = 0;

    float s = (float)S;
    float twos = 2 * s;
    const float w2threshold = exp(log(LARGE_MANTISSA) / 2);
    size_t MaxRefIteration = PerturbFloat.size - 1;

    T TwoFiftySix = T(256.0);

    while (iter < n_iterations) {
        const MattReferenceSingleIter<float> *curFloatIter = &PerturbFloat.iters[RefIteration];
        const MattReferenceSingleIter<T> *curDoubleIter = &PerturbDouble.iters[RefIteration];

        if (curFloatIter->bad == false) {
            const float DeltaSubNWXOrig = DeltaSubNWX;
            const float DeltaSubNWYOrig = DeltaSubNWY;

            DeltaSubNWX =
                DeltaSubNWXOrig * curFloatIter->x * 2 -
                DeltaSubNWYOrig * curFloatIter->y * 2 +
                s * DeltaSubNWXOrig * DeltaSubNWXOrig - s * DeltaSubNWYOrig * DeltaSubNWYOrig +
                DeltaSub0DX;

            DeltaSubNWY =
                DeltaSubNWXOrig * (curFloatIter->y * 2 + twos * DeltaSubNWYOrig) +
                DeltaSubNWYOrig * curFloatIter->x * 2 +
                DeltaSub0DY;

            ++RefIteration;
            curFloatIter = &PerturbFloat.iters[RefIteration];
            curDoubleIter = &PerturbDouble.iters[RefIteration];

            const float tempZX =
                curFloatIter->x + DeltaSubNWX * s; // Xxrd

            const float tempZY =
                curFloatIter->y + DeltaSubNWY * s; // Xxid

            const float zn_size =
                tempZX * tempZX + tempZY * tempZY;

            const float DeltaSubNWXSquared = DeltaSubNWX * DeltaSubNWX;
            const float DeltaSubNWYSquared = DeltaSubNWY * DeltaSubNWY;
            const float w2 = DeltaSubNWXSquared + DeltaSubNWYSquared;
            const float normDeltaSubN = w2 * s * s;

            T DoubleTempZX;
            T DoubleTempZY;

            const bool zn_size_OK = (zn_size < 256.0f);
            const bool test1a = (zn_size < normDeltaSubN);
            const bool test1b = (RefIteration == MaxRefIteration);
            const bool test1ab = test1a || (test1b && zn_size_OK);
            const bool testw2 = (w2 >= w2threshold) && zn_size_OK;
            const bool none = !test1ab && !testw2 && zn_size_OK;

            if (none) {
                ++iter;
                continue;
            } else if (test1ab) {
                DoubleTempZX = (curDoubleIter->x + (T)DeltaSubNWX * S); // Xxrd, xr
                //HdrReduce(DoubleTempZX);
                DoubleTempZY = (curDoubleIter->y + (T)DeltaSubNWY * S); // Xxid, xi
                //HdrReduce(DoubleTempZY);

                RefIteration = 0;
                S = HdrSqrt(DoubleTempZX * DoubleTempZX + DoubleTempZY * DoubleTempZY);
                HdrReduce(S);
                s = (float)S;
                twos = 2 * s;

                DeltaSub0DX = (float)(DeltaReal / S);
                DeltaSub0DY = (float)(DeltaImaginary / S);
                DeltaSubNWX = (float)(DoubleTempZX / S);
                DeltaSubNWY = (float)(DoubleTempZY / S);

                ++iter;
                continue;
            }
            else if (testw2)
            {
                DoubleTempZX = (T)DeltaSubNWX * S;
                //HdrReduce(DoubleTempZX);
                DoubleTempZY = (T)DeltaSubNWY * S;
                //HdrReduce(DoubleTempZY);

                S = HdrSqrt(DoubleTempZX * DoubleTempZX + DoubleTempZY * DoubleTempZY);
                HdrReduce(S);
                s = (float)S;
                twos = 2 * s;

                DeltaSub0DX = (float)(DeltaReal / S);
                DeltaSub0DY = (float)(DeltaImaginary / S);
                DeltaSubNWX = (float)(DoubleTempZX / S);
                DeltaSubNWY = (float)(DoubleTempZY / S);

                ++iter;
                continue;
            }
            else {
                // zn_size fail
                break;
            }
        } else {
            // Do full iteration at double precision
            T DeltaSubNWXOrig = (T)DeltaSubNWX;
            T DeltaSubNWYOrig = (T)DeltaSubNWY;

            T DoubleTempDeltaSubNWX = DeltaSubNWXOrig * curDoubleIter->x * 2;
            //HdrReduce(DoubleTempDeltaSubNWX);
            DoubleTempDeltaSubNWX -= DeltaSubNWYOrig * curDoubleIter->y * 2;
            //HdrReduce(DoubleTempDeltaSubNWX);
            DoubleTempDeltaSubNWX += S * DeltaSubNWXOrig * DeltaSubNWXOrig;
            //HdrReduce(DoubleTempDeltaSubNWX);
            DoubleTempDeltaSubNWX -= S * DeltaSubNWYOrig * DeltaSubNWYOrig;
            //HdrReduce(DoubleTempDeltaSubNWX);
            DoubleTempDeltaSubNWX += DeltaReal / S;
            HdrReduce(DoubleTempDeltaSubNWX);

            T DoubleTempDeltaSubNWY = DeltaSubNWXOrig * (curDoubleIter->y * 2 + T(2) * S * DeltaSubNWYOrig);
            //HdrReduce(DoubleTempDeltaSubNWY);
            DoubleTempDeltaSubNWY += DeltaSubNWYOrig * curDoubleIter->x * 2;
            //HdrReduce(DoubleTempDeltaSubNWY);
            DoubleTempDeltaSubNWY += DeltaImaginary / S;
            HdrReduce(DoubleTempDeltaSubNWY);

            ++RefIteration;
            curFloatIter = &PerturbFloat.iters[RefIteration];
            curDoubleIter = &PerturbDouble.iters[RefIteration];

            const T tempZX =
                curDoubleIter->x +
                DoubleTempDeltaSubNWX * S; // Xxrd

            const T tempZY =
                curDoubleIter->y +
                DoubleTempDeltaSubNWY * S; // Xxid

            T zn_size =
                tempZX * tempZX + tempZY * tempZY;
            HdrReduce(zn_size);

            if (zn_size > TwoFiftySix) {
                break;
            }

            const T TwoS = S * S;
            T normDeltaSubN =
                DoubleTempDeltaSubNWX * DoubleTempDeltaSubNWX * TwoS +
                DoubleTempDeltaSubNWY * DoubleTempDeltaSubNWY * TwoS;
            HdrReduce(normDeltaSubN);

            T DeltaSubNWXNew;
            T DeltaSubNWYNew;

            if (zn_size < normDeltaSubN ||
                RefIteration == MaxRefIteration) {
                DeltaSubNWXNew = (curDoubleIter->x + DoubleTempDeltaSubNWX * S); // Xxrd, xr
                DeltaSubNWYNew = (curDoubleIter->y + DoubleTempDeltaSubNWY * S); // Xxid, xi

                RefIteration = 0;
            }
            else {
                DeltaSubNWXNew = DoubleTempDeltaSubNWX * S;
                DeltaSubNWYNew = DoubleTempDeltaSubNWY * S;
            }

            S = HdrSqrt(DeltaSubNWXNew * DeltaSubNWXNew + DeltaSubNWYNew * DeltaSubNWYNew);
            HdrReduce(S);
            s = (float)S;
            twos = 2 * s;

            DeltaSub0DX = (float)(DeltaReal / S);
            DeltaSub0DY = (float)(DeltaImaginary / S);
            DeltaSubNWX = (float)(DeltaSubNWXNew / S);
            DeltaSubNWY = (float)(DeltaSubNWYNew / S);
        }

        ++iter;
    }

    iter_matrix[idx] = (uint32_t)iter;
}

template<int32_t LM2>
__global__
void mandel_1x_float_perturb_scaled_bla(uint32_t* iter_matrix,
    MattPerturbSingleResults<float> PerturbFloat,
    MattPerturbSingleResults<double> PerturbDouble,
    GPU_BLAS<double, BLA<double>, LM2> doubleBlas,
    int width,
    int height,
    double cx,
    double cy,
    double dx,
    double dy,
    double centerX,
    double centerY,
    uint32_t n_iterations)
{
    int X = blockIdx.x * blockDim.x + threadIdx.x;
    int Y = blockIdx.y * blockDim.y + threadIdx.y;
    const float LARGE_MANTISSA = 1e30;

    if (X >= width || Y >= height)
        return;

    size_t idx = width * Y + X;

    if (iter_matrix[idx] != 0) {
        return;
    }

    using GPUBLA_TYPE = BLA<double>;
    char __shared__ SharedMem[sizeof(SharedMemStruct<double>)];
    auto* shared =
        reinterpret_cast<SharedMemStruct<double>*>(SharedMem);

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        GPUBLA_TYPE** elts = doubleBlas.GetB();

        for (size_t i = 0; i < doubleBlas.m_NumLevels; i++) {
            shared->altB[i] = elts[i];
        }
    }

    __syncthreads();

    size_t iter = 0;
    size_t RefIteration = 0;
    const double DeltaReal = dx * X - centerX;
    const double DeltaImaginary = -dy * Y - centerY;
    double S = sqrt(DeltaReal * DeltaReal + DeltaImaginary * DeltaImaginary);
    float DeltaSub0DX = (float)(DeltaReal / S);
    float DeltaSub0DY = (float)(DeltaImaginary / S);
    float DeltaSubNWX = 0;
    float DeltaSubNWY = 0;

    double DeltaNormSquared = 0;
    
    float s = (float)S;
    float twos = 2 * s;
    const float w2threshold = exp(log(LARGE_MANTISSA) / 2);
    size_t MaxRefIteration = PerturbFloat.size - 1;

    while (iter < n_iterations) {
        const MattReferenceSingleIter<float> *curFloatIter = &PerturbFloat.iters[RefIteration];
        const MattReferenceSingleIter<double> *curDoubleIter = &PerturbDouble.iters[RefIteration];

        double DeltaSubNX = DeltaSubNWX * S;
        double DeltaSubNY = DeltaSubNWY * S;

        const BLA<double>* b = nullptr;

        b = doubleBlas.LookupBackwards(shared->altB, RefIteration, DeltaNormSquared);
        if (b != nullptr) {
            for (;;) {
                int l = b->getL();

                // TODO this first RefIteration + l check bugs me
                if (RefIteration + l >= PerturbDouble.size) {
                    break;
                }

                if (iter + l >= n_iterations) {
                    break;
                }

                iter += l;
                RefIteration += l;

                b->getValue(DeltaSubNX, DeltaSubNY, DeltaReal, DeltaImaginary);

                curDoubleIter = &PerturbDouble.iters[RefIteration];
                const double tempZX = curDoubleIter->x + DeltaSubNX;
                const double tempZY = curDoubleIter->y + DeltaSubNY;
                const double normSquared = tempZX * tempZX + tempZY * tempZY;
                DeltaNormSquared = DeltaSubNX * DeltaSubNX + DeltaSubNY * DeltaSubNY;

                if (normSquared > 256) {
                    break;
                }

                if (normSquared < DeltaNormSquared ||
                    RefIteration >= PerturbFloat.size - 1) {
                    DeltaSubNX = tempZX;
                    DeltaSubNY = tempZY;
                    DeltaNormSquared = normSquared;
                    RefIteration = 0;
                }

                b = doubleBlas.LookupBackwards(shared->altB, RefIteration, DeltaNormSquared);
                if (b == nullptr) {
                    double DoubleTempZX = DeltaSubNX;
                    double DoubleTempZY = DeltaSubNY;

                    DeltaNormSquared = DoubleTempZX * DoubleTempZX + DoubleTempZY * DoubleTempZY; // Yay
                    S = sqrt(DeltaNormSquared);
                    s = (float)S;
                    twos = 2 * s;

                    DeltaSub0DX = (float)(DeltaReal / S);
                    DeltaSub0DY = (float)(DeltaImaginary / S);
                    DeltaSubNWX = (float)(DoubleTempZX / S);
                    DeltaSubNWY = (float)(DoubleTempZY / S);
                    break;
                }
            }
        }

        if (iter >= n_iterations) {
            break;
        }

        curFloatIter = &PerturbFloat.iters[RefIteration];
        curDoubleIter = &PerturbDouble.iters[RefIteration];

        if (curFloatIter->bad == false) {
            const float DeltaSubNWXOrig = DeltaSubNWX;
            const float DeltaSubNWYOrig = DeltaSubNWY;

            DeltaSubNWX =
                DeltaSubNWXOrig * curFloatIter->x * 2 -
                DeltaSubNWYOrig * curFloatIter->y * 2 +
                s * DeltaSubNWXOrig * DeltaSubNWXOrig - s * DeltaSubNWYOrig * DeltaSubNWYOrig +
                DeltaSub0DX;

            DeltaSubNWY =
                DeltaSubNWXOrig * (curFloatIter->y * 2 + twos * DeltaSubNWYOrig) +
                DeltaSubNWYOrig * curFloatIter->x * 2 +
                DeltaSub0DY;

            ++RefIteration;
            curFloatIter = &PerturbFloat.iters[RefIteration];
            curDoubleIter = &PerturbDouble.iters[RefIteration];

            const float tempZX =
                curFloatIter->x + DeltaSubNWX * s; // Xxrd

            const float tempZY =
                curFloatIter->y + DeltaSubNWY * s; // Xxid

            const float zn_size =
                tempZX * tempZX + tempZY * tempZY;

            const float DeltaSubNWXSquared = DeltaSubNWX * DeltaSubNWX;
            const float DeltaSubNWYSquared = DeltaSubNWY * DeltaSubNWY;
            const float w2 = DeltaSubNWXSquared + DeltaSubNWYSquared;
            const float normDeltaSubN = w2 * s * s;
            DeltaNormSquared = normDeltaSubN; // Yay

            double DoubleTempZX;
            double DoubleTempZY;

            const bool zn_size_OK = (zn_size < 256.0f);
            const bool test1a = (zn_size < normDeltaSubN);
            const bool test1b = (RefIteration == MaxRefIteration);
            const bool test1ab = test1a || (test1b && zn_size_OK);
            const bool testw2 = (w2 >= w2threshold) && zn_size_OK;
            const bool none = !test1ab && !testw2 && zn_size_OK;

            if (none) {
                ++iter;
                continue;
            } else if (test1ab) {
                DoubleTempZX = (curDoubleIter->x + (double)DeltaSubNWX * S); // Xxrd, xr
                DoubleTempZY = (curDoubleIter->y + (double)DeltaSubNWY * S); // Xxid, xi

                RefIteration = 0;

                DeltaNormSquared = DoubleTempZX * DoubleTempZX + DoubleTempZY * DoubleTempZY; // Yay
                S = sqrt(DeltaNormSquared);
                s = (float)S;
                twos = 2 * s;

                DeltaSub0DX = (float)(DeltaReal / S);
                DeltaSub0DY = (float)(DeltaImaginary / S);
                DeltaSubNWX = (float)(DoubleTempZX / S);
                DeltaSubNWY = (float)(DoubleTempZY / S);

                ++iter;
                continue;
            }
            else if (testw2)
            {
                DoubleTempZX = DeltaSubNWX * S;
                DoubleTempZY = DeltaSubNWY * S;

                DeltaNormSquared = DoubleTempZX * DoubleTempZX + DoubleTempZY * DoubleTempZY; // Yay
                S = sqrt(DeltaNormSquared);
                s = (float)S;
                twos = 2 * s;

                DeltaSub0DX = (float)(DeltaReal / S);
                DeltaSub0DY = (float)(DeltaImaginary / S);
                DeltaSubNWX = (float)(DoubleTempZX / S);
                DeltaSubNWY = (float)(DoubleTempZY / S);

                ++iter;
                continue;
            }
            else {
                // zn_size fail
                break;
            }
        } else {
            // Do full iteration at double precision
            double DeltaSubNWXOrig = DeltaSubNWX;
            double DeltaSubNWYOrig = DeltaSubNWY;

            const double DoubleTempDeltaSubNWX =
                DeltaSubNWXOrig * curDoubleIter->x * 2 -
                DeltaSubNWYOrig * curDoubleIter->y * 2 +
                S * DeltaSubNWXOrig * DeltaSubNWXOrig - S * DeltaSubNWYOrig * DeltaSubNWYOrig +
                DeltaReal / S;

            const double DoubleTempDeltaSubNWY =
                DeltaSubNWXOrig * (curDoubleIter->y * 2 + 2 * S * DeltaSubNWYOrig) +
                DeltaSubNWYOrig * curDoubleIter->x * 2 +
                DeltaImaginary / S;

            ++RefIteration;
            curFloatIter = &PerturbFloat.iters[RefIteration];
            curDoubleIter = &PerturbDouble.iters[RefIteration];

            const double tempZX =
                curDoubleIter->x +
                DoubleTempDeltaSubNWX * S; // Xxrd

            const double tempZY =
                curDoubleIter->y +
                DoubleTempDeltaSubNWY * S; // Xxid

            const double zn_size =
                tempZX * tempZX + tempZY * tempZY;

            if (zn_size > 256.0) {
                break;
            }

            const double TwoS = S * S;
            const double normDeltaSubN =
                DoubleTempDeltaSubNWX * DoubleTempDeltaSubNWX * TwoS +
                DoubleTempDeltaSubNWY * DoubleTempDeltaSubNWY * TwoS;
            DeltaNormSquared = normDeltaSubN; // Yay

            double DeltaSubNWXNew;
            double DeltaSubNWYNew;

            if (zn_size < normDeltaSubN ||
                RefIteration == MaxRefIteration) {
                DeltaSubNWXNew = (curDoubleIter->x + DoubleTempDeltaSubNWX * S); // Xxrd, xr
                DeltaSubNWYNew = (curDoubleIter->y + DoubleTempDeltaSubNWY * S); // Xxid, xi

                RefIteration = 0;
            }
            else {
                DeltaSubNWXNew = DoubleTempDeltaSubNWX * S;
                DeltaSubNWYNew = DoubleTempDeltaSubNWY * S;
            }

            DeltaNormSquared = DeltaSubNWXNew * DeltaSubNWXNew + DeltaSubNWYNew * DeltaSubNWYNew; // Yay
            S = sqrt(DeltaNormSquared);
            s = (float)S;
            twos = 2 * s;

            DeltaSub0DX = (float)(DeltaReal / S);
            DeltaSub0DY = (float)(DeltaImaginary / S);
            DeltaSubNWX = (float)(DeltaSubNWXNew / S);
            DeltaSubNWY = (float)(DeltaSubNWYNew / S);
        }

        ++iter;
    }

    iter_matrix[idx] = (uint32_t)iter;
}

__global__
void mandel_2x_float_perturb_scaled(uint32_t* iter_matrix,
    MattPerturbSingleResults<dblflt> PerturbDoubleFlt,
    MattPerturbSingleResults<double> PerturbDouble,
    int width,
    int height,
    double cx,
    double cy,
    double dx,
    double dy,
    double centerX,
    double centerY,
    uint32_t n_iterations)
{
    int X = blockIdx.x * blockDim.x + threadIdx.x;
    int Y = blockIdx.y * blockDim.y + threadIdx.y;
    const float LARGE_MANTISSA = 1e30;

    if (X >= width || Y >= height)
        return;

    //size_t idx = width * (height - Y - 1) + X;
    size_t idx = width * Y + X;

    if (iter_matrix[idx] != 0) {
        return;
    }

    size_t iter = 0;
    size_t RefIteration = 0;
    const double DeltaReal = dx * X - centerX;
    const double DeltaImaginary = -dy * Y - centerY;

    // DeltaSubNWX = 2 * DeltaSubNWX * PerturbDoubleFlt.x[RefIteration] - 2 * DeltaSubNWY * PerturbDoubleFlt.y[RefIteration] +
    //               S * DeltaSubNWX * DeltaSubNWX - S * DeltaSubNWY * DeltaSubNWY +
    //               dX
    // DeltaSubNWY = 2 * DeltaSubNWX * PerturbDoubleFlt.y[RefIteration] + 2 * DeltaSubNWY * PerturbDoubleFlt.x[RefIteration] +
    //               2 * S * DeltaSubNWX * DeltaSubNWY +
    //               dY
    // 
    // wrn = (2 * Xr + wr * s) * wr - (2 * Xi + wi * s) * wi + ur;
    //     = 2 * Xr * wr + wr * wr * s - 2 * Xi * wi - wi * wi * s + ur;
    // win = 2 * ((Xr + wr * s) * wi + Xi * wr) + ui;
    //     = 2 * (Xr * wi + wr * s * wi + Xi * wr) + ui;
    //     = 2 * Xr * wi + 2 * wr * s * wi + 2 * Xi * wr + ui;

    double S = sqrt(DeltaReal * DeltaReal + DeltaImaginary * DeltaImaginary);

    //double S = 1;
    dblflt DeltaSub0DX = double_to_dblflt(DeltaReal / S);
    dblflt DeltaSub0DY = double_to_dblflt(DeltaImaginary / S);
    dblflt DeltaSubNWX = add_float_to_dblflt(0, 0);
    dblflt DeltaSubNWY = add_float_to_dblflt(0, 0);

    dblflt s = double_to_dblflt(S);
    dblflt twos = add_dblflt(s, s);
    const dblflt w2threshold = double_to_dblflt(exp(log(LARGE_MANTISSA) / 2));
    size_t MaxRefIteration = PerturbDoubleFlt.size - 1;

    while (iter < n_iterations) {
        const MattReferenceSingleIter<dblflt>* curDblFloatIter = &PerturbDoubleFlt.iters[RefIteration];
        const MattReferenceSingleIter<double>* curDoubleIter = &PerturbDouble.iters[RefIteration];

        if (curDblFloatIter->bad == false) {
            const dblflt DeltaSubNWXOrig = DeltaSubNWX;
            const dblflt DeltaSubNWYOrig = DeltaSubNWY;

            const dblflt Two = add_float_to_dblflt(0, 2);
            const dblflt tempX = mul_dblflt(curDblFloatIter->x, Two);
            const dblflt tempY = mul_dblflt(curDblFloatIter->y, Two);

            DeltaSubNWX = mul_dblflt(DeltaSubNWXOrig, tempX);
            DeltaSubNWX = sub_dblflt(DeltaSubNWX, mul_dblflt(DeltaSubNWYOrig, tempY));
            DeltaSubNWX = add_dblflt(DeltaSubNWX, mul_dblflt(mul_dblflt(s, DeltaSubNWXOrig), DeltaSubNWXOrig));
            DeltaSubNWX = sub_dblflt(DeltaSubNWX, mul_dblflt(mul_dblflt(s, DeltaSubNWYOrig), DeltaSubNWYOrig));
            DeltaSubNWX = add_dblflt(DeltaSubNWX, DeltaSub0DX);

            DeltaSubNWY = mul_dblflt(DeltaSubNWXOrig, (add_dblflt(tempY, mul_dblflt(twos, DeltaSubNWYOrig))));
            DeltaSubNWY = add_dblflt(DeltaSubNWY, mul_dblflt(DeltaSubNWYOrig, tempX));
            DeltaSubNWY = add_dblflt(DeltaSubNWY, DeltaSub0DY);

            ++RefIteration;
            curDblFloatIter = &PerturbDoubleFlt.iters[RefIteration];
            curDoubleIter = &PerturbDouble.iters[RefIteration];

            const dblflt tempZX =
                add_dblflt(curDblFloatIter->x, mul_dblflt(DeltaSubNWX, s)); // Xxrd

            const dblflt tempZY =
                add_dblflt(curDblFloatIter->y, mul_dblflt(DeltaSubNWY, s)); // Xxid

            const dblflt zn_size =
                add_dblflt(sqr_dblflt(tempZX), sqr_dblflt(tempZY));

            const dblflt DeltaSubNWXSquared = sqr_dblflt(DeltaSubNWX);
            const dblflt DeltaSubNWYSquared = sqr_dblflt(DeltaSubNWY);
            const dblflt w2 = add_dblflt(DeltaSubNWXSquared, DeltaSubNWYSquared);
            const dblflt normDeltaSubN = mul_dblflt(w2, sqr_dblflt(s));

            double DoubleTempZX;
            double DoubleTempZY;

            const bool zn_size_OK = (zn_size.y < 256.0f);
            const bool test1a = (zn_size.y < normDeltaSubN.y);
            const bool test1b = (RefIteration == MaxRefIteration);
            const bool test1ab = test1a || (test1b && zn_size_OK);
            const bool testw2 = (w2.y >= w2threshold.y) && zn_size_OK;
            const bool none = !test1ab && !testw2 && zn_size_OK;

            if (none) {
                ++iter;
                continue;
            }
            else if (test1ab) {
                DoubleTempZX = (curDoubleIter->x + dblflt_to_double(DeltaSubNWX) * S); // Xxrd, xr
                DoubleTempZY = (curDoubleIter->y + dblflt_to_double(DeltaSubNWY) * S); // Xxid, xi

                RefIteration = 0;

                S = sqrt(DoubleTempZX * DoubleTempZX + DoubleTempZY * DoubleTempZY);
                s = double_to_dblflt(S);
                twos = add_dblflt(s, s);

                DeltaSub0DX = double_to_dblflt(DeltaReal / S);
                DeltaSub0DY = double_to_dblflt(DeltaImaginary / S);
                DeltaSubNWX = double_to_dblflt(DoubleTempZX / S);
                DeltaSubNWY = double_to_dblflt(DoubleTempZY / S);

                ++iter;
                continue;
            }
            else if (testw2)
            {
                DoubleTempZX = dblflt_to_double(DeltaSubNWX) * S;
                DoubleTempZY = dblflt_to_double(DeltaSubNWY) * S;

                S = sqrt(DoubleTempZX * DoubleTempZX + DoubleTempZY * DoubleTempZY);
                s = double_to_dblflt(S);
                twos = add_dblflt(s, s);

                DeltaSub0DX = double_to_dblflt(DeltaReal / S);
                DeltaSub0DY = double_to_dblflt(DeltaImaginary / S);
                DeltaSubNWX = double_to_dblflt(DoubleTempZX / S);
                DeltaSubNWY = double_to_dblflt(DoubleTempZY / S);

                ++iter;
                continue;
            }
            else {
                // zn_size fail
                break;
            }
        }
        else {
            // Do full iteration at double precision
            double DeltaSubNWXOrig = dblflt_to_double(DeltaSubNWX);
            double DeltaSubNWYOrig = dblflt_to_double(DeltaSubNWY);

            const double DoubleTempDeltaSubNWX =
                DeltaSubNWXOrig * curDoubleIter->x * 2 -
                DeltaSubNWYOrig * curDoubleIter->y * 2 +
                S * DeltaSubNWXOrig * DeltaSubNWXOrig - S * DeltaSubNWYOrig * DeltaSubNWYOrig +
                DeltaReal / S;

            const double DoubleTempDeltaSubNWY =
                DeltaSubNWXOrig * (curDoubleIter->y * 2 + 2 * S * DeltaSubNWYOrig) +
                DeltaSubNWYOrig * curDoubleIter->x * 2 +
                DeltaImaginary / S;

            ++RefIteration;
            curDblFloatIter = &PerturbDoubleFlt.iters[RefIteration];
            curDoubleIter = &PerturbDouble.iters[RefIteration];

            const double tempZX =
                curDoubleIter->x +
                DoubleTempDeltaSubNWX * S; // Xxrd

            const double tempZY =
                curDoubleIter->y +
                DoubleTempDeltaSubNWY * S; // Xxid

            const double zn_size =
                tempZX * tempZX + tempZY * tempZY;

            if (zn_size > 256.0) {
                break;
            }

            const double TwoS = S * S;
            const double normDeltaSubN =
                DoubleTempDeltaSubNWX * DoubleTempDeltaSubNWX * TwoS +
                DoubleTempDeltaSubNWY * DoubleTempDeltaSubNWY * TwoS;

            double DeltaSubNWXNew;
            double DeltaSubNWYNew;

            if (zn_size < normDeltaSubN ||
                RefIteration == MaxRefIteration) {
                DeltaSubNWXNew = (curDoubleIter->x + DoubleTempDeltaSubNWX * S); // Xxrd, xr
                DeltaSubNWYNew = (curDoubleIter->y + DoubleTempDeltaSubNWY * S); // Xxid, xi

                RefIteration = 0;
            }
            else {
                DeltaSubNWXNew = DoubleTempDeltaSubNWX * S;
                DeltaSubNWYNew = DoubleTempDeltaSubNWY * S;
            }

            S = sqrt(DeltaSubNWXNew * DeltaSubNWXNew + DeltaSubNWYNew * DeltaSubNWYNew);
            s = double_to_dblflt(S);
            twos = add_dblflt(s, s);

            DeltaSub0DX = double_to_dblflt(DeltaReal / S);
            DeltaSub0DY = double_to_dblflt(DeltaImaginary / S);
            DeltaSubNWX = double_to_dblflt(DeltaSubNWXNew / S);
            DeltaSubNWY = double_to_dblflt(DeltaSubNWYNew / S);
        }

        ++iter;
    }

    iter_matrix[idx] = (uint32_t)iter;
}

GPURenderer::GPURenderer() {
    ClearLocals();
}

GPURenderer::~GPURenderer() {
    ResetMemory();
}

void GPURenderer::ResetMemory() {
    if (iter_matrix_cu != nullptr) {
        hipFree(iter_matrix_cu);
    }

    ClearLocals();
}

void GPURenderer::ClearLocals() {
    // Assumes memory is freed
    iter_matrix_cu = nullptr;

    width = 0;
    height = 0;
    local_width = 0;
    local_height = 0;
    w_block = 0;
    h_block = 0;
    N_cu = 0;
}

void GPURenderer::ClearMemory() {
    if (iter_matrix_cu == nullptr) {
        return;
    }

    hipMemset(iter_matrix_cu, 0, N_cu * sizeof(int));
}

uint32_t GPURenderer::InitializeMemory(
    size_t w,
    size_t h)
{
    if ((local_width == w) &&
        (local_height == h)) {
        return 0;
    }

    if (local_width % NB_THREADS_W != 0) {
        return 10000;
    }

    if (local_height % NB_THREADS_H != 0) {
        return 10001;
    }

    width = (uint32_t)w;
    height = (uint32_t)h;
    local_width = width;
    local_height = height;
    w_block = local_width / NB_THREADS_W;
    h_block = local_height / NB_THREADS_H;
    N_cu = w_block * NB_THREADS_W * h_block * NB_THREADS_H;

    if (iter_matrix_cu != nullptr) {
        hipFree(iter_matrix_cu);
    }

    hipError_t err = hipMallocManaged(&iter_matrix_cu, N_cu * sizeof(int), hipMemAttachGlobal);
    if (err != hipSuccess) {
        ClearLocals();
        return err;
    }

    ClearMemory();
    return 0;
}

uint32_t GPURenderer::Render(
    RenderAlgorithm algorithm,
    uint32_t* buffer,
    MattCoords cx,
    MattCoords cy,
    MattCoords dx,
    MattCoords dy,
    uint32_t n_iterations,
    int iteration_precision)
{
    if (iter_matrix_cu == nullptr) {
        return hipSuccess;
    }

    dim3 nb_blocks(w_block, h_block, 1);
    dim3 threads_per_block(NB_THREADS_W, NB_THREADS_H, 1);

    if (algorithm == RenderAlgorithm::Gpu1x64) {
        switch (iteration_precision) {
        case 1:
            mandel_1x_double<1> << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
                local_width, local_height, cx.doubleOnly, cy.doubleOnly, dx.doubleOnly, dy.doubleOnly,
                n_iterations);
            break;
        case 4:
            mandel_1x_double<4> << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
                local_width, local_height, cx.doubleOnly, cy.doubleOnly, dx.doubleOnly, dy.doubleOnly,
                n_iterations);
            break;
        case 8:
            mandel_1x_double<8> << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
                local_width, local_height, cx.doubleOnly, cy.doubleOnly, dx.doubleOnly, dy.doubleOnly,
                n_iterations);
            break;
        case 16:
            mandel_1x_double<16> << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
                local_width, local_height, cx.doubleOnly, cy.doubleOnly, dx.doubleOnly, dy.doubleOnly,
                n_iterations);
            break;
        default:
            break;
        }
    }
    else if (algorithm == RenderAlgorithm::Gpu2x64) {
        dbldbl cx2{ cx.dbl.head, cx.dbl.tail };
        dbldbl cy2{ cy.dbl.head, cy.dbl.tail };
        dbldbl dx2{ dx.dbl.head, dx.dbl.tail };
        dbldbl dy2{ dy.dbl.head, dy.dbl.tail };

        mandel_2x_double << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
            local_width, local_height, cx2, cy2, dx2, dy2,
            n_iterations);
    }
    else if (algorithm == RenderAlgorithm::Gpu4x64) {
        using namespace GQD;
        gqd_real cx2;
        cx2 = make_qd(cx.qdbl.v1, cx.qdbl.v2, cx.qdbl.v3, cx.qdbl.v4);

        gqd_real cy2;
        cy2 = make_qd(cy.qdbl.v1, cy.qdbl.v2, cy.qdbl.v3, cy.qdbl.v4);

        gqd_real dx2;
        dx2 = make_qd(dx.qdbl.v1, dx.qdbl.v2, dx.qdbl.v3, dx.qdbl.v4);

        gqd_real dy2;
        dy2 = make_qd(dy.qdbl.v1, dy.qdbl.v2, dy.qdbl.v3, dy.qdbl.v4);

        mandel_4x_double << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
            local_width, local_height, cx2, cy2, dx2, dy2,
            n_iterations);
    }
    else if (algorithm == RenderAlgorithm::Gpu1x32) {
        switch (iteration_precision) {
        case 1:
            mandel_1x_float<1> << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
                local_width, local_height, cx.floatOnly, cy.floatOnly, dx.floatOnly, dy.floatOnly,
                n_iterations);
            break;
        case 4:
            mandel_1x_float<4> << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
                local_width, local_height, cx.floatOnly, cy.floatOnly, dx.floatOnly, dy.floatOnly,
                n_iterations);
            break;
        case 8:
            mandel_1x_float<8> << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
                local_width, local_height, cx.floatOnly, cy.floatOnly, dx.floatOnly, dy.floatOnly,
                n_iterations);
            break;
        case 16:
            mandel_1x_float<16> << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
                local_width, local_height, cx.floatOnly, cy.floatOnly, dx.floatOnly, dy.floatOnly,
                n_iterations);
            break;
        default:
            break;
        }
    }
    else if (algorithm == RenderAlgorithm::Gpu2x32) {
        dblflt cx2{ cx.flt.x, cx.flt.y };
        dblflt cy2{ cy.flt.x, cy.flt.y };
        dblflt dx2{ dx.flt.x, dx.flt.y };
        dblflt dy2{ dy.flt.x, dy.flt.y };

        switch (iteration_precision) {
        case 1:
            mandel_2x_float<1> << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
                local_width, local_height, cx2, cy2, dx2, dy2,
                n_iterations);
            break;
        case 4:
            mandel_2x_float<4> << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
                local_width, local_height, cx2, cy2, dx2, dy2,
                n_iterations);
            break;
        case 8:
            mandel_2x_float<8> << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
                local_width, local_height, cx2, cy2, dx2, dy2,
                n_iterations);
            break;
        case 16:
            mandel_2x_float<16> << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
                local_width, local_height, cx2, cy2, dx2, dy2,
                n_iterations);
            break;
        default:
            break;
        }
    }
    else if (algorithm == RenderAlgorithm::Gpu4x32) {
        using namespace GQF;
        gqf_real cx2;
        cx2 = make_qf(cx.qflt.v1, cx.qflt.v2, cx.qflt.v3, cx.qflt.v4);

        gqf_real cy2;
        cy2 = make_qf(cy.qflt.v1, cy.qflt.v2, cy.qflt.v3, cy.qflt.v4);

        gqf_real dx2;
        dx2 = make_qf(dx.qflt.v1, dx.qflt.v2, dx.qflt.v3, dx.qflt.v4);

        gqf_real dy2;
        dy2 = make_qf(dy.qflt.v1, dy.qflt.v2, dy.qflt.v3, dy.qflt.v4);

        mandel_4x_float << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
            local_width, local_height, cx2, cy2, dx2, dy2,
            n_iterations);
    }
    else {
        return hipSuccess;
    }

    return ExtractIters(buffer);
}

uint32_t GPURenderer::RenderPerturbBLA(
    RenderAlgorithm algorithm,
    uint32_t* buffer,
    MattPerturbResults<float>* float_perturb,
    BLAS<float> * /*blas*/, // TODO
    MattCoords cx,
    MattCoords cy,
    MattCoords dx,
    MattCoords dy,
    MattCoords centerX,
    MattCoords centerY,
    uint32_t n_iterations,
    int /*iteration_precision*/)
{
    uint32_t result = hipSuccess;

    if (iter_matrix_cu == nullptr) {
        return result;
    }

    dim3 nb_blocks(w_block, h_block, 1);
    dim3 threads_per_block(NB_THREADS_W, NB_THREADS_H, 1);

    MattPerturbSingleResults<float> cudaResults(
        float_perturb->size,
        float_perturb->PeriodMaybeZero,
        float_perturb->iters);

    result = cudaResults.CheckValid();
    if (result != 0) {
        return result;
    }

    if (algorithm == RenderAlgorithm::Gpu1x32Perturbed) {
        mandel_1x_float_perturb<false> << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
            cudaResults,
            local_width, local_height, cx.floatOnly, cy.floatOnly, dx.floatOnly, dy.floatOnly,
            centerX.floatOnly, centerY.floatOnly,
            n_iterations);

        result = ExtractIters(buffer);
    }
    else if (algorithm == RenderAlgorithm::Gpu1x32PerturbedPeriodic) {
        mandel_1x_float_perturb<true> << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
            cudaResults,
            local_width, local_height, cx.floatOnly, cy.floatOnly, dx.floatOnly, dy.floatOnly,
            centerX.floatOnly, centerY.floatOnly,
            n_iterations);

        result = ExtractIters(buffer);
    }

    return result;
}

uint32_t GPURenderer::RenderPerturbLAv2(
    RenderAlgorithm algorithm,
    uint32_t* buffer,
    MattPerturbResults<HDRFloat<float>>* float_perturb,
    const LAReference &LaReference,
    MattCoords cx,
    MattCoords cy,
    MattCoords dx,
    MattCoords dy,
    MattCoords centerX,
    MattCoords centerY,
    uint32_t n_iterations)
{
    uint32_t result = hipSuccess;

    if (iter_matrix_cu == nullptr) {
        return result;
    }

    dim3 nb_blocks(w_block, h_block, 1);
    dim3 threads_per_block(NB_THREADS_W, NB_THREADS_H, 1);

    MattPerturbSingleResults<HDRFloat<float>> cudaResults(
        float_perturb->size,
        float_perturb->PeriodMaybeZero,
        float_perturb->iters);

    result = cudaResults.CheckValid();
    if (result != 0) {
        return result;
    }

    GPU_LAReference laReferenceCuda{LaReference};
    result = laReferenceCuda.CheckValid();
    if (result != 0) {
        return result;
    }

    if (algorithm == RenderAlgorithm::GpuHDRx32PerturbedLAv2) {
        mandel_1xHDR_InitStatics << <nb_blocks, threads_per_block >> > ();

        mandel_1xHDR_float_perturb_lav2<HDRFloat<float>> << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
            cudaResults, laReferenceCuda,
            local_width, local_height, cx.hdrflt, cy.hdrflt, dx.hdrflt, dy.hdrflt,
            centerX.hdrflt, centerY.hdrflt,
            n_iterations);

        result = ExtractIters(buffer);
    }

    return result;
}

uint32_t GPURenderer::RenderPerturbBLA(
    RenderAlgorithm algorithm,
    uint32_t* buffer,
    MattPerturbResults<double>* double_perturb,
    BLAS<double> *blas,
    MattCoords cx,
    MattCoords cy,
    MattCoords dx,
    MattCoords dy,
    MattCoords centerX,
    MattCoords centerY,
    uint32_t n_iterations,
    int /*iteration_precision*/)
{
    uint32_t result = hipSuccess;

    if (iter_matrix_cu == nullptr) {
        return result;
    }

    dim3 nb_blocks(w_block, h_block, 1);
    dim3 threads_per_block(NB_THREADS_W, NB_THREADS_H, 1);

    if (algorithm == RenderAlgorithm::Gpu1x64Perturbed) {
        MattPerturbSingleResults<double> cudaResults(
            double_perturb->size,
            double_perturb->PeriodMaybeZero,
            double_perturb->iters);

        result = cudaResults.CheckValid();
        if (result != 0) {
            return result;
        }

        mandel_1x_double_perturb << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
            cudaResults,
            local_width, local_height, cx.doubleOnly, cy.doubleOnly, dx.doubleOnly, dy.doubleOnly,
            centerX.doubleOnly, centerY.doubleOnly,
            n_iterations);

        result = ExtractIters(buffer);
    } else if (algorithm == RenderAlgorithm::Gpu1x64PerturbedBLA) {
        MattPerturbSingleResults<double> cudaResults(
            double_perturb->size,
            double_perturb->PeriodMaybeZero,
            double_perturb->iters);

        result = cudaResults.CheckValid();
        if (result != 0) {
            return result;
        }

        auto Run = [&]<int32_t LM2>() -> uint32_t {
            GPU_BLAS<double, BLA<double>, LM2> gpu_blas(blas->m_B);
            result = gpu_blas.CheckValid();
            if (result != 0) {
                return result;
            }

            mandel_1xHDR_InitStatics << <nb_blocks, threads_per_block >> > ();

            mandel_1x_double_perturb_bla<LM2> << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
                cudaResults,
                gpu_blas,
                local_width, local_height, cx.doubleOnly, cy.doubleOnly, dx.doubleOnly, dy.doubleOnly,
                centerX.doubleOnly, centerY.doubleOnly,
                n_iterations);

            return ExtractIters(buffer);
        };

        LargeSwitch
    }
    //else if (algorithm == RenderAlgorithm::Gpu2x32PerturbedScaled) {
    //    MattPerturbSingleResults<dblflt> cudaResults(
    //        Perturb->size,
    //        Perturb->PeriodMaybeZero,
    //        Perturb->iters);

    //    result = cudaResults.CheckValid();
    //    if (result != 0) {
    //        return result;
    //    }

    //    MattPerturbSingleResults<double> cudaResultsDouble(
    //        Perturb->size,
    //        Perturb->PeriodMaybeZero,
    //        Perturb->iters);

    //    result = cudaResultsDouble.CheckValid();
    //    if (result != 0) {
    //        return result;
    //    }

    //    mandel_2x_float_perturb_setup << <nb_blocks, threads_per_block >> > (cudaResults);

    //    mandel_2x_float_perturb_scaled << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
    //        cudaResults, cudaResultsDouble,
    //        local_width, local_height, cx.doubleOnly, cy.doubleOnly, dx.doubleOnly, dy.doubleOnly,
    //        centerX.doubleOnly, centerY.doubleOnly,
    //        n_iterations);

    //    result = ExtractIters(buffer);
    //}

    return result;
}

template<class T>
uint32_t GPURenderer::RenderPerturbBLA(
    RenderAlgorithm algorithm,
    uint32_t* buffer,
    MattPerturbResults<T>* double_perturb,
    MattPerturbResults<float>* float_perturb,
    BLAS<T>* blas,
    MattCoords cx,
    MattCoords cy,
    MattCoords dx,
    MattCoords dy,
    MattCoords centerX,
    MattCoords centerY,
    uint32_t n_iterations,
    int /*iteration_precision*/)
{
    uint32_t result = hipSuccess;

    if (iter_matrix_cu == nullptr) {
        return result;
    }

    dim3 nb_blocks(w_block, h_block, 1);
    dim3 threads_per_block(NB_THREADS_W, NB_THREADS_H, 1);

    MattPerturbSingleResults<float> cudaResults(
        float_perturb->size,
        float_perturb->PeriodMaybeZero,
        float_perturb->iters);

    result = cudaResults.CheckValid();
    if (result != 0) {
        return result;
    }

    MattPerturbSingleResults<T> cudaResultsDouble(
        double_perturb->size,
        float_perturb->PeriodMaybeZero,
        double_perturb->iters);

    result = cudaResultsDouble.CheckValid();
    if (result != 0) {
        return result;
    }

    if (algorithm == RenderAlgorithm::Gpu1x32PerturbedScaled) {
        if constexpr (std::is_same<T, double>::value) {
            mandel_1x_float_perturb_scaled<T> << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
                cudaResults, cudaResultsDouble,
                local_width, local_height, cx.doubleOnly, cy.doubleOnly, dx.doubleOnly, dy.doubleOnly,
                centerX.doubleOnly, centerY.doubleOnly,
                n_iterations);

            result = ExtractIters(buffer);
        }
    }
    else if (algorithm == RenderAlgorithm::GpuHDRx32PerturbedScaled) {
        if constexpr (std::is_same<T, HDRFloat<float>>::value) {
            mandel_1xHDR_InitStatics << <nb_blocks, threads_per_block >> > ();

            mandel_1x_float_perturb_scaled<T> << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
                cudaResults, cudaResultsDouble,
                local_width, local_height, cx.hdrflt, cy.hdrflt, dx.hdrflt, dy.hdrflt,
                centerX.hdrflt, centerY.hdrflt,
                n_iterations);

            result = ExtractIters(buffer);
        }
    } else if (algorithm == RenderAlgorithm::Gpu1x32PerturbedScaledBLA) {
        if constexpr (std::is_same<T, double>::value) {

            auto Run = [&]<int32_t LM2>() -> uint32_t {
                GPU_BLAS<double, BLA<double>, LM2> doubleGpuBlas(blas->m_B);
                result = doubleGpuBlas.CheckValid();
                if (result != 0) {
                    return result;
                }

                mandel_1xHDR_InitStatics << <nb_blocks, threads_per_block >> > ();

                mandel_1x_float_perturb_scaled_bla<LM2> << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
                    cudaResults, cudaResultsDouble, doubleGpuBlas,
                    local_width, local_height, cx.doubleOnly, cy.doubleOnly, dx.doubleOnly, dy.doubleOnly,
                    centerX.doubleOnly, centerY.doubleOnly,
                    n_iterations);

                return ExtractIters(buffer);
            };

            LargeSwitch
        }
    }

    return result;
}

template uint32_t GPURenderer::RenderPerturbBLA<double>(
    RenderAlgorithm algorithm,
    uint32_t* buffer,
    MattPerturbResults<double>* double_perturb,
    MattPerturbResults<float>* float_perturb,
    BLAS<double>* blas,
    MattCoords cx,
    MattCoords cy,
    MattCoords dx,
    MattCoords dy,
    MattCoords centerX,
    MattCoords centerY,
    uint32_t n_iterations,
    int /*iteration_precision*/
);

template uint32_t GPURenderer::RenderPerturbBLA<HDRFloat<float>>(
    RenderAlgorithm algorithm,
    uint32_t* buffer,
    MattPerturbResults<HDRFloat<float>>* double_perturb,
    MattPerturbResults<float>* float_perturb,
    BLAS<HDRFloat<float>>* blas,
    MattCoords cx,
    MattCoords cy,
    MattCoords dx,
    MattCoords dy,
    MattCoords centerX,
    MattCoords centerY,
    uint32_t n_iterations,
    int /*iteration_precision*/
);


uint32_t GPURenderer::RenderPerturbBLA(
    RenderAlgorithm algorithm,
    uint32_t* buffer,
    MattPerturbResults<dblflt>* dblflt_perturb,
    BLAS<dblflt>* /*blas*/,  // TODO
    MattCoords cx,
    MattCoords cy,
    MattCoords dx,
    MattCoords dy,
    MattCoords centerX,
    MattCoords centerY,
    uint32_t n_iterations,
    int /*iteration_precision*/)
{
    uint32_t result = hipSuccess;

    if (iter_matrix_cu == nullptr) {
        return result;
    }

    dim3 nb_blocks(w_block, h_block, 1);
    dim3 threads_per_block(NB_THREADS_W, NB_THREADS_H, 1);

    MattPerturbSingleResults<dblflt> cudaResults(
        dblflt_perturb->size,
        dblflt_perturb->PeriodMaybeZero,
        dblflt_perturb->iters);

    result = cudaResults.CheckValid();
    if (result != 0) {
        return result;
    }

    if (algorithm == RenderAlgorithm::Gpu2x32Perturbed) {
        dblflt cx2{ cx.flt.x, cx.flt.y };
        dblflt cy2{ cy.flt.x, cy.flt.y };
        dblflt dx2{ dx.flt.x, dx.flt.y };
        dblflt dy2{ dy.flt.x, dy.flt.y };
        dblflt centerX2{ centerX.flt.x, centerX.flt.y };
        dblflt centerY2{ centerY.flt.x, centerY.flt.y };

        mandel_2x_float_perturb_setup << <nb_blocks, threads_per_block >> > (cudaResults);

        mandel_2x_float_perturb << <nb_blocks, threads_per_block >> > (iter_matrix_cu,
            cudaResults,
            local_width, local_height, cx2, cy2, dx2, dy2,
            centerX2, centerY2,
            n_iterations);

        result = ExtractIters(buffer);
    }

    return result;
}

uint32_t GPURenderer::RenderPerturbBLA(
    RenderAlgorithm algorithm,
    uint32_t* buffer,
    MattPerturbResults<HDRFloat<float>>* perturb,
    BLAS<HDRFloat<float>>* blas,
    MattCoords cx,
    MattCoords cy,
    MattCoords dx,
    MattCoords dy,
    MattCoords centerX,
    MattCoords centerY,
    uint32_t n_iterations,
    int /*iteration_precision*/)
{
    uint32_t result = hipSuccess;

    if (iter_matrix_cu == nullptr) {
        return result;
    }

    dim3 nb_blocks(w_block, h_block, 1);
    dim3 threads_per_block(NB_THREADS_W, NB_THREADS_H, 1);

    if (algorithm == RenderAlgorithm::GpuHDRx32PerturbedBLA) {
        MattPerturbSingleResults<HDRFloat<float>> cudaResults(
            perturb->size,
            perturb->PeriodMaybeZero,
            perturb->iters);

        result = cudaResults.CheckValid();
        if (result != 0) {
            return result;
        }

        auto Run = [&]<int32_t LM2>() -> uint32_t {
            GPU_BLAS<HDRFloat<float>, BLA<HDRFloat<float>>, LM2> gpu_blas(blas->m_B);
            result = gpu_blas.CheckValid();
            if (result != 0) {
                return result;
            }

            mandel_1xHDR_InitStatics << <nb_blocks, threads_per_block >> > ();

            mandel_1xHDR_float_perturb_bla<HDRFloat<float>, LM2> << <nb_blocks, threads_per_block >> > (
                iter_matrix_cu,
                cudaResults,
                gpu_blas,
                local_width, local_height, cx.hdrflt, cy.hdrflt, dx.hdrflt, dy.hdrflt,
                centerX.hdrflt, centerY.hdrflt,
                n_iterations);

            return ExtractIters(buffer);
        };

        LargeSwitch
    }

    return result;
}

uint32_t GPURenderer::RenderPerturbBLA(
    RenderAlgorithm algorithm,
    uint32_t* buffer,
    MattPerturbResults<HDRFloat<double>>* perturb,
    BLAS<HDRFloat<double>>* blas,
    MattCoords cx,
    MattCoords cy,
    MattCoords dx,
    MattCoords dy,
    MattCoords centerX,
    MattCoords centerY,
    uint32_t n_iterations,
    int /*iteration_precision*/)
{
    uint32_t result = hipSuccess;

    if (iter_matrix_cu == nullptr) {
        return result;
    }

    dim3 nb_blocks(w_block, h_block, 1);
    dim3 threads_per_block(NB_THREADS_W, NB_THREADS_H, 1);

    if (algorithm == RenderAlgorithm::GpuHDRx64PerturbedBLA) {
        MattPerturbSingleResults<HDRFloat<double>> cudaResults(
            perturb->size,
            perturb->PeriodMaybeZero,
            perturb->iters);

        result = cudaResults.CheckValid();
        if (result != 0) {
            return result;
        }

        auto Run = [&]<int32_t LM2>() -> uint32_t {
            GPU_BLAS<HDRFloat<double>, BLA<HDRFloat<double>>, LM2> gpu_blas(blas->m_B);
            result = gpu_blas.CheckValid();
            if (result != 0) {
                return result;
            }

            mandel_1xHDR_InitStatics << <nb_blocks, threads_per_block >> > ();

            mandel_1xHDR_float_perturb_bla<HDRFloat<double>, LM2> << <nb_blocks, threads_per_block >> > (
                iter_matrix_cu,
                cudaResults,
                gpu_blas,
                local_width, local_height, cx.hdrflt, cy.hdrflt, dx.hdrflt, dy.hdrflt,
                centerX.hdrflt, centerY.hdrflt,
                n_iterations);

            return ExtractIters(buffer);
        };

        LargeSwitch
    }

    return result;
}

uint32_t GPURenderer::ExtractIters(uint32_t* buffer) {
    const size_t ERROR_COLOR = 255;
    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
        hipMemset(buffer, ERROR_COLOR, sizeof(int) * width * height);
        return result;
    }

    result = hipMemcpy(buffer,
                        iter_matrix_cu,
                        sizeof(int) * N_cu,
                        hipMemcpyDefault);
    if (result != hipSuccess) {
        return result;
    }

    return hipSuccess;
}

const char* GPURenderer::ConvertErrorToString(uint32_t err) {
    auto typeNotExposedOutSideHere = static_cast<hipError_t>(err);
    return hipGetErrorString(typeNotExposedOutSideHere);
}